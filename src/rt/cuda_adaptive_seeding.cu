#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2014 Nathaniel Jones
 * Massachusetts Institute of Technology
 */

#include <stdio.h>
#include <stdlib.h>

#include "optix_world.h"
#include "optix_common.h"

#include "kmeans.h"

//#define PRINT_CUDA

#ifdef __cplusplus
extern "C" {
#endif

// Ambient sample distribution based on Wang et al. (2009) "An efficient GPU-based approach for interactive global illumination"
__device__ inline static
PointDirection average_point_direction(const PointDirection& a, const PointDirection& b, const PointDirection& c, const PointDirection& d)
{
	PointDirection average;
	average.pos = (a.pos + b.pos + c.pos + d.pos) / 4.0f;
	//average.dir = optix::normalize(a.dir + b.dir + c.dir + d.dir);
	average.dir = a.dir + b.dir + c.dir + d.dir;
	const float length = optix::length(average.dir);
	if ( length > 0.0f )
		average.dir /= length;
	return average;
}

__device__ inline static
float geometric_error(const PointDirection& a, const PointDirection& b, const float alpha)
{
	return alpha * optix::length(a.pos - b.pos) + sqrtf(2.0f * fmaxf(1.0f - optix::dot(a.dir, b.dir), 0.0f));
}

__device__ inline static
void reduce(float *error, const int level, const int idX, const int idY, const int width)
{
	int tid = idX + idY * width;
	unsigned int stride = 1u;
	float err = error[tid];

	for (int i = 0; i < level; i++) {
		unsigned int stride2 = stride << 1;
		if (!(idX % stride2) && !(idY % stride2)) {
			err += error[tid + stride];
			err += error[tid + stride * width];
			err += error[tid + stride * (width + 1)];
		
			error[tid] = err;
		}
		stride = stride2;
		__syncthreads();
	}
}

// Ambient sample distribution
__global__ static
void geometric_variation(PointDirection *deviceHits, int *seed,
				   const unsigned int width, const unsigned int height, const unsigned int levels, const float alpha)
{
	extern __shared__ PointDirection blockSharedMemory[];

	unsigned int idX = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int idY = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int tid = idX + idY * width;
	unsigned int sid = blockDim.x * threadIdx.y + threadIdx.x;

	float *err = (float*)malloc(levels * sizeof(float));
	unsigned int stride = 1u;

	PointDirection hit = deviceHits[tid];
	unsigned int valid = optix::dot(hit.dir, hit.dir) > 0.0f && optix::dot(hit.pos, hit.pos) >= 0.0f;
	if (!valid)
		hit.pos.x = hit.pos.y = hit.pos.z = hit.dir.x = hit.dir.y = hit.dir.z = 0.0f;
	PointDirection accum = hit;
	blockSharedMemory[sid] = hit;

	__syncthreads();

	/* Calculate geometric error for each hit point to each quad-tree node. */
	for (int i = 0; i < levels; i++) {
		unsigned int stride2 = stride << 1;

		if (!(idX % stride2) && !(idY % stride2)) {
			accum = average_point_direction(
				accum,
				blockSharedMemory[sid + stride],
				blockSharedMemory[sid + stride * width],
				blockSharedMemory[sid + stride * (width + 1)]
			);

			blockSharedMemory[sid] = accum;
		}

		__syncthreads();

		err[i] = valid ? geometric_error(hit, blockSharedMemory[sid - idX % stride2 - (idY % stride2) * blockDim.x], alpha) : 0.0f;
		stride = stride2;
	}

	__syncthreads();

	float *error = (float *)blockSharedMemory;

	for (int i = levels; i--; ) {
		unsigned int stride2 = stride >> 1;

		/* Calculate geometric error average at each quad-tree node. */
		error[tid] = err[i];

		__syncthreads();

		if (i) 
			reduce(error, i, idX, idY, width); // sum errors at this quad tree node

		/* Divide the pool proportinally to error at each quad-tree node. */
		if (!(idX % stride) && !(idY % stride)) {
			float err0 = error[tid];
			float err1 = error[tid + stride2];
			float err2 = error[tid + stride2 * width];
			float err3 = error[tid + stride2 * (width + 1)];
			float errSum = err0 + err1 + err2 + err3;
			int seedSum = seed[tid];
			float scoreSum = errSum > 0.0f ? seedSum / errSum : 0.0f;

			int s[4];
			s[0] = scoreSum * err0 + 0.5f;
			s[1] = scoreSum * err1 + 0.5f;
			s[2] = scoreSum * err2 + 0.5f;
			s[3] = scoreSum * err3 + 0.5f;
			int diff = seedSum - s[0] - s[1] - s[2] - s[3];
			if (diff && errSum > 0.0f) {
				int maxIndex = err0 > err1 ?
								err0 > err2 ?
									err0 > err3 ? 0 : 3 :
									err2 > err3 ? 2 : 3 :
								err1 > err2 ?
									err1 > err3 ? 1 : 3 :
									err2 > err3 ? 2 : 3;
				s[maxIndex] += diff;
			}
			seed[tid]                         = s[0];
			seed[tid + stride2]               = s[1];
			seed[tid + stride2 * width]       = s[2];
			seed[tid + stride2 * (width + 1)] = s[3];
		}

		__syncthreads();

		stride = stride2;
	}

	free(err);
}

static unsigned int __cdecl calc_block_dim(const unsigned int maxThreadsPerBlock, const unsigned int levels)
{
	unsigned int blockDim = 1u;
	unsigned int size = maxThreadsPerBlock << 1;
	while ( size >>= 2 )
		blockDim <<= 1;
	if ( blockDim > (1u << levels) )
		blockDim = 1 << levels;
	return blockDim;
}

/* Score the relative need for an irradiance cache entry at each hit point */
void __cdecl cuda_score_hits(PointDirection *hits, int *seeds, const unsigned int width, const unsigned int height, const float weight, const unsigned int seed_count)
{
	PointDirection *deviceHits;
	int *deviceSeeds;
	
	/* Calculate number of levels */
	unsigned int levels = 0;
	unsigned int size = width > height ? width : height;
	while ( size >>= 1 )
		levels++;
	fprintf(stderr, "Levels: %i\n", levels);

	/* Determine block size */
	hipDeviceProp_t deviceProp;
	int deviceNum;
	hipGetDevice(&deviceNum);
	hipGetDeviceProperties(&deviceProp, deviceNum);

	/* To support reduction, blockDim *must* be a power of two. */
	const unsigned int blockDim = calc_block_dim(deviceProp.maxThreadsPerBlock, levels);
	const unsigned int blocksX = (width - 1) / blockDim + 1;
	const unsigned int blocksY = (height - 1) / blockDim + 1;
	const unsigned int blockSharedMemorySize = blockDim * blockDim * sizeof(PointDirection);

	if (blockSharedMemorySize > deviceProp.sharedMemPerBlock)
		err("WARNING: Your CUDA hardware has insufficient block shared memory %u (%u needed).\n", deviceProp.sharedMemPerBlock, blockSharedMemorySize);

	const dim3 dimGrid(blocksX, blocksY);
	const dim3 dimBlock(blockDim, blockDim);
	fprintf(stderr, "Block %i x %i, Grid %i x %i, Shared %i\n", blockDim, blockDim, blocksX, blocksY, blockSharedMemorySize);

	/* Allocate memory on the GPU */
	size = width * height;
	checkCuda(hipMalloc(&deviceHits, size * sizeof(PointDirection)));
	checkCuda(hipMalloc(&deviceSeeds, size * sizeof(int)));

	/* Copy data to GPU */
	seeds[0] = seed_count;
	fprintf(stderr, "Target total score: %i\n", seed_count);
	checkCuda(hipMemcpy(deviceHits, hits, size * sizeof(PointDirection), hipMemcpyHostToDevice));
	checkCuda(hipMemcpy(deviceSeeds, seeds, sizeof(int), hipMemcpyHostToDevice)); // transfer only first entry

	/* Run kernel */
	geometric_variation <<< dimGrid, dimBlock, blockSharedMemorySize >>>
			(deviceHits, deviceSeeds, width, height, levels, weight);
	
	hipDeviceSynchronize(); checkLastCudaError();

	/* Copy results from GPU */
	checkCuda(hipMemcpy(seeds, deviceSeeds, size * sizeof(int), hipMemcpyDeviceToHost));

	/* Free memory on the GPU */
	checkCuda(hipFree(deviceHits));
	checkCuda(hipFree(deviceSeeds));
}


// Ambient sample distribution for large images
__global__ static
void mip_map_hits(PointDirection *deviceHits, PointDirection *deviceMipMap,
				   const unsigned int width, const unsigned int height)
{
	extern __shared__ PointDirection blockSharedMemory[];

	unsigned int idX = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int idY = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int tid = idX + idY * width;
	unsigned int sid = blockDim.x * threadIdx.y + threadIdx.x;
	unsigned int offset = 0u;

	unsigned int stride = 1u;

	PointDirection hit = deviceHits[tid];
	unsigned int valid = optix::dot(hit.dir, hit.dir) > 0.0f && optix::dot(hit.pos, hit.pos) >= 0.0f;
	if (!valid)
		hit.pos.x = hit.pos.y = hit.pos.z = hit.dir.x = hit.dir.y = hit.dir.z = 0.0f;
	PointDirection accum = hit;
	blockSharedMemory[sid] = hit;

	__syncthreads();

	/* Calculate geometric error for each hit point to each quad-tree node. */
	while (stride < blockDim.x) {
#ifdef PRINT_CUDA
		if (!tid)
			printf("mip_map_hits stride=%i, offset=%i, accum=%g,%g,%g\n", stride, offset, accum.pos.x, accum.pos.y, accum.pos.z);
#endif
		unsigned int stride2 = stride << 1;

		if (!(idX % stride2) && !(idY % stride2)) {
			accum = average_point_direction(
				accum,
				blockSharedMemory[sid + stride],
				blockSharedMemory[sid + stride * blockDim.x],
				blockSharedMemory[sid + stride * (blockDim.x + 1)]
			);

			blockSharedMemory[sid] = accum;
			deviceMipMap[offset + (idX + idY * width / stride2) / stride2] = accum;
		}
#ifdef PRINT_CUDA
		if (!tid)
			printf("mip_map_hits accum=%g,%g,%g\n", accum.pos.x, accum.pos.y, accum.pos.z);
#endif

		__syncthreads();

		stride = stride2;
		offset += (width * height) / (stride2 * stride2);
	}
}

__global__ static
void calc_error(PointDirection *devicePointDirections, PointDirection *deviceMipMap, float *error,
				   const unsigned int width, const unsigned int height, const unsigned int levels, float alpha)
{
	unsigned int idX = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int idY = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int tid = idX + idY * width;

	unsigned int stride = 1u;

	PointDirection hit = devicePointDirections[tid];
	unsigned int valid = optix::dot(hit.dir, hit.dir) > 0.0f && optix::dot(hit.pos, hit.pos) >= 0.0f;
	if (!valid)
		hit.pos.x = hit.pos.y = hit.pos.z = hit.dir.x = hit.dir.y = hit.dir.z = 0.0f;

	PointDirection *mipMapLevel = deviceMipMap;

	/* Calculate geometric error for each hit point to each quad-tree node. */
	for (unsigned int i = 0u; i < levels; i++) {
#ifdef PRINT_CUDA
		if (!tid)
			printf("calc_error stride=%i, i=%i, valid=%i\n", stride, i, valid);
#endif
		stride <<= 1;

		error[tid + i * width * height] = valid ? geometric_error(hit, mipMapLevel[idX / stride + (idY / stride) * (width / stride)], alpha) : 0.0f;
		mipMapLevel += (width * height) / (stride * stride);
	}
}

__global__ static
void reduce_error(float *error, const unsigned int width, const unsigned int height, const unsigned int levels, const unsigned int scale)
{
	unsigned int idX = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int idY = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int tid = (idX + idY * width) * scale;

	for (unsigned int j = 1u; j < levels; j++) {
		tid += width * height;
		float err = error[tid];

		unsigned int stride = 1u;

		while (stride < (1 << j) && stride < blockDim.x) {
#ifdef PRINT_CUDA
			if (!(tid % (width * height)))
				printf("reduce_error stride=%i, j=%i, scale=%i, err=%g\n", stride, j, scale, err);
#endif
			unsigned int stride2 = stride << 1;
			if (!(idX % stride2) && !(idY % stride2)) {
				err += error[tid + stride * scale];
				err += error[tid + stride * scale * width];
				err += error[tid + stride * scale * (width + 1)];
		
				error[tid] = err;
			}
			stride = stride2;
			__syncthreads();
		}
	}
}

__global__ static
void calc_score(float *error, int *seed, const unsigned int width, const unsigned int height, const unsigned int levels, const unsigned int scale)
{
	unsigned int idX = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int idY = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int tid = (idX + idY * width) * scale;

	unsigned int stride = 1 << levels;

	for (int i = levels; i--; ) {
		unsigned int stride2 = stride >> 1;

		/* Divide the pool proportinally to error at each quad-tree node. */
		if (!(idX % stride) && !(idY % stride)) {
			unsigned int lid = tid + width * height * i;
			float err0 = error[lid];
			float err1 = error[lid + stride2 * scale];
			float err2 = error[lid + stride2 * scale * width];
			float err3 = error[lid + stride2 * scale * (width + 1)];
			float errSum = err0 + err1 + err2 + err3;
			int seedSum = seed[tid];
			float scoreSum = errSum > 0.0f ? seedSum / errSum : 0.0f;

			int s[4];
			s[0] = scoreSum * err0 + 0.5f;
			s[1] = scoreSum * err1 + 0.5f;
			s[2] = scoreSum * err2 + 0.5f;
			s[3] = scoreSum * err3 + 0.5f;
			int diff = seedSum - s[0] - s[1] - s[2] - s[3];
#ifdef PRINT_CUDA
			if (!tid)
				printf("calc_score stride=%i, i=%i, lid=%i, scale=%i, errSum=%g, seedSum=%i, scoreSum=%g, diff=%i\n", stride, i, lid, scale, errSum, seedSum, scoreSum, diff);
#endif
			if (diff && errSum > 0.0f) {
				int maxIndex = err0 > err1 ?
								err0 > err2 ?
									err0 > err3 ? 0 : 3 :
									err2 > err3 ? 2 : 3 :
								err1 > err2 ?
									err1 > err3 ? 1 : 3 :
									err2 > err3 ? 2 : 3;
				s[maxIndex] += diff;
			}
			seed[tid]                                 = s[0];
			seed[tid + stride2 * scale]               = s[1];
			seed[tid + stride2 * scale * width]       = s[2];
			seed[tid + stride2 * scale * (width + 1)] = s[3];
		}

		__syncthreads();

		stride = stride2;
	}
}

/* Calculate average of hits at each quad tree node */
static void __cdecl cuda_mip_map_hits_recursive(PointDirection *deviceHits, PointDirection *deviceMipMap,
	const unsigned int width, const unsigned int height, const unsigned int levels, const unsigned int maxThreadsPerBlock, dim3 dimGrid, dim3 dimBlock, unsigned int blockSharedMemorySize)
{
	/* Calculate average of hits at each quad tree node */
	mip_map_hits <<< dimGrid, dimBlock, blockSharedMemorySize >>>
			(deviceHits, deviceMipMap, width, height);

	hipDeviceSynchronize(); checkLastCudaError();

	if ( dimBlock.x < (1u << levels) ) {
		unsigned int complete = 1u;
		unsigned int offset = 0u;
		unsigned int nextOffset = (width * height) >> 2; // TODO assumes size is a power of 4
		for (unsigned int i = 1u; i < dimBlock.x / 2u; i <<= 1) {
			complete++;
			offset += nextOffset;
			nextOffset >>= 2;
		}

		const unsigned int blockDim = calc_block_dim(maxThreadsPerBlock, levels - complete);
		const unsigned int blocksX = (dimGrid.x - 1) / blockDim + 1;
		const unsigned int blocksY = (dimGrid.y - 1) / blockDim + 1;
		const dim3 dimSuperGrid(blocksX, blocksY);
		const dim3 dimSuperBlock(blockDim, blockDim);

		cuda_mip_map_hits_recursive(deviceMipMap + offset, deviceMipMap + offset + nextOffset, dimGrid.x, dimGrid.y, levels - complete, maxThreadsPerBlock, dimSuperGrid, dimSuperBlock, dimSuperBlock.x * dimSuperBlock.y * sizeof(PointDirection));
	}
}

/* Calculate average geometric variation for each quad tree node */
static void __cdecl cuda_score_hits_recursive(float *deviceError, int *deviceSeeds,
	const unsigned int width, const unsigned int height, unsigned int levels, const unsigned int scale, const unsigned int maxThreadsPerBlock, dim3 dimGrid, dim3 dimBlock)
{
	/* Perform reduction on error */
	reduce_error <<< dimGrid, dimBlock >>>
			(deviceError, width, height, levels, scale);

	hipDeviceSynchronize(); checkLastCudaError();

	/* Recruse if block not large enough for reduction */
	if ( dimBlock.x < (1u << levels) ) {
		unsigned int complete = 0u;
		for (unsigned int i = 1u; i < dimBlock.x; i <<= 1)
			complete++;

		const unsigned int blockDim = calc_block_dim(maxThreadsPerBlock, levels - complete);
		const unsigned int blocksX = (dimGrid.x - 1) / blockDim + 1;
		const unsigned int blocksY = (dimGrid.y - 1) / blockDim + 1;
		const dim3 dimSuperGrid(blocksX, blocksY);
		const dim3 dimSuperBlock(blockDim, blockDim);

		cuda_score_hits_recursive(deviceError + width * height * complete, deviceSeeds, width, height, levels - complete, scale * dimBlock.x, maxThreadsPerBlock, dimSuperGrid, dimSuperBlock);
		levels = complete;
	}

	/* Calculate score for each leaf node based on error */
	calc_score <<< dimGrid, dimBlock >>>
			(deviceError, deviceSeeds, width, height, levels, scale);

	hipDeviceSynchronize(); checkLastCudaError();
}

/* Score the relative need for an irradiance cache entry at each hit point */
void __cdecl cuda_score_hits_big(PointDirection *hits, int *seeds, const unsigned int width, const unsigned int height, const float weight, const unsigned int seed_count)
{
	PointDirection *deviceHits, *deviceMipMap;
	float *deviceError;
	int *deviceSeeds;
	
	/* Calculate number of levels */
	unsigned int levels = 0;
	unsigned int size = width > height ? width : height;
	while ( size >>= 1 )
		levels++;
	fprintf(stderr, "Levels: %i\n", levels);

	/* Determine block size */
	hipDeviceProp_t deviceProp;
	int deviceNum;
	hipGetDevice(&deviceNum);
	hipGetDeviceProperties(&deviceProp, deviceNum);

	/* To support reduction, blockDim *must* be a power of two. */
	const unsigned int blockDim = calc_block_dim(deviceProp.maxThreadsPerBlock, levels);
	const unsigned int blocksX = (width - 1) / blockDim + 1;
	const unsigned int blocksY = (height - 1) / blockDim + 1;
	const unsigned int blockSharedMemorySize = blockDim * blockDim * sizeof(PointDirection);

	if (blockSharedMemorySize > deviceProp.sharedMemPerBlock)
		err("WARNING: Your CUDA hardware has insufficient block shared memory %u (%u needed).\n", deviceProp.sharedMemPerBlock, blockSharedMemorySize);

	const dim3 dimGrid(blocksX, blocksY);
	const dim3 dimBlock(blockDim, blockDim);
	fprintf(stderr, "Block %i x %i, Grid %i x %i, Shared %i, Weight %g\n", blockDim, blockDim, blocksX, blocksY, blockSharedMemorySize, weight);

	/* Allocate memory on the GPU */
	size = width * height;
	checkCuda(hipMalloc(&deviceHits, size * sizeof(PointDirection)));
	checkCuda(hipMalloc(&deviceMipMap, size * sizeof(PointDirection) / 3u)); // Storage requirement for mip map is 1/3 or original data
	checkCuda(hipMalloc(&deviceError, size * levels * sizeof(float)));

	/* Copy data to GPU */
	checkCuda(hipMemcpy(deviceHits, hits, size * sizeof(PointDirection), hipMemcpyHostToDevice));

	/* Calculate average of hits at each quad tree node */
	cuda_mip_map_hits_recursive(deviceHits, deviceMipMap, width, height, levels, deviceProp.maxThreadsPerBlock, dimGrid, dimBlock, blockSharedMemorySize);

	/* Calculate geometric variation at each quad tree node */
	calc_error <<< dimGrid, dimBlock >>>
			(deviceHits, deviceMipMap, deviceError, width, height, levels, weight);

	hipDeviceSynchronize(); checkLastCudaError();

	/* Free memory on the GPU */
	checkCuda(hipFree(deviceHits));
	checkCuda(hipFree(deviceMipMap));

	/* Allocate memory on the GPU */
	checkCuda(hipMalloc(&deviceSeeds, size * sizeof(int)));

	/* Copy data to GPU */
	seeds[0] = seed_count;
	fprintf(stderr, "Target total score: %i\n", seed_count);
	checkCuda(hipMemcpy(deviceSeeds, seeds, sizeof(int), hipMemcpyHostToDevice)); // transfer only first entry

	/* Calculate average geometric variation for each quad tree node */
	cuda_score_hits_recursive(deviceError, deviceSeeds, width, height, levels, 1u, deviceProp.maxThreadsPerBlock, dimGrid, dimBlock);

	/* Copy results from GPU */
	checkCuda(hipMemcpy(seeds, deviceSeeds, size * sizeof(int), hipMemcpyDeviceToHost));

	/* Free memory on the GPU */
	checkCuda(hipFree(deviceError));
	checkCuda(hipFree(deviceSeeds));
}


#ifdef __cplusplus
}
#endif
