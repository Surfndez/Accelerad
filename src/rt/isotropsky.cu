#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2015 Nathaniel Jones
 * Massachusetts Institute of Technology
 */

#include <optix_world.h>
//#include "optix_shader_common.h"

struct Transform
{
	optix::Matrix<3,3> m;
};

/* Program variables */
rtDeclareVariable(float,      skybright, , ); /* isotropic sky radiance */
rtDeclareVariable(Transform,  transform, , ); /* transformation matrix, ignored */

// Calculate the isotropic sky value for the current ray direction.
// This function replicates the algorithm in isotrop_sky.cal distributed with Daysim.
RT_CALLABLE_PROGRAM float3 isotrop_sky(const float3 ignore0, const float3 ignore1)
{
	return make_float3(skybright);
}
