#include "hip/hip_runtime.h"
/*
* Copyright (c) 2013-2015 Nathaniel Jones
* Massachusetts Institute of Technology
*/

#include <optix_world.h>
#include "optix_shader_common.h"

#define GAMMA  2.2f
#define LUMINOUS_EFFICACY 179.0f

using namespace optix;

/* Program variables */
rtDeclareVariable(unsigned int, frame, , ); /* Current frame number, starting from zero */
rtDeclareVariable(unsigned int, camera, , ); /* Camera type (-vt) */
rtDeclareVariable(float3, eye, , ); /* Eye position (-vp) */
rtDeclareVariable(float3, U, , ); /* view.hvec */
rtDeclareVariable(float3, V, , ); /* view.vvec */
rtDeclareVariable(float3, W, , ); /* view.vdir */
rtDeclareVariable(float2, fov, , ); /* Field of view (-vh, -vv) */
rtDeclareVariable(float2, shift, , ); /* Camera shift (-vs, -vl) */
rtDeclareVariable(float2, clip, , ); /* Fore and aft clipping planes (-vo, -va) */
rtDeclareVariable(float, vdist, , ); /* Focal length */
rtDeclareVariable(float, dstrpix, , ); /* Pixel sample jitter (-pj) */
rtDeclareVariable(unsigned int, do_irrad, , ); /* Calculate irradiance (-i) */

rtDeclareVariable(float, exposure, , ) = 1.0f; /* Current exposure */
rtDeclareVariable(unsigned int, greyscale, , ) = 0u; /* Convert to monocrhome */
rtDeclareVariable(int, tonemap, , ) = RT_TEXTURE_ID_NULL; /* texture ID */
rtDeclareVariable(float, fc_scale, , ) = 1000.0f; /* Current exposure */
rtDeclareVariable(int, fc_log, , ) = 0; /* Current exposure */
rtDeclareVariable(float, fc_mask, , ) = 0.0f; /* Current exposure */

/* Contex variables */
rtBuffer<unsigned int, 2>        output_buffer;
rtBuffer<float3, 2>              direct_buffer; /* GPU storage for direct component */
rtBuffer<float3, 2>              diffuse_buffer; /* GPU storage for diffuse component */
#ifdef RAY_COUNT
rtBuffer<unsigned int, 2>        ray_count_buffer;
#endif
//rtBuffer<RayParams, 2>           last_view_buffer;
//rtBuffer<unsigned int, 2>        rnd_seeds;
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(unsigned int, radiance_primary_ray_type, , );
rtDeclareVariable(unsigned int, diffuse_ray_type, , );

/* OptiX variables */
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim, rtLaunchDim, );


// Pick the ray direction based on camera type as in image.c.
RT_PROGRAM void ray_generator()
{
	PerRayData_radiance prd;
	init_rand(&prd.state, launch_index.x + launch_dim.x * (launch_index.y + launch_dim.y * frame));

	float2 d = make_float2(hiprand_uniform(prd.state), hiprand_uniform(prd.state));
	d = 0.5f + dstrpix * (0.5f - d); // this is pixjitter() from rpict.c
	d = shift + (make_float2(launch_index) + d) / make_float2(launch_dim) - 0.5f;
	float3 ray_origin = eye;
	float z = 1.0f;

	// This is adapted from viewray() in image.c.
	if (camera == VT_PAR) { /* parallel view */
		ray_origin += d.x*U + d.y*V;
		d = make_float2(0.0f);
	}
	else if (camera == VT_HEM) { /* hemispherical fisheye */
		z = 1.0f - d.x*d.x * dot(U, U) - d.y*d.y * dot(V, V);
		if (z < 0.0f) {
			output_buffer[launch_index] = 0xff000000;//TODO throw an exception?
			return;
		}
		z = sqrtf(z);
	}
	else if (camera == VT_CYL) { /* cylindrical panorama */
		float dd = d.x * fov.x * (M_PIf / 180.0f);
		z = cosf(dd);
		d.x = sinf(dd);
	}
	else if (camera == VT_ANG) { /* angular fisheye */
		d *= fov / 180.0f;
		float dd = length(d);
		if (dd > 1.0f) {
			output_buffer[launch_index] = 0xff000000;//TODO throw an exception?
			return;
		}
		z = cosf(M_PIf * dd);
		d *= sqrtf(1.0f - z*z) / dd;
	}
	else if (camera == VT_PLS) { /* planispheric fisheye */
		d *= make_float2(length(U), length(V));
		float dd = dot(d, d);
		z = (1.0f - dd) / (1.0f + dd);
		d *= 1.0f + z;
	}

	float3 ray_direction = d.x*U + d.y*V + z*W;
	ray_origin += clip.x * ray_direction;
	ray_direction = normalize(ray_direction);

	// Zero or negative aft clipping distance indicates infinity
	float aft = clip.y - clip.x;
	if (aft <= FTINY) {
		aft = RAY_END;
	}

	Ray ray = make_Ray(ray_origin, ray_direction, frame ? diffuse_ray_type : do_irrad ? radiance_primary_ray_type : radiance_ray_type, 0.0f, aft);

	prd.weight = 1.0f;
	prd.depth = 0;
	prd.ambient_depth = 0;
	//prd.seed = rnd_seeds[launch_index];
	setupPayload(prd, 1);

	rtTrace(top_object, ray, prd);

	checkFinite(prd.result);

	float3 accum;
	if (frame)
		accum = direct_buffer[launch_index] + (diffuse_buffer[launch_index] = ((frame - 1.0f) / frame) * diffuse_buffer[launch_index] + (1.0f / frame) * prd.result);
	else
		accum = direct_buffer[launch_index] = prd.result;

	/* Tone map */
	if (tonemap == RT_TEXTURE_ID_NULL) {
		accum *= exposure;
		if (greyscale)
			accum = make_float3(bright(accum));
	}
	else {
		float luminance = bright(accum) * LUMINOUS_EFFICACY;
		if (luminance < fc_mask)
			accum = make_float3(0.0f);
		else if (fc_log > 0)
			accum = make_float3(rtTex1D<float4>(tonemap, log10f(luminance / fc_scale) / fc_log + 1.0f));
		else
			accum = make_float3(rtTex1D<float4>(tonemap, luminance / fc_scale));
	}
	accum = clamp(accum * 256.0f, 0.0f, 255.0f);

	output_buffer[launch_index] = 0xff000000 |
		((int)(256.0f * powf((accum.x + 0.5f) / 256.0f, 1.0f / GAMMA)) & 0xff) << 16 |
		((int)(256.0f * powf((accum.y + 0.5f) / 256.0f, 1.0f / GAMMA)) & 0xff) << 8 |
		((int)(256.0f * powf((accum.z + 0.5f) / 256.0f, 1.0f / GAMMA)) & 0xff);

#ifdef RAY_COUNT
	if (frame)
		ray_count_buffer[launch_index] += prd.ray_count;
	else
		ray_count_buffer[launch_index] = prd.ray_count;
#endif
}

RT_PROGRAM void exception()
{
	const unsigned int code = rtGetExceptionCode();
	rtPrintf("Caught exception 0x%X at launch index (%d,%d)\n", code, launch_index.x, launch_index.y);
	output_buffer[launch_index] = code;
}
