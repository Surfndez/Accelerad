#include "hip/hip_runtime.h"
/*
* Copyright (c) 2013-2015 Nathaniel Jones
* Massachusetts Institute of Technology
*/

#include <optix_world.h>
#include "optix_shader_common.h"

#define GAMMA  2.2f
#define LUMINOUS_EFFICACY 179.0f

using namespace optix;

/* Program variables */
rtDeclareVariable(unsigned int, frame, , ); /* Current frame number, starting from zero */
rtDeclareVariable(unsigned int, camera, , ); /* Camera type (-vt) */
rtDeclareVariable(float3, eye, , ); /* Eye position (-vp) */
rtDeclareVariable(float3, U, , ); /* view.hvec */
rtDeclareVariable(float3, V, , ); /* view.vvec */
rtDeclareVariable(float3, W, , ); /* view.vdir */
rtDeclareVariable(float2, fov, , ); /* Field of view (-vh, -vv) */
rtDeclareVariable(float2, shift, , ); /* Camera shift (-vs, -vl) */
rtDeclareVariable(float2, clip, , ); /* Fore and aft clipping planes (-vo, -va) */
rtDeclareVariable(float, vdist, , ); /* Focal length */
rtDeclareVariable(float, dstrpix, , ); /* Pixel sample jitter (-pj) */
rtDeclareVariable(unsigned int, do_irrad, , ); /* Calculate irradiance (-i) */

rtDeclareVariable(float, exposure, , ) = 1.0f; /* Current exposure */
rtDeclareVariable(unsigned int, greyscale, , ) = 0u; /* Convert to monocrhome */
rtDeclareVariable(int, tonemap, , ) = RT_TEXTURE_ID_NULL; /* texture ID */
rtDeclareVariable(float, fc_scale, , ) = 1000.0f; /* Current exposure */
rtDeclareVariable(int, fc_log, , ) = 0; /* Current exposure */
rtDeclareVariable(float, fc_mask, , ) = 0.0f; /* Current exposure */

/* Contex variables */
rtBuffer<unsigned int, 2>        color_buffer; /* Output RGBA colors */
rtBuffer<Metrics, 2>             metrics_buffer; /* Output metrics */
rtBuffer<float3, 2>              direct_buffer; /* GPU storage for direct component */
rtBuffer<float3, 2>              diffuse_buffer; /* GPU storage for diffuse component */
#ifdef RAY_COUNT
rtBuffer<unsigned int, 2>        ray_count_buffer;
#endif
//rtBuffer<RayParams, 2>           last_view_buffer;
//rtBuffer<unsigned int, 2>        rnd_seeds;
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(unsigned int, radiance_primary_ray_type, , );
rtDeclareVariable(unsigned int, diffuse_ray_type, , );

/* OptiX variables */
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim, rtLaunchDim, );


RT_METHOD float3 getViewDirection(const float &x, const float &y);
RT_METHOD int splane_normal(const float3 &e1, const float3 &e2, float3 &n);
RT_METHOD float getSolidAngle();
RT_METHOD float getPositionIndex(const float3 &dir);


// Pick the ray direction based on camera type as in image.c.
RT_PROGRAM void ray_generator()
{
	PerRayData_radiance prd;
	init_rand(&prd.state, launch_index.x + launch_dim.x * (launch_index.y + launch_dim.y * frame));

	float2 d = make_float2(hiprand_uniform(prd.state), hiprand_uniform(prd.state));
	d = 0.5f + dstrpix * (0.5f - d); // this is pixjitter() from rpict.c
	d = shift + (make_float2(launch_index) + d) / make_float2(launch_dim) - 0.5f;
	float3 ray_origin = eye;
	float z = 1.0f;

	// This is adapted from viewray() in image.c.
	if (camera == VT_PAR) { /* parallel view */
		ray_origin += d.x*U + d.y*V;
		d = make_float2(0.0f);
	}
	else if (camera == VT_HEM) { /* hemispherical fisheye */
		z = 1.0f - d.x*d.x * dot(U, U) - d.y*d.y * dot(V, V);
		if (z < 0.0f) {
			color_buffer[launch_index] = 0xff000000;//TODO throw an exception?
			return;
		}
		z = sqrtf(z);
	}
	else if (camera == VT_CYL) { /* cylindrical panorama */
		float dd = d.x * fov.x * (M_PIf / 180.0f);
		z = cosf(dd);
		d.x = sinf(dd);
	}
	else if (camera == VT_ANG) { /* angular fisheye */
		d *= fov / 180.0f;
		float dd = length(d);
		if (dd > 1.0f) {
			color_buffer[launch_index] = 0xff000000;//TODO throw an exception?
			return;
		}
		z = cosf(M_PIf * dd);
		d *= sqrtf(1.0f - z*z) / dd;
	}
	else if (camera == VT_PLS) { /* planispheric fisheye */
		d *= make_float2(length(U), length(V));
		float dd = dot(d, d);
		z = (1.0f - dd) / (1.0f + dd);
		d *= 1.0f + z;
	}

	float3 ray_direction = d.x*U + d.y*V + z*W;
	ray_origin += clip.x * ray_direction;
	ray_direction = normalize(ray_direction);

	// Zero or negative aft clipping distance indicates infinity
	float aft = clip.y - clip.x;
	if (aft <= FTINY) {
		aft = RAY_END;
	}

	Ray ray = make_Ray(ray_origin, ray_direction, frame ? diffuse_ray_type : do_irrad ? radiance_primary_ray_type : radiance_ray_type, 0.0f, aft);

	prd.result = make_float3(0.0f);
	prd.weight = 1.0f;
	prd.depth = 0;
	prd.ambient_depth = 0;
	//prd.seed = rnd_seeds[launch_index];
	setupPayload(prd, 1);

	rtTrace(top_object, ray, prd);

	checkFinite(prd.result);

	float3 accum;
	if (frame)
		accum = direct_buffer[launch_index] + (diffuse_buffer[launch_index] = ((frame - 1.0f) / frame) * diffuse_buffer[launch_index] + (1.0f / frame) * prd.result);
	else
		accum = direct_buffer[launch_index] = prd.result;

	/* Tone map */
	const float luminance = bright(accum) * LUMINOUS_EFFICACY;
	if (tonemap == RT_TEXTURE_ID_NULL) {
		accum *= exposure;
		if (greyscale)
			accum = make_float3(bright(accum));
	}
	else {
		if (luminance < fc_mask)
			accum = make_float3(0.0f);
		else if (fc_log > 0)
			accum = make_float3(rtTex1D<float4>(tonemap, log10f(luminance / fc_scale) / fc_log + 1.0f));
		else
			accum = make_float3(rtTex1D<float4>(tonemap, luminance / fc_scale));
	}
	accum = clamp(accum * 256.0f, 0.0f, 255.0f);

	/* Save pixel color */
	color_buffer[launch_index] = 0xff000000 |
		((int)(256.0f * powf((accum.x + 0.5f) / 256.0f, 1.0f / GAMMA)) & 0xff) << 16 |
		((int)(256.0f * powf((accum.y + 0.5f) / 256.0f, 1.0f / GAMMA)) & 0xff) << 8 |
		((int)(256.0f * powf((accum.z + 0.5f) / 256.0f, 1.0f / GAMMA)) & 0xff);

	/* Calculate metrics */
	float guth = getPositionIndex(ray.direction);
	Metrics metrics;
	metrics.omega = getSolidAngle(); //TODO what if negative or bad angle?
	metrics.ev = luminance * dot(W, ray.direction * metrics.omega);
	metrics.avlum = luminance * metrics.omega;
	metrics.dgp = luminance * luminance * metrics.omega / (guth * guth);
	metrics_buffer[launch_index] = metrics;

#ifdef RAY_COUNT
	if (frame)
		ray_count_buffer[launch_index] += prd.ray_count;
	else
		ray_count_buffer[launch_index] = prd.ray_count;
#endif
}

/* From viewray() in image.c */
RT_METHOD float3 getViewDirection(const float &x, const float &y)
{
	float2 d = make_float2(x, y) - 0.5f;
	float z = 1.0f;

	if (camera == VT_PAR) { /* parallel view */
		d = make_float2(0.0f);
	}
	else if (camera == VT_HEM) { /* hemispherical fisheye */
		z = 1.0f - d.x*d.x * dot(U, U) - d.y*d.y * dot(V, V);
		if (z < 0.0f)
			return make_float3(0.0f);
		z = sqrtf(z);
	}
	else if (camera == VT_CYL) { /* cylindrical panorama */
		float dd = d.x * fov.x * (M_PIf / 180.0f);
		z = cosf(dd);
		d.x = sinf(dd);
	}
	else if (camera == VT_ANG) { /* angular fisheye */
		d *= fov / 180.0f;
		float dd = length(d);
		if (dd > 1.0f)
			return make_float3(0.0f);
		z = cosf(M_PIf * dd);
		d *= sqrtf(1.0f - z*z) / dd;
	}
	else if (camera == VT_PLS) { /* planispheric fisheye */
		d *= make_float2(length(U), length(V));
		float dd = dot(d, d);
		z = (1.0f - dd) / (1.0f + dd);
		d *= 1.0f + z;
	}

	return d.x*U + d.y*V + z*W;
}

/* From splane_normal in pictool.c */
RT_METHOD int splane_normal(const float3 &e1, const float3 &e2, float3 &n)
{
	n = cross(e1, e2 - e1);
	if (dot(n, n) == 0.0f)
		return 0;
	n = normalize(n);
	return 1;
}

/* From pict_get_sangle in pictool.c */
RT_METHOD float getSolidAngle()
{
	const float2 min = make_float2(launch_index) / make_float2(launch_dim);
	const float2 max = (make_float2(launch_index) + 1.0f) / make_float2(launch_dim);
	const float3 minmin = getViewDirection(min.x, min.y);
	const float3 minmax = getViewDirection(min.x, max.y);
	const float3 maxmin = getViewDirection(max.x, min.y);
	const float3 maxmax = getViewDirection(max.x, max.y);

	float3 n[4] = { make_float3(0.0f), make_float3(0.0f), make_float3(0.0f), make_float3(0.0f) };

	int i = splane_normal(minmin, minmax, n[0]);
	i &= splane_normal(minmax, maxmax, n[1]);
	i &= splane_normal(maxmax, maxmin, n[2]);
	i &= splane_normal(maxmin, minmin, n[3]);

	if (!i)
		return 0.0f;
	float ang = 0.0f;
	for (i = 0; i < 4; i++) {
		float a = dot(n[i], n[(i + 1) % 4]);
		ang += M_PIf - fabsf(acosf(clamp(a, -1.0f, 1.0f)));
	}
	ang = ang - 2.0f * M_PIf;
	if ((ang > (2.0f * M_PIf)) || ang < 0) {
		//fprintf(stderr, "Normal error in pict_get_sangle %f %d %d\n", ang, x, y);
		return 0.0f;
	}
	return ang;
}

/* From get_posindex in evalglare.c */
RT_METHOD float getPositionIndex(const float3 &dir)
{
	float3 up = normalize(V); // TODO Not necessarily
	float3 hv = cross(W, up);
	float phi = acosf(dot(cross(W, hv), dir)) - M_PI_2f;
	float teta = M_PI_2f - acosf(dot(hv, dir));
	float sigma = acosf(dot(W, dir));
	hv = normalize(normalize(dir) / cosf(sigma) - W);
	float tau = acosf(dot(up, hv));
	tau *= 180.0f / M_PIf;
	sigma *= 180.0f / M_PIf;

	if (phi < FTINY)
		phi = FTINY;
	if (sigma <= 0)
		sigma = -sigma;
	if (teta < FTINY)
		teta = FTINY;

	float posindex = expf((35.2f - 0.31889f * tau - 1.22f * expf(-2.0f * tau / 9.0f)) / 1000.0f * sigma + (21.0f + 0.26667f * tau - 0.002963f * tau * tau) / 100000.0f * sigma * sigma);

	/* below line of sight, using Iwata model */
	if (phi < 0.0f) {
		float fact = 0.8f;
		float d = 1.0f / tanf(phi);
		float s = tanf(teta) / tanf(phi);
		float r = sqrtf((s * s + 1.0f) / (d * d));
		if (r > 0.6f)
			fact = 1.2f;
		if (r > 3.0f)
			r = 3.0f;

		posindex = 1.0f + fact * r;
	}
	if (posindex > 16.0f)
		posindex = 16.0f;

	return posindex;
}

RT_PROGRAM void exception()
{
	const unsigned int code = rtGetExceptionCode();
	rtPrintf("Caught exception 0x%X at launch index (%d,%d)\n", code, launch_index.x, launch_index.y);
	color_buffer[launch_index] = 0xffffffff;
	Metrics metrics;
	metrics.omega = -1.0f;
	metrics.ev = code;
	metrics.avlum = 0.0f;
	metrics.dgp = 0.0f;
	metrics_buffer[launch_index] = metrics;
}
