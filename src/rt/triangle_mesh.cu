#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_aabb_namespace.h>
#include "optix_shader_common.h"

using namespace optix;

// This is to be plugged into an RTgeometry object to represent
// a triangle mesh with a vertex buffer of triangle soup (triangle list)
// with an interleaved position, normal, texturecoordinate layout.

/* Program variables */
rtDeclareVariable(unsigned int, backvis, , ); /* backface visibility (bv) */

/* Contex variables */
rtBuffer<float3> vertex_buffer;
rtBuffer<float3> normal_buffer;
rtBuffer<float2> texcoord_buffer;
rtBuffer<uint3>  vindex_buffer;    // position indices 
//rtBuffer<uint3>  nindex_buffer;    // normal indices
//rtBuffer<uint3>  tindex_buffer;    // texcoord indices
rtBuffer<int>    material_buffer; // per-face material index
rtBuffer<int2>   material_alt_buffer; // per-material alternate material indices
rtDeclareVariable(unsigned int, radiance_primary_ray_type, , );
rtDeclareVariable(unsigned int, diffuse_primary_ray_type, , ) = ~0;	/* Not always defined */
rtDeclareVariable(unsigned int, shadow_ray_type, , );

/* OptiX variables */
rtDeclareVariable(Ray, ray, rtCurrentRay, );

/* Attributes */
rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(int, surface_id, attribute surface_id, );

RT_PROGRAM void mesh_intersect(unsigned int primIdx)
{
	uint3 v_idx = vindex_buffer[primIdx];

	float3 p0 = vertex_buffer[ v_idx.x ];
	float3 p1 = vertex_buffer[ v_idx.y ];
	float3 p2 = vertex_buffer[ v_idx.z ];

	// Intersect ray with triangle
	float3 n;
	float  t, beta, gamma;
	if( intersect_triangle( ray, p0, p1, p2, n, t, beta, gamma ) && ( backvis || dot( n, ray.direction ) < 0) ) {

		int mat = material_buffer[primIdx];
		if ( mat < 0 || mat >= material_alt_buffer.size() ) /* Material void or missing */
			return;
		if (ray.ray_type == radiance_primary_ray_type || ray.ray_type == diffuse_primary_ray_type) /* Lambert material for irradiance calculations */
			mat = material_alt_buffer[mat].x;
		else if (ray.ray_type != shadow_ray_type || ray.tmax - t > ray.tmax * 0.0002f) /* For materials whose type depends on ray type (such as illum and mirror) */
			mat = material_alt_buffer[mat].y;
		if ( mat < 0 || mat >= material_alt_buffer.size() ) /* Material void or missing */
			return;

		if ( rtPotentialIntersection( t ) ) {

			//int3 n_idx = nindex_buffer[ primIdx ];
			geometric_normal = normalize( n );
			if ( normal_buffer.size() == 0 ) { //|| n_idx.x < 0 || n_idx.y < 0 || n_idx.z < 0 ) {
				shading_normal = geometric_normal;
			} else {
				float3 n0 = normal_buffer[ v_idx.x ];
				float3 n1 = normal_buffer[ v_idx.y ];
				float3 n2 = normal_buffer[ v_idx.z ];
				shading_normal = normalize( n1*beta + n2*gamma + n0*(1.0f-beta-gamma) );
				if ( !isfinite( shading_normal ) )
					shading_normal = geometric_normal;
			}

			//int3 t_idx = tindex_buffer[ primIdx ];
			if ( texcoord_buffer.size() == 0 ) { //|| t_idx.x < 0 || t_idx.y < 0 || t_idx.z < 0 ) {
				texcoord = make_float3( 0.0f, 0.0f, 0.0f );
			} else {
				float2 t0 = texcoord_buffer[ v_idx.x ];
				float2 t1 = texcoord_buffer[ v_idx.y ];
				float2 t2 = texcoord_buffer[ v_idx.z ];
				texcoord = make_float3( t1*beta + t2*gamma + t0*(1.0f-beta-gamma) );
			}

			surface_id = v_idx.x; // Not necessarily unique per triangle, but different for each surface

			rtReportIntersection( mat );
		}
	}
}

RT_PROGRAM void mesh_bounds(unsigned int primIdx, float result[6])
{  
	const uint3 v_idx = vindex_buffer[primIdx];

	const float3 v0   = vertex_buffer[ v_idx.x ];
	const float3 v1   = vertex_buffer[ v_idx.y ];
	const float3 v2   = vertex_buffer[ v_idx.z ];
	const float  area = length(cross(v1-v0, v2-v0));

	Aabb* aabb = (Aabb*)result;
  
	if(area > 0.0f && !isinf(area)) {
		aabb->m_min = fminf( fminf( v0, v1), v2 );
		aabb->m_max = fmaxf( fmaxf( v0, v1), v2 );
	} else {
		aabb->invalidate();
	}
}

