#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2015 Nathaniel Jones
 * Massachusetts Institute of Technology
 */

//#define RT_USE_TEMPLATED_RTCALLABLEPROGRAM
#include <optix_world.h>
#include "optix_shader_common.h"

using namespace optix;

/* Program variables */
#ifdef HIT_TYPE
rtDeclareVariable(unsigned int, type, , ); /* The material type representing "source" */
#endif

/* Context variables */
rtBuffer<DistantLight> lights;
rtBuffer<rtCallableProgramId<float(const float3)> > functions;
//rtDeclareVariable(rtCallableProgramId<float(float3)>, func, , );
//rtDeclareVariable(rtCallableProgramX<float(float3)>, func, , );
#ifdef DAYSIM
rtDeclareVariable(int, daysimSortMode, , ); /* how the daylight coefficients are sorted */
#endif

/* OptiX variables */
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow,   rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );


#ifdef DAYSIM
RT_METHOD int daysimComputePatch(const float3 dir);
#endif

RT_PROGRAM void miss()
{
	prd_radiance.result = make_float3( 0.0f );
	prd_radiance.distance = ray.tmax;
	if ( ray.tmax < RAY_END ) // ray length was truncated
		return;

	const float3 H = optix::normalize(ray.direction);

	// compute direct lighting
	unsigned int num_lights = lights.size();
	for (int i = 0; i < num_lights; ++i) {
		DistantLight light = lights[i];

		// get the angle bwetween the light direction and the view
		float3 L = optix::normalize(light.pos);
		float lDh = optix::dot( L, H );
		float solid_angle = 2.0f * M_PIf * (1.0f - lDh);

		if (solid_angle <= light.solid_angle) {
			float3 color = light.color;
			if (light.function > -1) {
				//rtPrintf( "Sending (%f, %f, %f)\n", H.x, H.y, H.z);
				color *= functions[light.function]( H );
			}
			if (light.function > -1 || prd_radiance.ambient_depth == 0) { //TODO need a better test, see badcomponent() in source.c
				// no contribution to ambient calculation
				prd_radiance.result += color;
#ifdef DAYSIM
				if (daylightCoefficients >= 2) {
					int patch = DAYSIM_MAX_COEFS; // Ignore by default
					if (daysimSortMode == 1)
						patch = i; // TODO This assumes that all sources are sun positions in numerical order
					else if (daysimSortMode == 2)
						patch = daysimComputePatch(ray.direction);
					daysimAddCoef(prd_radiance.dc, patch, color.x);
				}
#endif /* DAYSIM */
			}
		}
	}

#ifdef HIT_TYPE
	prd_radiance.hit_type = type;
#endif
}

RT_PROGRAM void miss_shadow()
{
	float3 result = make_float3( 0.0f );

	const float3 H = optix::normalize(ray.direction);

	// compute direct lighting
	if ( prd_shadow.target >= 0 && prd_shadow.target < lights.size() ) {
		DistantLight light = lights[prd_shadow.target];
		if (light.casts_shadow) {

			// get the angle bwetween the light direction and the view
			float3 L = optix::normalize(light.pos);
			float lDh = optix::dot( L, H );
			float solid_angle = 2.0f * M_PIf * (1.0f - lDh);

			if (solid_angle <= light.solid_angle) {
				float3 color = light.color;
				if (light.function > -1) {
					color *= functions[light.function]( H );
				}
				result += color;
#ifdef DAYSIM
				if (daylightCoefficients >= 2) {
					int patch = DAYSIM_MAX_COEFS; // Ignore by default
					if (daysimSortMode == 1)
						patch = prd_shadow.target; // TODO This assumes that all sources are sun positions in numerical order
					else if (daysimSortMode == 2)
						patch = daysimComputePatch(ray.direction);
					daysimAddCoef(prd_shadow.dc, patch, color.x);
				}
#endif /* DAYSIM */
			}
		}
	}
	prd_shadow.result = result;
}

#ifdef DAYSIM
/*
* Computes the sky/ground patch hit by a ray in direction (dx,dy,dz)
* according to the Tregenza sky division.
*/
RT_METHOD int daysimComputePatch(const float3 dir)
{
	int patch;

	if (dir.z > 0.0f) { // sky
		const int number[8] = { 0, 30, 60, 84, 108, 126, 138, 144 };
		const float ring_division[8] = { 30.0f, 30.0f, 24.0f, 24.0f, 18.0f, 12.0f, 6.0f, 0.0f };
		int ringnumber = (int)(asinf(dir.z) * 15.0f * M_1_PIf);
		// origin of the number "15":
		// according to Tregenza, the celestial hemisphere is divided into 7 bands and
		// the zenith patch. The bands range from:
		//												altitude center
		// Band 1		0 to 12 Deg			30 patches	6
		// Band 2		12 to 24 Deg		30 patches	18
		// Band 3		24 to 36 Deg		24 patches	30
		// Band 4		36 to 48 Deg		24 patches	42
		// Band 5		48 to 60 Deg		18 patches	54
		// Band 6		60 to 72 Deg		12 patches	66
		// Band 7		72 to 84 Deg		 6 patches	78
		// Band 8		84 to 90 Deg		 1 patch 	90
		// since the zenith patch is only takes 6Deg instead of 12, the arc length
		// between 0 and 90 Deg (equlas o and Pi/2) is divided into 7.5 units:
		// Therefore, 7.5 units = (int) asin(z=1)/(Pi/2)
		//				1 unit = asin(z)*(2*7.5)/Pi)
		//				1 unit = asin(z)*(15)/Pi)
		// Note that (int) always rounds to the next lowest integer
		patch = number[ringnumber] + ring_division[ringnumber] * (atan2f(dir.y, dir.x) * 0.5f * M_1_PIf + (dir.y >= 0.0f ? 0.0f : 1.0f));
	} else { // ground
		if (dir.z >= -0.17365f)
			patch = 145;
		else if (dir.z >= -0.5f)
			patch = 146;
		else
			patch = 147;
	}

	return patch;
}
#endif /* DAYSIM */
