#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2015 Nathaniel Jones
 * Massachusetts Institute of Technology
 */

//#define RT_USE_TEMPLATED_RTCALLABLEPROGRAM
#include <optix_world.h>
#include "optix_shader_common.h"

using namespace optix;

/* Program variables */
#ifdef HIT_TYPE
rtDeclareVariable(unsigned int, type, , ); /* The material type representing "source" */
#endif

/* Context variables */
rtBuffer<DistantLight> lights;
#ifdef CALLABLE
rtBuffer<rtCallableProgramId<float(const float3)> > functions;
//rtDeclareVariable(rtCallableProgramId<float(float3)>, func, , );
//rtDeclareVariable(rtCallableProgramX<float(float3)>, func, , );
#else
rtBuffer<SkyBright> sky_brights;
rtBuffer<PerezLum> perez_lums;
#endif

/* OptiX variables */
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow,   rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

#ifndef CALLABLE
// Calculate the sky brightness function for sunny and cloudy skies.
// This function replicates the algorithm in skybright.cal.
RT_METHOD float sky_bright( SkyBright cie )
{
	float cosgamma = optix::dot( ray.direction, cie.sun ); // cosgamma = Dx*A8 + Dy*A9 + Dz*A10;
	float gamma = acosf(cosgamma); // gamma = Acos(cosgamma);		{ angle from sun to this point in sky }
	float dz = ray.direction.z;
	float select = cie.zenith; // unifsky = A2; select(A1, sunnysky, cloudysky, unifsky, intersky)

	if (cie.type == 1) { // CIE clear
		select *= ( 0.91f + 10.0f * expf( -3.0f * gamma ) + 0.45f * cosgamma * cosgamma ) / cie.factor;
		if ( dz > 0.01f ) {
			select *= 1.0f - expf(-0.32f / dz );
		}
		//sunnysky = A2 * (.91 + 10*exp(-3*gamma) + .45*cosgamma*cosgamma)
	    //  *  if( Dz - .01, 1.0 - exp(-.32/Dz), 1.0) / A4;
	} else if (cie.type == 2) { // CIE overcast
		select *= ( 1.0f + 2.0f * dz ) / 3.0f; // cloudysky = A2 * (1 + 2*Dz)/3;
	} else if (cie.type == 4) { // CIE intermediate
		float zt = acosf(cie.sun.z); // zt = Acos(A10);			{ angle from zenith to sun }
		float eta = acosf(dz); // eta = Acos(Dz);			{ angle from zenith to this point in sky }
		select *= ( ( 1.35f * sinf( 5.631f - 3.59f * eta ) + 3.12f ) * sinf( 4.396f - 2.6f * zt) + 6.37f - eta ) / 2.326f *
			expf( gamma * -0.563f * ( ( 2.629f - eta ) * ( 1.562f - zt ) + 0.812f ) ) / cie.factor;
		//intersky = A2 * ( (1.35*sin(5.631-3.59*eta)+3.12)*sin(4.396-2.6*zt)
		//	+ 6.37 - eta ) / 2.326 *
		// exp(gamma*-.563*((2.629-eta)*(1.562-zt)+.812)) / A4;
	}

	float a = powf(ray.direction.z + 1.01f, 10.0f);
	float b = powf(ray.direction.z + 1.01f, -10.0f);

	float skybr = (a * select + b * cie.ground) / (a + b); // wmean(a, x, b, y) = (a*x+b*y)/(a+b);
	return skybr;
}

// Calculate the All-weather Angular Sky Luminance Distribution value for the current ray direction.
// This function replicates the algorithm in perezlum.cal.
RT_METHOD float perez_lum( PerezLum perez )
{
	float cosgamma = optix::dot( ray.direction, perez.sun ); // cosgamma = Dx*A8 + Dy*A9 + Dz*A10;
	float gamma = acosf(cosgamma); // gamma = Acos(cosgamma);		{ angle from sun to this point in sky }
	//float zt = acos(perez.sun.z); // zt = Acos(A10);			{ angle from zenith to sun }
	//float eta = acos(ray.direction.z); // eta = Acos(Dz);			{ angle from zenith to this point in sky }

	float dz = ray.direction.z;
	if (dz < 0.01f) {
		dz = 0.01f;
	}

	float intersky = perez.diffuse * (1.0f + perez.coef[0] * expf( perez.coef[1] / dz ) ) * ( 1.0f + perez.coef[2] * expf(perez.coef[3] * gamma) + perez.coef[4] * cosgamma * cosgamma );
	//intersky = if( (Dz-0.01),  
	//		A1 * (1 + A3*Exp(A4/Dz) ) * ( 1 + A5*Exp(A6*gamma) + A7*cos(gamma)*cos(gamma) ),
	//		A1 * (1 + A3*Exp(A4/0.01) ) * ( 1 + A5*Exp(A6*gamma) + A7*cos(gamma)*cos(gamma) ) );

	float a = powf(ray.direction.z + 1.01f, 10.0f);
	float b = powf(ray.direction.z + 1.01f, -10.0f);

	float skybright = (a * intersky + b * perez.ground) / (a + b); // wmean(a, x, b, y) = (a*x+b*y)/(a+b);
	return skybright;
}
#endif

RT_PROGRAM void miss()
{
	prd_radiance.result = make_float3( 0.0f );
	prd_radiance.distance = ray.tmax;
	if ( ray.tmax < RAY_END ) // ray length was truncated
		return;

	const float3 H = optix::normalize(ray.direction);

	// compute direct lighting
	unsigned int num_lights = lights.size();
	for (int i = 0; i < num_lights; ++i) {
		DistantLight light = lights[i];

		// get the angle bwetween the light direction and the view
		float3 L = optix::normalize(light.pos);
		float lDh = optix::dot( L, H );
		float solid_angle = 2.0f * M_PIf * (1.0f - lDh);

		if (solid_angle <= light.solid_angle) {
			float3 color = light.color;
			if (light.function > -1) {
#ifdef CALLABLE
				//rtPrintf( "Sending (%f, %f, %f)\n", H.x, H.y, H.z);
				color *= functions[light.function]( H );
#else
				if (light.type == SKY_CIE) {
					color *= sky_bright( sky_brights[light.function] );
				} else if (light.type == SKY_PEREZ) {
					color *= perez_lum( perez_lums[light.function] );
				}
#endif
			}
			if ( light.function > -1 || prd_radiance.ambient_depth == 0 ) //TODO need a better test, see badcomponent() in source.c
				// no contribution to ambient calculation
				prd_radiance.result += color;
		}
	}

#ifdef HIT_TYPE
	prd_radiance.hit_type = type;
#endif
}

RT_PROGRAM void miss_shadow()
{
	float3 result = make_float3( 0.0f );

	const float3 H = optix::normalize(ray.direction);

	// compute direct lighting
	if ( prd_shadow.target >= 0 && prd_shadow.target < lights.size() ) {
		DistantLight light = lights[prd_shadow.target];
		if (light.casts_shadow) {

			// get the angle bwetween the light direction and the view
			float3 L = optix::normalize(light.pos);
			float lDh = optix::dot( L, H );
			float solid_angle = 2.0f * M_PIf * (1.0f - lDh);

			if (solid_angle <= light.solid_angle) {
				float3 color = light.color;
				if (light.function > -1) {
#ifdef CALLABLE
					color *= functions[light.function]( H );
#else
					if (light.type == SKY_CIE) {
						color *= sky_bright( sky_brights[light.function] );
					} else if (light.type == SKY_PEREZ) {
						color *= perez_lum( perez_lums[light.function] );
					}
#endif
				}
				result += color;
			}
		}
	}
	prd_shadow.result = result;
}
