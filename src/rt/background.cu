#include "hip/hip_runtime.h"
/*
 *  background.cu - miss program for ray tracing on GPUs.
 */

#include "accelerad_copyright.h"

//#define RT_USE_TEMPLATED_RTCALLABLEPROGRAM
#include <optix_world.h>
#include "optix_shader_common.h"
#include "optix_shader_ray.h"
#ifdef CONTRIB_DOUBLE
#include "optix_double.h"
#endif

using namespace optix;

/* Program variables */
#ifdef HIT_TYPE
rtDeclareVariable(unsigned int, type, , ); /* The material type representing "source" */
#endif

/* Context variables */
rtBuffer<DistantLight> lights;
#ifdef CONTRIB
rtBuffer<contrib4, 3> contrib_buffer; /* accumulate contributions */
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(unsigned int, contrib, , ) = 0u;		/* Boolean switch for computing contributions (V) */
#endif
//rtBuffer<rtCallableProgramId<float(const float3)> > functions;
//rtDeclareVariable(rtCallableProgramId<float(float3)>, func, , );
//rtDeclareVariable(rtCallableProgramX<float(float3)>, func, , );
rtDeclareVariable(int, directvis, , );		/* Boolean switch for light source visibility (dv) */

/* OptiX variables */
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow,   rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );


#ifdef DAYSIM_COMPATIBLE
RT_METHOD unsigned int daysimComputePatch(const float3 dir);
#endif

RT_PROGRAM void miss()
{
	prd_radiance.result = make_float3( 0.0f );
	prd_radiance.distance = ray.tmax;
	if ( ray.tmax < RAY_END ) // ray length was truncated
		return;

	const float3 H = optix::normalize(ray.direction);

	// compute direct lighting
	unsigned int num_lights = lights.size();
	for (int i = 0; i < num_lights; ++i) {
		DistantLight light = lights[i];

		// no contribution to ambient calculation
		if (prd_radiance.ambient_depth && light.casts_shadow) // badcomponent() in source.c
			continue; // TODO also no contribution from specular

		// get the angle bwetween the light direction and the view
		float3 L = optix::normalize(light.pos);
		float lDh = optix::dot( L, H );
		float solid_angle = 2.0f * M_PIf * (1.0f - lDh);

		if (solid_angle <= light.solid_angle) {
			if (!directvis && light.casts_shadow) { // srcignore() in source.c
				prd_radiance.result = make_float3(0.0f);
				break;
			}
			float3 color = light.color;
			if (light.function != RT_PROGRAM_ID_NULL)
				color *= ((rtCallableProgramId<float3(const float3, const float3)>)light.function)(H, -H);
			prd_radiance.result += color;
#ifdef DAYSIM_COMPATIBLE
			if (daylightCoefficients >= 2) {
				daysimAddCoef(prd_radiance.dc, daysimComputePatch(ray.direction), color.x);
			}
#endif /* DAYSIM_COMPATIBLE */
#ifdef CONTRIB
			if (light.contrib_index >= 0) {
				contrib3 contr = prd_radiance.rcoef;
				if (contrib)
					contr *= color;
				int contr_index = light.contrib_index;
				if (light.contrib_function != RT_PROGRAM_ID_NULL)
					contr_index += ((rtCallableProgramId<int(const float3)>)light.contrib_function)(H);
				if (contr_index >= light.contrib_index)
					contrib_buffer[make_uint3(contr_index, launch_index.x, launch_index.y)] += make_contrib4(contr);
			}
#endif /* CONTRIB */
		}
	}

#ifdef HIT_TYPE
	prd_radiance.hit_type = type;
#endif
}

RT_PROGRAM void miss_shadow()
{
	prd_shadow.result = make_float3(0.0f);
	if (ray.tmax < RAY_END) // ray length was truncated
		return;

	const float3 H = optix::normalize(ray.direction);

	// compute direct lighting
	if ( prd_shadow.target >= 0 && prd_shadow.target < lights.size() ) {
		DistantLight light = lights[prd_shadow.target];
		if (light.casts_shadow) {

			// get the angle bwetween the light direction and the view
			float3 L = optix::normalize(light.pos);
			float lDh = optix::dot( L, H );
			float solid_angle = 2.0f * M_PIf * (1.0f - lDh);

			if (solid_angle <= light.solid_angle) {
				float3 color = light.color;
				if (light.function != RT_PROGRAM_ID_NULL)
					color *= ((rtCallableProgramId<float3(const float3, const float3)>)light.function)(H, -H);
				prd_shadow.result = color;
#ifdef DAYSIM_COMPATIBLE
				if (daylightCoefficients >= 2) {
					// TODO This assumes that all sources are sun positions in numerical order
					// TODO If files are merged, add 148 to prd_shadow.target
					daysimAddCoef(prd_shadow.dc, prd_shadow.target, color.x);
				}
#endif /* DAYSIM_COMPATIBLE */
#ifdef CONTRIB
				if (light.contrib_index >= 0) {
					contrib3 contr = prd_shadow.rcoef;
					if (contrib)
						contr *= color;
					int contr_index = light.contrib_index;
					if (light.contrib_function != RT_PROGRAM_ID_NULL)
						contr_index += ((rtCallableProgramId<int(const float3)>)light.contrib_function)(H);
					if (contr_index >= light.contrib_index)
						contrib_buffer[make_uint3(contr_index, launch_index.x, launch_index.y)] += make_contrib4(contr);
				}
#endif /* CONTRIB */
			}
		}
	}
}

#ifdef DAYSIM_COMPATIBLE
/*
* Computes the sky/ground patch hit by a ray in direction (dx,dy,dz)
* according to the Tregenza sky division.
*/
RT_METHOD unsigned int daysimComputePatch(const float3 dir)
{
	if (dir.z > 0.0f) { // sky
		const unsigned int number[8] = { 0, 30, 60, 84, 108, 126, 138, 144 };
		const float ring_division[8] = { 30.0f, 30.0f, 24.0f, 24.0f, 18.0f, 12.0f, 6.0f, 0.0f };
		int ringnumber = (int)(asinf(dir.z) * 15.0f * M_1_PIf);
		// origin of the number "15":
		// according to Tregenza, the celestial hemisphere is divided into 7 bands and
		// the zenith patch. The bands range from:
		//												altitude center
		// Band 1		0 to 12 Deg			30 patches	6
		// Band 2		12 to 24 Deg		30 patches	18
		// Band 3		24 to 36 Deg		24 patches	30
		// Band 4		36 to 48 Deg		24 patches	42
		// Band 5		48 to 60 Deg		18 patches	54
		// Band 6		60 to 72 Deg		12 patches	66
		// Band 7		72 to 84 Deg		 6 patches	78
		// Band 8		84 to 90 Deg		 1 patch 	90
		// since the zenith patch is only takes 6Deg instead of 12, the arc length
		// between 0 and 90 Deg (equlas o and Pi/2) is divided into 7.5 units:
		// Therefore, 7.5 units = (int) asin(z=1)/(Pi/2)
		//				1 unit = asin(z)*(2*7.5)/Pi)
		//				1 unit = asin(z)*(15)/Pi)
		// Note that (int) always rounds to the next lowest integer
		return number[ringnumber] + ring_division[ringnumber] * (atan2f(dir.y, dir.x) * 0.5f * M_1_PIf + (dir.y >= 0.0f ? 0.0f : 1.0f));
	}
	// ground
	if (dir.z >= -0.17365f)
		return 145;
	if (dir.z >= -0.5f)
		return 146;
	return 147;
}
#endif /* DAYSIM_COMPATIBLE */
