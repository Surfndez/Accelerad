#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2015 Nathaniel Jones
 * Massachusetts Institute of Technology
 */

//#define RT_USE_TEMPLATED_RTCALLABLEPROGRAM
#include <optix_world.h>
#include "optix_shader_common.h"

using namespace optix;

/* Program variables */
#ifdef HIT_TYPE
rtDeclareVariable(unsigned int, type, , ); /* The material type representing "source" */
#endif

/* Context variables */
rtBuffer<DistantLight> lights;
rtBuffer<rtCallableProgramId<float(const float3)> > functions;
//rtDeclareVariable(rtCallableProgramId<float(float3)>, func, , );
//rtDeclareVariable(rtCallableProgramX<float(float3)>, func, , );

/* OptiX variables */
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow,   rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );


RT_PROGRAM void miss()
{
	prd_radiance.result = make_float3( 0.0f );
	prd_radiance.distance = ray.tmax;
	if ( ray.tmax < RAY_END ) // ray length was truncated
		return;

	const float3 H = optix::normalize(ray.direction);

	// compute direct lighting
	unsigned int num_lights = lights.size();
	for (int i = 0; i < num_lights; ++i) {
		DistantLight light = lights[i];

		// get the angle bwetween the light direction and the view
		float3 L = optix::normalize(light.pos);
		float lDh = optix::dot( L, H );
		float solid_angle = 2.0f * M_PIf * (1.0f - lDh);

		if (solid_angle <= light.solid_angle) {
			float3 color = light.color;
			if (light.function > -1) {
				//rtPrintf( "Sending (%f, %f, %f)\n", H.x, H.y, H.z);
				color *= functions[light.function]( H );
			}
			if ( light.function > -1 || prd_radiance.ambient_depth == 0 ) //TODO need a better test, see badcomponent() in source.c
				// no contribution to ambient calculation
				prd_radiance.result += color;
		}
	}

#ifdef HIT_TYPE
	prd_radiance.hit_type = type;
#endif
}

RT_PROGRAM void miss_shadow()
{
	float3 result = make_float3( 0.0f );

	const float3 H = optix::normalize(ray.direction);

	// compute direct lighting
	if ( prd_shadow.target >= 0 && prd_shadow.target < lights.size() ) {
		DistantLight light = lights[prd_shadow.target];
		if (light.casts_shadow) {

			// get the angle bwetween the light direction and the view
			float3 L = optix::normalize(light.pos);
			float lDh = optix::dot( L, H );
			float solid_angle = 2.0f * M_PIf * (1.0f - lDh);

			if (solid_angle <= light.solid_angle) {
				float3 color = light.color;
				if (light.function > -1) {
					color *= functions[light.function]( H );
				}
				result += color;
			}
		}
	}
	prd_shadow.result = result;
}
