#include "hip/hip_runtime.h"
/*
 *  sensor.cu - entry point for individual ray tracing on GPUs.
 */

#include "accelerad_copyright.h"

#include <optix_world.h>
#include "optix_shader_common.h"
#include "optiX_shader_ray.h"
#ifdef CONTRIB_DOUBLE
#include "optix_double.h"
#endif

using namespace optix;

/* Program variables */
rtDeclareVariable(unsigned int,  do_irrad, , ) = 0u; /* Calculate irradiance (-i) */

/* Contex variables */
rtBuffer<RayData, 2>             ray_buffer;
#ifdef DAYSIM_COMPATIBLE
rtBuffer<DC, 3>                  dc_buffer;
#endif
//rtBuffer<unsigned int, 2>        rnd_seeds;
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(rtObject,      top_irrad, , );
rtDeclareVariable(unsigned int,  radiance_ray_type, , );
rtDeclareVariable(unsigned int,  radiance_primary_ray_type, , );
rtDeclareVariable(unsigned int,  imm_irrad, , ) = 0u; /* Immediate irradiance (-I) */

/* OptiX variables */
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );
rtDeclareVariable(float, time_view_scale, , ) = 1e-6f;

//#define TIME_VIEW


RT_PROGRAM void ray_generator()
{
#ifdef TIME_VIEW
	clock_t t0 = clock();
	ray_buffer[launch_index].val = make_float3( t0 );
#endif
	PerRayData_radiance prd;
	init_rand(&prd.state, launch_index.x + launch_dim.x * launch_index.y);
	prd.weight = 1.0f;
	prd.depth = 0;
	prd.ambient_depth = 0;
	//prd.seed = rnd_seeds[launch_index];
#ifdef CONTRIB
	prd.rcoef = make_contrib3(1.0f); //Probably not necessary
#endif
#ifdef ANTIMATTER
	prd.mask = 0u;
	prd.inside = 0;
#endif
#ifdef DAYSIM_COMPATIBLE
	prd.dc = make_uint3(0, launch_index.x, launch_index.y);
#endif
	setupPayload(prd);

	const float tmin = ray_start( ray_buffer[launch_index].origin, RAY_START );
	if ( imm_irrad ) {
		Ray ray = make_Ray(ray_buffer[launch_index].origin, ray_buffer[launch_index].dir, radiance_ray_type, -tmin, tmin);
		rtTrace(top_irrad, ray, prd);
	} else {
		// Zero or negative aft clipping distance indicates infinity
		float aft = ray_buffer[launch_index].max;
		if (aft <= FTINY) {
			aft = RAY_END;
		}

		Ray ray = make_Ray(ray_buffer[launch_index].origin, ray_buffer[launch_index].dir, do_irrad ? radiance_primary_ray_type : radiance_ray_type, tmin, aft);
		rtTrace(top_object, ray, prd);
	}

	checkFinite(prd.result);

#ifdef TIME_VIEW
	clock_t t1 = clock();
 
	float expected_fps   = 1.0f;
	float pixel_time     = ( t1 - t0 ) * time_view_scale * expected_fps;
	ray_buffer[launch_index].val = make_float3( pixel_time );
#else
	ray_buffer[launch_index].val = prd.result;
#endif
	//ray_buffer[launch_index].hit = ray_buffer[launch_index].origin + prd.distance * ray_buffer[launch_index].dir;
	ray_buffer[launch_index].weight = prd.weight;
	ray_buffer[launch_index].length = prd.distance;
	//ray_buffer[launch_index].t = prd.distance;
#ifdef RAY_COUNT
	ray_buffer[launch_index].ray_count = prd.ray_count;
#endif
#ifdef DAYSIM_COMPATIBLE
	if (dc_buffer.size().x)
		daysimCopy(&dc_buffer[prd.dc], prd.dc);
#endif
}

RT_PROGRAM void exception()
{
#ifdef PRINT_OPTIX
	rtPrintExceptionDetails();
#endif
#ifdef TIME_VIEW
	clock_t t1 = clock();
 
	float expected_fps   = 1.0f;
	float ray_time       = ( t1 - ray_buffer[launch_index].val.x ) * time_view_scale * expected_fps;
	ray_buffer[index].val = make_float3( ray_time );
#else
	ray_buffer[launch_index].val = exceptionToFloat3(rtGetExceptionCode());
	ray_buffer[launch_index].weight = -1.0f;
#endif
}
