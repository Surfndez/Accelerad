#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2014 Nathaniel Jones
 * Massachusetts Institute of Technology
 */

#include <optix_world.h>
#include "optix_shader_common.h"

using namespace optix;

/* Program variables */
rtDeclareVariable(unsigned int,  do_irrad, , ); /* Calculate irradiance (-i) */

/* Contex variables */
rtBuffer<RayData, 2>             ray_buffer;
//rtBuffer<unsigned int, 2>        rnd_seeds;
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(unsigned int,  radiance_ray_type, , );
rtDeclareVariable(unsigned int,  radiance_primary_ray_type, , );

/* OptiX variables */
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );
rtDeclareVariable(float, time_view_scale, , ) = 1e-6f;

//#define TIME_VIEW

// Initialize the random state
RT_METHOD void init_state( PerRayData_radiance* prd )
{
	rand_state state;
	prd->state = &state;
	hiprand_init( launch_index.x + launch_dim.x * launch_index.y, 0, 0, prd->state );
}

RT_PROGRAM void ray_generator()
{
#ifdef TIME_VIEW
	clock_t t0 = clock();
	ray_buffer[launch_index].val = make_float3( t0 );
#endif
	PerRayData_radiance prd;
	init_state( &prd );

	// Zero or negative aft clipping distance indicates infinity
	float aft = ray_buffer[launch_index].max;
	if (aft <= FTINY) {
		aft = RAY_END;
	}

	Ray ray = make_Ray(ray_buffer[launch_index].origin, ray_buffer[launch_index].dir, do_irrad ? radiance_primary_ray_type : radiance_ray_type, FTINY, aft);

	prd.weight = 1.0f;
	prd.depth = 0;
	prd.ambient_depth = 0;
	//prd.seed = rnd_seeds[launch_index];
#ifdef FILL_GAPS
	prd.primary = 1;
#endif
#ifdef RAY_COUNT
	prd.ray_count = 1;
#endif

	rtTrace(top_object, ray, prd);

#ifdef TIME_VIEW
	clock_t t1 = clock();
 
	float expected_fps   = 1.0f;
	float pixel_time     = ( t1 - t0 ) * time_view_scale * expected_fps;
	ray_buffer[launch_index].val = make_float3( pixel_time );
#elif defined RAY_COUNT
	ray_buffer[launch_index].val = make_float3( prd.ray_count );
#else
	ray_buffer[launch_index].val = prd.result;
	ray_buffer[launch_index].length = prd.distance;
	ray_buffer[launch_index].weight = prd.weight;
#endif
}

RT_PROGRAM void exception()
{
	const unsigned int code = rtGetExceptionCode();
	rtPrintf( "Caught exception 0x%X at launch index (%d,%d)\n", code, launch_index.x, launch_index.y );
#ifdef TIME_VIEW
	clock_t t1 = clock();
 
	float expected_fps   = 1.0f;
	float ray_time       = ( t1 - ray_buffer[launch_index].val.x ) * time_view_scale * expected_fps;
	ray_buffer[index].val = make_float3( ray_time );
#else
	ray_buffer[launch_index].val = exceptionToFloat3( code );
	ray_buffer[launch_index].weight = -1.0f;
#endif
}
