#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2015 Nathaniel Jones
 * Massachusetts Institute of Technology
 */

#include <optix_world.h>
#include "optix_shader_common.h"

using namespace optix;

/* Program variables */
rtDeclareVariable(unsigned int,  do_irrad, , ); /* Calculate irradiance (-i) */

/* Contex variables */
rtBuffer<RayData, 2>             ray_buffer;
#ifdef DAYSIM
rtBuffer<DC, 3>                  dc_buffer;
#endif
//rtBuffer<unsigned int, 2>        rnd_seeds;
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(rtObject,      top_irrad, , );
rtDeclareVariable(unsigned int,  radiance_ray_type, , );
rtDeclareVariable(unsigned int,  radiance_primary_ray_type, , );
rtDeclareVariable(unsigned int,  imm_irrad, , ); /* Immediate irradiance (-I) */

/* OptiX variables */
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );
rtDeclareVariable(float, time_view_scale, , ) = 1e-6f;

//#define TIME_VIEW

// Initialize the random state
RT_METHOD void init_state( PerRayData_radiance* prd )
{
	rand_state state;
	prd->state = &state;
	hiprand_init( launch_index.x + launch_dim.x * launch_index.y, 0, 0, prd->state );
}

RT_PROGRAM void ray_generator()
{
#ifdef TIME_VIEW
	clock_t t0 = clock();
	ray_buffer[launch_index].val = make_float3( t0 );
#endif
	PerRayData_radiance prd;
	init_state( &prd );
	prd.weight = 1.0f;
	prd.depth = 0;
	prd.ambient_depth = 0;
	//prd.seed = rnd_seeds[launch_index];
#ifdef DAYSIM
	prd.dc = make_uint3(0, launch_index.x, launch_index.y);
#endif
	setupPayload(prd, 1);

	const float tmin = ray_start( ray_buffer[launch_index].origin, RAY_START );
	if ( imm_irrad ) {
		Ray ray = make_Ray(ray_buffer[launch_index].origin, ray_buffer[launch_index].dir, radiance_ray_type, -tmin, tmin);
		rtTrace(top_irrad, ray, prd);
	} else {
		// Zero or negative aft clipping distance indicates infinity
		float aft = ray_buffer[launch_index].max;
		if (aft <= FTINY) {
			aft = RAY_END;
		}

		Ray ray = make_Ray(ray_buffer[launch_index].origin, ray_buffer[launch_index].dir, do_irrad ? radiance_primary_ray_type : radiance_ray_type, tmin, aft);
		rtTrace(top_object, ray, prd);
	}

#ifdef TIME_VIEW
	clock_t t1 = clock();
 
	float expected_fps   = 1.0f;
	float pixel_time     = ( t1 - t0 ) * time_view_scale * expected_fps;
	ray_buffer[launch_index].val = make_float3( pixel_time );
#else
	ray_buffer[launch_index].val = prd.result;
#endif
	//ray_buffer[launch_index].hit = ray_buffer[launch_index].origin + prd.distance * ray_buffer[launch_index].dir;
	ray_buffer[launch_index].weight = prd.weight;
	ray_buffer[launch_index].length = prd.distance;
	//ray_buffer[launch_index].t = prd.distance;
#ifdef RAY_COUNT
	ray_buffer[launch_index].ray_count = prd.ray_count;
#endif
#ifdef DAYSIM
	daysimCopy(&dc_buffer[prd.dc], prd.dc);
#endif
}

RT_PROGRAM void exception()
{
	const unsigned int code = rtGetExceptionCode();
	rtPrintf( "Caught exception 0x%X at launch index (%d,%d)\n", code, launch_index.x, launch_index.y );
#ifdef TIME_VIEW
	clock_t t1 = clock();
 
	float expected_fps   = 1.0f;
	float ray_time       = ( t1 - ray_buffer[launch_index].val.x ) * time_view_scale * expected_fps;
	ray_buffer[index].val = make_float3( ray_time );
#else
	ray_buffer[launch_index].val = exceptionToFloat3( code );
	ray_buffer[launch_index].weight = -1.0f;
#endif
}
