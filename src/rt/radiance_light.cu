#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2014 Nathaniel Jones
 * Massachusetts Institute of Technology
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "optix_shader_common.h"

using namespace optix;

/* Material variables */
rtDeclareVariable(float3,       color, , );
rtDeclareVariable(float,        maxrad, , ) = RAY_END;
rtDeclareVariable(float,        siz, , ) = -1.0f;		/* output solid angle or area */
rtDeclareVariable(float,        flen, , );				/* focal length (negative if distant source) */
rtDeclareVariable(float3,       aim, , );				/* aim direction or center */

/* OptiX variables */
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(PerRayData_radiance, prd, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );

/* Attributes */
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 


static __device__ __inline__ int spotout();


RT_PROGRAM void closest_hit_shadow()
{
	if ( t_hit > maxrad || spotout() )
		prd_shadow.result = make_float3( 0.0f );
	else
		prd_shadow.result = color;
}

RT_PROGRAM void closest_hit_radiance()
{
	// no contribution to ambient calculation
	if ( t_hit > maxrad || prd.ambient_depth > 0 || spotout() ) //TODO need a better ambient test and handle maxrad < 0
		prd.result = make_float3( 0.0f );
	else
		prd.result = color;
	prd.distance = t_hit;

#ifdef HIT_TYPE
	prd.hit_type = MAT_LIGHT;
#endif
}

static __device__ __inline__ int spotout()
{
	if ( siz < -FTINY )
		return(0); /* Not a spotlight */
	if ( flen < -FTINY ) {		/* distant source */
		const float3 vd = aim - ray.origin;
		float d = dot( ray.direction, vd );
		/*			wrong side?
		if (d <= FTINY)
			return(1);	*/
		d = dot( vd, vd ) - d * d;
		if ( M_PIf * d > siz )
			return(1);	/* out */
		return(0);	/* OK */
	}
					/* local source */
	if ( siz < 2.0f * M_PIf * ( 1.0f + dot( aim, ray.direction ) ) )
		return(1);	/* out */
	return(0);	/* OK */
}
