#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2014 Nathaniel Jones
 * Massachusetts Institute of Technology
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "optix_shader_common.h"

using namespace optix;

/* Material variables */
rtDeclareVariable(float3,       color, , );

/* OptiX variables */
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(PerRayData_radiance, prd, rtPayload, );

/* Attributes */
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 

RT_PROGRAM void closest_hit_radiance()
{
	// no contribution to ambient calculation
	if ( prd.ambient_depth > 0 ) //TODO need a better test
		prd.result = make_float3( 0.0f );
	else
		prd.result = color;
	prd.distance = t_hit;

#ifdef HIT_TYPE
	prd.hit_type = MAT_LIGHT;
#endif
}
