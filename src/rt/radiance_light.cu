#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2014 Nathaniel Jones
 * Massachusetts Institute of Technology
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "optix_shader_common.h"

using namespace optix;

/* Material variables */
#ifdef HIT_TYPE
rtDeclareVariable(unsigned int, type, , ); /* The material type representing "light", "illum", "glow", or "spot" */
#endif
rtDeclareVariable(float3,       color, , );
rtDeclareVariable(float,        maxrad, , ) = RAY_END;
rtDeclareVariable(float,        siz, , ) = -1.0f;		/* output solid angle or area */
rtDeclareVariable(float,        flen, , );				/* focal length (negative if distant source) */
rtDeclareVariable(float3,       aim, , );				/* aim direction or center */
#ifdef CALLABLE
rtDeclareVariable(rtCallableProgramId<float(float3)>, function, , );		/* function or texture modifier */
#else
rtDeclareVariable(int,          lindex, , ) = -1;		/* function or texture modifier */

/* Geometry instance variables */
rtBuffer<Light> light_sources;
#endif

/* Context variables */
rtDeclareVariable(int,          directvis, , );		/* Boolean switch for light source visibility (dv) */

/* OptiX variables */
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(PerRayData_radiance, prd, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );

/* Attributes */
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 


RT_METHOD int spotout();
#ifndef CALLABLE
RT_METHOD float texture_function( const float3& normal );
#endif


RT_PROGRAM void closest_hit_shadow()
{
	float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
	//float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );

	//float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

	if ( t_hit > maxrad || spotout() || dot( world_shading_normal, ray.direction ) > 0.0f )
		prd_shadow.result = make_float3( 0.0f );
#ifdef CALLABLE
	else if ( function > RT_PROGRAM_ID_NULL )
		prd_shadow.result = color * function( ray.direction );
#else
	else if ( lindex > -1 )
		prd_shadow.result = color * texture_function( world_shading_normal );
#endif
	else
		prd_shadow.result = color;
}

RT_PROGRAM void closest_hit_radiance()
{
	float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
	//float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );

	//float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

	// no contribution to ambient calculation
	if ( !directvis || t_hit > maxrad || prd.ambient_depth > 0 || spotout() || dot( world_shading_normal, ray.direction ) > 0.0f ) //TODO need a better ambient test and handle maxrad < 0
		prd.result = make_float3( 0.0f );
#ifdef CALLABLE
	else if ( function > RT_PROGRAM_ID_NULL )
		prd.result = color * function( ray.direction );
#else
	else if ( lindex > -1 )
		prd.result = color * texture_function( world_shading_normal );
#endif
	else
		prd.result = color;
	prd.distance = t_hit;

#ifdef HIT_TYPE
	prd.hit_type = type;
#endif
}

RT_METHOD int spotout()
{
	if ( siz < -FTINY )
		return(0); /* Not a spotlight */
	if ( flen < -FTINY ) {		/* distant source */
		const float3 vd = aim - ray.origin;
		float d = dot( ray.direction, vd );
		/*			wrong side?
		if (d <= FTINY)
			return(1);	*/
		d = dot( vd, vd ) - d * d;
		if ( M_PIf * d > siz )
			return(1);	/* out */
		return(0);	/* OK */
	}
					/* local source */
	if ( siz < 2.0f * M_PIf * ( 1.0f + dot( aim, ray.direction ) ) )
		return(1);	/* out */
	return(0);	/* OK */
}

#ifndef CALLABLE
RT_METHOD float texture_function( const float3& normal )
{
	const Light light_source = light_sources[lindex];

	float phi = acosf( dot( ray.direction, normalize( light_source.w ) ) );
	float theta = atan2f( -dot( ray.direction, normalize( light_source.v ) ), -dot( ray.direction, normalize( light_source.u ) ) );
	theta += 2.0f * M_PIf * ( theta < 0.0f );

	/* Normalize to [0, 1] within range */
	phi = ( 180.0f * M_1_PIf * phi - light_source.min.x ) / ( light_source.max.x - light_source.min.x );
	theta = ( 180.0f * M_1_PIf * theta - light_source.min.y ) / ( light_source.max.y - light_source.min.y );

	float rdot = dot( ray.direction, normal );
	return light_source.multiplier * rtTex2D<float>( light_source.texture, phi, theta ) / fabsf( rdot ); // this is flatcorr from source.cal
}
#endif /* CALLLABLE */
