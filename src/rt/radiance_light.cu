#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2014 Nathaniel Jones
 * Massachusetts Institute of Technology
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "optix_shader_common.h"

using namespace optix;

/* Material variables */
#ifdef HIT_TYPE
rtDeclareVariable(unsigned int, type, , ); /* The material type representing "light", "glow", or "spot" */
#endif
rtDeclareVariable(float3,       color, , );
rtDeclareVariable(float,        maxrad, , ) = RAY_END;
rtDeclareVariable(float,        siz, , ) = -1.0f;		/* output solid angle or area */
rtDeclareVariable(float,        flen, , );				/* focal length (negative if distant source) */
rtDeclareVariable(float3,       aim, , );				/* aim direction or center */
rtDeclareVariable(unsigned int, function, , ) = RT_PROGRAM_ID_NULL;		/* function or texture modifier */

/* OptiX variables */
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(PerRayData_radiance, prd, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );

/* Attributes */
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 


static __device__ __inline__ int spotout();
static __device__ __inline__ float texture_function( const float3& normal );


RT_PROGRAM void closest_hit_shadow()
{
	float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
	//float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );

	//float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

	if ( t_hit > maxrad || spotout() )
		prd_shadow.result = make_float3( 0.0f );
	else if ( function > RT_PROGRAM_ID_NULL )
		prd.result = color * texture_function( world_shading_normal );
	else
		prd_shadow.result = color;
}

RT_PROGRAM void closest_hit_radiance()
{
	float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
	//float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );

	//float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

	// no contribution to ambient calculation
	if ( t_hit > maxrad || prd.ambient_depth > 0 || spotout() ) //TODO need a better ambient test and handle maxrad < 0
		prd.result = make_float3( 0.0f );
	else if ( function > RT_PROGRAM_ID_NULL )
		prd.result = color * texture_function( world_shading_normal );
	else
		prd.result = color;
	prd.distance = t_hit;

#ifdef HIT_TYPE
	prd.hit_type = type;
#endif
}

static __device__ __inline__ int spotout()
{
	if ( siz < -FTINY )
		return(0); /* Not a spotlight */
	if ( flen < -FTINY ) {		/* distant source */
		const float3 vd = aim - ray.origin;
		float d = dot( ray.direction, vd );
		/*			wrong side?
		if (d <= FTINY)
			return(1);	*/
		d = dot( vd, vd ) - d * d;
		if ( M_PIf * d > siz )
			return(1);	/* out */
		return(0);	/* OK */
	}
					/* local source */
	if ( siz < 2.0f * M_PIf * ( 1.0f + dot( aim, ray.direction ) ) )
		return(1);	/* out */
	return(0);	/* OK */
}

static __device__ __inline__ float texture_function( const float3& normal )
{
#ifdef CALLABLE
	return function( ray.direction );
#else /* CALLLABLE */
	const float3 u = make_float3( 1.0f, 0.0f, 0.0f ); //TODO use orientation of fixture
	const float3 v = make_float3( 0.0f, 1.0f, 0.0f );
	const float3 w = make_float3( 0.0f, 0.0f, 1.0f );

	float phi = acosf( dot( ray.direction, w ) );
	float theta = atan2f( dot( ray.direction, v ), dot( ray.direction, u ) );
	theta += 2.0f * M_PIf * ( theta < 0.0f );

	/* Normalize to [0, 1] */
	phi *= M_1_PIf * 2.0f; //TODO some IES files are 0-180 degrees rather than 0-90 degrees
	theta *= M_1_PIf * 0.5f;

	float rdot = dot( ray.direction, normal );
	return rtTex2D<float>( function, phi, theta ) / fabsf( rdot ); // this is flatcorr from source.cal
#endif /* CALLLABLE */
}
