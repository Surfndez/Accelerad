#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2014 Nathaniel Jones
 * Massachusetts Institute of Technology
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "optix_shader_common.h"

using namespace optix;

/* Material variables */
#ifdef HIT_TYPE
rtDeclareVariable(unsigned int, type, , ); /* The material type representing "light", "glow", or "spot" */
#endif
rtDeclareVariable(float3,       color, , );
rtDeclareVariable(float,        maxrad, , ) = RAY_END;
rtDeclareVariable(float,        siz, , ) = -1.0f;		/* output solid angle or area */
rtDeclareVariable(float,        flen, , );				/* focal length (negative if distant source) */
rtDeclareVariable(float3,       aim, , );				/* aim direction or center */
#ifdef CALLABLE
rtDeclareVariable(unsigned int, function, , ) = RT_PROGRAM_ID_NULL;		/* function or texture modifier */
#else
rtDeclareVariable(int,          lindex, , ) = -1;		/* function or texture modifier */

/* Geometry instance variables */
rtBuffer<Light> light_sources;
#endif

/* Context variables */
rtDeclareVariable(int,          directvis, , );		/* Boolean switch for light source visibility (dv) */

/* OptiX variables */
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(PerRayData_radiance, prd, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );

/* Attributes */
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 


static __device__ __inline__ int spotout();
#ifndef CALLABLE
static __device__ __inline__ float texture_function( const float3& normal );
#endif


RT_PROGRAM void closest_hit_shadow()
{
	float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
	//float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );

	//float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

	if ( t_hit > maxrad || spotout() )
		prd_shadow.result = make_float3( 0.0f );
#ifdef CALLABLE
	else if ( function > RT_PROGRAM_ID_NULL )
		prd_shadow.result = color * function( ray.direction );
#else
	else if ( lindex > -1 )
		prd_shadow.result = color * texture_function( world_shading_normal );
#endif
	else
		prd_shadow.result = color;
}

RT_PROGRAM void closest_hit_radiance()
{
	float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
	//float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );

	//float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

	// no contribution to ambient calculation
	if ( !directvis || t_hit > maxrad || prd.ambient_depth > 0 || spotout() ) //TODO need a better ambient test and handle maxrad < 0
		prd.result = make_float3( 0.0f );
#ifdef CALLABLE
	else if ( function > RT_PROGRAM_ID_NULL )
		prd.result = color * function( ray.direction );
#else
	else if ( lindex > -1 )
		prd.result = color * texture_function( world_shading_normal );
#endif
	else
		prd.result = color;
	prd.distance = t_hit;

#ifdef HIT_TYPE
	prd.hit_type = type;
#endif
}

static __device__ __inline__ int spotout()
{
	if ( siz < -FTINY )
		return(0); /* Not a spotlight */
	if ( flen < -FTINY ) {		/* distant source */
		const float3 vd = aim - ray.origin;
		float d = dot( ray.direction, vd );
		/*			wrong side?
		if (d <= FTINY)
			return(1);	*/
		d = dot( vd, vd ) - d * d;
		if ( M_PIf * d > siz )
			return(1);	/* out */
		return(0);	/* OK */
	}
					/* local source */
	if ( siz < 2.0f * M_PIf * ( 1.0f + dot( aim, ray.direction ) ) )
		return(1);	/* out */
	return(0);	/* OK */
}

#ifndef CALLABLE
static __device__ __inline__ float texture_function( const float3& normal )
{
	const Light light_source = light_sources[lindex];

	const float3 u = make_float3( 1.0f, 0.0f, 0.0f ); //TODO use orientation of fixture
	const float3 v = make_float3( 0.0f, 1.0f, 0.0f );
	const float3 w = make_float3( 0.0f, 0.0f, 1.0f );

	float phi = acosf( dot( ray.direction, w ) );
	float theta = atan2f( -dot( ray.direction, v ), -dot( ray.direction, u ) );
	theta += 2.0f * M_PIf * ( theta < 0.0f );

	/* Normalize to [0, 1] within range */
	phi = ( 180.0f * M_1_PIf * phi - light_source.min.x ) / ( light_source.max.x - light_source.min.x );
	theta = ( 180.0f * M_1_PIf * theta - light_source.min.y ) / ( light_source.max.y - light_source.min.y );

	float rdot = dot( ray.direction, normal );
	return rtTex2D<float>( light_source.texture, phi, theta ) / fabsf( rdot ); // this is flatcorr from source.cal
}
#endif /* CALLLABLE */
