#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2014 Nathaniel Jones
 * Massachusetts Institute of Technology
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "optix_shader_common.h"

using namespace optix;

/* Material variables */
rtDeclareVariable(float3,       color, , );
rtDeclareVariable(float,        maxrad, , ) = RAY_END;

/* OptiX variables */
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(PerRayData_radiance, prd, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );

/* Attributes */
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 

RT_PROGRAM void closest_hit_shadow()
{
	if ( t_hit > maxrad )
		prd_shadow.result = make_float3( 0.0f );
	else
		prd_shadow.result = color;
}

RT_PROGRAM void closest_hit_radiance()
{
	// no contribution to ambient calculation
	if ( t_hit > maxrad || prd.ambient_depth > 0 ) //TODO need a better ambient test and handle maxrad < 0
		prd.result = make_float3( 0.0f );
	else
		prd.result = color;
	prd.distance = t_hit;

#ifdef HIT_TYPE
	prd.hit_type = MAT_LIGHT;
#endif
}
