#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2016 Nathaniel Jones
 * Massachusetts Institute of Technology
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "optix_shader_common.h"
#ifdef CONTRIB
#include "optix_shader_contrib.h"
#endif

using namespace optix;

/* Material variables */
#ifdef HIT_TYPE
rtDeclareVariable(unsigned int, type, , ); /* The material type representing "light", "illum", "glow", or "spot" */
#endif
rtDeclareVariable(float3,       color, , );
rtDeclareVariable(float,        maxrad, , ) = RAY_END;
rtDeclareVariable(float,        siz, , ) = -1.0f;		/* output solid angle or area */
rtDeclareVariable(float,        flen, , );				/* focal length (negative if distant source) */
rtDeclareVariable(float3,       aim, , );				/* aim direction or center */
rtDeclareVariable(rtCallableProgramId<float3(const float3, const float3)>, function, , );		/* function or texture modifier */

/* Context variables */
rtDeclareVariable(int,          directvis, , );		/* Boolean switch for light source visibility (dv) */
#ifdef ANTIMATTER
rtDeclareVariable(rtObject, top_object, , );
#endif

/* OptiX variables */
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(PerRayData_radiance, prd, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );

/* Attributes */
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(int, surface_id, attribute surface_id, );
#ifdef ANTIMATTER
rtDeclareVariable(int, mat_id, attribute mat_id, );
#endif


RT_METHOD int spotout();


RT_PROGRAM void closest_hit_shadow()
{
#ifdef ANTIMATTER
	if (prd_shadow.mask & (1 << mat_id)) {
		prd_shadow.result = make_float3(0.0f);
		return;
	}
#endif /* ANTIMATTER */

	float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));

	if ( t_hit > maxrad || spotout() || dot( world_shading_normal, ray.direction ) > 0.0f || surface_id != -prd_shadow.target - 1 )
		prd_shadow.result = make_float3( 0.0f );
	else if ( function > RT_PROGRAM_ID_NULL )
		prd_shadow.result = color * function( ray.direction, world_shading_normal );
	else
		prd_shadow.result = color;
}

RT_PROGRAM void closest_hit_radiance()
{
#ifdef ANTIMATTER
	if (prd.mask & (1 << mat_id)) {
		float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
		prd.inside += dot(world_geometric_normal, ray.direction) < 0.0f ? 1 : -1;

		/* Continue the ray */
		float3 snormal = faceforward(world_geometric_normal, -ray.direction, world_geometric_normal);
		Ray new_ray = make_Ray(ray.origin, ray.direction, ray.ray_type, ray_start(ray.origin + t_hit * ray.direction, ray.direction, snormal, RAY_START) + t_hit, RAY_END);
		rtTrace(top_object, new_ray, prd);
		return;
	}
#endif /* ANTIMATTER */

	float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));

	// no contribution to ambient calculation
	if ( !directvis || 0.0f > maxrad && prd.depth > 0 || prd.ambient_depth > 0 || spotout() || dot( world_shading_normal, ray.direction ) > 0.0f ) //TODO need a better ambient test
		prd.result = make_float3( 0.0f );
	else if ( function > RT_PROGRAM_ID_NULL )
		prd.result = color * function( ray.direction, world_shading_normal );
	else
		prd.result = color;
	prd.distance = t_hit;

#ifdef HIT_TYPE
	prd.hit_type = type;
#endif
#ifdef CONTRIB
	contribution(prd.rcoef, prd.result, ray.direction);
#endif
}

RT_METHOD int spotout()
{
	if ( siz < -FTINY )
		return(0); /* Not a spotlight */
	if ( flen < -FTINY ) {		/* distant source */
		const float3 vd = aim - ray.origin;
		float d = dot( ray.direction, vd );
		/*			wrong side?
		if (d <= FTINY)
			return(1);	*/
		d = dot( vd, vd ) - d * d;
		return (M_PIf * d > siz); /* If true then out */
	}
					/* local source */
	return (siz < 2.0f * M_PIf * (1.0f + dot(aim, ray.direction)));	/* If true then out */
}
