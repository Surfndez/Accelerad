#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2015 Nathaniel Jones
 * Massachusetts Institute of Technology
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "optix_shader_common.h"

using namespace optix;

/* Material variables */
#ifdef HIT_TYPE
rtDeclareVariable(unsigned int, type, , ); /* The material type representing "light", "illum", "glow", or "spot" */
#endif
rtDeclareVariable(float3,       color, , );
rtDeclareVariable(float,        maxrad, , ) = RAY_END;
rtDeclareVariable(float,        siz, , ) = -1.0f;		/* output solid angle or area */
rtDeclareVariable(float,        flen, , );				/* focal length (negative if distant source) */
rtDeclareVariable(float3,       aim, , );				/* aim direction or center */
rtDeclareVariable(rtCallableProgramId<float3(float3,float3)>, function, , );		/* function or texture modifier */

/* Context variables */
rtDeclareVariable(int,          directvis, , );		/* Boolean switch for light source visibility (dv) */

/* OptiX variables */
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(PerRayData_radiance, prd, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );

/* Attributes */
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(int, surface_id, attribute surface_id, );



RT_METHOD int spotout();


RT_PROGRAM void closest_hit_shadow()
{
	float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
	//float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );

	//float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

	if ( t_hit > maxrad || spotout() || dot( world_shading_normal, ray.direction ) > 0.0f || surface_id != -prd_shadow.target - 1 )
		prd_shadow.result = make_float3( 0.0f );
	else if ( function > RT_PROGRAM_ID_NULL )
		prd_shadow.result = color * function( ray.direction, world_shading_normal );
	else
		prd_shadow.result = color;
}

RT_PROGRAM void closest_hit_radiance()
{
	float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
	//float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );

	//float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

	// no contribution to ambient calculation
	if ( !directvis || 0.0f > maxrad && prd.depth > 0 || prd.ambient_depth > 0 || spotout() || dot( world_shading_normal, ray.direction ) > 0.0f ) //TODO need a better ambient test
		prd.result = make_float3( 0.0f );
	else if ( function > RT_PROGRAM_ID_NULL )
		prd.result = color * function( ray.direction, world_shading_normal );
	else
		prd.result = color;
	prd.distance = t_hit;

#ifdef HIT_TYPE
	prd.hit_type = type;
#endif
}

RT_METHOD int spotout()
{
	if ( siz < -FTINY )
		return(0); /* Not a spotlight */
	if ( flen < -FTINY ) {		/* distant source */
		const float3 vd = aim - ray.origin;
		float d = dot( ray.direction, vd );
		/*			wrong side?
		if (d <= FTINY)
			return(1);	*/
		d = dot( vd, vd ) - d * d;
		return (M_PIf * d > siz); /* If true then out */
	}
					/* local source */
	return (siz < 2.0f * M_PIf * (1.0f + dot(aim, ray.direction)));	/* If true then out */
}
