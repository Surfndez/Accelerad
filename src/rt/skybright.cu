#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2014 Nathaniel Jones
 * Massachusetts Institute of Technology
 */

#include <optix_world.h>
//#include "optix_shader_common.h"

/* Program variables */
rtDeclareVariable(unsigned int, type, , ); /* 1 for CIE clear, 2 for CIE overcast, 3 for uniform, 4 for CIE intermediate */
rtDeclareVariable(float,        zenith, , ); /* zenith brightness */
rtDeclareVariable(float,        ground, , ); /* ground plane brightness */
rtDeclareVariable(float,        factor, , ); /* normalization factor based on sun direction */
rtDeclareVariable(float3,       sun, , ); /* sun direction */

// Calculate the sky brightness function for sunny and cloudy skies.
// This function replicates the algorithm in skybright.cal.
RT_CALLABLE_PROGRAM float sky_bright( const float3 direction )
{
	rtPrintf("SkyBright Recieved (%f, %f, %f)\n", direction.x, direction.y, direction.z);
	float cosgamma = optix::dot( direction, sun ); // cosgamma = Dx*A8 + Dy*A9 + Dz*A10;
	float gamma = acosf(cosgamma); // gamma = Acos(cosgamma);		{ angle from sun to this point in sky }
	//float dz = direction.z;
	float sky = zenith; // unifsky = A2; select(A1, sunnysky, cloudysky, unifsky, intersky)

	if (type == 1u) { // CIE clear
		sky *= ( 0.91f + 10.0f * expf( -3.0f * gamma ) + 0.45f * cosgamma * cosgamma ) / factor;
		if ( direction.z > 0.01f ) {
			sky *= 1.0f - expf(-0.32f / direction.z );
		}
		//sunnysky = A2 * (.91 + 10*exp(-3*gamma) + .45*cosgamma*cosgamma)
	    //  *  if( Dz - .01, 1.0 - exp(-.32/Dz), 1.0) / A4;
	} else if (type == 2u) { // CIE overcast
		sky *= ( 1.0f + 2.0f * direction.z ) / 3.0f; // cloudysky = A2 * (1 + 2*Dz)/3;
	} else if (type == 4u) { // CIE intermediate
		float zt = acosf(sun.z); // zt = Acos(A10);			{ angle from zenith to sun }
		float eta = acosf(direction.z); // eta = Acos(Dz);			{ angle from zenith to this point in sky }
		sky *= ( ( 1.35f * sinf( 5.631f - 3.59f * eta ) + 3.12f ) * sinf( 4.396f - 2.6f * zt) + 6.37f - eta ) / 2.326f *
			expf( gamma * -0.563f * ( ( 2.629f - eta ) * ( 1.562f - zt ) + 0.812f ) ) / factor;
		//intersky = A2 * ( (1.35*sin(5.631-3.59*eta)+3.12)*sin(4.396-2.6*zt)
		//	+ 6.37 - eta ) / 2.326 *
		// exp(gamma*-.563*((2.629-eta)*(1.562-zt)+.812)) / A4;
	}

	float a = powf(direction.z + 1.01f, 10.0f);
	float b = powf(direction.z + 1.01f, -10.0f);

	float skybr = (a * sky + b * ground) / (a + b); // wmean(a, x, b, y) = (a*x+b*y)/(a+b);
	return skybr;
}