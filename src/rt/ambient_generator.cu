#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2015 Nathaniel Jones
 * Massachusetts Institute of Technology
 */

#include <optix_world.h>
#include "optix_shader_common.h"

using namespace optix;

/* Program variables */
rtDeclareVariable(unsigned int,  camera, , );
rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , ); /* view.hvec */
rtDeclareVariable(float3,        V, , ); /* view.vvec */
rtDeclareVariable(float3,        W, , ); /* view.vdir */
rtDeclareVariable(float2,        fov, , );
rtDeclareVariable(float2,        shift, , );
rtDeclareVariable(float2,        clip, , );
//rtDeclareVariable(float,         dstrpix, , ); /* Pixel sample jitter (-pj) */

/* Contex variables */
rtBuffer<AmbientRecord, 3>       ambient_record_buffer; /* output */
//rtBuffer<unsigned int, 2>        rnd_seeds;
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(unsigned int,  ambient_record_ray_type, , );
rtDeclareVariable(unsigned int,  segments, , );
rtDeclareVariable(unsigned int,  level, , ) = 0u;

/* OptiX variables */
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );

// Initialize the random state
RT_METHOD void init_state( PerRayData_ambient_record* prd )
{
	rand_state state;
	prd->state = &state;
	hiprand_init( launch_index.x + launch_dim.x * launch_index.y, 0, 0, prd->state );
}

RT_METHOD float2 get_offset( unsigned int segment )
{
	float2 offset = make_float2( 0.5f );
	float delta = 0.5f;

	for ( ; segment > 0u; segment >>= 2 ) {
		unsigned int x = segment & 1u;
		unsigned int y = ( segment >> 1 ) & 1u;
		y = x ^ y;
		if ( x ) offset.x += delta;
		if ( y ) offset.y += delta;
		delta *= -0.5f;
	}

	return offset;
}

// Pick the ray direction based on camera type as in image.c.
RT_PROGRAM void ambient_camera()
{
	PerRayData_ambient_record prd;
	init_state( &prd );
	prd.parent = NULL;
	prd.result.pos = prd.result.val = make_float3( 0.0f );
	prd.result.lvl = level;
	prd.result.weight = 1.0f;
	for ( int i = level; i--; )
		prd.result.weight *= AVGREFL; // Compute weight as in makeambient() from ambient.c

	uint3 index = make_uint3( launch_index, 0u );
	unsigned int power_of_two = 0u; // always a power of 2

	// Zero or negative aft clipping distance indicates infinity
	float aft = clip.y - clip.x;
	if (aft <= FTINY) {
		aft = RAY_END;
	}

	for ( unsigned int segment = 0u; segment < segments; segment++ ) {
		// Choose the parent
		if ( segment ) {
			// Level up if we've reached a new power of 2
			if ( !( segment & ( segment - 1u ) ) )
				power_of_two = segment;

			index.z = segment - power_of_two;
			AmbientRecord record = ambient_record_buffer[index];
			prd.parent = &record;
			// Check that the parent was computed
			if ( prd.parent->weight < FTINY )
				continue;
		}

		//float2 d = make_float2( hiprand_uniform( state ), hiprand_uniform( state ) );
		//d = 0.5f + dstrpix * ( 0.5f - d ); // this is pixjitter() from rpict.c
		float2 d = get_offset( segment );
		d = shift + ( make_float2( launch_index ) + d ) / make_float2( launch_dim ) - 0.5f;
		float3 ray_origin = eye;
		float z = 1.0f;

		// This is adapted from viewray() in image.c.
  		if( camera == VT_PAR ) { /* parallel view */
			ray_origin += d.x*U + d.y*V;
			d = make_float2( 0.0f );
		} else if ( camera == VT_HEM ) { /* hemispherical fisheye */
			z = 1.0f - d.x*d.x * dot( U, U ) - d.y*d.y * dot( V, V );
			if (z < 0.0f) {
				//ambient_record_buffer[launch_index] = make_float4( 0.0f );//TODO throw an exception?
				return;
			}
			z = sqrtf(z);
		} else if ( camera == VT_CYL ) { /* cylindrical panorama */
			float dd = d.x * fov.x * ( M_PIf / 180.0f );
			z = cosf( dd );
			d.x = sinf( dd );
		} else if ( camera == VT_ANG ) { /* angular fisheye */
			d *= fov / 180.0f;
			float dd = sqrtf( dot( d, d ) );
			if (dd > 1.0f) {
				//ambient_record_buffer[launch_index] = make_float4( 0.0f );//TODO throw an exception?
				return;
			}
			z = cosf( M_PIf * dd );
			d *= sqrtf( 1.0f - z*z ) / dd;
		} else if ( camera == VT_PLS ) { /* planispheric fisheye */
			d *= make_float2( sqrtf( dot( U, U ) ), sqrtf( dot( V, V ) ) );
			float dd = dot( d, d );
			z = ( 1.0f - dd ) / ( 1.0f + dd );
			d *= 1.0f + z;
		}

		float3 ray_direction = d.x*U + d.y*V + z*W;
		ray_origin += clip.x * ray_direction;
		ray_direction = normalize(ray_direction);

		Ray ray = make_Ray(ray_origin, ray_direction, ambient_record_ray_type, 0.0f, aft);

#ifndef OLDAMB
		prd.result.rad = make_float2( 0.0f );
#else
		prd.result.rad = 0.0f;
#endif
#ifdef RAY_COUNT
		prd.result.ray_count = 0;
#endif
#ifdef HIT_COUNT
		prd.result.hit_count = 0;
#endif

		rtTrace(top_object, ray, prd);

		checkFinite(prd.result.val);

		index.z = segment;
		ambient_record_buffer[index] = prd.result;
	}
}

RT_PROGRAM void exception()
{
	const unsigned int code = rtGetExceptionCode();
	rtPrintf( "Caught exception 0x%X at launch index (%d,%d)\n", code, launch_index.x, launch_index.y );
	uint3 index = make_uint3( launch_index, segments - 1u ); // record error to last segment
	ambient_record_buffer[index].lvl = level;
	ambient_record_buffer[index].val = exceptionToFloat3( code );
	ambient_record_buffer[index].weight = -1.0f;
}
