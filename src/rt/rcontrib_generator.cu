#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2016 Nathaniel Jones
 * Massachusetts Institute of Technology
 */

#include <optix_world.h>
#include "optix_shader_common.h"
#include "optiX_shader_ray.h"
#ifdef CONTRIB_DOUBLE
#include "optix_double.h"
#endif

using namespace optix;

/* Program variables */
rtDeclareVariable(unsigned int, do_irrad, , ) = 0u; /* Calculate irradiance (-i) */

/* Contex variables */
rtBuffer<float3, 2>              origin_buffer;
rtBuffer<float3, 2>              direction_buffer;
rtBuffer<contrib4, 3>            contrib_buffer;
#ifdef RAY_COUNT
rtBuffer<unsigned int, 2>        ray_count_buffer;
#endif
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(rtObject, top_irrad, , );
rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(unsigned int, radiance_primary_ray_type, , );
rtDeclareVariable(unsigned int, imm_irrad, , ) = 0u; /* Immediate irradiance (-I) */
rtDeclareVariable(unsigned int, lim_dist, , ) = 0u; /* Limit ray distance (-ld) */
rtDeclareVariable(unsigned int, contrib_segment, , ) = 0u; /* Start row for large outputs */

/* OptiX variables */
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim, rtLaunchDim, );


RT_PROGRAM void ray_generator()
{
	const uint2 index = make_uint2(launch_index.x, launch_index.y + contrib_segment);
	PerRayData_radiance prd;
	init_rand(&prd.state, index.x + launch_dim.x * index.y);
	prd.result = make_float3(0.0f); // Probably not necessary
	prd.weight = 1.0f;
	prd.depth = 0;
	prd.ambient_depth = 0;
	//prd.seed = rnd_seeds[launch_index];
#ifdef CONTRIB
	prd.rcoef = make_contrib3(1.0f);
#endif
#ifdef ANTIMATTER
	prd.mask = 0u;
	prd.inside = 0;
#endif
	setupPayload(prd);

	/* Zero the output */
	for (int i = 0; i < contrib_buffer.size().x; i++)
		contrib_buffer[make_uint3(i, launch_index.x, launch_index.y)] = make_contrib4(0.0f);

	float3 org = origin_buffer[index];
	float3 dir = direction_buffer[index];

	if (dot(dir, dir) > 0.0f) {
		const float tmin = ray_start(org, RAY_START);
		if (imm_irrad) {
			Ray ray = make_Ray(org, -normalize(dir), radiance_ray_type, -tmin, tmin);
			rtTrace(top_irrad, ray, prd);
		}
		else {
			Ray ray = make_Ray(org, normalize(dir), do_irrad ? radiance_primary_ray_type : radiance_ray_type, tmin, lim_dist ? length(dir) : RAY_END);
			rtTrace(top_object, ray, prd);
		}
	}

	checkFinite(prd.result); // Probably not necessary

#ifdef RAY_COUNT
	ray_count_buffer[launch_index] = prd.ray_count;
#endif
}

RT_PROGRAM void exception()
{
#ifdef PRINT_OPTIX
	rtPrintExceptionDetails();
#endif
#ifdef CONTRIB_DOUBLE
	contrib_buffer[make_uint3(0, launch_index.x, launch_index.y)] = make_contrib4(rtGetExceptionCode(), 0.0f, 0.0f, -1.0f);
#else
	contrib_buffer[make_uint3(0, launch_index.x, launch_index.y)] = exceptionToFloat4(rtGetExceptionCode());
#endif
}
