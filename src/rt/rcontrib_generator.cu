#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2016 Nathaniel Jones
 * Massachusetts Institute of Technology
 */

#include <optix_world.h>
#include "optix_shader_common.h"

using namespace optix;

/* Program variables */
rtDeclareVariable(unsigned int, do_irrad, , ) = 0u; /* Calculate irradiance (-i) */

/* Contex variables */
rtBuffer<float3, 2>              origin_buffer;
rtBuffer<float3, 2>              direction_buffer;
rtBuffer<float3, 3>              contrib_buffer;
#ifdef RAY_COUNT
rtBuffer<unsigned int, 2>        ray_count_buffer;
#endif
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(rtObject, top_irrad, , );
rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(unsigned int, radiance_primary_ray_type, , );
rtDeclareVariable(unsigned int, imm_irrad, , ) = 0u; /* Immediate irradiance (-I) */
rtDeclareVariable(unsigned int, lim_dist, , ) = 0u; /* Limit ray distance (-ld) */

/* OptiX variables */
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim, rtLaunchDim, );


RT_PROGRAM void ray_generator()
{
	PerRayData_radiance prd;
	init_rand(&prd.state, launch_index.x + launch_dim.x * launch_index.y);
	prd.weight = 1.0f;
	prd.depth = 0;
	prd.ambient_depth = 0;
	//prd.seed = rnd_seeds[launch_index];
#ifdef ANTIMATTER
	prd.mask = 0u;
	prd.inside = 0;
#endif
	setupPayload(prd);

	/* Zero the output */
	for (int i = 0; i < contrib_buffer.size().z; i++)
		contrib_buffer[make_uint3(launch_index.x, launch_index.y, i)] = make_float3(0.0f);

	float3 org = origin_buffer[launch_index];
	float3 dir = direction_buffer[launch_index];

	const float tmin = ray_start(org, RAY_START);
	if (imm_irrad) {
		Ray ray = make_Ray(org, -normalize(dir), radiance_ray_type, -tmin, tmin);
		rtTrace(top_irrad, ray, prd);
	}
	else {
		Ray ray = make_Ray(org, normalize(dir), do_irrad ? radiance_primary_ray_type : radiance_ray_type, tmin, lim_dist ? length(dir) : RAY_END);
		rtTrace(top_object, ray, prd);
	}

	checkFinite(prd.result);

#ifdef RAY_COUNT
	ray_count_buffer[launch_index] = prd.ray_count;
#endif
}

RT_PROGRAM void exception()
{
	const unsigned int code = rtGetExceptionCode();
	rtPrintf("Caught exception 0x%X at launch index (%d,%d)\n", code, launch_index.x, launch_index.y);
	contrib_buffer[make_uint3(launch_index.x, launch_index.y, 0)] = make_float3(code, 0.0f, -1.0f);;
}
