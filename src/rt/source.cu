#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2014 Nathaniel Jones
 * Massachusetts Institute of Technology
 */

#include <optix_world.h>
//#include "optix_shader_common.h"

using namespace optix;

/* Program variables */
rtDeclareVariable(int, data, , ); /* texture ID */
rtDeclareVariable(int, type, , ); /* type of data (true for float) */
rtDeclareVariable(float3, minimum, , ); /* texture minimum coordinates */
rtDeclareVariable(float3, maximum, , ); /* texture maximum coordinates */
rtDeclareVariable(float3, u, , ); /* transform matrix u-direction */
rtDeclareVariable(float3, v, , ); /* transform matrix v-direction */
rtDeclareVariable(float3, w, , ); /* transform matrix w-direction */
rtDeclareVariable(float, multiplier, , ) = 1.0f; /* multiplier for light source intensity */

// Calculate source distribution with correction for flat sources.
RT_CALLABLE_PROGRAM float3 flatcorr( const float3 direction, const float3 normal )
{
	//rtPrintf("FlatCorr Recieved (%f, %f, %f) (%f, %f, %f)\n", direction.x, direction.y, direction.z, normal.x, normal.y, normal.z);
	float phi = acosf( dot( direction, normalize( w ) ) );
	float theta = atan2f( -dot( direction, normalize( v ) ), -dot( direction, normalize( u ) ) );
	theta += 2.0f * M_PIf * ( theta < 0.0f );

	/* Normalize to [0, 1] within range */
	phi = ( 180.0f * M_1_PIf * phi - minimum.x ) / ( maximum.x - minimum.x );
	theta = ( 180.0f * M_1_PIf * theta - minimum.y ) / ( maximum.y - minimum.y );

	float rdot = dot( direction, normal );
	if ( type )
		return make_float3( multiplier * rtTex2D<float>( data, phi, theta ) / fabsf( rdot ) ); // this is flatcorr from source.cal
	return multiplier * make_float3( rtTex2D<float4>( data, phi, theta ) ) / fabsf( rdot );
}