#include "hip/hip_runtime.h"
/*
* Copyright (c) 2013-2015 Nathaniel Jones
* Massachusetts Institute of Technology
*/

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "optix_shader_common.h"

using namespace optix;

#define  AMBIENT
#define  TRANSMISSION

#ifndef  MAXITER
#define  MAXITER	10		/* maximum # specular ray attempts */
#endif
#define  MAXSPART	64		/* maximum partitions per source */
//#define frandom()	(rnd( prd.seed )/float(RAND_MAX))
//#define frandom()	(rnd( prd.seed ))

/* specularity flags */
#define  SP_REFL	01		/* has reflected specular component */
#define  SP_TRAN	02		/* has transmitted specular */
#define  SP_PURE	04		/* purely specular (zero roughness) */
#define  SP_FLAT	010		/* flat reflecting surface */
#define  SP_RBLT	020		/* reflection below sample threshold */
#define  SP_TBLT	040		/* transmission below threshold */

typedef struct {
	unsigned int specfl;		/* specularity flags, defined above */
	float3 mcolor;		/* color of this material */
	float3 scolor;		/* color of specular component */
	//float3 vrefl;		/* vector in direction of reflected ray */
	float3 prdir;		/* vector in transmitted direction */
	float3 normal;
	float3 hit;
	float  alpha2;		/* roughness squared */
	float  rdiff, rspec;	/* reflected specular, diffuse */
	float  trans;		/* transmissivity */
	float  tdiff, tspec;	/* transmitted specular, diffuse */
	float3 pnorm;		/* perturbed surface normal */
	float  pdot;		/* perturbed dot product */
}  NORMDAT;		/* normal material data */

/* Context variables */
rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(unsigned int, ambient_ray_type, , );
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(rtObject, top_ambient, , );

#ifdef AMBIENT
rtDeclareVariable(float3, ambval, , );	/* This is the final value used in place of an indirect light calculation */
rtDeclareVariable(int, ambvwt, , );	/* As new indirect irradiances are computed, they will modify the default ambient value in a moving average, with the specified weight assigned to the initial value given on the command and all other weights set to 1 */
rtDeclareVariable(int, ambounce, , );	/* Ambient bounces (ab) */
//rtDeclareVariable(int,          ambres, , );	/* Ambient resolution (ar) */
rtDeclareVariable(float, ambacc, , );	/* Ambient accuracy (aa). This value will approximately equal the error from indirect illuminance interpolation */
rtDeclareVariable(int, ambdiv, , );	/* Ambient divisions (ad) */
rtDeclareVariable(int, ambdiv_final, , ); /* Number of ambient divisions for final-pass fill (ag) */
rtDeclareVariable(int, ambssamp, , );	/* Ambient super-samples (as) */
#ifdef OLDAMB
rtDeclareVariable(float, maxarad, , );	/* maximum ambient radius */
rtDeclareVariable(float, minarad, , );	/* minimum ambient radius */
#endif /* OLDAMB */
rtDeclareVariable(float, avsum, , );		/* computed ambient value sum (log) */
rtDeclareVariable(unsigned int, navsum, , );	/* number of values in avsum */
#endif /* AMBIENT */

rtDeclareVariable(float, minweight, , );	/* minimum ray weight (lw) */
rtDeclareVariable(int, maxdepth, , );	/* maximum recursion depth (lr) */

rtBuffer<DistantLight> lights;

/* Material variables */
rtDeclareVariable(unsigned int, type, , );	/* The material type representing "plastic", "metal", or "trans" */
rtDeclareVariable(float3, color, , );	/* The material color given by the rad file "plastic", "metal", or "trans" object */
rtDeclareVariable(float, spec, , );	/* The material specularity given by the rad file "plastic", "metal", or "trans" object */
rtDeclareVariable(float, rough, , );	/* The material roughness given by the rad file "plastic", "metal", or "trans" object */

/* OptiX variables */
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(PerRayData_radiance, prd, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );

/* Attributes */
//rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );


//RT_METHOD float3 dirnorm(Ray *shadow_ray, PerRayData_shadow *shadow_prd, const NORMDAT *nd, const float& omega);
//RT_METHOD float3 gaussamp(const NORMDAT *nd);
#ifdef AMBIENT
RT_METHOD float3 multambient(float3 aval, const float3& normal, const float3& pnormal, const float3& hit);
#ifdef DAYSIM_COMPATIBLE
RT_METHOD int doambient(float3 *rcol, const float3& normal, const float3& pnormal, const float3& hit, DaysimCoef dc);
#else
RT_METHOD int doambient(float3 *rcol, const float3& normal, const float3& pnormal, const float3& hit);
#endif
//RT_METHOD int ambsample( AMBHEMI *hp, const int& i, const int& j, const float3 normal, const float3 hit );
#endif /* AMBIENT */


RT_PROGRAM void closest_hit_radiance()
{
	NORMDAT nd;

	float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));

	/* check for back side */
	nd.pnorm = faceforward(world_shading_normal, -ray.direction, world_geometric_normal);
	nd.normal = faceforward(world_geometric_normal, -ray.direction, world_geometric_normal);

	float3 result = make_float3(0.0f);
	nd.hit = ray.origin + t_hit * ray.direction;
	nd.mcolor = color;
	nd.scolor = make_float3(0.0f);
	nd.rspec = spec;
	nd.alpha2 = rough * rough;
	nd.specfl = 0u; /* specularity flags */

	/* get roughness */
	if (nd.alpha2 <= FTINY) {
		nd.specfl |= SP_PURE; // label this as a purely specular reflection
	}

	/* perturb normal */
	float3 pert = nd.normal - nd.pnorm;
	int hastexture = dot(pert, pert) > FTINY * FTINY;
	nd.pdot = -dot(ray.direction, nd.pnorm);
	if (nd.pdot < 0.0f) {		/* fix orientation from raynormal in raytrace.c */
		nd.pnorm += 2.0f * nd.pdot * ray.direction;
		nd.pdot = -nd.pdot;
	}
	if (nd.pdot < 0.001f)
		nd.pdot = 0.001f;			/* non-zero for dirnorm() */

	// if it's a face or a ring label as flat (currently we only support triangles, so everything is flat)
	nd.specfl |= SP_FLAT;

	/* modify material color */
	//nd.mcolor *= rtTex3D(rtTextureId id, texcoord.x, texcoord.y, texcoord.z).xyz;

	/* compute Fresnel approx. */
	float fest = 0.0f;
	if (nd.specfl & SP_PURE && nd.rspec >= FRESTHRESH) {
		fest = FRESNE(nd.pdot);
		nd.rspec += fest * (1.0f - nd.rspec);
	}

	/* compute transmission */
	nd.tdiff = nd.tspec = nd.trans = 0.0f; // because it's opaque

	/* diffuse reflection */
	nd.rdiff = 1.0f - nd.trans - nd.rspec;

	if (!(nd.specfl & SP_PURE && nd.rdiff <= FTINY && nd.tdiff <= FTINY)) { /* not 100% pure specular */
		/* ambient from this side */
		if (nd.rdiff > FTINY) {
			float3 aval = nd.mcolor * nd.rdiff;	/* modified by material color */
			if (nd.specfl & SP_RBLT)	/* add in specular as well? */
				aval += nd.scolor;
			result += multambient(aval, nd.normal, nd.pnorm, nd.hit);	/* add to returned color */
		}

#ifdef TRANSMISSION
		/* ambient from other side */
		if (nd.tdiff > FTINY) {
			float3 aval = nd.mcolor;	/* modified by material color */
			if (nd.specfl & SP_TBLT)
				aval *= nd.trans;
			else
				aval *= nd.tdiff;
			result += multambient(aval, -nd.normal, -nd.pnorm, nd.hit);	/* add to returned color */
		}
#endif /* TRANSMISSION */
	}

	prd.distance = t_hit;

	// pass the color back up the tree
	prd.result = result;

#ifdef HIT_TYPE
	prd.hit_type = type;
#endif
}


#ifdef AMBIENT
// Compute the ambient component and multiply by the coefficient.
RT_METHOD float3 multambient(float3 aval, const float3& normal, const float3& pnormal, const float3& hit)
{
	int do_ambient = 1;
	float 	d;

	if (ambdiv <= 0)			/* no ambient calculation */
		goto dumbamb;
	/* check number of bounces */
	if (prd.ambient_depth >= ambounce)
		goto dumbamb;
	/* check ambient list */
	//if (ambincl != -1 && r->ro != NULL && ambincl != inset(ambset, r->ro->omod))
	//	goto dumbamb;

	if (ambacc > FTINY && navsum != 0) {			/* ambient storage */
		//if (tracktime)				/* sort to minimize thrashing */
		//	sortambvals(0);

		/* interpolate ambient value */
		//acol = make_float3( 0.0f );
		//d = sumambient(acol, r, normal, rdepth, &atrunk, thescene.cuorg, thescene.cusize);
		PerRayData_ambient ambient_prd;
		ambient_prd.result = make_float3(0.0f);
		ambient_prd.surface_normal = pnormal;
		ambient_prd.ambient_depth = prd.ambient_depth;
		ambient_prd.wsum = 0.0f;
		ambient_prd.weight = prd.weight;
#ifdef OLDAMB
		ambient_prd.state = prd.state;
#endif
#ifdef DAYSIM_COMPATIBLE
		ambient_prd.dc = daysimNext(prd.dc);
		daysimSet(ambient_prd.dc, 0.0f);
#endif
#ifdef HIT_COUNT
		ambient_prd.hit_count = 0;
#endif
		const float tmin = ray_start(hit, AMBIENT_RAY_LENGTH);
		Ray ambient_ray = make_Ray(hit, normal, ambient_ray_type, -tmin, tmin);
		rtTrace(top_ambient, ambient_ray, ambient_prd);
#ifdef HIT_COUNT
		prd.hit_count += ambient_prd.hit_count;
#endif
		if (ambient_prd.wsum > FTINY) { // TODO if miss program is called, set wsum = 1.0f or place this before ambacc == 0.0f
			ambient_prd.result *= 1.0f / ambient_prd.wsum;
#ifdef DAYSIM_COMPATIBLE
			daysimAddScaled(prd.dc, ambient_prd.dc, aval.x / ambient_prd.wsum);
#endif
			return aval * ambient_prd.result;
		}
		//rdepth++;				/* need to cache new value */
		//d = makeambient(acol, r, normal, rdepth-1); //TODO implement as miss program for ambient ray
		//rdepth--;
		//if ( dot( ambient_prd.result, ambient_prd.result) > FTINY) { // quick check to see if a value was returned by miss program
		//	return aval * ambient_prd.result;		/* got new value */
		//}

#ifdef FILL_GAPS
		do_ambient = prd.primary && ambdiv_final;
#else
		do_ambient = !prd.ambient_depth && ambdiv_final;
#endif
	}
	if (do_ambient) {			/* no ambient storage */
		/* Option to show error if nothing found */
		if (ambdiv_final < 0)
			rtThrow(RT_EXCEPTION_USER - ambdiv_final);

		float3 acol = aval;
#ifdef DAYSIM_COMPATIBLE
		DaysimCoef dc = daysimNext(prd.dc);
		daysimSet(dc, 0.0f);
		d = doambient(&acol, normal, pnormal, hit, dc);
		if (d > FTINY)
			daysimAdd(prd.dc, dc);
#else
		d = doambient(&acol, normal, pnormal, hit);
#endif
		if (d > FTINY)
			return acol;
	}
dumbamb:					/* return global value */
	if ((ambvwt <= 0) || (navsum == 0)) {
#ifdef DAYSIM_COMPATIBLE
		daysimAdd(prd.dc, aval.x * ambval.x);
#endif
		return aval * ambval;
	}
	float l = bright(ambval);			/* average in computations */
	if (l > FTINY) {
		d = (logf(l)*(float)ambvwt + avsum) / (float)(ambvwt + navsum);
		d = expf(d) / l;
		aval *= ambval;	/* apply color of ambval */
#ifdef DAYSIM_COMPATIBLE
		daysimAdd(prd.dc, aval.x * ambval.x * d);
#endif
	}
	else {
		d = expf(avsum / (float)navsum);
#ifdef DAYSIM_COMPATIBLE
		daysimAdd(prd.dc, aval.x * d);
#endif
	}
	return aval * d;
}


/* sample indirect hemisphere, based on samp_hemi in ambcomp.c */
#ifdef DAYSIM_COMPATIBLE
RT_METHOD int doambient(float3 *rcol, const float3& normal, const float3& pnormal, const float3& hit, DaysimCoef dc)
#else
RT_METHOD int doambient(float3 *rcol, const float3& normal, const float3& pnormal, const float3& hit)
#endif
{
	float	d;
	float wt = prd.weight;

	/* set number of divisions */
	if (ambacc <= FTINY && wt > (d = 0.8f * fmaxf(*rcol) * wt / (ambdiv_final * minweight)))
		wt = d;			/* avoid ray termination */
	int n = sqrtf(ambdiv_final * wt) + 0.5f;
	int i = 1 + 8 * (ambacc > FTINY);	/* minimum number of samples */
	if (n < i)
		n = i;
	const int nn = n * n;
	float3 acol = make_float3(0.0f);
	unsigned int sampOK = 0u;
	/* assign coefficient */
	float3 acoef = *rcol / nn;

	/* Setup from ambsample in ambcomp.c */
	PerRayData_radiance new_prd;
	/* generate hemispherical sample */
	/* ambient coefficient for weight */
	if (ambacc > FTINY)
		d = AVGREFL; // Reusing this variable
	else
		d = fmaxf(acoef);
	new_prd.weight = prd.weight * d;
	if (new_prd.weight < minweight) //if (rayorigin(&ar, AMBIENT, r, ar.rcoef) < 0)
		return(0);

	new_prd.depth = prd.depth + 1;
	new_prd.ambient_depth = prd.ambient_depth + 1;
	//new_prd.seed = prd.seed;//lcg( prd.seed );
	new_prd.state = prd.state;
#ifdef DAYSIM_COMPATIBLE
	new_prd.dc = daysimNext(dc);
#endif

	Ray amb_ray = make_Ray(hit, pnormal, radiance_ray_type, RAY_START, RAY_END); // Use normal point as temporary direction
	/* End ambsample setup */

	/* make tangent plane axes */
	float3 ux = getperpendicular(pnormal, prd.state);
	float3 uy = cross(pnormal, ux);
	/* sample divisions */
	for (i = n; i--;)
		for (int j = n; j--;) {
			//hp.sampOK += ambsample( &hp, i, j, normal, hit );
			/* ambsample in ambcomp.c */
			float2 spt = 0.1f + 0.8f * make_float2(hiprand_uniform(prd.state), hiprand_uniform(prd.state));
			SDsquare2disk(spt, (j + spt.y) / n, (i + spt.x) / n);
			float zd = sqrtf(1.0f - dot(spt, spt));
			amb_ray.direction = normalize(spt.x*ux + spt.y*uy + zd*pnormal);
			if (dot(amb_ray.direction, normal) <= 0) /* Prevent light leaks */
				continue;
			amb_ray.tmin = ray_start(hit, amb_ray.direction, normal, RAY_START);
			//dimlist[ndims++] = AI(hp,i,j) + 90171;

			setupPayload(new_prd, 0);
			//Ray amb_ray = make_Ray( hit, rdir, radiance_ray_type, RAY_START, RAY_END );
			rtTrace(top_object, amb_ray, new_prd);
			resolvePayload(prd, new_prd);

			//ndims--;
			if (isnan(new_prd.result)) // TODO How does this happen?
				continue;
			if (new_prd.distance <= FTINY)
				continue;		/* should never happen */
			acol += new_prd.result * acoef;	/* add to our sum */
#ifdef DAYSIM_COMPATIBLE
			daysimAddScaled(dc, new_prd.dc, acoef.x);
#endif
			sampOK++;
		}
	*rcol = acol;
	if (!sampOK) {		/* utter failure? */
		return(0);
	}
	if (sampOK < nn) {
		//hp.sampOK *= -1;	/* soft failure */
		return(1);
	}
	//n = ambssamp * wt + 0.5f;
	//if (n > 8) {			/* perform super-sampling? */
	//	ambsupersamp(hp, n);
	//	*rcol = hp.acol;
	//}
	return(1);			/* all is well */
}
#endif /* AMBIENT */
