#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2016 Nathaniel Jones
 * Massachusetts Institute of Technology
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "optix_shader_common.h"

using namespace optix;

#define  TRANSMISSION

#ifndef  MAXITER
#define  MAXITER	10		/* maximum # specular ray attempts */
#endif
#define  MAXSPART	64		/* maximum partitions per source */
//#define frandom()	(rnd( prd.seed )/float(RAND_MAX))
//#define frandom()	(rnd( prd.seed ))

/* specularity flags */
#define  SP_REFL	01		/* has reflected specular component */
#define  SP_TRAN	02		/* has transmitted specular */
#define  SP_PURE	04		/* purely specular (zero roughness) */
#define  SP_FLAT	010		/* flat reflecting surface */
#define  SP_RBLT	020		/* reflection below sample threshold */
#define  SP_TBLT	040		/* transmission below threshold */

typedef struct {
	unsigned int specfl;		/* specularity flags, defined above */
	float3 mcolor;		/* color of this material */
	float3 scolor;		/* color of specular component */
	//float3 vrefl;		/* vector in direction of reflected ray */
	float3 prdir;		/* vector in transmitted direction */
	float3 normal;
	float3 hit;
	float  alpha2;		/* roughness squared */
	float  rdiff, rspec;	/* reflected specular, diffuse */
	float  trans;		/* transmissivity */
	float  tdiff, tspec;	/* transmitted specular, diffuse */
	float3 pnorm;		/* perturbed surface normal */
	float  pdot;		/* perturbed dot product */
}  NORMDAT;		/* normal material data */

/* Context variables */
rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(unsigned int, ambient_ray_type, , );
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(rtObject, top_ambient, , );

rtDeclareVariable(float3, ambval, , );	/* This is the final value used in place of an indirect light calculation */
rtDeclareVariable(int, ambvwt, , );	/* As new indirect irradiances are computed, they will modify the default ambient value in a moving average, with the specified weight assigned to the initial value given on the command and all other weights set to 1 */
rtDeclareVariable(int, ambounce, , );	/* Ambient bounces (ab) */
//rtDeclareVariable(int,          ambres, , );	/* Ambient resolution (ar) */
rtDeclareVariable(float, ambacc, , );	/* Ambient accuracy (aa). This value will approximately equal the error from indirect illuminance interpolation */
rtDeclareVariable(int, ambdiv, , );	/* Ambient divisions (ad) */
rtDeclareVariable(int, ambdiv_final, , ); /* Number of ambient divisions for final-pass fill (ag) */
rtDeclareVariable(int, ambssamp, , );	/* Ambient super-samples (as) */
#ifdef OLDAMB
rtDeclareVariable(float, maxarad, , );	/* maximum ambient radius */
rtDeclareVariable(float, minarad, , );	/* minimum ambient radius */
#endif /* OLDAMB */
rtDeclareVariable(float, avsum, , );		/* computed ambient value sum (log) */
rtDeclareVariable(unsigned int, navsum, , );	/* number of values in avsum */

rtDeclareVariable(float, minweight, , );	/* minimum ray weight (lw) */
rtDeclareVariable(int, maxdepth, , );	/* maximum recursion depth (lr) */

rtBuffer<DistantLight> lights;

/* Material variables */
rtDeclareVariable(unsigned int, type, , );	/* The material type representing "plastic", "metal", or "trans" */
rtDeclareVariable(float3, color, , );	/* The material color given by the rad file "plastic", "metal", or "trans" object */
rtDeclareVariable(float, spec, , );	/* The material specularity given by the rad file "plastic", "metal", or "trans" object */
rtDeclareVariable(float, rough, , );	/* The material roughness given by the rad file "plastic", "metal", or "trans" object */
rtDeclareVariable(unsigned int, ambincl, , ) = 1u;	/* Flag to skip ambient calculation and use default (ae, aE, ai, aI) */

/* OptiX variables */
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(PerRayData_radiance, prd, rtPayload, );

/* Attributes */
//rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
#ifdef ANTIMATTER
rtDeclareVariable(int, mat_id, attribute mat_id, );
#endif


RT_METHOD float3 multambient(float3 aval, const float3& normal, const float3& pnormal, const float3& hit);
#ifdef DAYSIM_COMPATIBLE
RT_METHOD int doambient(float3 *rcol, const float3& normal, const float3& pnormal, const float3& hit, DaysimCoef dc);
#else
RT_METHOD int doambient(float3 *rcol, const float3& normal, const float3& pnormal, const float3& hit);
#endif


RT_PROGRAM void closest_hit_radiance()
{
	NORMDAT nd;

	float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));

	/* check for back side */
	nd.pnorm = faceforward(world_shading_normal, -ray.direction, world_geometric_normal);
	nd.normal = faceforward(world_geometric_normal, -ray.direction, world_geometric_normal);

	float3 result = make_float3(0.0f);
	nd.hit = ray.origin + t_hit * ray.direction;
	nd.mcolor = color;
	nd.scolor = make_float3(0.0f);
	nd.rspec = spec;
	nd.alpha2 = rough * rough;
	nd.specfl = 0u; /* specularity flags */

#ifdef ANTIMATTER
	if (prd.mask & (1 << mat_id)) {
		prd.inside += dot(world_geometric_normal, ray.direction) < 0.0f ? 1 : -1;

		/* Continue the ray */
		Ray new_ray = make_Ray(ray.origin, ray.direction, ray.ray_type, ray_start(nd.hit, ray.direction, nd.normal, RAY_START) + t_hit, RAY_END);
		rtTrace(top_object, new_ray, prd);
		return;
	}
#endif /* ANTIMATTER */

	/* get roughness */
	if (nd.alpha2 <= FTINY) {
		nd.specfl |= SP_PURE; // label this as a purely specular reflection
	}

	/* perturb normal */
	float3 pert = nd.normal - nd.pnorm;
	int hastexture = dot(pert, pert) > FTINY * FTINY;
	nd.pdot = -dot(ray.direction, nd.pnorm);
	if (nd.pdot < 0.0f) {		/* fix orientation from raynormal in raytrace.c */
		nd.pnorm += 2.0f * nd.pdot * ray.direction;
		nd.pdot = -nd.pdot;
	}
	if (nd.pdot < 0.001f)
		nd.pdot = 0.001f;			/* non-zero for dirnorm() */

	// if it's a face or a ring label as flat (currently we only support triangles, so everything is flat)
	nd.specfl |= SP_FLAT;

	/* modify material color */
	//nd.mcolor *= rtTex3D(rtTextureId id, texcoord.x, texcoord.y, texcoord.z).xyz;

	/* compute Fresnel approx. */
	float fest = 0.0f;
	if (nd.specfl & SP_PURE && nd.rspec >= FRESTHRESH) {
		fest = FRESNE(nd.pdot);
		nd.rspec += fest * (1.0f - nd.rspec);
	}

	/* compute transmission */
	nd.tdiff = nd.tspec = nd.trans = 0.0f; // because it's opaque

	/* diffuse reflection */
	nd.rdiff = 1.0f - nd.trans - nd.rspec;

	if (!(nd.specfl & SP_PURE && nd.rdiff <= FTINY && nd.tdiff <= FTINY)) { /* not 100% pure specular */
		/* ambient from this side */
		if (nd.rdiff > FTINY) {
			float3 aval = nd.mcolor * nd.rdiff;	/* modified by material color */
			if (nd.specfl & SP_RBLT)	/* add in specular as well? */
				aval += nd.scolor;
			result += multambient(aval, nd.normal, nd.pnorm, nd.hit);	/* add to returned color */
		}

#ifdef TRANSMISSION
		/* ambient from other side */
		if (nd.tdiff > FTINY) {
			float3 aval = nd.mcolor;	/* modified by material color */
			if (nd.specfl & SP_TBLT)
				aval *= nd.trans;
			else
				aval *= nd.tdiff;
			result += multambient(aval, -nd.normal, -nd.pnorm, nd.hit);	/* add to returned color */
		}
#endif /* TRANSMISSION */
	}

	prd.distance = t_hit;

	// pass the color back up the tree
	prd.result = result;

#ifdef HIT_TYPE
	prd.hit_type = type;
#endif
}


// Compute the ambient component and multiply by the coefficient.
RT_METHOD float3 multambient(float3 aval, const float3& normal, const float3& pnormal, const float3& hit)
{
	float 	d;

	/* ambient calculation */
	if (ambdiv > 0 && prd.ambient_depth < ambounce && ambincl) {
		float3 acol = aval;
	#ifdef DAYSIM_COMPATIBLE
		DaysimCoef dc = daysimNext(prd.dc);
		daysimSet(dc, 0.0f);
		d = doambient(&acol, normal, pnormal, hit, dc);
		if (d > FTINY)
			daysimAdd(prd.dc, dc);
	#else
		d = doambient(&acol, normal, pnormal, hit);
	#endif
		if (d > FTINY)
			return acol;
	}
					/* return global value */
	if ((ambvwt <= 0) || (navsum == 0)) {
#ifdef DAYSIM_COMPATIBLE
		daysimAdd(prd.dc, aval.x * ambval.x);
#endif
		return aval * ambval;
	}
	float l = bright(ambval);			/* average in computations */
	if (l > FTINY) {
		d = (logf(l)*(float)ambvwt + avsum) / (float)(ambvwt + navsum);
		d = expf(d) / l;
		aval *= ambval;	/* apply color of ambval */
#ifdef DAYSIM_COMPATIBLE
		daysimAdd(prd.dc, aval.x * ambval.x * d);
#endif
	}
	else {
		d = expf(avsum / (float)navsum);
#ifdef DAYSIM_COMPATIBLE
		daysimAdd(prd.dc, aval.x * d);
#endif
	}
	return aval * d;
}


/* sample indirect hemisphere, based on samp_hemi in ambcomp.c */
#ifdef DAYSIM_COMPATIBLE
RT_METHOD int doambient(float3 *rcol, const float3& normal, const float3& pnormal, const float3& hit, DaysimCoef dc)
#else
RT_METHOD int doambient(float3 *rcol, const float3& normal, const float3& pnormal, const float3& hit)
#endif
{
	float	d;
	float wt = prd.weight;

	/* set number of divisions */
	if (ambacc <= FTINY && wt > (d = 0.8f * fmaxf(*rcol) * wt / (ambdiv_final * minweight)))
		wt = d;			/* avoid ray termination */
	float3 acol = make_float3(0.0f);
	float3 acoef = *rcol;

	/* Setup from ambsample in ambcomp.c */
	PerRayData_radiance new_prd;
	/* generate hemispherical sample */
	/* ambient coefficient for weight */
	if (ambacc > FTINY)
		d = AVGREFL; // Reusing this variable
	else
		d = fmaxf(acoef);
	new_prd.weight = prd.weight * d;
	if (new_prd.weight < minweight) //if (rayorigin(&ar, AMBIENT, r, ar.rcoef) < 0)
		return(0);

	new_prd.depth = prd.depth + 1;
	new_prd.ambient_depth = prd.ambient_depth + 1;
	//new_prd.seed = prd.seed;//lcg( prd.seed );
	new_prd.state = prd.state;
#ifdef ANTIMATTER
	new_prd.mask = prd.mask;
	new_prd.inside = prd.inside;
#endif
#ifdef DAYSIM_COMPATIBLE
	new_prd.dc = daysimNext(dc);
#endif

	Ray amb_ray = make_Ray(hit, pnormal, radiance_ray_type, RAY_START, RAY_END); // Use normal point as temporary direction
	/* End ambsample setup */

	/* make tangent plane axes */
	float3 ux = getperpendicular(pnormal, prd.state);
	float3 uy = cross(pnormal, ux);

	/* ambsample in ambcomp.c */
	float2 spt = 0.01f + 0.98f * make_float2(hiprand_uniform(prd.state), hiprand_uniform(prd.state));
	SDsquare2disk(spt, spt.y, spt.x);
	float zd = sqrtf(1.0f - dot(spt, spt));
	amb_ray.direction = normalize(spt.x*ux + spt.y*uy + zd*pnormal);
	if (dot(amb_ray.direction, normal) <= 0) /* Prevent light leaks */
		return(0);
	amb_ray.tmin = ray_start(hit, amb_ray.direction, normal, RAY_START);

	setupPayload(new_prd);
	//Ray amb_ray = make_Ray( hit, rdir, radiance_ray_type, RAY_START, RAY_END );
	rtTrace(top_object, amb_ray, new_prd);
	resolvePayload(prd, new_prd);

	if (isnan(new_prd.result)) // TODO How does this happen?
		return(0);
	if (new_prd.distance <= FTINY)
		return(0);		/* should never happen */
	acol += new_prd.result * acoef;	/* add to our sum */
#ifdef DAYSIM_COMPATIBLE
	daysimAddScaled(dc, new_prd.dc, acoef.x);
#endif
	*rcol = acol;
	return(1);			/* all is well */
}
