#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2014 Nathaniel Jones
 * Massachusetts Institute of Technology
 */

#include <optix_world.h>
#include "optix_shader_common.h"

using namespace optix;

/* Program variables */
rtDeclareVariable(unsigned int,  camera, , ) = 0u;
rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , ); /* view.hvec */
rtDeclareVariable(float3,        V, , ); /* view.vvec */
rtDeclareVariable(float3,        W, , ); /* view.vdir */
rtDeclareVariable(float2,        fov, , );
rtDeclareVariable(float2,        shift, , );
rtDeclareVariable(float2,        clip, , );
rtDeclareVariable(float,         dstrpix, , ); /* Pixel sample jitter (-pj) */

/* Contex variables */
rtBuffer<PointDirection, 3>      seed_buffer;
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(unsigned int,  point_cloud_ray_type, , );
rtDeclareVariable(unsigned int,  seeds, , ) = 1u; /* number of seed points to discover per thread */

/* OptiX variables */
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );

static __device__ float3 uniform_solid_angle( float2 in )
{
	float2 d = 2.0f * in - 1.0f; // map input from [0, 1] to [-1, 1]
	float z = d.y > 0.0f ? 1.0f : -1.0f;
	if ( ( abs( d.x ) < FTINY ) && ( abs( d.y ) < FTINY ) )
		return make_float3( 0.0f, 0.0f, z );

	d.y = 2.0f * d.y - z;
	float s, x, y;
	if ( abs( d.x ) > abs( d.y ) ) {
		float angle = M_PI_4f * d.y / d.x;
		s = d.x;
		x = cosf( angle );
		y = sinf( angle );
	} else {
		float angle = M_PI_4f * d.x / d.y;
		s = d.y;
		x = sinf( angle );
		y = cosf( angle );
	}
	float r = s * sqrtf( 2.0f - s * s );
	return make_float3( r * x, r * y, z - z * s * s );
}

RT_PROGRAM void point_cloud_camera()
{
	PerRayData_point_cloud prd;
	prd.backup.pos = make_float3( 0.0f );
	prd.backup.dir = make_float3( 0.0f );

	// Init random state
	rand_state state;
	hiprand_init( launch_index.x + launch_dim.x * launch_index.y, 0, 0, &state );

	uint3 index = make_uint3( launch_index, 0u );

	float3 direction;
	float2 d = make_float2( hiprand_uniform( &state ), hiprand_uniform( &state ) );
	d = 0.5f + dstrpix * ( 0.5f - d ); // this is pixjitter() from rpict.c

	// Set initial ray direction
	if ( camera ) { // using the camera viewport
		d = shift + ( make_float2( launch_index ) + d ) / make_float2( launch_dim ) - 0.5f;
		float3 ray_origin = eye;
		float z = 1.0f;

		// This is adapted from viewray() in image.c.
  		if( camera == VT_PAR ) { /* parallel view */
			ray_origin += d.x*U + d.y*V;
			d = make_float2( 0.0f );
		} else if ( camera == VT_HEM ) { /* hemispherical fisheye */
			z = 1.0f - d.x*d.x * dot( U, U ) - d.y*d.y * dot( V, V );
			if (z < 0.0f) {
				seed_buffer[index] = prd.backup;//TODO throw an exception?
				return;
			}
			z = sqrtf(z);
		} else if ( camera == VT_CYL ) { /* cylindrical panorama */
			float dd = d.x * fov.x * ( M_PIf / 180.0f );
			z = cosf( dd );
			d.x = sinf( dd );
		} else if ( camera == VT_ANG ) { /* angular fisheye */
			d *= fov / 180.0f;
			float dd = sqrtf( dot( d, d ) );
			if (dd > 1.0f) {
				seed_buffer[index] = prd.backup;//TODO throw an exception?
				return;
			}
			z = cosf( M_PIf * dd );
			d *= sqrtf( 1.0f - z*z ) / dd;
		} else if ( camera == VT_PLS ) { /* planispheric fisheye */
			d *= make_float2( sqrtf( dot( U, U ) ), sqrtf( dot( V, V ) ) );
			float dd = dot( d, d );
			z = ( 1.0f - dd ) / ( 1.0f + dd );
			d *= 1.0f + z;
		}

		direction = normalize(d.x*U + d.y*V + z*W);
	} else { // using a sphere with equal solid angle divisions
		d = ( make_float2( launch_index ) + d ) / make_float2( launch_dim );// - 0.5f;

		// Get the position and normal of the first ray
		direction = uniform_solid_angle( d );
	}

	Ray ray = make_Ray(eye, direction, point_cloud_ray_type, RAY_START, RAY_END);

	unsigned int loop = 2u * seeds; // Prevent infinite looping
	while ( index.z < seeds && loop-- ) {
		// Trace the current ray
		rtTrace(top_object, ray, prd);

		// Check for a valid result
		if ( isfinite( prd.result.pos ) && dot( prd.result.dir, prd.result.dir ) > FTINY ) { // NaN values will be false
			seed_buffer[index] = prd.result; // This could contain points on glass materials
			index.z++;
		} else {
			prd.result.pos = eye;
			prd.result.dir = direction;
		}

		// Prepare for next ray
		ray.origin = prd.result.pos;
		//ray.direction = reflect( ray.direction, prd.result.dir );

		float3 uz = normalize( prd.result.dir );
		float3 uy = cross_direction( uz );
		float3 ux = normalize( cross( uy, uz ) );
		uy = normalize( cross( uz, ux ) );

		float zd = sqrtf( hiprand_uniform( &state ) );
		float phi = 2.0f*M_PIf * hiprand_uniform( &state );
		float xd = cosf(phi) * zd;
		float yd = sinf(phi) * zd;
		zd = sqrtf(1.0f - zd*zd);
		ray.direction = normalize( xd*ux + yd*uy + zd*uz );
	}

	// If outdoors, there are no bounces, but we need to prevent junk data
	prd.backup.pos = make_float3( 0.0f );
	prd.backup.dir = make_float3( 0.0f );
	while ( index.z < seeds ) {
		seed_buffer[index] = prd.backup;
		index.z++;
	}
}

RT_PROGRAM void exception()
{
	const unsigned int code = rtGetExceptionCode();
	rtPrintf( "Caught exception 0x%X at launch index (%d,%d)\n", code, launch_index.x, launch_index.y );
	uint3 index = make_uint3( launch_index, seeds - 1u ); // record error to last segment
	seed_buffer[index].pos = exceptionToFloat3( code );
	seed_buffer[index].dir = make_float3( 0.0f );
}
