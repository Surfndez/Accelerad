#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2015 Nathaniel Jones
 * Massachusetts Institute of Technology
 */

#include <optix_world.h>
#include "optix_shader_common.h"

using namespace optix;

/* Program variables */
rtDeclareVariable(unsigned int,  camera, , ); /* Camera type (-vt) */
rtDeclareVariable(float3,        eye, , ); /* Eye position (-vp) */
rtDeclareVariable(float3,        U, , ); /* view.hvec */
rtDeclareVariable(float3,        V, , ); /* view.vvec */
rtDeclareVariable(float3,        W, , ); /* view.vdir */
rtDeclareVariable(float2,        fov, , ); /* Field of view (-vh, -vv) */
rtDeclareVariable(float2,        shift, , ); /* Camera shift (-vs, -vl) */
rtDeclareVariable(float2,        clip, , ); /* Fore and aft clipping planes (-vo, -va) */
rtDeclareVariable(float,         dstrpix, , ); /* Pixel sample jitter (-pj) */

/* Contex variables */
rtBuffer<PointDirection, 3>      seed_buffer;
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(unsigned int,  point_cloud_ray_type, , );

/* OptiX variables */
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );

RT_METHOD float3 uniform_solid_angle( float2 in )
{
	float2 d = 2.0f * in - 1.0f; // map input from [0, 1] to [-1, 1]
	float z = d.y > 0.0f ? 1.0f : -1.0f;
	if ( ( fabsf( d.x ) < FTINY ) && ( fabsf( d.y ) < FTINY ) )
		return make_float3( 0.0f, 0.0f, z );

	d.y = 2.0f * d.y - z;
	float s, x, y;
	if ( fabsf( d.x ) > fabsf( d.y ) ) {
		float angle = M_PI_4f * d.y / d.x;
		s = d.x;
		x = cosf( angle );
		y = sinf( angle );
	} else {
		float angle = M_PI_4f * d.x / d.y;
		s = d.y;
		x = sinf( angle );
		y = cosf( angle );
	}
	float r = s * sqrtf( 2.0f - s * s );
	return make_float3( r * x, r * y, z - z * s * s );
}

RT_PROGRAM void point_cloud_camera()
{
	PerRayData_point_cloud prd;

	// Init random state
	rand_state* state;
	init_rand(&state, launch_index.x + launch_dim.x * launch_index.y);

	uint3 index = make_uint3( launch_index, 0u );
	const unsigned int seeds = seed_buffer.size().z;
	unsigned int loop = 2u * seeds; // Prevent infinite looping

	Ray ray;
	ray.origin = eye;
	ray.ray_type = point_cloud_ray_type;
	ray.tmin = 0.0f;

	float2 d = make_float2( hiprand_uniform( state ), hiprand_uniform( state ) );
	d = 0.5f + dstrpix * ( 0.5f - d ); // this is pixjitter() from rpict.c

	// Set initial ray direction
	if ( camera ) { // using the camera viewport
		d = shift + ( make_float2( launch_index ) + d ) / make_float2( launch_dim ) - 0.5f;
		float z = 1.0f;

		// This is adapted from viewray() in image.c.
  		if( camera == VT_PAR ) { /* parallel view */
			ray.origin += d.x*U + d.y*V;
			d = make_float2( 0.0f );
		} else if ( camera == VT_HEM ) { /* hemispherical fisheye */
			z = 1.0f - d.x*d.x * dot( U, U ) - d.y*d.y * dot( V, V );
			if (z < 0.0f)
				goto clearout;
			z = sqrtf(z);
		} else if ( camera == VT_CYL ) { /* cylindrical panorama */
			float dd = d.x * fov.x * ( M_PIf / 180.0f );
			z = cosf( dd );
			d.x = sinf( dd );
		} else if ( camera == VT_ANG ) { /* angular fisheye */
			d *= fov / 180.0f;
			float dd = length(d);
			if (dd > 1.0f)
				goto clearout;
			z = cosf( M_PIf * dd );
			d *= sqrtf( 1.0f - z*z ) / dd;
		} else if ( camera == VT_PLS ) { /* planispheric fisheye */
			d *= make_float2(length(U), length(V));
			float dd = dot( d, d );
			z = ( 1.0f - dd ) / ( 1.0f + dd );
			d *= 1.0f + z;
		}

		ray.direction = d.x*U + d.y*V + z*W;
		ray.direction += clip.x * ray.direction;
		ray.direction = normalize(ray.direction);

		// Zero or negative aft clipping distance indicates infinity
		ray.tmax = clip.y - clip.x;
		if (ray.tmax <= FTINY) {
			ray.tmax = RAY_END;
		}
	} else { // using a sphere with equal solid angle divisions
		d = ( make_float2( launch_index ) + d ) / make_float2( launch_dim );// - 0.5f;

		// Get the position and normal of the first ray
		ray.direction = uniform_solid_angle(d);
		ray.tmax = RAY_END;
	}

	while ( index.z < seeds && loop-- ) {
		clear(prd.backup);

		// Trace the current ray
		rtTrace(top_object, ray, prd);

		// Check for a valid result
		if ( isfinite( prd.result.pos ) && dot( prd.result.dir, prd.result.dir ) > FTINY ) { // NaN values will be false
			seed_buffer[index] = prd.result; // This could contain points on glass materials
			index.z++;
#ifndef AMBIENT_CELL
		} else {
			prd.result.pos = eye;
			prd.result.dir = ray.direction;
#endif /* AMBIENT_CELL */
		}

#ifdef AMBIENT_CELL
		if (!(isfinite(prd.backup.pos) && dot(prd.backup.dir, prd.backup.dir) > FTINY)) // NaN values will be false
			break;

		// Prepare for next ray
		ray.origin = prd.backup.pos;
		ray.direction = reflect(ray.direction, prd.backup.dir);
		ray.tmin = RAY_START;// ray_start(ray.origin, ray.direction, prd.backup.dir, RAY_START);
#else /* AMBIENT_CELL */
		// Prepare for next ray
		ray.origin = prd.result.pos;

		float3 uz = normalize( prd.result.dir );
		float3 ux = getperpendicular(uz);
		float3 uy = normalize(cross(uz, ux));

		float zd = sqrtf( hiprand_uniform( state ) );
		float phi = 2.0f*M_PIf * hiprand_uniform( state );
		float xd = cosf(phi) * zd;
		float yd = sinf(phi) * zd;
		zd = sqrtf(1.0f - zd*zd);
		ray.direction = normalize( xd*ux + yd*uy + zd*uz );

		ray.tmin = ray_start(ray.origin, RAY_START);
#endif /* AMBIENT_CELL */
		ray.tmax = RAY_END;
	}

clearout:
	// If outdoors, there are no bounces, but we need to prevent junk data
	while ( index.z < seeds ) {
		clear(seed_buffer[index]);
		index.z++;
	}
}

RT_PROGRAM void exception()
{
	const unsigned int code = rtGetExceptionCode();
	rtPrintf( "Caught exception 0x%X at launch index (%d,%d)\n", code, launch_index.x, launch_index.y );
	uint3 index = make_uint3(launch_index, seed_buffer.size().z - 1u); // record error to last segment
	seed_buffer[index].pos = exceptionToFloat3( code );
	seed_buffer[index].dir = make_float3( 0.0f );
#ifdef AMBIENT_CELL
	seed_buffer[index].cell = make_uint2(0);
#endif
}
