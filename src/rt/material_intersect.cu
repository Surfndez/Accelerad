#include "hip/hip_runtime.h"
/*
 *  material_intersect.cu - hit programs for the material on GPUs.
 */

#include "accelerad_copyright.h"

#include "otypes.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "optix_shader_ray.h"
#include "optix_point_common.h"
#ifdef CONTRIB
#include "optix_shader_contrib.h"
#endif

using namespace optix;


/* Program variables */
rtDeclareVariable(unsigned int, backvis, , ) = 1u; /* backface visibility (bv) */

/* Context variables */
rtDeclareVariable(unsigned int, do_irrad, , ) = 0u;	/* Calculate irradiance (-i) */
rtDeclareVariable(unsigned int, frame, , ) = 0u;	/* Current frame number, starting from zero, for rvu only */

rtBuffer<MaterialData> material_data;	/* One entry per Radiance material. */

/* OptiX variables */
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(PerRayData_radiance, prd, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );
rtDeclareVariable(PerRayData_point_cloud, prd_point_cloud, rtPayload, );

/* Attributes */
//rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(int, surface_id, attribute surface_id, );
rtDeclareVariable(int, mat_id, attribute mat_id, );

#ifdef ANTIMATTER
/* Context variables */
rtDeclareVariable(rtObject, top_object, , );
#endif


RT_PROGRAM void any_hit()
{
	if (mat_id < 0 || mat_id >= material_data.size()) {
		rtIgnoreIntersection();
	}
	else {
		const MaterialData mat = material_data[mat_id];

		// Backface visibility
		if (mat.type != MAT_CLIP && !backvis && dot(geometric_normal, ray.direction) > 0) {
			rtIgnoreIntersection();
		}
	}
}

RT_PROGRAM void closest_hit_radiance()
{
	IntersectData data;
	data.mat = material_data[mat_id];
	data.ray_type = ray.ray_type;
	data.t = t_hit;
	data.ray_direction = ray.direction;
	data.hit = ray.origin + t_hit * ray.direction;

	data.world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	data.world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));

	data.surface_id = surface_id;

#ifdef ANTIMATTER
	bool continue_ray = false;
	if (data.mat.type == MAT_CLIP) {
		if (dot(data.world_geometric_normal, ray.direction) < 0.0f) {
			/* Entering a volume */
			prd.mask |= data.mat.params.mask;
			continue_ray = true;
		}
		else if ((prd.mask & data.mat.params.mask) && prd.inside > 0 && data.mat.proxy > -1) {
			/* Leaving a volume and rendering the alternate material */
			data.mat = material_data[data.mat.proxy]; // TODO this will produce odd results if the proxy material is transparent
		}
		else {
			/* Just leave the volume */
			prd.mask &= ~data.mat.params.mask;
			continue_ray = true;
		}
	}
	else if (prd.mask & (1 << mat_id)) {
		/* Entering or leaving the material while in antimatter. */
		prd.inside += dot(data.world_geometric_normal, ray.direction) < 0.0f ? 1 : -1;
		continue_ray = true;
	}
	if (continue_ray) {
		/* Continue the ray */
		const float3 normal = faceforward(data.world_geometric_normal, -ray.direction, data.world_geometric_normal);
		Ray new_ray = make_Ray(ray.origin, ray.direction, RADIANCE_RAY, ray_start(data.hit, ray.direction, normal, RAY_START) + t_hit, ray.tmax);
		rtTrace(top_object, new_ray, prd);
		return;
	}
#endif /* ANTIMATTER */

	if (data.mat.type == MAT_ILLUM) {
		if (data.mat.proxy < 0) return;
		data.mat = material_data[data.mat.proxy];
	}
	if (prd.depth == 0 && do_irrad)
		if (data.mat.type == MAT_PLASTIC || data.mat.type == MAT_METAL || data.mat.type == MAT_TRANS) {
			data.mat = material_data[0];
		}
	int radiance_program_id = data.mat.radiance_program_id;
	if (prd.depth == 0 && frame)
		radiance_program_id = data.mat.diffuse_program_id;

	/* Call the material's callable program. */
	if (radiance_program_id != RT_PROGRAM_ID_NULL)
		prd = rtMarkedCallableProgramId<PerRayData_radiance(IntersectData const&, PerRayData_radiance)>(radiance_program_id, "closest_hit_radiance_call_site")(data, prd);

#ifdef HIT_TYPE
	prd.hit_type = data.mat.type;
#endif
#ifdef CONTRIB
	contribution(prd.rcoef, prd.result, ray.direction, data.mat.contrib_index, data.mat.contrib_function);
#endif
}

RT_PROGRAM void closest_hit_shadow()
{
	IntersectData data;
	data.mat = material_data[mat_id];
	data.ray_type = ray.ray_type;
	data.t = t_hit;
	data.ray_direction = ray.direction;
	data.hit = ray.origin + t_hit * ray.direction;

	data.world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	data.world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));

	data.surface_id = surface_id;

#ifdef ANTIMATTER
	bool continue_ray = false;
	if (data.mat.type == MAT_CLIP) {
		if (dot(data.world_geometric_normal, ray.direction) < 0.0f) {
			/* Entering a volume */
			prd_shadow.mask |= data.mat.params.mask;
			continue_ray = true;
		}
		else if ((prd_shadow.mask & data.mat.params.mask) && prd_shadow.inside > 0 && data.mat.proxy > -1) {
			/* Leaving a volume and rendering the alternate material */
			data.mat = material_data[data.mat.proxy]; // TODO this will produce odd results if the proxy material is transparent
		}
		else {
			/* Just leave the volume */
			prd_shadow.mask &= ~data.mat.params.mask;
			continue_ray = true;
		}
	}
	else if (prd_shadow.mask & (1 << mat_id)) {
		/* Entering or leaving the material while in antimatter. */
		prd_shadow.inside += dot(data.world_geometric_normal, ray.direction) < 0.0f ? 1 : -1;
		continue_ray = true;
	}
	if (continue_ray) {
		/* Continue the ray */
		const float3 normal = faceforward(data.world_geometric_normal, -ray.direction, data.world_geometric_normal);
		Ray new_ray = make_Ray(ray.origin, ray.direction, SHADOW_RAY, ray_start(data.hit, ray.direction, normal, RAY_START) + t_hit, ray.tmax);
		rtTrace(top_object, new_ray, prd_shadow);
		return;
	}
#endif /* ANTIMATTER */

	if (data.mat.shadow_program_id != RT_PROGRAM_ID_NULL)
		prd_shadow = rtMarkedCallableProgramId<PerRayData_shadow(IntersectData const&, PerRayData_shadow)>(data.mat.shadow_program_id, "closest_hit_shadow_call_site")(data, prd_shadow);

	//#ifdef CONTRIB
	//	contribution(prd_shadow.rcoef, prd_shadow.result, ray.direction, data.mat.contrib_index, data.mat.contrib_function); //TODO calculate contribution of shadow?
	//#endif
}

RT_PROGRAM void closest_hit_point_cloud()
{
	IntersectData data;
	data.mat = material_data[mat_id];
	data.ray_type = ray.ray_type;
	data.t = t_hit;
	data.ray_direction = ray.direction;
	data.hit = ray.origin + t_hit * ray.direction;

	data.world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	data.world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));

	data.surface_id = surface_id;

#ifdef ANTIMATTER
	bool continue_ray = false;
	if (data.mat.type == MAT_CLIP) {
		if (dot(data.world_geometric_normal, ray.direction) < 0.0f) {
			/* Entering a volume */
			prd_shadow.mask |= data.mat.params.mask;
			continue_ray = true;
		}
		else if ((prd_shadow.mask & data.mat.params.mask) && prd_shadow.inside > 0 && data.mat.proxy > -1) {
			/* Leaving a volume and rendering the alternate material */
			data.mat = material_data[data.mat.proxy]; // TODO this will produce odd results if the proxy material is transparent
		}
		else {
			/* Just leave the volume */
			prd_shadow.mask &= ~data.mat.params.mask;
			continue_ray = true;
		}
	}
	else if (prd_shadow.mask & (1 << mat_id)) {
		/* Entering or leaving the material while in antimatter. */
		prd_shadow.inside += dot(data.world_geometric_normal, ray.direction) < 0.0f ? 1 : -1;
		continue_ray = true;
	}
	if (continue_ray) {
		/* Continue the ray */
		const float3 normal = faceforward(data.world_geometric_normal, -ray.direction, data.world_geometric_normal);
		Ray new_ray = make_Ray(ray.origin, ray.direction, POINT_CLOUD_RAY, ray_start(data.hit, ray.direction, normal, RAY_START) + t_hit, ray.tmax);
		rtTrace(top_object, new_ray, prd_point_cloud);
		return;
	}
#endif /* ANTIMATTER */

	if (data.mat.point_cloud_program_id != RT_PROGRAM_ID_NULL)
		prd_point_cloud = rtMarkedCallableProgramId<PerRayData_point_cloud(IntersectData const&, PerRayData_point_cloud)>(data.mat.point_cloud_program_id, "closest_hit_point_cloud_call_site")(data, prd_point_cloud);
}
