#include "hip/hip_runtime.h"
/*
* Copyright (c) 2013-2016 Nathaniel Jones
* Massachusetts Institute of Technology
*/

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "optix_shader_common.h"
#include "optix_point_common.h"

using namespace optix;

#ifdef ANTIMATTER

/* Material variables */
#ifdef HIT_TYPE
rtDeclareVariable(unsigned int, type, , ); /* The material type representing "antimatter" */
#endif
rtDeclareVariable(unsigned int, mask, , ) = 0u; /* Bitmask of materials to be clipped. */

/* Context variables */
rtDeclareVariable(rtObject, top_object, , );

/* OptiX variables */
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(PerRayData_radiance, prd, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );
rtDeclareVariable(PerRayData_ambient_record, prd_ambient, rtPayload, );
rtDeclareVariable(PerRayData_point_cloud, prd_point_cloud, rtPayload, );

/* Attributes */
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );


RT_PROGRAM void closest_hit_radiance()
{
	float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));

	if (dot(world_geometric_normal, ray.direction) < 0.0f) {
		/* Entering a volume */
		prd.mask |= mask;
	}
	//else if (prd_point_cloud.mask && prd_point_cloud.inside > 0 && alt_mat > -1) {
	//	/* Leaving a volume and rendering the alternate material */
	// TODO implement this
	//	return;
	//}
	else {
		/* Just leave the volume */
		prd.mask &= ~mask;
	}

	/* Continue the ray */
	float3 hit_point = ray.origin + t_hit * ray.direction;
	float3 snormal = faceforward(world_geometric_normal, -ray.direction, world_geometric_normal);
	Ray new_ray = make_Ray(ray.origin, ray.direction, ray.ray_type, ray_start(hit_point, ray.direction, snormal, RAY_START) + t_hit, RAY_END);
	rtTrace(top_object, new_ray, prd);
}


RT_PROGRAM void closest_hit_shadow()
{
	float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));

	if (dot(world_geometric_normal, ray.direction) < 0.0f) {
		/* Entering a volume */
		prd_shadow.mask |= mask;
	}
	//else if (prd_point_cloud.mask && prd_point_cloud.inside > 0 && alt_mat > -1) {
	//	/* Leaving a volume and rendering the alternate material */
	// TODO implement this
	//	return;
	//}
	else {
		/* Just leave the volume */
		prd_shadow.mask &= ~mask;
	}

	/* Continue the ray */
	float3 hit_point = ray.origin + t_hit * ray.direction;
	float3 snormal = faceforward(world_geometric_normal, -ray.direction, world_geometric_normal);
	Ray new_ray = make_Ray(ray.origin, ray.direction, ray.ray_type, ray_start(hit_point, ray.direction, snormal, RAY_START) + t_hit, RAY_END);
	rtTrace(top_object, new_ray, prd_shadow);
}


RT_PROGRAM void closest_hit_point_cloud()
{
	float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));

	if (dot(world_geometric_normal, ray.direction) < 0.0f) {
		/* Entering a volume */
		prd_point_cloud.mask |= mask;
	}
	//else if (prd_point_cloud.mask && prd_point_cloud.inside > 0 && alt_mat > -1) {
	//	/* Leaving a volume and rendering the alternate material */
	// TODO implement this
	//	return;
	//}
	else {
		/* Just leave the volume */
		prd_point_cloud.mask &= ~mask;
	}

	/* Continue the ray */
	float3 hit_point = ray.origin + t_hit * ray.direction;
	float3 snormal = faceforward(world_geometric_normal, -ray.direction, world_geometric_normal);
	Ray new_ray = make_Ray(ray.origin, ray.direction, ray.ray_type, ray_start(hit_point, ray.direction, snormal, RAY_START) + t_hit, RAY_END);
	rtTrace(top_object, new_ray, prd_point_cloud);
}

#endif /* ANTIMATTER */
