#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2014 Nathaniel Jones
 * Massachusetts Institute of Technology
 */

#include <optix_world.h>
#include "optix_shader_common.h"

using namespace optix;

/* Contex variables */
rtBuffer<PointDirection, 1>      cluster_buffer; /* input */
rtBuffer<PointDirection, 3>      seed_buffer; /* output */
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(unsigned int,  point_cloud_ray_type, , );

/* OptiX variables */
rtDeclareVariable(uint3, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint3, launch_dim,   rtLaunchDim, );

RT_PROGRAM void hemisphere_camera()
{
	PerRayData_point_cloud prd;
	prd.backup.pos = make_float3( 0.0f );
	prd.backup.dir = make_float3( 0.0f );

	PointDirection eye = cluster_buffer[launch_index.x];

	// Check for valid input
	if ( isfinite( eye.pos ) && isfinite( eye.dir ) && dot( eye.dir, eye.dir ) > FTINY ) { // NaN values will be false
		// Init random state
		rand_state state;
		hiprand_init( launch_index.x + launch_dim.x * ( launch_index.y + launch_dim.y * launch_index.z ), 0, 0, &state );

		// Make axes
		float3 uz = normalize( eye.dir );
		float3 uy = cross_direction( uz );
		float3 ux = normalize( cross( uy, uz ) );
		uy = normalize( cross( uz, ux ) );

		// Set ray direction
		float zd = sqrtf( ( launch_index.y + hiprand_uniform( &state ) ) / launch_dim.y );
		float phi = 2.0f*M_PIf * ( launch_index.z + hiprand_uniform( &state ) ) / launch_dim.z;
		float xd = cosf(phi) * zd;
		float yd = sinf(phi) * zd;
		zd = sqrtf(1.0f - zd*zd);
		float3 rdir = normalize( xd * ux + yd * uy + zd * uz );

		// Trace the current ray
		Ray ray = make_Ray(eye.pos, rdir, point_cloud_ray_type, ray_start( eye.pos, rdir, eye.dir, RAY_START ), RAY_END);
		rtTrace(top_object, ray, prd);

		// Check for a valid result
		if ( isfinite( prd.result.pos ) && dot( prd.result.dir, prd.result.dir ) > FTINY ) { // NaN values will be false
			seed_buffer[launch_index] = prd.result; // This could contain points on glass materials
			return;
		}
	}
	seed_buffer[launch_index] = prd.backup;
}

RT_PROGRAM void exception()
{
	const unsigned int code = rtGetExceptionCode();
	rtPrintf( "Caught exception 0x%X at launch index (%d,%d,%d)\n", code, launch_index.x, launch_index.y, launch_index.z );
	seed_buffer[launch_index].pos = exceptionToFloat3( code );
	seed_buffer[launch_index].dir = make_float3( 0.0f );
}
