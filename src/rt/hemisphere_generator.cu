#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2015 Nathaniel Jones
 * Massachusetts Institute of Technology
 */

#include <optix_world.h>
#include "optix_shader_common.h"

using namespace optix;

/* Contex variables */
rtBuffer<PointDirection, 1>      cluster_buffer; /* input */
rtBuffer<PointDirection, 3>      seed_buffer; /* output */
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(unsigned int,  point_cloud_ray_type, , );

/* OptiX variables */
rtDeclareVariable(uint3, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint3, launch_dim,   rtLaunchDim, );

RT_PROGRAM void hemisphere_camera()
{
	PerRayData_point_cloud prd;
	clear(prd.backup);

	PointDirection eye = cluster_buffer[launch_index.z];

	// Check for valid input
	if ( isfinite( eye.pos ) && isfinite( eye.dir ) && dot( eye.dir, eye.dir ) > FTINY ) { // NaN values will be false
		// Init random state
		rand_state* state;
		init_rand(&state, launch_index.x + launch_dim.x * (launch_index.y + launch_dim.y * launch_index.z));

		// Make axes
		float3 uz = normalize(eye.dir);
		float3 ux = getperpendicular(uz, state);
		float3 uy = cross(uz, ux);
						/* avoid coincident samples */
		float2 spt = 0.1f + 0.8f * make_float2(hiprand_uniform(state), hiprand_uniform(state));
		SDsquare2disk(spt, (launch_index.y + spt.y) / launch_dim.y, (launch_index.x + spt.x) / launch_dim.x);
		float zd = sqrtf(1.0f - dot(spt, spt));
		float3 rdir = normalize(spt.x * ux + spt.y * uy + zd * uz);

		// Trace the current ray
		Ray ray = make_Ray(eye.pos, rdir, point_cloud_ray_type, ray_start( eye.pos, rdir, uz, RAY_START ), RAY_END);
		rtTrace(top_object, ray, prd);

		// Check for a valid result
		if ( isfinite( prd.result.pos ) && dot( prd.result.dir, prd.result.dir ) > FTINY ) { // NaN values will be false
			seed_buffer[launch_index] = prd.result; // This could contain points on glass materials
			return;
		}
	}
	seed_buffer[launch_index] = prd.backup;
}

RT_PROGRAM void exception()
{
	const unsigned int code = rtGetExceptionCode();
	rtPrintf( "Caught exception 0x%X at launch index (%d,%d,%d)\n", code, launch_index.x, launch_index.y, launch_index.z );
	seed_buffer[launch_index].pos = exceptionToFloat3( code );
	seed_buffer[launch_index].dir = make_float3( 0.0f );
#ifdef AMBIENT_CELL
	seed_buffer[launch_index].cell = make_uint2(0);
#endif
}
