#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2015 Nathaniel Jones
 * Massachusetts Institute of Technology
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "optix_shader_common.h"

using namespace optix;

#define  AMBIENT
#define  TRANSMISSION
//#define  ITERATIVE

#ifndef  MAXITER
#define  MAXITER	10		/* maximum # specular ray attempts */
#endif
#define  MAXSPART	64		/* maximum partitions per source */
//#define frandom()	(rnd( prd.seed )/float(RAND_MAX))
//#define frandom()	(rnd( prd.seed ))

				/* specularity flags */
#define  SP_REFL	01		/* has reflected specular component */
#define  SP_TRAN	02		/* has transmitted specular */
#define  SP_PURE	04		/* purely specular (zero roughness) */
#define  SP_FLAT	010		/* flat reflecting surface */
#define  SP_RBLT	020		/* reflection below sample threshold */
#define  SP_TBLT	040		/* transmission below threshold */

typedef struct {
	unsigned int specfl;		/* specularity flags, defined above */
	float3 mcolor;		/* color of this material */
	float3 scolor;		/* color of specular component */
	//float3 vrefl;		/* vector in direction of reflected ray */
	float3 prdir;		/* vector in transmitted direction */
	float3 normal;
	float3 hit;
	float  alpha2;		/* roughness squared */
	float  rdiff, rspec;	/* reflected specular, diffuse */
	float  trans;		/* transmissivity */
	float  tdiff, tspec;	/* transmitted specular, diffuse */
	float3 pnorm;		/* perturbed surface normal */
	float  pdot;		/* perturbed dot product */
}  NORMDAT;		/* normal material data */

/* Context variables */
rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(unsigned int, shadow_ray_type, , );
rtDeclareVariable(unsigned int, ambient_ray_type, , );
rtDeclareVariable(rtObject,     top_object, , );
rtDeclareVariable(rtObject,     top_ambient, , );

rtDeclareVariable(float,        specthresh, , );	/* This is the minimum fraction of reflection or transmission, under which no specular sampling is performed */
rtDeclareVariable(float,        specjitter, , );	/* specular sampling (ss) */

#ifdef LIGHTS
rtDeclareVariable(float,        dstrsrc, , ); /* direct jitter (dj) */
rtDeclareVariable(float,        srcsizerat, , );	/* direct sampling ratio (ds) */
//rtDeclareVariable(float,        shadthresh, , );	/* direct threshold (dt) */
//rtDeclareVariable(float,        shadcert, , );	/* direct certainty (dc) */
//rtDeclareVariable(int,          directrelay, , );	/* direct relays for secondary sources (dr) */
//rtDeclareVariable(int,          vspretest, , );	/* direct presampling density for secondary sources (dp) */
#endif /* LIGHTS */

#ifdef AMBIENT
rtDeclareVariable(float3,       ambval, , );	/* This is the final value used in place of an indirect light calculation */
rtDeclareVariable(int,          ambvwt, , );	/* As new indirect irradiances are computed, they will modify the default ambient value in a moving average, with the specified weight assigned to the initial value given on the command and all other weights set to 1 */
rtDeclareVariable(int,          ambounce, , );	/* Ambient bounces (ab) */
//rtDeclareVariable(int,          ambres, , );	/* Ambient resolution (ar) */
rtDeclareVariable(float,        ambacc, , );	/* Ambient accuracy (aa). This value will approximately equal the error from indirect illuminance interpolation */
rtDeclareVariable(int,          ambdiv, , );	/* Ambient divisions (ad) */
rtDeclareVariable(int,          ambdiv_final, , ); /* Number of ambient divisions for final-pass fill (ag) */
rtDeclareVariable(int,          ambssamp, , );	/* Ambient super-samples (as) */
#ifdef OLDAMB
rtDeclareVariable(float,        maxarad, , );	/* maximum ambient radius */
rtDeclareVariable(float,        minarad, , );	/* minimum ambient radius */
#endif /* OLDAMB */
rtDeclareVariable(float,        avsum, , );		/* computed ambient value sum (log) */
rtDeclareVariable(unsigned int, navsum, , );	/* number of values in avsum */
#endif /* AMBIENT */

rtDeclareVariable(float,        minweight, , );	/* minimum ray weight (lw) */
rtDeclareVariable(int,          maxdepth, , );	/* maximum recursion depth (lr) */

rtBuffer<DistantLight> lights;

/* Program variables */
rtDeclareVariable(unsigned int, metal, , );	/* The material type representing "metal" */

/* Material variables */
rtDeclareVariable(unsigned int, type, , );	/* The material type representing "plastic", "metal", or "trans" */
rtDeclareVariable(float3,       color, , );	/* The material color given by the rad file "plastic", "metal", or "trans" object */
rtDeclareVariable(float,        spec, , );	/* The material specularity given by the rad file "plastic", "metal", or "trans" object */
rtDeclareVariable(float,        rough, , );	/* The material roughness given by the rad file "plastic", "metal", or "trans" object */
rtDeclareVariable(float,        transm, , ) = 0.0f;	/* The material transmissivity given by the rad file "trans" object */
rtDeclareVariable(float,        tspecu, , ) = 0.0f;	/* The material transmitted specular component given by the rad file "trans" object */

/* Geometry instance variables */
#ifdef LIGHTS
rtBuffer<float3> vertex_buffer;
rtBuffer<uint3>  lindex_buffer;    // position indices
#endif

/* OptiX variables */
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(PerRayData_radiance, prd, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow, rtPayload, );

/* Attributes */
//rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );


RT_METHOD float3 dirnorm(Ray *shadow_ray, PerRayData_shadow *shadow_prd, const NORMDAT *nd, const float& omega);
RT_METHOD float3 gaussamp(const NORMDAT *nd);
#ifdef AMBIENT
RT_METHOD float3 multambient(float3 aval, const float3& normal, const float3& pnormal, const float3& hit);
#ifndef OLDAMB
#ifdef DAYSIM_COMPATIBLE
RT_METHOD int doambient(float3 *rcol, const float3& normal, const float3& pnormal, const float3& hit, DaysimCoef dc);
#else
RT_METHOD int doambient(float3 *rcol, const float3& normal, const float3& pnormal, const float3& hit);
#endif
//RT_METHOD int ambsample( AMBHEMI *hp, const int& i, const int& j, const float3 normal, const float3 hit );
#else /* OLDAMB */
#ifdef DAYSIM_COMPATIBLE
RT_METHOD float doambient( float3 *rcol, const float3& normal, const float3& pnormal, const float3& hit, DaysimCoef dc );
RT_METHOD int divsample( AMBSAMP  *dp, AMBHEMI  *h, const float3& normal, const float3& hit, DaysimCoef dc );
#else
RT_METHOD float doambient( float3 *rcol, const float3& normal, const float3& pnormal, const float3& hit );
RT_METHOD int divsample( AMBSAMP  *dp, AMBHEMI  *h, const float3& normal, const float3& hit );
#endif
RT_METHOD void inithemi( AMBHEMI  *hp, float3 ac, const float3& normal );
//RT_METHOD void comperrs( AMBSAMP *da, AMBHEMI *hp );
//RT_METHOD int ambcmp( const void *p1, const void *p2 );
#endif /* OLDAMB */
#endif /* AMBIENT */
#ifdef LIGHTS
RT_METHOD unsigned int flatpart( const float3& v, const float3& r0, const float3& r1, const float3& r2 );
RT_METHOD float solid_angle( const float3& r0, const float3& r1, const float3& r2 );
RT_METHOD float3 barycentric( float2& lambda, const float3& r0, const float3& r1, const float3& r2, const int flip );
#endif /* LIGHTS */
//RT_METHOD float2 multisamp2(float r);
//RT_METHOD int ilhash(int3 d);


#ifndef LIGHTS
RT_PROGRAM void any_hit_shadow()
{
	// this material is opaque, so it fully attenuates all shadow rays
	prd_shadow.result = make_float3(0.0f);
	rtTerminateRay();
}
#endif


RT_PROGRAM void closest_hit_radiance()
{
	NORMDAT nd;

	float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
	float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );

	/* check for back side */
	nd.pnorm = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );
	nd.normal = faceforward( world_geometric_normal, -ray.direction, world_geometric_normal );

	PerRayData_radiance new_prd;
	float3 result = make_float3( 0.0f );
	nd.hit = ray.origin + t_hit * ray.direction;
	nd.mcolor = color;
	nd.scolor = make_float3(0.0f);
	nd.rspec = spec;
	nd.alpha2 = rough * rough;
	nd.specfl = 0u; /* specularity flags */

	/* get roughness */
	if (nd.alpha2 <= FTINY) {
		nd.specfl |= SP_PURE; // label this as a purely specular reflection
	}

	/* perturb normal */
	float3 pert = nd.normal - nd.pnorm;
	int hastexture = dot(pert, pert) > FTINY * FTINY;
	nd.pdot = -dot(ray.direction, nd.pnorm);
	if (nd.pdot < 0.0f) {		/* fix orientation from raynormal in raytrace.c */
		nd.pnorm += 2.0f * nd.pdot * ray.direction;
		nd.pdot = -nd.pdot;
	}
	if (nd.pdot < 0.001f)
		nd.pdot = 0.001f;			/* non-zero for dirnorm() */

	// if it's a face or a ring label as flat (currently we only support triangles, so everything is flat)
	nd.specfl |= SP_FLAT;

	/* modify material color */
	//nd.mcolor *= rtTex3D(rtTextureId id, texcoord.x, texcoord.y, texcoord.z).xyz;

	/* compute Fresnel approx. */
	float fest = 0.0f;
	if (nd.specfl & SP_PURE && nd.rspec >= FRESTHRESH) {
		fest = FRESNE(nd.pdot);
		nd.rspec += fest * (1.0f - nd.rspec);
	}

	/* compute transmission */
	nd.tdiff = nd.tspec = nd.trans = 0.0f; // because it's opaque
#ifdef TRANSMISSION
	float transtest = 0.0f, transdist = t_hit;
	nd.prdir = ray.direction;
	if (transm > 0.0f) { // type == MAT_TRANS
		nd.trans = transm * (1.0f - nd.rspec);
		nd.tspec = nd.trans * tspecu;
		nd.tdiff = nd.trans - nd.tspec;
		if (nd.tspec > FTINY) {
			nd.specfl |= SP_TRAN;

							/* check threshold */
			if (!(nd.specfl & SP_PURE) && specthresh >= nd.tspec - FTINY)
				nd.specfl |= SP_TBLT;
			if (prd.ambient_depth || !hastexture) {
				transtest = 2.0f;
			} else {
				if (dot(nd.prdir - pert, nd.normal) < -FTINY)
					nd.prdir = normalize(nd.prdir - pert);	/* OK */
			}
		}
	}

	/* transmitted ray */
	if ((nd.specfl&(SP_TRAN | SP_PURE | SP_TBLT)) == (SP_TRAN | SP_PURE)) {
		new_prd.weight = prd.weight * fmaxf(nd.mcolor) * nd.tspec;
		if (new_prd.weight >= minweight) {
			new_prd.depth = prd.depth;
			new_prd.ambient_depth = prd.ambient_depth;
			new_prd.state = prd.state;
#ifdef DAYSIM_COMPATIBLE
			new_prd.dc = daysimNext(prd.dc);
#endif
			setupPayload(new_prd, 0);
			Ray trans_ray = make_Ray(nd.hit, nd.prdir, radiance_ray_type, ray_start(nd.hit, nd.prdir, nd.normal, RAY_START), RAY_END);
			rtTrace(top_object, trans_ray, new_prd);
			float3 rcol = new_prd.result * nd.mcolor * nd.tspec;
			result += rcol;
#ifdef DAYSIM_COMPATIBLE
			daysimAddScaled(prd.dc, new_prd.dc, nd.mcolor.x * nd.tspec);
#endif
			transtest *= bright(rcol);
			transdist = t_hit + new_prd.distance;
			resolvePayload(prd, new_prd);
		}
	}
	else
		transtest = 0.0f;
#endif

	// return if it's a shadow ray, which it isn't

	/* get specular reflection */
	if (nd.rspec > FTINY) {
		nd.specfl |= SP_REFL;

		/* compute specular color */
		if (type != metal) {
			nd.scolor = make_float3(nd.rspec);
		} else {
			if (fest > FTINY) {
				float d = spec * (1.0f - fest);
				nd.scolor = fest + nd.mcolor * d;
			} else {
				nd.scolor = nd.mcolor * nd.rspec;
			}
		}

		/* check threshold */
		if (!(nd.specfl & SP_PURE) && specthresh >= nd.rspec - FTINY) {
			nd.specfl |= SP_RBLT;
		}
	}

	/* reflected ray */
	float mirtest = 0.0f, mirdist = t_hit;
	if ((nd.specfl&(SP_REFL | SP_PURE | SP_RBLT)) == (SP_REFL | SP_PURE)) {
		new_prd.weight = prd.weight * fmaxf(nd.scolor);
		new_prd.depth = prd.depth + 1;
		if (new_prd.weight >= minweight && new_prd.depth <= abs(maxdepth)) {
			new_prd.ambient_depth = prd.ambient_depth;
			new_prd.state = prd.state;
#ifdef DAYSIM_COMPATIBLE
			new_prd.dc = daysimNext(prd.dc);
#endif
			setupPayload(new_prd, 0);
			float3 vrefl = reflect(ray.direction, nd.pnorm);
			Ray refl_ray = make_Ray(nd.hit, vrefl, radiance_ray_type, ray_start(nd.hit, vrefl, nd.normal, RAY_START), RAY_END);
			rtTrace(top_object, refl_ray, new_prd);
			float3 rcol = new_prd.result * nd.scolor;
			result += rcol;
#ifdef DAYSIM_COMPATIBLE
			daysimAddScaled(prd.dc, new_prd.dc, nd.scolor.x);
#endif
			if (nd.specfl & SP_FLAT && (prd.ambient_depth || !hastexture)) {
				mirtest = 2.0f * bright(rcol);
				mirdist = t_hit + new_prd.distance;
			}
			resolvePayload(prd, new_prd);
		}
	}

	/* diffuse reflection */
	nd.rdiff = 1.0f - nd.trans - nd.rspec;

	if (!(nd.specfl & SP_PURE && nd.rdiff <= FTINY && nd.tdiff <= FTINY)) { /* not 100% pure specular */

		/* checks *BLT flags */
		if (!(nd.specfl & SP_PURE))
			result += gaussamp(&nd);

#ifdef AMBIENT
		/* ambient from this side */
		if (nd.rdiff > FTINY) {
			float3 aval = nd.mcolor * nd.rdiff;	/* modified by material color */
			if (nd.specfl & SP_RBLT)	/* add in specular as well? */
				aval += nd.scolor;
			result += multambient(aval, nd.normal, nd.pnorm, nd.hit);	/* add to returned color */
		}

#ifdef TRANSMISSION
		/* ambient from other side */
		if (nd.tdiff > FTINY) {
			float3 aval = nd.mcolor;	/* modified by material color */
			if (nd.specfl & SP_TBLT)
				aval *= nd.trans;
			else
				aval *= nd.tdiff;
			result += multambient(aval, -nd.normal, -nd.pnorm, nd.hit);	/* add to returned color */
		}
#endif /* TRANSMISSION */
#endif /* AMBIENT */

		/* add direct component */
		// This is the call to direct() in source.c
		// Let's start at line 447, and not bother with sorting for now

		// compute direct lighting
		PerRayData_shadow shadow_prd;
#ifdef DAYSIM_COMPATIBLE
		shadow_prd.dc = daysimNext(prd.dc);
#endif
		Ray shadow_ray = make_Ray(nd.hit, nd.pnorm, shadow_ray_type, RAY_START, RAY_END);

		/* contributions from distant lights (mainly the sun) */
		unsigned int num_lights = lights.size();
		for (unsigned int i = 0u; i < num_lights; i++) {
			const DistantLight light = lights[i];
			if ( light.casts_shadow ) {
				shadow_prd.target = i;
				shadow_ray.direction = normalize( light.pos );
				shadow_ray.tmin = ray_start(nd.hit, shadow_ray.direction, nd.normal, RAY_START);
				shadow_ray.tmax = RAY_END;
				result += dirnorm(&shadow_ray, &shadow_prd, &nd, light.solid_angle);
			}
		}

#ifdef LIGHTS
		/* contributions from nearby lights */
		num_lights = lindex_buffer.size();
		for (unsigned int i = 0u; i < num_lights; i++) {
			const uint3 v_idx = lindex_buffer[i];

			const float3 r0 = vertex_buffer[v_idx.x] - nd.hit;
			const float3 r1 = vertex_buffer[v_idx.y] - nd.hit;
			const float3 r2 = vertex_buffer[v_idx.z] - nd.hit;
			float3 rdir = ( r0 + r1 + r2 ) / 3.0f;

			const unsigned int divs = flatpart( rdir, r0, r1, r2 ); //TODO divisions should be smaller closer to the light source
			const float step = 1.0f / divs;

			for ( int j = 0; j < divs; j++ )
				for ( int k = 0; k < divs; k++ ) {
					float2 lambda = make_float2( step * j, step * k );
					const float3 p0 = barycentric( lambda, r0, r1, r2, k + j >= divs );

					lambda = make_float2( step * ( j + 1 ), step * k );
					const float3 p1 = barycentric( lambda, r0, r1, r2, k + j >= divs );

					lambda = make_float2( step * j, step * ( k + 1 ) );
					const float3 p2 = barycentric( lambda, r0, r1, r2, k + j >= divs );

					const float omega = solid_angle( p0, p1, p2 );

					if ( omega > FTINY ) {
						/* from nextssamp in srcsamp.c */
						rdir = ( p0 + p1 + p2 ) / 3.0f;
						if ( dstrsrc > FTINY ) {
							/* jitter sample using random barycentric coordinates */
							lambda = make_float2( hiprand_uniform( prd.state ), hiprand_uniform( prd.state ) );
							float3 vpos = barycentric( lambda, p0, p1, p2, lambda.x + lambda.y >= 1.0f );
							rdir += dstrsrc * ( vpos - rdir );
						}

						shadow_prd.target = -v_idx.x - 1; //TODO find a better way to identify surface
						shadow_ray.direction = normalize( rdir );
						shadow_ray.tmin = ray_start(nd.hit, shadow_ray.direction, nd.normal, RAY_START);
						shadow_ray.tmax = length(rdir) * 1.0001f;
						result += dirnorm(&shadow_ray, &shadow_prd, &nd, omega);
					}
				}
		}
#endif /* LIGHTS */
	}

	/* check distance */
	float d = bright( result );
#ifdef TRANSMISSION
	if (transtest > d)
		prd.distance = transdist;
	else
#endif
	if (mirtest > d)
		prd.distance = mirdist;
	else
		prd.distance = t_hit;

	// pass the color back up the tree
	prd.result = result;

#ifdef HIT_TYPE
	prd.hit_type = type;
#endif
}

/* compute source contribution */
RT_METHOD float3 dirnorm(Ray *shadow_ray, PerRayData_shadow *shadow_prd, const NORMDAT *nd, const float& omega)
{
	float3 cval = make_float3( 0.0f );
	float ldot = dot(nd->pnorm, shadow_ray->direction);

#ifdef TRANSMISSION
	if (ldot < 0.0f ? nd->trans <= FTINY : nd->trans >= 1.0f - FTINY)
#else
	if ( ldot <= FTINY )
#endif
		return cval;

	// cast shadow ray
	shadow_prd->result = make_float3( 0.0f );
#ifdef DAYSIM_COMPATIBLE
	daysimSet(shadow_prd->dc, 0.0f);
#endif
	rtTrace( top_object, *shadow_ray, *shadow_prd );
	if( fmaxf( shadow_prd->result ) <= 0.0f )
		return cval;
	
	/* Fresnel estimate */
	float lrdiff = nd->rdiff;
	float ltdiff = nd->tdiff;
	if (nd->specfl & SP_PURE && nd->rspec >= FRESTHRESH && (lrdiff > FTINY) | (ltdiff > FTINY)) {
		float dtmp = 1.0f - FRESNE(fabs(ldot));
		lrdiff *= dtmp;
		ltdiff *= dtmp;
	}

	if (ldot > FTINY && lrdiff > FTINY) {
		/*
		 *  Compute and add diffuse reflected component to returned
		 *  color.  The diffuse reflected component will always be
		 *  modified by the color of the material.
		 */
		float dtmp = ldot * omega * lrdiff * M_1_PIf;
		cval += nd->mcolor * dtmp;
	}
#ifdef TRANSMISSION
	if (ldot < -FTINY && ltdiff > FTINY) {
		/*
		 *  Compute diffuse transmission.
		 */
		float dtmp = -ldot * omega * ltdiff * M_1_PIf;
		cval += nd->mcolor * dtmp;
	}
#endif
	if (ldot > FTINY && (nd->specfl&(SP_REFL | SP_PURE)) == SP_REFL) {
		/*
		 *  Compute specular reflection coefficient using
		 *  Gaussian distribution model.
		 */
		/* roughness */
		float dtmp = nd->alpha2;
		/* + source if flat */
		if (nd->specfl & SP_FLAT)
			dtmp += omega * 0.25f * M_1_PIf;
		/* half vector */
		float3 vtmp = shadow_ray->direction - ray.direction;
		float d2 = dot(vtmp, nd->pnorm);
		d2 *= d2;
		float d3 = dot( vtmp, vtmp );
		float d4 = (d3 - d2) / d2;
		/* new W-G-M-D model */
		dtmp = expf(-d4/dtmp) * d3 / (M_PIf * d2*d2 * dtmp);
		/* worth using? */
		if (dtmp > FTINY) {
			dtmp *= ldot * omega;
			cval += nd->scolor * dtmp;
		}
	}
#ifdef TRANSMISSION
	if (ldot < -FTINY && (nd->specfl&(SP_TRAN | SP_PURE)) == SP_TRAN) {
		/*
		 *  Compute specular transmission.  Specular transmission
		 *  is always modified by material color.
		 */
						/* roughness + source */
		float dtmp = nd->alpha2 + omega * M_1_PIf;
						/* Gaussian */
		dtmp = expf((2.0f * dot(nd->prdir, shadow_ray->direction) - 2.0f) / dtmp) / (M_PIf * dtmp); // may need to perturb direction
						/* worth using? */
		if (dtmp > FTINY) {
			dtmp *= nd->tspec * omega * sqrtf(-ldot / nd->pdot);
			cval += nd->mcolor * dtmp;
		}
	}
#endif
#ifdef DAYSIM_COMPATIBLE
	daysimAddScaled(prd.dc, shadow_prd->dc, cval.x);
#endif
	return cval * shadow_prd->result;
}

// sample Gaussian specular
RT_METHOD float3 gaussamp(const NORMDAT *nd)
{
	float3 rcol = make_float3( 0.0f );

	/* This section is based on the gaussamp method in normal.c */
	if ((nd->specfl & (SP_REFL | SP_RBLT)) != SP_REFL && (nd->specfl & (SP_TRAN | SP_TBLT)) != SP_TRAN)
		return rcol;

	PerRayData_radiance gaus_prd;
	gaus_prd.depth = prd.depth + 1;
	if ( gaus_prd.depth > abs(maxdepth) )
		return rcol;
	gaus_prd.ambient_depth = prd.ambient_depth + 1; //TODO the increment is a hack to prevent the sun from affecting specular values
	//gaus_prd.seed = prd.seed;//lcg( prd.seed );
	gaus_prd.state = prd.state;
	Ray gaus_ray = make_Ray(nd->hit, nd->pnorm, radiance_ray_type, RAY_START, RAY_END);

	float d;

	/* set up sample coordinates */
	float3 u = getperpendicular(nd->pnorm); // prd.state?
	float3 v = cross(nd->pnorm, u);

	unsigned int nstarget, nstaken, ntrials;

	/* compute reflection */
	gaus_prd.weight = prd.weight * fmaxf(nd->scolor);
	if ((nd->specfl & (SP_REFL | SP_RBLT)) == SP_REFL && gaus_prd.weight >= minweight) {
		float3 scolor = nd->scolor;
		nstarget = 1;
		if (specjitter > 1.5f) {	/* multiple samples? */ // By default it's 1.0
			nstarget = specjitter * prd.weight + 0.5f;
			if ( gaus_prd.weight <= minweight * nstarget )
				nstarget = gaus_prd.weight / minweight;
			if ( nstarget > 1 ) {
				d = 1.0f / nstarget;
				scolor *= d; //scolor, stored as ray rcoef
				gaus_prd.weight *= d; // TODO make sure weight isn't changed by hit programs
			} else
				nstarget = 1;
		}
		float3 scol = make_float3( 0.0f );
#ifdef DAYSIM_COMPATIBLE
		DaysimCoef dc = daysimNext(prd.dc);
		if (nstarget > 1) {
			daysimSet(dc, 0.0f);
			gaus_prd.dc = daysimNext(dc);
		} else
			gaus_prd.dc = dc;
#endif
		//dimlist[ndims++] = (int)(size_t)np->mp;
		unsigned int maxiter = MAXITER * nstarget;
		for (nstaken = ntrials = 0; nstaken < nstarget && ntrials < maxiter; ntrials++) {
			float2 rv = make_float2( hiprand_uniform( prd.state ), hiprand_uniform( prd.state ) ); // should be evenly distributed in both dimensions
			d = 2.0f * M_PIf * rv.x;
			float cosp = cosf( d );
			float sinp = sinf( d );
			if ( ( 0.0f <= specjitter ) && ( specjitter < 1.0f ) )
				rv.y = 1.0f - specjitter * rv.y;
			if ( rv.y <= FTINY )
				d = 1.0f;
			else
				d = sqrtf(nd->alpha2 * -logf(rv.y));
			float3 h = nd->pnorm + d * (cosp * u + sinp * v);
			d = -2.0f * dot( h, ray.direction ) / ( 1.0f + d*d );
			gaus_ray.direction = ray.direction + h * d;

			/* sample rejection test */
			if ((d = dot(gaus_ray.direction, nd->normal)) <= FTINY)
				continue;

			gaus_ray.direction = normalize( gaus_ray.direction );
			gaus_ray.tmin = ray_start(nd->hit, gaus_ray.direction, nd->normal, RAY_START);
			setupPayload(gaus_prd, 0);
			//if (nstaken) // check for prd data that needs to be cleared
			rtTrace(top_object, gaus_ray, gaus_prd);
			resolvePayload(prd, gaus_prd);

			/* W-G-M-D adjustment */
			if (nstarget > 1) {	
				d = 2.0f / (1.0f - dot(ray.direction, nd->normal) / d);
				scol += gaus_prd.result * d;
#ifdef DAYSIM_COMPATIBLE
				daysimAddScaled(dc, gaus_prd.dc, d);
#endif
			} else {
				rcol += gaus_prd.result * scolor;
#ifdef DAYSIM_COMPATIBLE
				daysimAddScaled(prd.dc, gaus_prd.dc, scolor.x);
#endif
			}

			++nstaken;
		}
		/* final W-G-M-D weighting */
		if (nstarget > 1) {
			scol *= scolor;
			d = (float)nstarget / ntrials;
			rcol += scol * d;
#ifdef DAYSIM_COMPATIBLE
			daysimAddScaled(prd.dc, dc, scolor.x * d);
#endif
		}
		//ndims--;
	}

#ifdef TRANSMISSION
	/* compute transmission */
	float3 mcolor = nd->mcolor * nd->tspec;	/* modified by color */
	gaus_prd.weight = prd.weight * fmaxf(mcolor);
	gaus_prd.ambient_depth = prd.ambient_depth;
	if ((nd->specfl & (SP_TRAN | SP_TBLT)) == SP_TRAN && gaus_prd.weight >= minweight) {
		nstarget = 1;
		if (specjitter > 1.5f) {	/* multiple samples? */ // By default it's 1.0
			nstarget = specjitter * prd.weight + 0.5f;
			if ( gaus_prd.weight <= minweight * nstarget )
				nstarget = gaus_prd.weight / minweight;
			if ( nstarget > 1 ) {
				d = 1.0f / nstarget;
				mcolor *= d; //mcolor, stored as ray rcoef
				gaus_prd.weight *= d; // TODO make sure weight isn't changed by hit programs
			} else
				nstarget = 1;
		}
		//dimlist[ndims++] = (int)(size_t)np->mp;
		unsigned int maxiter = MAXITER * nstarget;
		for (nstaken = ntrials = 0; nstaken < nstarget && ntrials < maxiter; ntrials++) {
			float2 rv = make_float2( hiprand_uniform( prd.state ), hiprand_uniform( prd.state ) ); // should be evenly distributed in both dimensions
			d = 2.0f * M_PIf * rv.x;
			float cosp = cosf( d );
			float sinp = sinf( d );
			if ( ( 0.0f <= specjitter ) && ( specjitter < 1.0f ) )
				rv.y = 1.0f - specjitter * rv.y;
			if ( rv.y <= FTINY )
				d = 1.0f;
			else
				d = sqrtf(nd->alpha2 * -logf(rv.y));
			gaus_ray.direction = nd->prdir + d * (cosp * u + sinp * v); // ray direction is perturbed

			/* sample rejection test */
			if (dot(gaus_ray.direction, nd->normal) >= -FTINY)
				continue;

			gaus_ray.direction = normalize( gaus_ray.direction );
			gaus_ray.tmin = ray_start(nd->hit, gaus_ray.direction, nd->normal, RAY_START);
#ifdef DAYSIM_COMPATIBLE
			gaus_prd.dc = daysimNext(prd.dc);
#endif
			setupPayload(gaus_prd, 0);
			//if (nstaken) // check for prd data that needs to be cleared
			rtTrace(top_object, gaus_ray, gaus_prd);
			resolvePayload(prd, gaus_prd);
			rcol += gaus_prd.result * mcolor;
			++nstaken;
#ifdef DAYSIM_COMPATIBLE
			daysimAddScaled(prd.dc, gaus_prd.dc, mcolor.x);
#endif
		}
		//ndims--;
	}
#endif
	//return make_float3(0.0f);
	return rcol;
}

#ifdef AMBIENT
// Compute the ambient component and multiply by the coefficient.
RT_METHOD float3 multambient(float3 aval, const float3& normal, const float3& pnormal, const float3& hit)
{
	int do_ambient = 1;
	float 	d;

	if (ambdiv <= 0)			/* no ambient calculation */
		goto dumbamb;
						/* check number of bounces */
	if (prd.ambient_depth >= ambounce)
		goto dumbamb;
						/* check ambient list */
	//if (ambincl != -1 && r->ro != NULL && ambincl != inset(ambset, r->ro->omod))
	//	goto dumbamb;

#ifdef ITERATIVE
	if (!prd.ambient_depth)
		return make_float3(0.0f);
#else /* ITERATIVE */
	if (ambacc > FTINY && navsum != 0) {			/* ambient storage */
		//if (tracktime)				/* sort to minimize thrashing */
		//	sortambvals(0);

		/* interpolate ambient value */
		//acol = make_float3( 0.0f );
		//d = sumambient(acol, r, normal, rdepth, &atrunk, thescene.cuorg, thescene.cusize);
		PerRayData_ambient ambient_prd;
		ambient_prd.result = make_float3( 0.0f );
		ambient_prd.surface_normal = pnormal;
		ambient_prd.ambient_depth = prd.ambient_depth;
		ambient_prd.wsum = 0.0f;
		ambient_prd.weight = prd.weight;
#ifdef OLDAMB
		ambient_prd.state = prd.state;
#endif
#ifdef DAYSIM_COMPATIBLE
		ambient_prd.dc = daysimNext(prd.dc);
		daysimSet(ambient_prd.dc, 0.0f);
#endif
#ifdef HIT_COUNT
		ambient_prd.hit_count = 0;
#endif
		const float tmin = ray_start( hit, AMBIENT_RAY_LENGTH );
		Ray ambient_ray = make_Ray( hit, normal, ambient_ray_type, -tmin, tmin );
		rtTrace(top_ambient, ambient_ray, ambient_prd);
#ifdef HIT_COUNT
		prd.hit_count += ambient_prd.hit_count;
#endif
		if (ambient_prd.wsum > FTINY) { // TODO if miss program is called, set wsum = 1.0f or place this before ambacc == 0.0f
			ambient_prd.result *= 1.0f / ambient_prd.wsum;
#ifdef DAYSIM_COMPATIBLE
			daysimAddScaled(prd.dc, ambient_prd.dc, aval.x / ambient_prd.wsum);
#endif
			return aval * ambient_prd.result;
		}
		//rdepth++;				/* need to cache new value */
		//d = makeambient(acol, r, normal, rdepth-1); //TODO implement as miss program for ambient ray
		//rdepth--;
		//if ( dot( ambient_prd.result, ambient_prd.result) > FTINY) { // quick check to see if a value was returned by miss program
		//	return aval * ambient_prd.result;		/* got new value */
		//}

#ifdef FILL_GAPS
		do_ambient = prd.primary && ambdiv_final;
#else
		do_ambient = !prd.ambient_depth && ambdiv_final;
#endif
	}
#endif /* ITERATIVE */
	if (do_ambient) {			/* no ambient storage */
		/* Option to show error if nothing found */
		if (ambdiv_final < 0)
			rtThrow(RT_EXCEPTION_USER - ambdiv_final);

		float3 acol = aval;
#ifdef DAYSIM_COMPATIBLE
		DaysimCoef dc = daysimNext(prd.dc);
		daysimSet(dc, 0.0f);
		d = doambient(&acol, normal, pnormal, hit, dc);
		if (d > FTINY)
			daysimAdd(prd.dc, dc);
#else
		d = doambient(&acol, normal, pnormal, hit);
#endif
		if (d > FTINY)
			return acol;
	}
dumbamb:					/* return global value */
	if ((ambvwt <= 0) || (navsum == 0)) {
#ifdef DAYSIM_COMPATIBLE
		daysimAdd(prd.dc, aval.x * ambval.x);
#endif
		return aval * ambval;
	}
	float l = bright(ambval);			/* average in computations */
	if (l > FTINY) {
		d = (logf(l)*(float)ambvwt + avsum) / (float)(ambvwt + navsum);
		d = expf(d) / l;
		aval *= ambval;	/* apply color of ambval */
#ifdef DAYSIM_COMPATIBLE
		daysimAdd(prd.dc, aval.x * ambval.x * d);
#endif
	} else {
		d = expf( avsum / (float)navsum );
#ifdef DAYSIM_COMPATIBLE
		daysimAdd(prd.dc, aval.x * d);
#endif
	}
	return aval * d;
}

#ifndef OLDAMB
/* sample indirect hemisphere, based on samp_hemi in ambcomp.c */
#ifdef DAYSIM_COMPATIBLE
RT_METHOD int doambient(float3 *rcol, const float3& normal, const float3& pnormal, const float3& hit, DaysimCoef dc)
#else
RT_METHOD int doambient(float3 *rcol, const float3& normal, const float3& pnormal, const float3& hit)
#endif
{
	float	d;
	float wt = prd.weight;

					/* set number of divisions */
	if (ambacc <= FTINY && wt > (d = 0.8f * fmaxf(*rcol) * wt / (ambdiv_final * minweight)))
		wt = d;			/* avoid ray termination */
#ifdef ITERATIVE
	int i, n = 1;
#else /* ITERATIVE */
	int n = sqrtf(ambdiv_final * wt) + 0.5f;
	int i = 1 + 8 * (ambacc > FTINY);	/* minimum number of samples */
	if (n < i)
		n = i;
#endif /* ITERATIVE */
	const int nn = n * n;
	float3 acol = make_float3( 0.0f );
	unsigned int sampOK = 0u;
					/* assign coefficient */
	float3 acoef = *rcol / nn;

	/* Setup from ambsample in ambcomp.c */
	PerRayData_radiance new_prd;
					/* generate hemispherical sample */
					/* ambient coefficient for weight */
	if (ambacc > FTINY)
		d = AVGREFL; // Reusing this variable
	else
		d = fmaxf( acoef );
	new_prd.weight = prd.weight * d;
	if (new_prd.weight < minweight) //if (rayorigin(&ar, AMBIENT, r, ar.rcoef) < 0)
		return(0);

	new_prd.depth = prd.depth + 1;
	new_prd.ambient_depth = prd.ambient_depth + 1;
	//new_prd.seed = prd.seed;//lcg( prd.seed );
	new_prd.state = prd.state;
#ifdef DAYSIM_COMPATIBLE
	new_prd.dc = daysimNext(dc);
#endif

	Ray amb_ray = make_Ray( hit, pnormal, radiance_ray_type, RAY_START, RAY_END ); // Use normal point as temporary direction
	/* End ambsample setup */

					/* make tangent plane axes */
	float3 ux = getperpendicular( pnormal, prd.state );
	float3 uy = cross( pnormal, ux );
					/* sample divisions */
	for (i = n; i--; )
	    for (int j = n; j--; ) {
			//hp.sampOK += ambsample( &hp, i, j, normal, hit );
			/* ambsample in ambcomp.c */
#ifdef ITERATIVE
			float2 spt = 0.01f + 0.98f * make_float2(hiprand_uniform(prd.state), hiprand_uniform(prd.state));
#else /* ITERATIVE */
			float2 spt = 0.1f + 0.8f * make_float2( hiprand_uniform( prd.state ), hiprand_uniform( prd.state ) );
#endif /* ITERATIVE */
			SDsquare2disk( spt, (j+spt.y) / n, (i+spt.x) / n );
			float zd = sqrtf( 1.0f - dot( spt, spt ) );
			amb_ray.direction = normalize( spt.x*ux + spt.y*uy + zd*pnormal );
			if (dot(amb_ray.direction, normal) <= 0) /* Prevent light leaks */
				continue;
			amb_ray.tmin = ray_start( hit, amb_ray.direction, normal, RAY_START );
			//dimlist[ndims++] = AI(hp,i,j) + 90171;

			setupPayload(new_prd, 0);
			//Ray amb_ray = make_Ray( hit, rdir, radiance_ray_type, RAY_START, RAY_END );
			rtTrace(top_object, amb_ray, new_prd);
			resolvePayload(prd, new_prd);

			//ndims--;
			if ( isnan( new_prd.result ) ) // TODO How does this happen?
				continue;
			if ( new_prd.distance <= FTINY )
				continue;		/* should never happen */
			acol += new_prd.result * acoef;	/* add to our sum */
#ifdef DAYSIM_COMPATIBLE
			daysimAddScaled(dc, new_prd.dc, acoef.x);
#endif
			sampOK++;
		}
	*rcol = acol;
	if ( !sampOK ) {		/* utter failure? */
		return( 0 );
	}
	if ( sampOK < nn ) {
		//hp.sampOK *= -1;	/* soft failure */
		return( 1 );
	}
	//n = ambssamp * wt + 0.5f;
	//if (n > 8) {			/* perform super-sampling? */
	//	ambsupersamp(hp, n);
	//	*rcol = hp.acol;
	//}
	return( 1 );			/* all is well */
}
#else /* OLDAMB */
#ifdef DAYSIM_COMPATIBLE
RT_METHOD float doambient( float3 *rcol, const float3& normal, const float3& pnormal, const float3& hit, DaysimCoef dc )
#else
RT_METHOD float doambient( float3 *rcol, const float3& normal, const float3& pnormal, const float3& hit )
#endif
{
	float  b;//, d;
	AMBHEMI  hemi;
	AMBSAMP  *div;
	AMBSAMP  dnew;
	float3  acol;
	AMBSAMP  *dp;
	float  arad;
	int  divcnt;
	int  i, j;
					/* initialize hemisphere */
	inithemi(&hemi, *rcol, pnormal);
	divcnt = hemi.nt * hemi.np;
					/* initialize */
	//if (pg != NULL)
	//	pg[0] = pg[1] = pg[2] = 0.0;
	//if (dg != NULL)
	//	dg[0] = dg[1] = dg[2] = 0.0;
	*rcol = make_float3( 0.0f );
	if (divcnt == 0)
		return(0.0f); //TODO does this change the value of rcol in the calling method?
					/* allocate super-samples */
	//if (hemi.ns > 0) {// || pg != NULL || dg != NULL) {
	//	div = (AMBSAMP *)malloc(divcnt*sizeof(AMBSAMP));
	//	//if (div == NULL) // This is 0
	//	//	error(SYSTEM, "out of memory in doambient");
	//} else
		div = NULL; // This is 0
					/* sample the divisions */
	arad = 0.0f;
	acol = make_float3( 0.0f );
	if ((dp = div) == NULL)
		dp = &dnew;
	divcnt = 0;
	for (i = 0; i < hemi.nt; i++)
		for (j = 0; j < hemi.np; j++) {
			dp->t = i; dp->p = j;
			dp->v = make_float3( 0.0f );
			dp->r = 0.0f;
			dp->n = 0;
#ifdef DAYSIM_COMPATIBLE
			if (divsample( dp, &hemi, normal, hit, dc ) < 0) {
#else
			if (divsample( dp, &hemi, normal, hit ) < 0) {
#endif
				if (div != NULL)
					dp++;
				continue;
			}
			arad += dp->r;
			divcnt++;
			if (div != NULL)
				dp++;
			else
				acol += dp->v;
		}
	if (!divcnt) {
		//if (div != NULL)
		//	free((void *)div);
		return(0.0f);		/* no samples taken */
	}
	if (divcnt < hemi.nt*hemi.np) {
		//pg = dg = NULL;		/* incomplete sampling */
		hemi.ns = 0;
	} else if (arad > FTINY && divcnt/arad < minarad) {
		hemi.ns = 0;		/* close enough */
	} else if (hemi.ns > 0) {	/* else perform super-sampling? */
		//comperrs(div, &hemi);			/* compute errors */
		//qsort(div, divcnt, sizeof(AMBSAMP), ambcmp);	/* sort divs */ TODO necessary?
						/* super-sample */
		for (i = hemi.ns; i > 0; i--) {
			dnew = *div;
#ifdef DAYSIM_COMPATIBLE
			if (divsample( &dnew, &hemi, normal, hit, dc ) < 0) {
#else
			if (divsample( &dnew, &hemi, normal, hit ) < 0) {
#endif
				dp++;
				continue;
			}
			dp = div;		/* reinsert */
			j = divcnt < i ? divcnt : i;
			while (--j > 0 && dnew.k < dp[1].k) {
				*dp = *(dp+1);
				dp++;
			}
			*dp = dnew;
		}
		//if (pg != NULL || dg != NULL)	/* restore order */
		//	qsort(div, divcnt, sizeof(AMBSAMP), ambnorm);
	}
					/* compute returned values */
	if (div != NULL) {
		arad = 0.0f;		/* note: divcnt may be < nt*np */
		for (i = hemi.nt*hemi.np, dp = div; i-- > 0; dp++) {
			arad += dp->r;
			if (dp->n > 1) {
				b = 1.0f/dp->n;
				dp->v *= b;
				dp->r *= b;
				dp->n = 1;
			}
			acol += dp->v;
		}
		//b = bright(acol);
		//if (b > FTINY) {
		//	b = 1.0f/b;	/* compute & normalize gradient(s) */
		//	//if (pg != NULL) {
		//	//	posgradient(pg, div, &hemi);
		//	//	for (i = 0; i < 3; i++)
		//	//		pg[i] *= b;
		//	//}
		//	//if (dg != NULL) {
		//	//	dirgradient(dg, div, &hemi);
		//	//	for (i = 0; i < 3; i++)
		//	//		dg[i] *= b;
		//	//}
		//}
		//free((void *)div);
	}
	*rcol = acol;
	if (arad <= FTINY)
		arad = maxarad;
	else
		arad = (divcnt+hemi.ns)/arad;
	//if (pg != NULL) {		/* reduce radius if gradient large */
	//	d = DOT(pg,pg);
	//	if (d*arad*arad > 1.0f)
	//		arad = 1.0f/sqrtf(d);
	//}
	if (arad < minarad) {
		arad = minarad;
		//if (pg != NULL && d*arad*arad > 1.0f) {	/* cap gradient */
		//	d = 1.0f/arad/sqrtf(d);
		//	for (i = 0; i < 3; i++)
		//		pg[i] *= d;
		//}
	}
	if ((arad /= sqrtf(prd.weight)) > maxarad)
		arad = maxarad;
	return(arad);
}

/* initialize sampling hemisphere */
RT_METHOD void inithemi( AMBHEMI  *hp, float3 ac, const float3& normal )
{
	float	d;
	int  i;
	float wt = prd.weight;
					/* set number of divisions */
	if (ambacc <= FTINY && wt > (d = 0.8f * fmaxf(ac) * wt / (ambdiv_final * minweight)))
		wt = d;			/* avoid ray termination */
	hp->nt = sqrtf(ambdiv_final * wt * M_1_PIf) + 0.5f;
	i = ambacc > FTINY ? 3 : 1;	/* minimum number of samples */
	if (hp->nt < i)
		hp->nt = i;
	hp->np = M_PIf * hp->nt + 0.5f;
					/* set number of super-samples */
	hp->ns = ambssamp * wt + 0.5f;
					/* assign coefficient */
	hp->acoef = ac;
	d = 1.0f/(hp->nt*hp->np);
	hp->acoef *= d;
					/* make axes */
	hp->uz = normal;
	hp->ux = getperpendicular(hp->uz);
	hp->uy = cross(hp->uz, hp->ux);
}

/* sample a division */
#ifdef DAYSIM_COMPATIBLE
RT_METHOD int divsample( AMBSAMP  *dp, AMBHEMI  *h, const float3& normal, const float3& hit, DaysimCoef dc )
#else
RT_METHOD int divsample( AMBSAMP  *dp, AMBHEMI  *h, const float3& normal, const float3& hit )
#endif
{
	PerRayData_radiance new_prd;
	//RAY  ar;
	//float3 rcoef; /* contribution coefficient w.r.t. parent */
	//int3  hlist;
	float2  spt;
	float  xd, yd, zd;
	float  b2;
	float  phi;
					/* ambient coefficient for weight */
	if (ambacc > FTINY)
		b2 = AVGREFL; // Reusing this variable
	else
		b2 = fmaxf(h->acoef);
	new_prd.weight = prd.weight * b2;
	if (new_prd.weight < minweight) //if (rayorigin(&ar, AMBIENT, r, ar.rcoef) < 0)
		return(-1);
	//if (ambacc > FTINY) {
	//	rcoef *= h->acoef;
	//	rcoef *= 1.0f / AVGREFL; // This all seems unnecessary
	//}
	//hlist = make_int3( prd.seed, dp->t, dp->p );
	//multisamp(spt, 2, urand(ilhash(hlist,3)+dp->n));//TODO implement
	//spt = multisamp2( frandom() );
	//int il = ilhash( hlist );
	//spt = make_float2( rnd( il ) );
	//spt = make_float2( rnd( prd.seed ) );
	spt = make_float2( hiprand_uniform( prd.state ), hiprand_uniform( prd.state ) );
	zd = sqrtf((dp->t + spt.x)/h->nt);
	phi = 2.0f*M_PIf * (dp->p + spt.y)/h->np;
	xd = cosf(phi) * zd;
	yd = sinf(phi) * zd;
	zd = sqrtf(1.0f - zd*zd);
	float3 rdir = normalize( xd*h->ux + yd*h->uy + zd*h->uz );
	//dimlist[ndims++] = dp->t*h->np + dp->p + 90171;

	new_prd.depth = prd.depth + 1;
	new_prd.ambient_depth = prd.ambient_depth + 1;
	//new_prd.seed = prd.seed;//lcg( prd.seed );
	new_prd.state = prd.state;
#ifdef DAYSIM_COMPATIBLE
	new_prd.dc = daysimNext(dc);
#endif
	setupPayload(new_prd, 0);
	Ray amb_ray = make_Ray( hit, rdir, radiance_ray_type, ray_start( hit, rdir, normal, RAY_START ), RAY_END );
	rtTrace(top_object, amb_ray, new_prd);
	resolvePayload(prd, new_prd);

	//ndims--;
	if ( isnan( new_prd.result ) ) // TODO How does this happen?
		return(-1);
	new_prd.result *= h->acoef;	/* apply coefficient */
#ifdef DAYSIM_COMPATIBLE
	daysimAddScaled(dc, new_prd.dc, h->acoef.x);
#endif
	dp->v += new_prd.result;
					/* use rt to improve gradient calc */
	if (new_prd.distance > FTINY && new_prd.distance < RAY_END)
		dp->r += 1.0f/new_prd.distance;

					/* (re)initialize error */
	if (dp->n++) {
		b2 = bright(dp->v)/dp->n - bright(new_prd.result);
		b2 = b2*b2 + dp->k*((dp->n-1)*(dp->n-1));
		dp->k = b2/(dp->n*dp->n);
	} else
		dp->k = 0.0f;
	return(0);
}

/* compute initial error estimates */
//RT_METHOD void comperrs( AMBSAMP *da, AMBHEMI *hp )
//{
//	float  b, b2;
//	int  i, j;
//	AMBSAMP  *dp;
//				/* sum differences from neighbors */
//	dp = da;
//	for (i = 0; i < hp->nt; i++)
//		for (j = 0; j < hp->np; j++) {
////#ifdef  DEBUG
////			if (dp->t != i || dp->p != j)
////				error(CONSISTENCY,
////					"division order in comperrs");
////#endif
//			b = bright(dp[0].v);
//			if (i > 0) {		/* from above */
//				b2 = bright(dp[-hp->np].v) - b;
//				b2 *= b2 * 0.25f;
//				dp[0].k += b2;
//				dp[-hp->np].k += b2;
//			}
//			if (j > 0) {		/* from behind */
//				b2 = bright(dp[-1].v) - b;
//				b2 *= b2 * 0.25f;
//				dp[0].k += b2;
//				dp[-1].k += b2;
//			} else {		/* around */
//				b2 = bright(dp[hp->np-1].v) - b;
//				b2 *= b2 * 0.25f;
//				dp[0].k += b2;
//				dp[hp->np-1].k += b2;
//			}
//			dp++;
//		}
//				/* divide by number of neighbors */
//	dp = da;
//	for (j = 0; j < hp->np; j++)		/* top row */
//		(dp++)->k *= 1.0f/3.0f;
//	if (hp->nt < 2)
//		return;
//	for (i = 1; i < hp->nt-1; i++)		/* central region */
//		for (j = 0; j < hp->np; j++)
//			(dp++)->k *= 0.25f;
//	for (j = 0; j < hp->np; j++)		/* bottom row */
//		(dp++)->k *= 1.0f/3.0f;
//}

/* decreasing order */
//RT_METHOD int ambcmp( const void *p1, const void *p2 )
//{
//	const AMBSAMP	*d1 = (const AMBSAMP *)p1;
//	const AMBSAMP	*d2 = (const AMBSAMP *)p2;
//
//	if (d1->k < d2->k)
//		return(1);
//	if (d1->k > d2->k)
//		return(-1);
//	return(0);
//}
#endif /* OLDAMB */
#endif /* AMBIENT */

#ifdef LIGHTS
/* partition a flat source */
RT_METHOD unsigned int flatpart( const float3& v, const float3& r0, const float3& r1, const float3& r2 )
{
	//float3 vp = source[si->sn].snorm;
	//if ( dot( v, vp ) <= 0.0f )		/* behind source */
	//	return 0u;

	if ( srcsizerat <= FTINY )
		return 1u;

	float d;

	/* Find longest edge */
	float3 vp = r1 - r0;
	float d2 = dot( vp, vp );
	vp = r2 - r1;
	if ( ( d = dot( vp, vp ) ) > d2 )
		d2 = d;
	vp = r2 - r0;
	if ( ( d = dot( vp, vp ) ) > d2 )
		d2 = d;

	/* Find minimum partition size */
	d = srcsizerat / prd.weight;
	d *= d * dot( v, v );

	/* Find number of partions */
	d2 /= d;
	if ( d2 < 1.0f )
		return 1u;
	if ( d2 > ( d = MAXSPART >> 1 ) ) // Divide maximum partitions by two going from rectangle to triangle
		d2 = d;
	return (unsigned int)sqrtf( d2 );
}

/* Solid angle calculation from "The solid angle of a plane triangle", A van Oosterom and J Strackee */
RT_METHOD float solid_angle( const float3& r0, const float3& r1, const float3& r2 )
{
	const float l0 = length( r0 );
	const float l1 = length( r1 );
	const float l2 = length( r2 );

	const float numerator = dot( r0, cross( r1, r2 ) );
	const float denominator = l0 * l1 * l2 + dot( r0, r1 ) * l2 + dot( r0, r2 ) * l1 + dot( r1, r2 ) * l0;
	return 2.0f * fabsf( atan2( numerator, denominator ) );
}

/* Compute point from barycentric coordinates and flip if outside triangle */
RT_METHOD float3 barycentric( float2& lambda, const float3& r0, const float3& r1, const float3& r2, const int flip )
{
	if ( flip )
		lambda = 1.0f - lambda;
	return r0 * ( 1.0f - lambda.x - lambda.y ) + r1 * lambda.x + r2 * lambda.y;
}
#endif /* LIGHTS */

/* convert 1-dimensional sample to 2 dimensions, based on multisamp.c */
//RT_METHOD float2 multisamp2(float r)	/* 1-dimensional sample [0,1) */
//{
//	int	j;
//	register int	k;
//	int2	ti;
//	float	s;
//
//	ti = make_int2( 0 );
//	j = 8;
//	while (j--) {
//		k = s = r*(1<<2);
//		r = s - k;
//		ti += ti + make_int2( ((k>>2) & 1), ((k>>1) & 1) );
//	}
//	ti += make_int2( frandom() );
//	ti *= 1.0f/256.0f;
//}

/* hash a set of integer values */
//RT_METHOD int ilhash(int3 d)
//{
//	register int  hval;
//
//	hval = 0;
//	hval ^= d.x * 73771;
//	hval ^= d.y * 96289;
//	hval ^= d.z * 103699;
//	return(hval & 0x7fffffff);
//}
