#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2014 Nathaniel Jones
 * Massachusetts Institute of Technology
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "optix_shader_common.h"

using namespace optix;

#define  AMBIENT
//#define  TRANSMISSION
//#define  FILL_GAPS_LAST_ONLY

#ifndef  MAXITER
#define  MAXITER	10		/* maximum # specular ray attempts */
#endif
#define  FRESNE(ci)	(expf(-5.85f*(ci)) - 0.00287989916f) /* estimate of Fresnel function */
#define  FRESTHRESH	0.017999f	/* minimum specularity for approx. */
#define  MAXSPART	64		/* maximum partitions per source */
//#define frandom()	(rnd( prd.seed )/float(RAND_MAX))
//#define frandom()	(rnd( prd.seed ))

				/* specularity flags */
#define  SP_REFL	01		/* has reflected specular component */
#define  SP_TRAN	02		/* has transmitted specular */
#define  SP_PURE	04		/* purely specular (zero roughness) */
#define  SP_FLAT	010		/* flat reflecting surface */
#define  SP_RBLT	020		/* reflection below sample threshold */
#define  SP_TBLT	040		/* transmission below threshold */

/* Context variables */
rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(unsigned int, shadow_ray_type, , );
rtDeclareVariable(unsigned int, ambient_ray_type, , );
rtDeclareVariable(rtObject,     top_object, , );
rtDeclareVariable(rtObject,     top_shadower, , );
rtDeclareVariable(rtObject,     top_ambient, , );

rtDeclareVariable(float3,       CIE_rgbf, , ); /* This is the value [ CIE_rf, CIE_gf, CIE_bf ] from color.h */

rtDeclareVariable(float,        specthresh, , ); /* This is the minimum fraction of reflection or transmission, under which no specular sampling is performed */
rtDeclareVariable(float,        specjitter, , );

#ifdef AMBIENT
rtDeclareVariable(float3,       ambval, , ); /* This is the final value used in place of an indirect light calculation */
rtDeclareVariable(int,          ambvwt, , ); /* As new indirect irradiances are computed, they will modify the default ambient value in a moving average, with the specified weight assigned to the initial value given on the command and all other weights set to 1 */
rtDeclareVariable(int,          ambounce, , ); /* Ambient bounces (ab) */
//rtDeclareVariable(int,          ambres, , ); /* Ambient resolution (ar) */
rtDeclareVariable(float,        ambacc, , ); /* Ambient accuracy (aa). This value will approximately equal the error from indirect illuminance interpolation */
rtDeclareVariable(int,          ambdiv, , ); /* Ambient divisions (ad) */
rtDeclareVariable(int,          ambssamp, , ); /* Ambient super-samples (as) */
rtDeclareVariable(float,        maxarad, , ); /* maximum ambient radius */
rtDeclareVariable(float,        minarad, , ); /* minimum ambient radius */
rtDeclareVariable(float,        avsum, , ); /* computed ambient value sum (log) */
rtDeclareVariable(unsigned int, navsum, , ); /* number of values in avsum */
#endif /* AMBIENT */

rtDeclareVariable(float,        minweight, , ); /* minimum ray weight (lw) */
rtDeclareVariable(int,          maxdepth, , ); /* maximum recursion depth (lr) */

//#ifdef FILL_GAPS_LAST_ONLY
//rtDeclareVariable(unsigned int, level, , ) = 0u;
//#endif

rtBuffer<DistantLight> lights;

/* Program variables */
rtDeclareVariable(unsigned int, metal, , ); /* The material type representing "metal" */

/* Material variables */
rtDeclareVariable(unsigned int, type, , ); /* The material type representing "plastic", "metal", or "trans" */
rtDeclareVariable(float3,       color, , ); /* The material color given by the rad file "plastic", "metal", or "trans" object */
rtDeclareVariable(float,        spec, , ); /* The material specularity given by the rad file "plastic", "metal", or "trans" object */
rtDeclareVariable(float,        rough, , ); /* The material roughness given by the rad file "plastic", "metal", or "trans" object */
rtDeclareVariable(float,        transm, , ) = 0.0f; /* The material transmissivity given by the rad file "trans" object */
rtDeclareVariable(float,        tspecu, , ) = 0.0f; /* The material transmitted specular component given by the rad file "trans" object */

/* OptiX variables */
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(PerRayData_radiance, prd, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow, rtPayload, );

/* Attributes */
//rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );


static __device__ float3 gaussamp( const unsigned int& specfl, float3 scolor, float3 mcolor, const float3& normal, const float3& hit, const float& alpha2, const float& tspec );
#ifdef AMBIENT
static __device__ float3 multambient( float3 aval, const float3& normal, const float3& hit );
#ifndef OLDAMB
static __device__ int doambient( float3 *rcol, const float3& normal, const float3& hit );
//static __device__ __inline__ int ambsample( AMBHEMI *hp, const int& i, const int& j, const float3 normal, const float3 hit );
#else /* OLDAMB */
static __device__ float doambient( float3 *rcol, const float3& normal, const float3& hit );
static __device__ __inline__ void inithemi( AMBHEMI  *hp, float3 ac, const float3& normal );
static __device__ int divsample( AMBSAMP  *dp, AMBHEMI  *h, const float3& hit );
//static __device__ void comperrs( AMBSAMP *da, AMBHEMI *hp );
//static __device__ int ambcmp( const void *p1, const void *p2 );
#endif /* OLDAMB */
#endif /* AMBIENT */
//static __device__ float2 multisamp2(float r);
//static __device__ __inline__ int ilhash(int3 d);
static __device__ __inline__ float bright( const float3 &rgb );


RT_PROGRAM void any_hit_shadow()
{
	// this material is opaque, so it fully attenuates all shadow rays
	prd_shadow.result = make_float3(0.0f);
	rtTerminateRay();
}


RT_PROGRAM void closest_hit_radiance()
{
	/* easy shadow test */
	// if this is a shadow ray and not a trans material, return

	float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
	float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );

	/* check for back side */
	// if backvis is false, create a new ray starting from the hit point (i.e., ignore this hit)
	float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

	PerRayData_radiance new_prd;
	float3 result = make_float3( 0.0f );
	float3 hit_point = ray.origin + t_hit * ray.direction;
	float3 mcolor = color;
	float3 scolor = make_float3( 0.0f );
	float rspec = spec;
	float alpha2 = rough * rough;
	unsigned int specfl = 0u; /* specularity flags */

	/* get modifiers */
	// we'll skip this for now

	/* get roughness */
	if (alpha2 <= FTINY) {
		specfl |= SP_PURE; // label this as a purely specular reflection
	}

	/* perturb normal */
	// if there's a bump map, we use that, else
	float pdot = -dot( ray.direction, ffnormal );

	// if it's a face or a ring (which it is currently) label as flat
	specfl |= SP_FLAT;

	/* modify material color */
	//mcolor *= rtTex3D(rtTextureId id, texcoord.x, texcoord.y, texcoord.z).xyz;

	/* compute Fresnel approx. */
	float fest = 0.0f;
	if (specfl & SP_PURE && rspec >= FRESTHRESH) {
		fest = FRESNE( pdot );
		rspec += fest * ( 1.0f - rspec );
	}

	/* compute transmission */
	float tdiff = 0.0f, tspec = 0.0f, trans = 0.0f; // because it's opaque
#ifdef TRANSMISSION
	float transtest = 0.0f, transdist = 0.0f;
	if (transm > 0.0f) { // type == MAT_TRANS
		trans = transm * (1.0f - rspec);
		tspec = trans * tspecu;
		tdiff = trans - tspec;
		if (tspec > FTINY) {
			specfl |= SP_TRAN;
							/* check threshold */
			if (!(specfl & SP_PURE) && specthresh >= tspec - FTINY)
				specfl |= SP_TBLT;
			//if (!hastexture || r->crtype & SHADOW) {
			//	VCOPY(nd.prdir, r->rdir);
			//	transtest = 2;
			//} else {
			//	for (i = 0; i < 3; i++)		/* perturb */
			//		nd.prdir[i] = r->rdir[i] - r->pert[i];
			//	if (DOT(nd.prdir, r->ron) < -FTINY)
			//		nd.prdir = normalize(nd.prdir);	/* OK */
			//	else
			//		VCOPY(nd.prdir, r->rdir);
			//}
		}
	}

	/* transmitted ray */
	if ((specfl&(SP_TRAN|SP_PURE|SP_TBLT)) == (SP_TRAN|SP_PURE)) {
		new_prd.weight = prd.weight * fmaxf(mcolor) * tspec;
		if (new_prd.weight >= minweight) {
			new_prd.depth = prd.depth;
			new_prd.ambient_depth = prd.ambient_depth;
			new_prd.state = prd.state;
#ifdef FILL_GAPS
			new_prd.primary = 0;
#endif
#ifdef RAY_COUNT
			new_prd.ray_count = 1;
#endif
#ifdef HIT_COUNT
			new_prd.hit_count = 0;
#endif
			float3 R = ray.direction; //TODO may need to perturb
			Ray trans_ray = make_Ray( hit_point, R, radiance_ray_type, RAY_START, RAY_END );
			rtTrace(top_object, trans_ray, new_prd);
			float3 rcol = new_prd.result * mcolor * tspec;
			result += rcol;
			transtest = 2.0f * bright( rcol );
			transdist = t_hit + new_prd.distance;
#ifdef RAY_COUNT
			prd.ray_count += new_prd.ray_count;
#endif
#ifdef HIT_COUNT
			prd.hit_count += new_prd.hit_count;
#endif
		}
	}
#endif

	// return if it's a shadow ray, which it isn't

	/* get specular reflection */
	if (rspec > FTINY) {
		specfl |= SP_REFL;

		/* compute specular color */
		if (type != metal) {
			scolor = make_float3( rspec );
		} else {
			if (fest > FTINY) {
				float d = spec * (1.0f - fest);
				scolor = fest + mcolor * d;
			} else {
				scolor = mcolor * rspec;
			}
		}

		/* check threshold */
		if (!(specfl & SP_PURE) && specthresh >= rspec - FTINY) {
			specfl |= SP_RBLT;
		}
	}

	/* reflected ray */
	float mirtest = 0.0f, mirdist = 0.0f;
	if ((specfl&(SP_REFL|SP_PURE|SP_RBLT)) == (SP_REFL|SP_PURE)) {
		new_prd.weight = prd.weight * fmaxf(scolor);
		new_prd.depth = prd.depth + 1;
		if (new_prd.weight >= minweight && new_prd.depth <= abs(maxdepth)) {
			new_prd.ambient_depth = prd.ambient_depth;
			new_prd.state = prd.state;
#ifdef FILL_GAPS
			new_prd.primary = 0;
#endif
#ifdef RAY_COUNT
			new_prd.ray_count = 1;
#endif
#ifdef HIT_COUNT
			new_prd.hit_count = 0;
#endif
			float3 R = reflect( ray.direction, ffnormal );
			Ray refl_ray = make_Ray( hit_point, R, radiance_ray_type, RAY_START, RAY_END );
			rtTrace(top_object, refl_ray, new_prd);
			float3 rcol = new_prd.result * scolor;
			result += rcol;
			mirtest = 2.0f * bright( rcol );
			mirdist = t_hit + new_prd.distance;
#ifdef RAY_COUNT
			prd.ray_count += new_prd.ray_count;
#endif
#ifdef HIT_COUNT
			prd.hit_count += new_prd.hit_count;
#endif
		}
	}

	/* diffuse reflection */
	float rdiff = 1.0f - trans - rspec;

	if (!(specfl & SP_PURE && rdiff <= FTINY && tdiff <= FTINY)) { /* not 100% pure specular */

		/* checks *BLT flags */
		if ( !(specfl & SP_PURE) )
			result += gaussamp( specfl, scolor, mcolor, ffnormal, hit_point, alpha2, tspec );

#ifdef AMBIENT
		/* ambient from this side */
		if (rdiff > FTINY) {
			float3 aval = mcolor * rdiff;	/* modified by material color */
			if (specfl & SP_RBLT)	/* add in specular as well? */
				aval += scolor;
			result += multambient(aval, ffnormal, hit_point);	/* add to returned color */
		}

#ifdef TRANSMISSION
		/* ambient from other side */
		if (tdiff > FTINY) {
			float3 aval = mcolor;	/* modified by material color */
			if (specfl & SP_TBLT)
				aval *= trans;
			else
				aval *= tdiff;
			result += multambient(aval, -ffnormal, hit_point);	/* add to returned color */
		}
#endif
#endif

		/* add direct component */
		// This is the call to direct() in source.c
		// Let's start at line 447, and not bother with sorting for now

		// compute direct lighting
		unsigned int num_lights = lights.size();
		PerRayData_shadow shadow_prd;
		Ray shadow_ray = make_Ray( hit_point, ffnormal, shadow_ray_type, RAY_START, RAY_END );
		for(int i = 0; i < num_lights; ++i) {
			DistantLight light = lights[i];
			//float Ldist = optix::length(light.pos - hit_point);
			//float3 L = optix::normalize(light.pos - hit_point);
			shadow_ray.direction = normalize(light.pos);
			float ldot = dot( ffnormal, shadow_ray.direction );

			// cast shadow ray
#ifdef TRANSMISSION
			if ( light.casts_shadow ) {
#else
			if ( ldot > 0.0f && light.casts_shadow ) { // assuming it's not a TRANS material
#endif
				shadow_prd.result = make_float3(0.0f);
				rtTrace(top_shadower, shadow_ray, shadow_prd);
				if( fmaxf(shadow_prd.result) > 0.0f ) {

					/* This comes from direct() in normal.c */
					float3 cval = make_float3( 0.0f );

					/* Fresnel estimate */
					float lrdiff = rdiff;
					float ltdiff = tdiff;
					if (specfl & SP_PURE && rspec >= FRESTHRESH && (lrdiff > FTINY) | (ltdiff > FTINY)) {
						float dtmp = 1.0f - FRESNE(fabs(ldot));
						lrdiff *= dtmp;
						ltdiff *= dtmp;
					}

					if (ldot > FTINY && lrdiff > FTINY) {
						/*
						 *  Compute and add diffuse reflected component to returned
						 *  color.  The diffuse reflected component will always be
						 *  modified by the color of the material.
						 */
						float dtmp = ldot * light.solid_angle * lrdiff * (1.0f/M_PIf);
						cval += mcolor * dtmp;
					}
					if (ldot > FTINY && (specfl&(SP_REFL|SP_PURE)) == SP_REFL) {
						/*
						 *  Compute specular reflection coefficient using
						 *  Gaussian distribution model.
						 */
						/* roughness */
						float dtmp = alpha2;
						/* + source if flat */
						if (specfl & SP_FLAT)
							dtmp += light.solid_angle * (0.25f/M_PIf);
						/* half vector */
						float3 vtmp = shadow_ray.direction - ray.direction;
						float d2 = dot( vtmp, ffnormal );
						d2 *= d2;
						float d3 = dot( vtmp, vtmp );
						float d4 = (d3 - d2) / d2;
						/* new W-G-M-D model */
						dtmp = expf(-d4/dtmp) * d3 / (M_PIf * d2*d2 * dtmp);
						/* worth using? */
						if (dtmp > FTINY) {
							dtmp *= ldot * light.solid_angle;
							cval += scolor * dtmp;
						}
					}
#ifdef TRANSMISSION
					if (ldot < -FTINY && ltdiff > FTINY) {
						/*
						 *  Compute diffuse transmission.
						 */
						float dtmp = -ldot * light.solid_angle * ltdiff * (1.0f/M_PIf);
						cval += mcolor * dtmp;
					}
					if (ldot < -FTINY && (specfl&(SP_TRAN|SP_PURE)) == SP_TRAN) {
						/*
						 *  Compute specular transmission.  Specular transmission
						 *  is always modified by material color.
						 */
										/* roughness + source */
						float dtmp = alpha2 + light.solid_angle * (1.0f/M_PIf);
										/* Gaussian */
						dtmp = expf( ( 2.0f * dot( ray.direction, shadow_ray.direction ) - 2.0f ) / dtmp ) / ( M_PIf * dtmp ); // may need to perturb direction
										/* worth using? */
						if (dtmp > FTINY) {
							dtmp *= tspec * light.solid_angle * sqrtf( -ldot / pdot );
							cval += mcolor * dtmp;
						}
					}
#endif
					result += cval * shadow_prd.result;
				} /* End direct() in normal.c */
			}
		}

	}

	/* check distance */
	float d = bright( result );
#ifdef TRANSMISSION
	if (transtest > d)
		prd.distance = transdist;
	else
#endif
	if (mirtest > d)
		prd.distance = mirdist;
	else
		prd.distance = t_hit;

	// pass the color back up the tree
	prd.result = result;

#ifdef HIT_TYPE
	prd.hit_type = type;
#endif
}

// sample Gaussian specular
static __device__ float3 gaussamp( const unsigned int& specfl, float3 scolor, float3 mcolor, const float3& normal, const float3& hit, const float& alpha2, const float& tspec )
{
	float3 rcol = make_float3( 0.0f );

	/* This section is based on the gaussamp method in normal.c */
	if ((specfl & (SP_REFL|SP_RBLT)) != SP_REFL && (specfl & (SP_TRAN|SP_TBLT)) != SP_TRAN)
		return rcol;

	PerRayData_radiance gaus_prd;
	gaus_prd.depth = prd.depth + 1;
	if ( gaus_prd.depth > abs(maxdepth) )
		return rcol;
	gaus_prd.ambient_depth = prd.ambient_depth + 1; //TODO the increment is a hack to prevent the sun from affecting specular values
	//gaus_prd.seed = prd.seed;//lcg( prd.seed );
	gaus_prd.state = prd.state;
	Ray gaus_ray = make_Ray( hit, normal, radiance_ray_type, RAY_START, RAY_END );

	float d;

	/* set up sample coordinates */
	float3 v = cross_direction( normal ); // should be using perturned normal, but currently using ffnormal
	float3 u = normalize( cross( v, normal ) );
	v = normalize( cross( normal, u ) );

	unsigned int nstarget, nstaken, ntrials;

	/* compute reflection */
	gaus_prd.weight = prd.weight * fmaxf(scolor);
	if ( (specfl & (SP_REFL|SP_RBLT)) == SP_REFL && gaus_prd.weight >= minweight ) {
		nstarget = 1;
		if (specjitter > 1.5f) {	/* multiple samples? */ // By default it's 1.0
			nstarget = specjitter * prd.weight + 0.5f;
			if ( gaus_prd.weight <= minweight * nstarget )
				nstarget = gaus_prd.weight / minweight;
			if ( nstarget > 1 ) {
				d = 1.0f / nstarget;
				scolor *= d; //scolor, stored as ray rcoef
				gaus_prd.weight *= d; // TODO make sure weight isn't changed by hit programs
			} else
				nstarget = 1;
		}
		float3 scol = make_float3( 0.0f );
		//dimlist[ndims++] = (int)(size_t)np->mp;
		unsigned int maxiter = MAXITER * nstarget;
		for (nstaken = ntrials = 0; nstaken < nstarget && ntrials < maxiter; ntrials++) {
			float2 rv = make_float2( hiprand_uniform( prd.state ), hiprand_uniform( prd.state ) ); // should be evenly distributed in both dimensions
			d = 2.0f * M_PIf * rv.x;
			float cosp = cosf( d );
			float sinp = sinf( d );
			if ( ( 0.0f <= specjitter ) && ( specjitter < 1.0f ) )
				rv.y = 1.0f - specjitter * rv.y;
			if ( rv.y <= FTINY )
				d = 1.0f;
			else
				d = sqrtf( alpha2 * -logf( rv.y ) ); // alpha2
			float3 h = normal + d * ( cosp * u + sinp * v ); // normal is perturbed
			d = -2.0f * dot( h, ray.direction ) / ( 1.0f + d*d );
			gaus_ray.direction = ray.direction + h * d;

			/* sample rejection test */
			if ( ( d = dot( gaus_ray.direction, normal ) ) <= FTINY) // this is ron, is this perturbed?
				continue;

			gaus_ray.direction = normalize( gaus_ray.direction );
#ifdef FILL_GAPS
			gaus_prd.primary = 0;
#endif
#ifdef RAY_COUNT
			gaus_prd.ray_count = 1;
#endif
#ifdef HIT_COUNT
			gaus_prd.hit_count = 0;
#endif
			//if (nstaken) // check for prd data that needs to be cleared
			rtTrace(top_object, gaus_ray, gaus_prd);
#ifdef RAY_COUNT
			prd.ray_count += gaus_prd.ray_count;
#endif
#ifdef HIT_COUNT
			prd.hit_count += gaus_prd.hit_count;
#endif

			/* W-G-M-D adjustment */
			if (nstarget > 1) {	
				d = 2.0f / ( 1.0f + dot( ray.direction, normal ) / d );
				scol += gaus_prd.result * d;
			} else {
				rcol += gaus_prd.result * scolor;
			}

			++nstaken;
		}
		/* final W-G-M-D weighting */
		if (nstarget > 1) {
			scol *= scolor;
			d = (float)nstarget / ntrials;
			rcol += scol * d;
		}
		//ndims--;
	}

#ifdef TRANSMISSION
	/* compute transmission */
	mcolor *= tspec;	/* modified by color */
	gaus_prd.weight = prd.weight * fmaxf(mcolor);
	if ( ( specfl & (SP_TRAN|SP_TBLT)) == SP_TRAN && gaus_prd.weight >= minweight ) {
		nstarget = 1;
		if (specjitter > 1.5f) {	/* multiple samples? */ // By default it's 1.0
			nstarget = specjitter * prd.weight + 0.5f;
			if ( gaus_prd.weight <= minweight * nstarget )
				nstarget = gaus_prd.weight / minweight;
			if ( nstarget > 1 ) {
				d = 1.0f / nstarget;
				scolor *= d; //scolor, stored as ray rcoef
				gaus_prd.weight *= d; // TODO make sure weight isn't changed by hit programs
			} else
				nstarget = 1;
		}
		//dimlist[ndims++] = (int)(size_t)np->mp;
		unsigned int maxiter = MAXITER * nstarget;
		for (nstaken = ntrials = 0; nstaken < nstarget && ntrials < maxiter; ntrials++) {
			float2 rv = make_float2( hiprand_uniform( prd.state ), hiprand_uniform( prd.state ) ); // should be evenly distributed in both dimensions
			d = 2.0f * M_PIf * rv.x;
			float cosp = cosf( d );
			float sinp = sinf( d );
			if ( ( 0.0f <= specjitter ) && ( specjitter < 1.0f ) )
				rv.y = 1.0f - specjitter * rv.y;
			if ( rv.y <= FTINY )
				d = 1.0f;
			else
				d = sqrtf( alpha2 * -logf( rv.y ) );
			float3 h = ray.direction + d * ( cosp * u + sinp * v ); // ray direction is perturbed
			d = -2.0f * dot( h, ray.direction ) / ( 1.0f + d*d );
			gaus_ray.direction = ray.direction + h * d;

			/* sample rejection test */
			if ( ( d = dot( gaus_ray.direction, normal ) ) >= -FTINY) // this is ron, is this perturbed?
				continue;

			gaus_ray.direction = normalize( gaus_ray.direction );
#ifdef RAY_COUNT
			gaus_prd.ray_count = 1;
#endif
			//if (nstaken) // check for prd data that needs to be cleared
			rtTrace(top_object, gaus_ray, gaus_prd);
#ifdef RAY_COUNT
			prd.ray_count += gaus_prd.ray_count;
#endif
			rcol += gaus_prd.result * mcolor;
			++nstaken;
		}
		//ndims--;
	}
#endif
	//return make_float3(0.0f);
	return rcol;
}

#ifdef AMBIENT
// Compute the ambient component and multiply by the coefficient.
static __device__ float3 multambient( float3 aval, const float3& normal, const float3& hit )
{
	//static int  rdepth = 0;			/* ambient recursion */ //This is part of the ray for parallelism
	float 	d;

	if (ambdiv <= 0)			/* no ambient calculation */
		goto dumbamb;
						/* check number of bounces */
	if (prd.ambient_depth >= ambounce)
		goto dumbamb;
						/* check ambient list */
	//if (ambincl != -1 && r->ro != NULL && ambincl != inset(ambset, r->ro->omod))
	//	goto dumbamb;

	if ( ambacc <= FTINY || navsum == 0 ) {			/* no ambient storage */
		float3 acol = aval;
		//rdepth++;
		d = doambient( &acol, normal, hit );
		//rdepth--;
		if (d > FTINY)
			return acol;
	} else {
		//if (tracktime)				/* sort to minimize thrashing */
		//	sortambvals(0);

		/* interpolate ambient value */
		//acol = make_float3( 0.0f );
		//d = sumambient(acol, r, normal, rdepth, &atrunk, thescene.cuorg, thescene.cusize);
		PerRayData_ambient ambient_prd;
		ambient_prd.result = make_float3( 0.0f );
		ambient_prd.surface_normal = normal;
		ambient_prd.weight = prd.weight;
		ambient_prd.wsum = 0.0f;
		ambient_prd.ambient_depth = prd.ambient_depth;
		ambient_prd.state = prd.state;
#ifdef HIT_COUNT
		ambient_prd.hit_count = 0;
#endif
		Ray ambient_ray = make_Ray( hit, normal, ambient_ray_type, -AMBIENT_RAY_LENGTH, AMBIENT_RAY_LENGTH );
		rtTrace(top_ambient, ambient_ray, ambient_prd);
#ifdef HIT_COUNT
		prd.hit_count += ambient_prd.hit_count;
#endif
		if (ambient_prd.wsum > FTINY) { // TODO if miss program is called, set wsum = 1.0f or place this before ambacc == 0.0f
			ambient_prd.result *= 1.0f / ambient_prd.wsum;
			return aval * ambient_prd.result;
		}
		//rdepth++;				/* need to cache new value */
		//d = makeambient(acol, r, normal, rdepth-1); //TODO implement as miss program for ambient ray
		//rdepth--;
		//if ( dot( ambient_prd.result, ambient_prd.result) > FTINY) { // quick check to see if a value was returned by miss program
		//	return aval * ambient_prd.result;		/* got new value */
		//}

#ifdef FILL_GAPS
		if ( prd.primary ) {
			float3 acol = aval;
			//rdepth++;
			d = doambient( &acol, normal, hit );
			//rdepth--;
			if (d > FTINY)
				return acol;
		}
#elif defined FILL_GAPS_LAST_ONLY
		//if ( prd.ambient_depth == level ) {
		if ( prd.ambient_depth == 0 ) {
			float3 acol = aval;
			//rdepth++;
			d = doambient( &acol, normal, hit );
			//rdepth--;
			if (d > FTINY)
				return acol;
		}
#endif
	}
dumbamb:					/* return global value */
	if ((ambvwt <= 0) || (navsum == 0)) {
		return aval * ambval;
	}
	float l = bright(ambval);			/* average in computations */
	if (l > FTINY) {
		d = (logf(l)*(float)ambvwt + avsum) / (float)(ambvwt + navsum);
		d = expf(d) / l;
		aval *= ambval;	/* apply color of ambval */
	} else {
		d = expf( avsum / (float)navsum );
	}
	return aval * d;
}

#ifndef OLDAMB
/* sample indirect hemisphere, based on samp_hemi in ambcomp.c */
static __device__ int doambient( float3 *rcol, const float3& normal, const float3& hit )
{
	float	d;
	int	j;
	float wt = prd.weight;

					/* set number of divisions */
	if (ambacc <= FTINY && wt > (d = 0.8f * fmaxf(*rcol) * wt / (ambdiv*minweight)))
		wt = d;			/* avoid ray termination */
	int n = sqrtf(ambdiv * wt) + 0.5f;
	int i = 1 + 5 * (ambacc > FTINY);	/* minimum number of samples */
	if (n < i)
		n = i;
	const int nn = n * n;
	float3 acol = make_float3( 0.0f );
	unsigned int sampOK = 0u;
					/* assign coefficient */
	float3 acoef = *rcol / nn;

	/* Setup from ambsample in ambcomp.c */
	PerRayData_radiance new_prd;
					/* generate hemispherical sample */
					/* ambient coefficient for weight */
	if (ambacc > FTINY)
		d = AVGREFL; // Reusing this variable
	else
		d = fmaxf( acoef );
	new_prd.weight = prd.weight * d;
	if (new_prd.weight < minweight) //if (rayorigin(&ar, AMBIENT, r, ar.rcoef) < 0)
		return(0);

	new_prd.depth = prd.depth + 1;
	new_prd.ambient_depth = prd.ambient_depth + 1;
	//new_prd.seed = prd.seed;//lcg( prd.seed );
	new_prd.state = prd.state;

	Ray amb_ray = make_Ray( hit, hit, radiance_ray_type, RAY_START, RAY_END ); // Use hit point as temporary direction
	/* End ambsample setup */

					/* make tangent plane axes */
	float3 uy = make_float3( hiprand_uniform( prd.state ), hiprand_uniform( prd.state ), hiprand_uniform( prd.state ) ) - 0.5f;
	uy = fmaxf( cross_direction( normal ) * 2.0f - 1.0f, uy );
	float3 ux = cross( uy, normal );
	ux = normalize( ux );
	uy = cross( normal, ux );
					/* sample divisions */
	for (i = n; i--; )
	    for (j = n; j--; ) {
			//hp.sampOK += ambsample( &hp, i, j, normal, hit );
			/* ambsample in ambcomp.c */
			float2 spt = 0.1f + 0.8f * make_float2( hiprand_uniform( prd.state ), hiprand_uniform( prd.state ) );
			SDsquare2disk( spt, (j+spt.y) / n, (i+spt.x) / n );
			float zd = sqrtf( 1.0f - dot( spt, spt ) );
			amb_ray.direction = normalize( spt.x*ux + spt.y*uy + zd*normal );
			//dimlist[ndims++] = AI(hp,i,j) + 90171;

#ifdef FILL_GAPS
			new_prd.primary = 0;
#endif
#ifdef RAY_COUNT
			new_prd.ray_count = 1;
#endif
			//Ray amb_ray = make_Ray( hit, rdir, radiance_ray_type, RAY_START, RAY_END );
			rtTrace(top_object, amb_ray, new_prd);
#ifdef RAY_COUNT
			prd.ray_count += new_prd.ray_count;
#endif

			//ndims--;
			if ( isnan( new_prd.result ) ) // TODO How does this happen?
				continue;
			if ( new_prd.distance <= FTINY )
				continue;		/* should never happen */
			acol += new_prd.result * acoef;	/* add to our sum */
			sampOK++;
		}
	*rcol = acol;
	if ( !sampOK ) {		/* utter failure? */
		return( 0 );
	}
	if ( sampOK < nn ) {
		//hp.sampOK *= -1;	/* soft failure */
		return( 1 );
	}
	//n = ambssamp * wt + 0.5f;
	//if (n > 8) {			/* perform super-sampling? */
	//	ambsupersamp(hp, n);
	//	*rcol = hp.acol;
	//}
	return( 1 );			/* all is well */
}
#else /* OLDAMB */
static __device__ float doambient( float3 *rcol, const float3& normal, const float3& hit )
{
	float  b;//, d;
	AMBHEMI  hemi;
	AMBSAMP  *div;
	AMBSAMP  dnew;
	float3  acol;
	AMBSAMP  *dp;
	float  arad;
	int  divcnt;
	int  i, j;
					/* initialize hemisphere */
	inithemi(&hemi, *rcol, normal);
	divcnt = hemi.nt * hemi.np;
					/* initialize */
	//if (pg != NULL)
	//	pg[0] = pg[1] = pg[2] = 0.0;
	//if (dg != NULL)
	//	dg[0] = dg[1] = dg[2] = 0.0;
	*rcol = make_float3( 0.0f );
	if (divcnt == 0)
		return(0.0f); //TODO does this change the value of rcol in the calling method?
					/* allocate super-samples */
	//if (hemi.ns > 0) {// || pg != NULL || dg != NULL) {
	//	div = (AMBSAMP *)malloc(divcnt*sizeof(AMBSAMP));
	//	//if (div == NULL) // This is 0
	//	//	error(SYSTEM, "out of memory in doambient");
	//} else
		div = NULL; // This is 0
					/* sample the divisions */
	arad = 0.0f;
	acol = make_float3( 0.0f );
	if ((dp = div) == NULL)
		dp = &dnew;
	divcnt = 0;
	for (i = 0; i < hemi.nt; i++)
		for (j = 0; j < hemi.np; j++) {
			dp->t = i; dp->p = j;
			dp->v = make_float3( 0.0f );
			dp->r = 0.0f;
			dp->n = 0;
			if (divsample( dp, &hemi, hit ) < 0) {
				if (div != NULL)
					dp++;
				continue;
			}
			arad += dp->r;
			divcnt++;
			if (div != NULL)
				dp++;
			else
				acol += dp->v;
		}
	if (!divcnt) {
		//if (div != NULL)
		//	free((void *)div);
		return(0.0f);		/* no samples taken */
	}
	if (divcnt < hemi.nt*hemi.np) {
		//pg = dg = NULL;		/* incomplete sampling */
		hemi.ns = 0;
	} else if (arad > FTINY && divcnt/arad < minarad) {
		hemi.ns = 0;		/* close enough */
	} else if (hemi.ns > 0) {	/* else perform super-sampling? */
		//comperrs(div, &hemi);			/* compute errors */
		//qsort(div, divcnt, sizeof(AMBSAMP), ambcmp);	/* sort divs */ TODO necessary?
						/* super-sample */
		for (i = hemi.ns; i > 0; i--) {
			dnew = *div;
			if (divsample( &dnew, &hemi, hit ) < 0) {
				dp++;
				continue;
			}
			dp = div;		/* reinsert */
			j = divcnt < i ? divcnt : i;
			while (--j > 0 && dnew.k < dp[1].k) {
				*dp = *(dp+1);
				dp++;
			}
			*dp = dnew;
		}
		//if (pg != NULL || dg != NULL)	/* restore order */
		//	qsort(div, divcnt, sizeof(AMBSAMP), ambnorm);
	}
					/* compute returned values */
	if (div != NULL) {
		arad = 0.0f;		/* note: divcnt may be < nt*np */
		for (i = hemi.nt*hemi.np, dp = div; i-- > 0; dp++) {
			arad += dp->r;
			if (dp->n > 1) {
				b = 1.0f/dp->n;
				dp->v *= b;
				dp->r *= b;
				dp->n = 1;
			}
			acol += dp->v;
		}
		//b = bright(acol);
		//if (b > FTINY) {
		//	b = 1.0f/b;	/* compute & normalize gradient(s) */
		//	//if (pg != NULL) {
		//	//	posgradient(pg, div, &hemi);
		//	//	for (i = 0; i < 3; i++)
		//	//		pg[i] *= b;
		//	//}
		//	//if (dg != NULL) {
		//	//	dirgradient(dg, div, &hemi);
		//	//	for (i = 0; i < 3; i++)
		//	//		dg[i] *= b;
		//	//}
		//}
		//free((void *)div);
	}
	*rcol = acol;
	if (arad <= FTINY)
		arad = maxarad;
	else
		arad = (divcnt+hemi.ns)/arad;
	//if (pg != NULL) {		/* reduce radius if gradient large */
	//	d = DOT(pg,pg);
	//	if (d*arad*arad > 1.0f)
	//		arad = 1.0f/sqrt(d);
	//}
	if (arad < minarad) {
		arad = minarad;
		//if (pg != NULL && d*arad*arad > 1.0f) {	/* cap gradient */
		//	d = 1.0f/arad/sqrt(d);
		//	for (i = 0; i < 3; i++)
		//		pg[i] *= d;
		//}
	}
	if ((arad /= sqrt(prd.weight)) > maxarad)
		arad = maxarad;
	return(arad);
}

/* initialize sampling hemisphere */
static __device__ __inline__ void inithemi( AMBHEMI  *hp, float3 ac, const float3& normal )
{
	float	d;
	int  i;
	float wt = prd.weight;
					/* set number of divisions */
	if (ambacc <= FTINY && wt > (d = 0.8f * fmaxf(ac) * wt / (ambdiv*minweight)))
		wt = d;			/* avoid ray termination */
	hp->nt = sqrtf(ambdiv * wt / M_PIf) + 0.5f;
	i = ambacc > FTINY ? 3 : 1;	/* minimum number of samples */
	if (hp->nt < i)
		hp->nt = i;
	hp->np = M_PIf * hp->nt + 0.5f;
					/* set number of super-samples */
	hp->ns = ambssamp * wt + 0.5f;
					/* assign coefficient */
	hp->acoef = ac;
	d = 1.0f/(hp->nt*hp->np);
	hp->acoef *= d;
					/* make axes */
	hp->uz = normal;
	hp->uy = cross_direction( hp->uz );
	hp->ux = normalize( cross(hp->uy, hp->uz) );
	hp->uy = normalize( cross(hp->uz, hp->ux) );
}

/* sample a division */
static __device__ int divsample( AMBSAMP  *dp, AMBHEMI  *h, const float3& hit )
{
	PerRayData_radiance new_prd;
	//RAY  ar;
	//float3 rcoef; /* contribution coefficient w.r.t. parent */
	//int3  hlist;
	float2  spt;
	float  xd, yd, zd;
	float  b2;
	float  phi;
					/* ambient coefficient for weight */
	if (ambacc > FTINY)
		b2 = AVGREFL; // Reusing this variable
	else
		b2 = fmaxf(h->acoef);
	new_prd.weight = prd.weight * b2;
	if (new_prd.weight < minweight) //if (rayorigin(&ar, AMBIENT, r, ar.rcoef) < 0)
		return(-1);
	//if (ambacc > FTINY) {
	//	rcoef *= h->acoef;
	//	rcoef *= 1.0f / AVGREFL; // This all seems unnecessary
	//}
	//hlist = make_int3( prd.seed, dp->t, dp->p );
	//multisamp(spt, 2, urand(ilhash(hlist,3)+dp->n));//TODO implement
	//spt = multisamp2( frandom() );
	//int il = ilhash( hlist );
	//spt = make_float2( rnd( il ) );
	//spt = make_float2( rnd( prd.seed ) );
	spt = make_float2( hiprand_uniform( prd.state ), hiprand_uniform( prd.state ) );
	zd = sqrtf((dp->t + spt.x)/h->nt);
	phi = 2.0f*M_PIf * (dp->p + spt.y)/h->np;
	xd = cosf(phi) * zd;
	yd = sinf(phi) * zd;
	zd = sqrtf(1.0f - zd*zd);
	float3 rdir = normalize( xd*h->ux + yd*h->uy + zd*h->uz );
	//dimlist[ndims++] = dp->t*h->np + dp->p + 90171;

	new_prd.depth = prd.depth + 1;
	new_prd.ambient_depth = prd.ambient_depth + 1;
	//new_prd.seed = prd.seed;//lcg( prd.seed );
	new_prd.state = prd.state;
#ifdef FILL_GAPS
	new_prd.primary = 0;
#endif
#ifdef RAY_COUNT
	new_prd.ray_count = 1;
#endif
	Ray amb_ray = make_Ray( hit, rdir, radiance_ray_type, RAY_START, RAY_END );
	rtTrace(top_object, amb_ray, new_prd);
#ifdef RAY_COUNT
	prd.ray_count += new_prd.ray_count;
#endif

	//ndims--;
	if ( isnan( new_prd.result ) ) // TODO How does this happen?
		return(-1);
	new_prd.result *= h->acoef;	/* apply coefficient */
	dp->v += new_prd.result;
					/* use rt to improve gradient calc */
	if (new_prd.distance > FTINY && new_prd.distance < RAY_END)
		dp->r += 1.0f/new_prd.distance; //TODO should this be sum of distances?

					/* (re)initialize error */
	if (dp->n++) {
		b2 = bright(dp->v)/dp->n - bright(new_prd.result);
		b2 = b2*b2 + dp->k*((dp->n-1)*(dp->n-1));
		dp->k = b2/(dp->n*dp->n);
	} else
		dp->k = 0.0f;
	return(0);
}

/* compute initial error estimates */
//static __device__ void comperrs( AMBSAMP *da, AMBHEMI *hp )
//{
//	float  b, b2;
//	int  i, j;
//	AMBSAMP  *dp;
//				/* sum differences from neighbors */
//	dp = da;
//	for (i = 0; i < hp->nt; i++)
//		for (j = 0; j < hp->np; j++) {
////#ifdef  DEBUG
////			if (dp->t != i || dp->p != j)
////				error(CONSISTENCY,
////					"division order in comperrs");
////#endif
//			b = bright(dp[0].v);
//			if (i > 0) {		/* from above */
//				b2 = bright(dp[-hp->np].v) - b;
//				b2 *= b2 * 0.25f;
//				dp[0].k += b2;
//				dp[-hp->np].k += b2;
//			}
//			if (j > 0) {		/* from behind */
//				b2 = bright(dp[-1].v) - b;
//				b2 *= b2 * 0.25f;
//				dp[0].k += b2;
//				dp[-1].k += b2;
//			} else {		/* around */
//				b2 = bright(dp[hp->np-1].v) - b;
//				b2 *= b2 * 0.25f;
//				dp[0].k += b2;
//				dp[hp->np-1].k += b2;
//			}
//			dp++;
//		}
//				/* divide by number of neighbors */
//	dp = da;
//	for (j = 0; j < hp->np; j++)		/* top row */
//		(dp++)->k *= 1.0f/3.0f;
//	if (hp->nt < 2)
//		return;
//	for (i = 1; i < hp->nt-1; i++)		/* central region */
//		for (j = 0; j < hp->np; j++)
//			(dp++)->k *= 0.25f;
//	for (j = 0; j < hp->np; j++)		/* bottom row */
//		(dp++)->k *= 1.0f/3.0f;
//}

/* decreasing order */
//static __device__ int ambcmp( const void *p1, const void *p2 )
//{
//	const AMBSAMP	*d1 = (const AMBSAMP *)p1;
//	const AMBSAMP	*d2 = (const AMBSAMP *)p2;
//
//	if (d1->k < d2->k)
//		return(1);
//	if (d1->k > d2->k)
//		return(-1);
//	return(0);
//}
#endif /* OLDAMB */
#endif /* AMBIENT */

//static __device__ float nextssamp( float3 *rdir, const DistantLight* light, const float3 hit,			/* compute sample for source, rtn. distance */
//	RAY  *r,		/* origin is read, direction is set */
//	SRCINDEX  *si		/* source index (modified to current) */\
//)
//{
//	int3  cent, size;
//	int2  parr;
////	SRCREC  *srcp;
//	float3  vpos;
//	float  d;
//	//int  i;
//nextsample:
//	//while (++si->sp >= si->np) {	/* get next sample */
//	//	if (++si->sn >= nsources)
//	//		return(0.0);	/* no more */
//	//	if (srcskip(source+si->sn, r->rorg))
//	//		si->np = 0;
//	//	else if (srcsizerat <= FTINY)
//	//		nopart(si, r);
//	//	else {
//	//		for (i = si->sn; source[i].sflags & SVIRTUAL;
//	//				i = source[i].sa.sv.sn)
//	//			;		/* partition source */
//	//		(*sfun[source[i].so->otype].of->partit)(si, r);
//	//	}
//	//	si->sp = -1;
//	//}
//					/* get partition */
//	cent = make_int3( 0 );
//	size = make_int3( MAXSPART );
//	parr = make_int2( 0, si->sp );
//	//if (!skipparts(cent, size, parr, si->spt)) //TODO implement this
//	//	error(CONSISTENCY, "bad source partition in nextssamp");
//					/* compute sample */
//	//srcp = source + si->sn;
//	if (dstrsrc > FTINY) {			/* jitter sample */
//		//dimlist[ndims] = si->sn + 8831;
//		//dimlist[ndims+1] = si->sp + 3109;
//		//d = urand(ilhash(dimlist,ndims+2)+samplendx);
//		if (srcp->sflags & SFLAT) {
//			//multisamp(vpos, 2, d);
//			vpos = make_float3( hiprand_uniform( prd.state ), hiprand_uniform( prd.state ), 0.5f );
//		} else
//			//multisamp(vpos, 3, d);
//			vpos = make_float3( hiprand_uniform( prd.state ), hiprand_uniform( prd.state ), hiprand_uniform( prd.state ) );
//		vpos = dstrsrc * ( 1.0f - 2.0f * vpos ) * (float3)size * ( 1.0f / MAXSPART );
//	} else
//		vpos = make_float3( 0.0f );
//
//	vpos += cent * ( 1.0f / MAXSPART );
//					/* avoid circular aiming failures */
//	if ((srcp->sflags & SCIR) && (si->np > 1 || dstrsrc > 0.7f)) {
//		float3 trim = make_float3( 0.0f );
//		if (srcp->sflags & (SFLAT|SDISTANT)) {
//			d = 1.12837917f;		/* correct setflatss() */
//			trim.x = d * sqrtf( 1.0f - 0.5f * vpos.y*vpos.y );
//			trim.y = d * sqrtf( 1.0f - 0.5f * vpos.x*vpos.x );
//		} else {
//			trim.z = trim.x = vpos.x*vpos.x;
//			d = vpos.y*vpos.y;
//			if (d > trim.z) trim.z = d;
//			trim.x += d;
//			d = vpos.z*vpos.z;
//			if (d > trim.z) trim.z = d;
//			trim.x += d;
//			if (trim.x > FTINY*FTINY) {
//				d = 1.0f / 0.7236f;	/* correct sphsetsrc() */
//				trim = make_float3( d * sqrtf( trim.z / trim.x ) );
//			} else
//				trim = make_float3( 0.0f );
//		}
//		vpos *= trim;
//	}
//					/* compute direction */
//	*rdir = light.pos + //TODO
//	for (i = 0; i < 3; i++)
//		r->rdir[i] = srcp->sloc[i] +
//				vpos[SU]*srcp->ss[SU][i] +
//				vpos[SV]*srcp->ss[SV][i] +
//				vpos[SW]*srcp->ss[SW][i];
//
//	if (!(srcp->sflags & SDISTANT))
//		*rdir -= hit;
//					/* compute distance */
//	if ((d = normalize(r->rdir)) == 0.0)
//		goto nextsample;		/* at source! */
//
//					/* compute sample size */
//	if (srcp->sflags & SFLAT) {
//		si->dom = sflatform(si->sn, r->rdir);
//		si->dom *= size.x * size.y * ( 1.0f / ( MAXSPART * MAXSPART ) );
//	} else if (srcp->sflags & SCYL) {
//		si->dom = scylform(si->sn, r->rdir);
//		si->dom *= size.x * ( 1.0f / MAXSPART );
//	} else {
//		si->dom = size.x * size.y * size.z * ( 1.0f / ( MAXSPART * MAXSPART * MAXSPART ) );
//	}
//	if (srcp->sflags & SDISTANT) {
//		si->dom *= srcp->ss2;
//		return(FHUGE);
//	}
//	if (si->dom <= 1e-4)
//		goto nextsample;		/* behind source? */
//	si->dom *= srcp->ss2/(d*d);
//	return(d);		/* sample OK, return distance */
//}

/* convert 1-dimensional sample to 2 dimensions, based on multisamp.c */
//static __device__ __inline__ float2 multisamp2(float r)	/* 1-dimensional sample [0,1) */
//{
//	int	j;
//	register int	k;
//	int2	ti;
//	float	s;
//
//	ti = make_int2( 0 );
//	j = 8;
//	while (j--) {
//		k = s = r*(1<<2);
//		r = s - k;
//		ti += ti + make_int2( ((k>>2) & 1), ((k>>1) & 1) );
//	}
//	ti += make_int2( frandom() );
//	ti *= 1.0f/256.0f;
//}

/* hash a set of integer values */
//static __device__ __inline__ int ilhash(int3 d)
//{
//	register int  hval;
//
//	hval = 0;
//	hval ^= d.x * 73771;
//	hval ^= d.y * 96289;
//	hval ^= d.z * 103699;
//	return(hval & 0x7fffffff);
//}

static __device__ __inline__ float bright( const float3 &rgb )
{
	return dot( rgb, CIE_rgbf );
}

