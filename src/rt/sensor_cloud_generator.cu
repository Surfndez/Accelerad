#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2014 Nathaniel Jones
 * Massachusetts Institute of Technology
 */

#include <optix_world.h>
#include "optix_shader_common.h"

using namespace optix;

/* Program variables */
rtDeclareVariable(unsigned int,  do_irrad, , ); /* Calculate irradiance (-i) */

/* Contex variables */
rtBuffer<RayData, 2>             ray_buffer;
rtBuffer<PointDirection, 3>      seed_buffer;
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(rtObject,      top_irrad, , );
rtDeclareVariable(unsigned int,  point_cloud_ray_type, , );
rtDeclareVariable(unsigned int,  seeds, , ) = 1u; /* number of seed points to discover per thread */
rtDeclareVariable(unsigned int,  imm_irrad, , ); /* Immediate irradiance (-I) */

/* OptiX variables */
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );

RT_PROGRAM void cloud_generator()
{
	PerRayData_point_cloud prd;
	prd.backup.pos = make_float3( 0.0f );
	prd.backup.dir = make_float3( 0.0f );

	// Init random state
	rand_state state;
	hiprand_init( launch_index.x + launch_dim.x * launch_index.y, 0, 0, &state );

	uint3 index = make_uint3( launch_index, 0u );

	float tmin = ray_start( ray_buffer[launch_index].origin, RAY_START );
	float tmax;
	if ( imm_irrad ) {
		tmax = tmin;
		tmin = -tmin;
	} else {
		// Zero or negative aft clipping distance indicates infinity
		tmax = ray_buffer[launch_index].max;
		if (tmax <= FTINY) {
			tmax = RAY_END;
		}
	}

	Ray ray = make_Ray(ray_buffer[launch_index].origin, ray_buffer[launch_index].dir, point_cloud_ray_type, tmin, tmax);

	unsigned int loop = 2u * seeds; // Prevent infinite looping
	while ( index.z < seeds && loop-- ) {
		// Trace the current ray
		if ( imm_irrad )
			rtTrace(top_irrad, ray, prd);
		else
			rtTrace(top_object, ray, prd);

		// Check for a valid result
		if ( isfinite( prd.result.pos ) && dot( prd.result.dir, prd.result.dir ) > FTINY ) { // NaN values will be false
			seed_buffer[index] = prd.result; // This could contain points on glass materials
			index.z++;
		} else {
			prd.result.pos = ray_buffer[launch_index].origin;
			prd.result.dir = ray_buffer[launch_index].dir;
		}

		// Prepare for next ray
		ray.origin = prd.result.pos;
		//ray.direction = reflect( ray.direction, prd.result.dir );

		float3 uz = normalize( prd.result.dir );
		float3 uy = cross_direction( uz );
		float3 ux = normalize( cross( uy, uz ) );
		uy = normalize( cross( uz, ux ) );

		float zd = sqrtf( hiprand_uniform( &state ) );
		float phi = 2.0f*M_PIf * hiprand_uniform( &state );
		float xd = cosf(phi) * zd;
		float yd = sinf(phi) * zd;
		zd = sqrtf(1.0f - zd*zd);
		ray.direction = normalize( xd*ux + yd*uy + zd*uz );
	}

	// If outdoors, there are no bounces, but we need to prevent junk data
	prd.backup.pos = make_float3( 0.0f );
	prd.backup.dir = make_float3( 0.0f );
	while ( index.z < seeds ) {
		seed_buffer[index] = prd.backup;
		index.z++;
	}
}

RT_PROGRAM void exception()
{
	const unsigned int code = rtGetExceptionCode();
	rtPrintf( "Caught exception 0x%X at launch index (%d,%d)\n", code, launch_index.x, launch_index.y );
	uint3 index = make_uint3( launch_index, seeds - 1u ); // record error to last segment
	seed_buffer[index].pos = exceptionToFloat3( code );
	seed_buffer[index].dir = make_float3( 0.0f );
}
