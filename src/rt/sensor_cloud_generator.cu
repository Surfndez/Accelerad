#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2015 Nathaniel Jones
 * Massachusetts Institute of Technology
 */

#include <optix_world.h>
#include "optix_shader_common.h"

using namespace optix;

/* Program variables */
rtDeclareVariable(unsigned int,  do_irrad, , ); /* Calculate irradiance (-i) */

/* Contex variables */
rtBuffer<RayData, 2>             ray_buffer;
rtBuffer<PointDirection, 3>      seed_buffer;
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(rtObject,      top_irrad, , );
rtDeclareVariable(unsigned int,  point_cloud_ray_type, , );
rtDeclareVariable(unsigned int,  imm_irrad, , ) = 0u; /* Immediate irradiance (-I) */

/* OptiX variables */
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );

RT_PROGRAM void cloud_generator()
{
	PerRayData_point_cloud prd;

	// Init random state
	rand_state* state;
	init_rand(&state, launch_index.x + launch_dim.x * launch_index.y);

	uint3 index = make_uint3( launch_index, 0u );

	float tmin = ray_start( ray_buffer[launch_index].origin, RAY_START );
	float tmax;
	if ( imm_irrad ) {
		tmax = tmin;
		tmin = -tmin;
	} else {
		// Zero or negative aft clipping distance indicates infinity
		tmax = ray_buffer[launch_index].max;
		if (tmax <= FTINY) {
			tmax = RAY_END;
		}
	}

	Ray ray = make_Ray(ray_buffer[launch_index].origin, ray_buffer[launch_index].dir, point_cloud_ray_type, tmin, tmax);

	const unsigned int seeds = seed_buffer.size().z;
	unsigned int loop = 2u * seeds; // Prevent infinite looping
	while ( index.z < seeds && loop-- ) {
		clear(prd.backup);
#ifdef ANTIMATTER
		prd.mask = 0u;
		prd.inside = 0;
#endif

		// Trace the current ray
		if ( imm_irrad )
			rtTrace(top_irrad, ray, prd);
		else
			rtTrace(top_object, ray, prd);

		// Check for a valid result
		if ( isfinite( prd.result.pos ) && dot( prd.result.dir, prd.result.dir ) > FTINY ) { // NaN values will be false
			seed_buffer[index] = prd.result; // This could contain points on glass materials
			index.z++;
#ifndef AMBIENT_CELL
		} else {
			prd.result.pos = ray_buffer[launch_index].origin;
			prd.result.dir = ray_buffer[launch_index].dir;
#endif /* AMBIENT_CELL */
		}

#ifdef AMBIENT_CELL
		if (!(isfinite(prd.backup.pos) && dot(prd.backup.dir, prd.backup.dir) > FTINY)) // NaN values will be false
			break;

		// Prepare for next ray
		ray.origin = prd.backup.pos;
		ray.direction = reflect(ray.direction, prd.backup.dir);
		ray.tmin = RAY_START;// ray_start(ray.origin, ray.direction, prd.backup.dir, RAY_START);
#else /* AMBIENT_CELL */
		// Prepare for next ray
		ray.origin = prd.result.pos;

		float3 uz = normalize( prd.result.dir );
		float3 ux = getperpendicular(uz);
		float3 uy = normalize(cross(uz, ux));

		float zd = sqrtf( hiprand_uniform( state ) );
		float phi = 2.0f*M_PIf * hiprand_uniform( state );
		float xd = cosf(phi) * zd;
		float yd = sinf(phi) * zd;
		zd = sqrtf(1.0f - zd*zd);
		ray.direction = normalize( xd*ux + yd*uy + zd*uz );

		ray.tmin = ray_start(ray.origin, RAY_START);
#endif /* AMBIENT_CELL */
		ray.tmax = RAY_END;
	}

	// If outdoors, there are no bounces, but we need to prevent junk data
	while ( index.z < seeds ) {
		clear(seed_buffer[index]);
		index.z++;
	}
}

RT_PROGRAM void exception()
{
	const unsigned int code = rtGetExceptionCode();
	rtPrintf( "Caught exception 0x%X at launch index (%d,%d)\n", code, launch_index.x, launch_index.y );
	uint3 index = make_uint3(launch_index, seed_buffer.size().z - 1u); // record error to last segment
	seed_buffer[index].pos = exceptionToFloat3( code );
	seed_buffer[index].dir = make_float3( 0.0f );
#ifdef AMBIENT_CELL
	seed_buffer[index].cell = make_uint2(0);
#endif
}
