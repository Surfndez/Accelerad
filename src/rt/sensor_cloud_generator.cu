#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2015 Nathaniel Jones
 * Massachusetts Institute of Technology
 */

#include <optix_world.h>
#include "optix_shader_common.h"

using namespace optix;

/* Program variables */
rtDeclareVariable(unsigned int,  do_irrad, , ); /* Calculate irradiance (-i) */

/* Contex variables */
rtBuffer<RayData, 2>             ray_buffer;
rtBuffer<PointDirection, 3>      seed_buffer;
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(rtObject,      top_irrad, , );
rtDeclareVariable(unsigned int,  point_cloud_ray_type, , );
rtDeclareVariable(unsigned int,  imm_irrad, , ) = 0u; /* Immediate irradiance (-I) */

/* OptiX variables */
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );

RT_PROGRAM void cloud_generator()
{
	PerRayData_point_cloud prd;
	clear(prd.backup);

	// Init random state
	rand_state* state;
	init_rand(&state, launch_index.x + launch_dim.x * launch_index.y);

	uint3 index = make_uint3( launch_index, 0u );

	float tmin = ray_start( ray_buffer[launch_index].origin, RAY_START );
	float tmax;
	if ( imm_irrad ) {
		tmax = tmin;
		tmin = -tmin;
	} else {
		// Zero or negative aft clipping distance indicates infinity
		tmax = ray_buffer[launch_index].max;
		if (tmax <= FTINY) {
			tmax = RAY_END;
		}
	}

	Ray ray = make_Ray(ray_buffer[launch_index].origin, ray_buffer[launch_index].dir, point_cloud_ray_type, tmin, tmax);

	unsigned int seeds = seed_buffer.size().z;
	unsigned int loop = 2u * seeds; // Prevent infinite looping
	while ( index.z < seeds && loop-- ) {
		// Trace the current ray
		if ( imm_irrad )
			rtTrace(top_irrad, ray, prd);
		else
			rtTrace(top_object, ray, prd);

		// Check for a valid result
		if ( isfinite( prd.result.pos ) && dot( prd.result.dir, prd.result.dir ) > FTINY ) { // NaN values will be false
			seed_buffer[index] = prd.result; // This could contain points on glass materials
			index.z++;
		} else {
			prd.result.pos = ray_buffer[launch_index].origin;
			prd.result.dir = ray_buffer[launch_index].dir;
		}

		// Prepare for next ray
		ray.origin = prd.result.pos;
		//ray.direction = reflect( ray.direction, prd.result.dir );

		float3 uz = normalize( prd.result.dir );
		float3 ux = getperpendicular(uz);
		float3 uy = normalize(cross(uz, ux));

		float zd = sqrtf( hiprand_uniform( state ) );
		float phi = 2.0f*M_PIf * hiprand_uniform( state );
		float xd = cosf(phi) * zd;
		float yd = sinf(phi) * zd;
		zd = sqrtf(1.0f - zd*zd);
		ray.direction = normalize( xd*ux + yd*uy + zd*uz );
	}

	// If outdoors, there are no bounces, but we need to prevent junk data
	while ( index.z < seeds ) {
		clear(seed_buffer[index]);
		index.z++;
	}
}

RT_PROGRAM void exception()
{
	const unsigned int code = rtGetExceptionCode();
	rtPrintf( "Caught exception 0x%X at launch index (%d,%d)\n", code, launch_index.x, launch_index.y );
	uint3 index = make_uint3(launch_index, seed_buffer.size().z - 1u); // record error to last segment
	seed_buffer[index].pos = exceptionToFloat3( code );
	seed_buffer[index].dir = make_float3( 0.0f );
#ifdef AMBIENT_CELL
	seed_buffer[index].cell = make_uint2(0);
#endif
}
