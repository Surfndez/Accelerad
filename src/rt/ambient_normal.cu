#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2015 Nathaniel Jones
 * Massachusetts Institute of Technology
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include "optix_shader_common.h"
#include "optix_ambient_common.h"

using namespace optix;

#define threadIndex()	launch_index.y / stride
//#define threadIndex()	(launch_index.x + launch_dim.x * launch_index.y) / stride
#ifndef OLDAMB
#define CORRAL
#define hessrow(i)	hess_row_buffer[make_uint2(i, threadIndex())]
#define gradrow(i)	grad_row_buffer[make_uint2(i, threadIndex())]
#ifdef AMB_SAVE_MEM
#define prevrow(i)	amb_samp_buffer[make_uint2(i, threadIndex())]
#define corral_u(i)	corral_u_buffer[make_uint2(i, threadIndex())]
#define corral_d(i)	corral_d_buffer[make_uint2(i, threadIndex())]
#else /* AMB_SAVE_MEM */
#define ambsam(i,j)	amb_samp_buffer[make_uint3(i, j, threadIndex())]
#ifdef AMB_SUPER_SAMPLE
#define earr(i,j)	earr_buffer[make_uint3(i, j, threadIndex())]
#endif
#endif /* AMB_SAVE_MEM */

typedef struct {
	int	ns;		/* number of samples per axis */
	int	sampOK;		/* acquired full sample set? */
	float3	acoef;		/* division contribution coefficient */
	float3	acol;		/* accumulated color */
	float3	ux, uy;		/* tangent axis unit vectors */
} AMBHEMI;		/* ambient sample hemisphere */

typedef struct {
	float3 r_i, r_i1, e_i, rcp, rI2_eJ2;
	float I1, I2;
} FFTRI;		/* vectors and coefficients for Hessian calculation */
#else /* OLDAMB */
#define rprevrow(i)	rprevrow_buffer[make_uint2(i, threadIndex())]
#define bprevrow(i)	bprevrow_buffer[make_uint2(i, threadIndex())]
#endif /* OLDAMB */

/* Context variables */
rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(rtObject,     top_object, , );
#ifndef OLDAMB
rtDeclareVariable(unsigned int, shadow_ray_type, , );
#endif /* OLDAMB */
rtDeclareVariable(unsigned int, stride, , ) = 1u; /* Spacing between used threads in warp. */

//rtDeclareVariable(float,        specthresh, , ); /* This is the minimum fraction of reflection or transmission, under which no specular sampling is performed */
//rtDeclareVariable(float,        specjitter, , );

//rtDeclareVariable(float3,       ambval, , ); /* This is the final value used in place of an indirect light calculation */
//rtDeclareVariable(int,          ambvwt, , ); /* As new indirect irradiances are computed, they will modify the default ambient value in a moving average, with the specified weight assigned to the initial value given on the command and all other weights set to 1 */
//rtDeclareVariable(int,          ambounce, , ); /* Ambient bounces (ab) */
//rtDeclareVariable(int,          ambres, , ); /* Ambient resolution (ar) */
rtDeclareVariable(float,        ambacc, , ); /* Ambient accuracy (aa). This value will approximately equal the error from indirect illuminance interpolation */
rtDeclareVariable(int,          ambdiv, , ); /* Ambient divisions (ad) */
rtDeclareVariable(int,          ambssamp, , ); /* Ambient super-samples (as) */
rtDeclareVariable(float,        maxarad, , ); /* maximum ambient radius */
rtDeclareVariable(float,        minarad, , ); /* minimum ambient radius */
rtDeclareVariable(float,        avsum, , ); /* computed ambient value sum (log) */
rtDeclareVariable(unsigned int, navsum, , ); /* number of values in avsum */

rtDeclareVariable(float,        minweight, , ); /* minimum ray weight (lw) */
rtDeclareVariable(int,          maxdepth, , ); /* maximum recursion depth (lr) */

/* Material variables */
rtDeclareVariable(unsigned int, type, , ); /* The material type representing "plastic", "metal", or "trans" */
rtDeclareVariable(float3,       color, , ); /* The material color given by the rad file "plastic", "metal", or "trans" object */
rtDeclareVariable(float,        spec, , ); /* The material specularity given by the rad file "plastic", "metal", or "trans" object */
rtDeclareVariable(float,        rough, , ); /* The material roughness given by the rad file "plastic", "metal", or "trans" object */
rtDeclareVariable(float,        trans, , ) = 0.0f; /* The material transmissivity given by the rad file "trans" object */
rtDeclareVariable(float,        tspec, , ) = 0.0f; /* The material transmitted specular component given by the rad file "trans" object */

/* Program variables */
#ifndef OLDAMB
rtBuffer<optix::Matrix<3, 3>, 2> hess_row_buffer;
rtBuffer<float3, 2>              grad_row_buffer;
#ifdef AMB_SAVE_MEM
rtBuffer<AmbientSample, 2>       amb_samp_buffer;
rtBuffer<float2, 2>              corral_u_buffer;
rtBuffer<float, 2>               corral_d_buffer;
#else /* AMB_SAVE_MEM */
rtBuffer<AmbientSample, 3>       amb_samp_buffer;
#ifdef AMB_SUPER_SAMPLE
rtBuffer<float, 3>               earr_buffer;
#endif
#endif /* AMB_SAVE_MEM */
#else /* OLDAMB */
rtBuffer<float, 2>               rprevrow_buffer;
rtBuffer<float, 2>               bprevrow_buffer;
#endif /* OLDAMB */

/* OptiX variables */
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(PerRayData_ambient_record, prd, rtPayload, );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim, rtLaunchDim, );

/* Attributes */
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );


RT_METHOD int check_overlap( const float3& normal, const float3& hit );
#ifndef OLDAMB
RT_METHOD int plugaleak( const AmbientRecord* record, const float3& anorm, const float3& normal, const float3& hit, float ang );
RT_METHOD int doambient( float3 *rcol, optix::Matrix<2,3> *uv, float2 *ra, float2 *pg, float2 *dg, unsigned int *crlp, const float3& normal, const float3& hit );
RT_METHOD int ambsample(AMBHEMI *hp, AmbientSample *ap, const int& i, const int& j, const int& n, const float3& normal, const float3& hit);
#ifdef AMB_SAVE_MEM
RT_METHOD int samp_hemi(AMBHEMI *hp, float3 *rcol, float wt, optix::Matrix<2, 3> *uv, float2 *ra, float2 *pg, float2 *dg, unsigned int *crlp, const float3& normal, const float3& hit);
#else /* AMB_SAVE_MEM */
#ifdef AMB_SUPER_SAMPLE
RT_METHOD void getambdiffs(AMBHEMI *hp);
RT_METHOD void ambsupersamp(AMBHEMI *hp, int cnt, const float3& normal, const float3& hit);
#endif /* AMB_SUPER_SAMPLE */
RT_METHOD int samp_hemi( AMBHEMI *hp, float3 *rcol, float wt, const float3& normal, const float3& hit );
RT_METHOD void ambHessian( AMBHEMI *hp, optix::Matrix<2,3> *uv, float2 *ra, float2 *pg, const float3& normal, const float3& hit );
RT_METHOD void ambdirgrad( AMBHEMI *hp, const float3& u, const float3& v, float2 *dg, const float3& normal, const float3& hit );
RT_METHOD unsigned int ambcorral( AMBHEMI *hp, optix::Matrix<2,3> *uv, const float2& r, const float3& normal, const float3& hit );
#endif /* AMB_SAVE_MEM */
RT_METHOD float back_ambval( const AmbientSample *n1, const AmbientSample *n2, const AmbientSample *n3 );
RT_METHOD void comp_fftri( FFTRI *ftp, const AmbientSample *n0, const AmbientSample *n1, const float3& hit );
RT_METHOD optix::Matrix<3,3> compose_matrix( const float3& va, const float3& vb );
RT_METHOD optix::Matrix<3,3> comp_hessian( FFTRI *ftp, const float3& normal );
RT_METHOD float3 comp_gradient( FFTRI *ftp, const float3& normal );
RT_METHOD optix::Matrix<2,2> eigenvectors( optix::Matrix<2,3> *uv, float2 *ra, optix::Matrix<3,3> *hessian );
#else /* OLDAMB */
RT_METHOD float doambient( float3 *rcol, float3 *pg, float3 *dg, const float3& normal, const float3& hit );
RT_METHOD int divsample( AMBSAMP  *dp, AMBHEMI  *h, const float3& hit, const float3& normal );
RT_METHOD void inithemi( AMBHEMI  *hp, const float3& ac, const float3& normal );
//RT_METHOD void comperrs( AMBSAMP *da, AMBHEMI *hp );
//RT_METHOD int ambcmp( const void *p1, const void *p2 );
#endif /* OLDAMB */
//RT_METHOD float2 multisamp2(float r);
//RT_METHOD int ilhash(int3 d);

RT_PROGRAM void any_hit_ambient_glass()
{
	rtIgnoreIntersection();
}

RT_PROGRAM void closest_hit_ambient()
{
	float3 ffnormal = -ray.direction;
	float3 hit_point = ray.origin + t_hit * ray.direction;

	// Check that this is not covered by parent
	if ( check_overlap( ffnormal, hit_point ) )
		return;

	/* compute weight */
	//float weight = 1.0f;
	//for (int i = prd.ambient_depth; i-- > 0; ) //TODO start at i-1?
	//	weight *= AVGREFL;
	//if (prd.weight < 0.1f * weight)	/* heuristic override */
	//	weight = 1.25f * prd.weight;
	float3 acol = make_float3( AVGREFL );
#ifdef DAYSIM_COMPATIBLE
	daysimSet(prd.dc, 0.0f);
#endif
#ifndef OLDAMB
	optix::Matrix<2,3> uv;
	float2 pg = make_float2( 0.0f );
	float2 dg = make_float2( 0.0f );
	float2 rad = make_float2( 0.0f );
	unsigned int corral = 0u;

	/* compute ambient */
	int i = doambient( &acol, &uv, &rad, &pg, &dg, &corral, ffnormal, hit_point );
	if ( !i || rad.x <= FTINY )	/* no Hessian or zero radius */
		return;
#else
	float3 pg = make_float3( 0.0f );
	float3 dg = make_float3( 0.0f );

	/* compute ambient */
	float rad = doambient( &acol, &pg, &dg, ffnormal, hit_point );
	if ( rad <= FTINY )
		return;
#endif

	acol *= 1.0f / AVGREFL;		/* undo assumed reflectance */

	//if (rn != r->ron)
	//	extambient(acol, &amb, r->rop, rn);	/* texture */

	// pass the color back up the tree
	prd.result.pos = hit_point;
	prd.result.val = acol;
#ifndef OLDAMB
	prd.result.gpos = pg;
	prd.result.gdir = dg;
	prd.result.rad = rad;
	prd.result.ndir = encodedir( ffnormal );
	prd.result.udir = encodedir( uv.getRow(0) );
	prd.result.corral = corral;
#else
	prd.result.dir = ffnormal;
	prd.result.gpos = pg;
	prd.result.gdir = dg;
	prd.result.rad = rad;
#endif
	//prd.result.lvl = lvl;
	//prd.result.weight = weight;
#ifdef DAYSIM_COMPATIBLE
	daysimScale(prd.dc, 1.0f / AVGREFL); // TODO Scaling should be done before extambient if textured
#endif
}

// based on sumambient from ambient.c
RT_METHOD int check_overlap( const float3& normal, const float3& hit )
{
	// Check that there is a parent
	if ( !prd.parent )
		return( 0 );

#ifndef OLDAMB
	/* Direction test using unperturbed normal */
	float3 w = decodedir( prd.parent->ndir );
	float d = dot( w, normal );
	if ( d <= 0.0f )		/* >= 90 degrees */
		return( 0 );

	float delta_r2 = 2.0f - 2.0f * d;	/* approx. radians^2 */
	const float minangle = 10.0f * M_PIf / 180.0f;
	float maxangle = minangle + ambacc;
					/* adjust maximum angle */
	//if (at->alist != NULL && (at->alist->lvl <= al) & (r->rweight < 0.6))
	//	maxangle = (maxangle - PI/2.)*pow(r->rweight,0.13) + PI/2.;
	if ( delta_r2 >= maxangle * maxangle )
		return( 0 );

	/* Modified ray behind test */
	float3 ck0 = hit - prd.parent->pos;
	d = dot( ck0, w );
	if ( d < -minarad * ambacc - 0.001f )
		return( 0 );
	d /= prd.parent->rad.x;
	float delta_t2 = d * d;
	if ( delta_t2 >= ambacc * ambacc )
		return( 0 );
	
	/* Elliptical radii test based on Hessian */
	float3 u = decodedir( prd.parent->udir );
	float3 v = cross( w, u );
	float uu, vv;
	d = (uu = dot( ck0, u )) / prd.parent->rad.x;
	delta_t2 += d * d;
	d = (vv = dot( ck0, v )) / prd.parent->rad.y;
	delta_t2 += d * d;
	if ( delta_t2 >= ambacc * ambacc )
		return( 0 );
	
	/* Test for potential light leak */
	if ( prd.parent->corral && plugaleak( prd.parent, w, normal, hit, atan2f( vv, uu ) ) )
		return( 0 );
	return( 1 );
#else /* OLDAMB */
	/* Ambient radius test. */
	float3 ck0 = prd.parent->pos - hit;
	float rad = prd.parent->rad;
	float e1 = optix::dot( ck0, ck0 ) / ( rad * rad );
	float acc = ambacc * ambacc * 1.21f;
	if ( e1 > acc )
		return( 0 );

	/* Direction test using closest normal. */
	float d = optix::dot( prd.parent->dir, normal );
	//if (rn != r->ron) {
	//	rn_dot = DOT(av->dir, rn);
	//	if (rn_dot > 1.0-FTINY)
	//		rn_dot = 1.0-FTINY;
	//	if (rn_dot >= d-FTINY) {
	//		d = rn_dot;
	//		rn_dot = -2.0;
	//	}
	//}
	float e2 = (1.0f - d) * prd.result.weight;
	return( e2 < 0.0f || e1 + e2 <= acc );
#endif /* OLDAMB */
}

#ifndef OLDAMB
/* Plug a potential leak where ambient cache value is occluded */
RT_METHOD int plugaleak( const AmbientRecord* record, const float3& anorm, const float3& normal, const float3& hit, float ang )
{
	const float cost70sq = 0.1169778f;	/* cos(70deg)^2 */
	float2 t;

	ang += 2.0f * M_PIf * (ang < 0);			/* check direction flags */
	if ( !(record->corral>>(int)( ang * 16.0f * M_1_PIf ) & 1) )
		return(0);
	/*
	 * Generate test ray, targeting 20 degrees above sample point plane
	 * along surface normal from cache position.  This should be high
	 * enough to miss local geometry we don't really care about.
	 */
	float3 vdif = record->pos - hit;
	float normdot = dot( anorm, normal );
	float ndotd = dot( vdif, normal );
	float nadotd = dot( vdif, anorm );
	float a = normdot * normdot - cost70sq;
	float b = 2.0f * ( normdot * ndotd - nadotd * cost70sq );
	float c = ndotd * ndotd - dot( vdif, vdif ) * cost70sq;
	if ( quadratic( &t, a, b, c ) != 2 )
		return(1);			/* should rarely happen */
	if ( t.y <= FTINY )
		return(0);			/* should fail behind test */

	float3 rdir = vdif + anorm * t.y;	/* further dist. > plane */
	Ray shadow_ray = make_Ray( ray.origin, normalize( rdir ), shadow_ray_type, RAY_START, length( rdir ) );
	PerRayData_shadow shadow_prd;
	shadow_prd.target = 0;
	shadow_prd.result = make_float3( 1.0f );
#ifdef ANTIMATTER
	shadow_prd.mask = 0u; //TODO check if we are in an antimatter volume
	shadow_prd.inside = 0;
#endif
	rtTrace( top_object, shadow_ray, shadow_prd );
	return( dot( shadow_prd.result, shadow_prd.result ) < 1.0f );	/* check for occluder */
}

RT_METHOD int doambient( float3 *rcol, optix::Matrix<2,3> *uv, float2 *ra, float2 *pg, float2 *dg, unsigned int *crlp, const float3& normal, const float3& hit )
{
	const float wt = prd.result.weight;
	AMBHEMI hp;

	if (!samp_hemi(&hp, rcol, wt,
#ifdef AMB_SAVE_MEM
		uv, ra, pg, dg, crlp,
#endif
		normal, hit))
		return(0);

	/* clear return values */
	//if (u != NULL)
	//	*u = make_float3( 0.0f );
	//if (v != NULL)
	//	*v = make_float3( 0.0f );
	//if (ra != NULL)
	//	*ra = make_float2( 0.0f );
	//if (pg != NULL)
	//	*pg = make_float2( 0.0f );
	//if (dg != NULL)
	//	*dg = make_float2( 0.0f );
	//if (crlp != NULL)
	//	*crlp = 0u;
	//if (hp == NULL)			/* sampling falure? */
	//	return(0);

	if ((ra == NULL) & (pg == NULL) & (dg == NULL) || (hp.sampOK < 0) | (hp.ns < 6)) { /* Hessian not requested/possible */
		return(-1);		/* value-only return value */
	}
#ifndef AMB_SAVE_MEM
	float	d, K;
	if ((d = bright(*rcol)) > FTINY) {	/* normalize Y values */
		d = 0.99f * ( hp.ns * hp.ns ) / d;
		K = 0.01f;
	} else {			/* or fall back on geometric Hessian */
		K = 1.0f;
		pg = NULL;
		dg = NULL;
		crlp = NULL;
	}
				/* relative Y channel from here on... */
	for (int i = 0; i < hp.ns; i++)
		for (int j = 0; j < hp.ns; j++) {
			AmbientSample *ap = &ambsam(i, j);
			ap->v.y = bright(ap->v) * d + K;
		}

	//if (uv == NULL)			/* make sure we have axis pointers */
	//	uv = my_uv;
					/* compute radii & pos. gradient */
	ambHessian( &hp, uv, ra, pg, normal, hit );

	if (dg != NULL)			/* compute direction gradient */
		ambdirgrad( &hp, uv->getRow(0), uv->getRow(1), dg, normal, hit );

	if (ra != NULL) {		/* scale/clamp radii */
		if (pg != NULL) {
			if ( ra->x * (d = fabsf( pg->x ) ) > 1.0f )
				ra->x = 1.0f / d;
			if ( ra->y * (d = fabsf( pg->y ) ) > 1.0f )
				ra->y = 1.0f / d;
			if ( ra->x > ra->y )
				ra->x = ra->y;
		}
		if ( ra->x < minarad ) {
			ra->x = minarad;
			if ( ra->y < minarad )
				ra->y = minarad;
		}
		*ra *= 1.0f / sqrtf( wt );
		if ( ra->y > 2.0f * ra->x )
			ra->y = 2.0f * ra->x;
		if ( ra->y > maxarad ) {
			ra->y = maxarad;
			if ( ra->x > maxarad )
				ra->x = maxarad;
		}
#ifdef CORRAL
					/* flag encroached directions */
		if ( (wt >= 0.89f * AVGREFL) & (crlp != NULL) )
			*crlp = ambcorral( &hp, uv, *ra * ambacc, normal, hit );
#endif /* CORRAL */
		if (pg != NULL) {	/* cap gradient if necessary */
			d = pg->x*pg->x * ra->x*ra->x + pg->y*pg->y * ra->y*ra->y;
			if ( d > 1.0f ) {
				*pg *= 1.0f / sqrtf(d);
			}
		}
	}
#endif /* AMB_SAVE_MEM */
	//free(hp);			/* clean up and return */
	return(1);
}

/* sample indirect hemisphere, based on samp_hemi in ambcomp.c */
RT_METHOD int samp_hemi(
	AMBHEMI *hp,
	float3 *rcol,
	float wt,
#ifdef AMB_SAVE_MEM
	optix::Matrix<2, 3> *uv,
	float2 *ra,
	float2 *pg,
	float2 *dg,
	unsigned int *crlp,
#endif /* AMB_SAVE_MEM */
	const float3& normal,
	const float3& hit
)
{
	float	d;
	int	j;
					/* set number of divisions */
#ifndef AMB_PARALLEL
	if (ambacc <= FTINY && wt > (d = 0.8f * fmaxf(*rcol) * wt / (ambdiv*minweight))) //TODO second wt should be radiance ray weight
		wt = d;			/* avoid ray termination */
#endif
	int n = sqrtf(ambdiv * wt) + 0.5f;
	int i = 1 + 5 * (ambacc > FTINY);	/* minimum number of samples */
	if (n < i)
		n = i;
					/* allocate sampling array */
	hp->ns = n;
	hp->acol = make_float3( 0.0f );
	hp->sampOK = 0;
					/* assign coefficient */
	hp->acoef = *rcol;
	d = 1.0f / (n*n);
	hp->acoef *= d;
					/* make tangent plane axes */
#ifdef AMB_PARALLEL
	hp->ux = getperpendicular(normal);
#else
	hp->ux = getperpendicular( normal, prd.state );
#endif
	hp->uy = cross( normal, hp->ux );

#ifdef AMB_SAVE_MEM
	AmbientSample current, prev;

	/* ambHessian from ambcomp.c */
	optix::Matrix<3,3> hessian;
	float3 gradient = make_float3( 0.0f );
	hessian.setRow( 0, gradient ); // Set zero matrix
	hessian.setRow( 1, gradient );
	hessian.setRow( 2, gradient );
	optix::Matrix<3,3> hessianY;
	float3 gradientY = make_float3( 0.0f );
	hessianY.setRow( 0, gradientY ); // Set zero matrix
	hessianY.setRow( 1, gradientY );
	hessianY.setRow( 2, gradientY );

	FFTRI fftr;
					/* be sure to assign unit vectors */
	uv->setRow( 0, hp->ux );
	uv->setRow( 1, hp->uy );

	/* ambdirgrad from ambcomp.c */
	float3 dgsum = make_float3( 0.0f );	/* sum values times -tan(theta) */

#ifdef CORRAL
	/* ambcorral from ambcomp.c */
	const float max_d = 1.0f / ( minarad * ambacc + 0.001f );
	const float ang_res = M_PI_2f / hp->ns;
	const float ang_step = ang_res / ( (int)( 16.0f * M_1_PIf * ang_res ) + 1.01f );
	float avg_d = 0.0f;
	unsigned int corral_count = 0u;
#endif /* CORRAL */

					/* sample divisions */
	for ( i = 0; i < hp->ns; i++ ) {
		optix::Matrix<3,3> hesscol;	/* compute first vertical edge */
		float3 gradcol;

	    for ( j = 0; j < hp->ns; j++ ) {
			hp->sampOK += ambsample(hp, &current, i, j, 0, normal, hit);
			current.v.y = bright( current.v ); /* relative Y channel from here on... */

			/* ambHessian from ambcomp.c */
			if ( i ) {
				if ( j ) {
					optix::Matrix<3,3> hessdia;	/* compute triangle contributions */
					float3 graddia;
					optix::Matrix<3,3> hesstmp;
					float3 gradtmp;

					float backg = back_ambval( &prevrow(j - 1), &prevrow(j), &prev );
								/* diagonal (inner) edge */
					comp_fftri(&fftr, &prevrow(j), &prev, hit);
					hessdia = comp_hessian( &fftr, normal );
					hessian += ( hesstmp = hessrow(j - 1) + hessdia - hesscol );
					hessianY += backg * hesstmp;
					graddia = comp_gradient( &fftr, normal );
					gradient += ( gradtmp = gradrow(j - 1) + graddia - gradcol );
					gradientY += backg * gradtmp;
								/* initialize edge in next row */
					comp_fftri( &fftr, &current, &prev, hit );
					hessrow(j - 1) = comp_hessian( &fftr, normal );
					gradrow(j - 1) = comp_gradient( &fftr, normal );
								/* new column edge & paired triangle */
					backg = back_ambval( &current, &prev, &prevrow(j) );
					comp_fftri( &fftr, &prevrow(j), &current, hit );
					hesscol = comp_hessian( &fftr, normal );
					hessian += ( hesstmp = hessrow(j - 1) - hessdia + hesscol );
					hessianY += backg * hesstmp;
					gradcol = comp_gradient( &fftr, normal );
					gradient += ( gradtmp = gradrow(j - 1) - graddia + gradcol );
					gradientY += backg * gradtmp;
					if ( i < hp->ns-1 ) {
						hessrow(j - 1) *= -1.0f;
						gradrow(j - 1) = -gradrow(j - 1);
					}

#ifdef CORRAL
					/* ambcorral from ambcomp.c */
					if ( ( i < hp->ns * 3 / 4 ) && ( i >= hp->ns>>2 ) )
						if ( ( j < hp->ns * 3 / 4 ) && ( j >= hp->ns>>2 ) )
							avg_d += current.d;
#endif /* CORRAL */
				} else {
					comp_fftri(&fftr, &prevrow(0), &current, hit);
					hesscol = comp_hessian( &fftr, normal );
					gradcol = comp_gradient( &fftr, normal );
				}
			} else if ( j ) {
					/* compute first row of edges */
				comp_fftri( &fftr, &prev, &current, hit );
				hessrow(j - 1) = comp_hessian(&fftr, normal);
				gradrow(j - 1) = comp_gradient(&fftr, normal);
			}

			/* ambdirgrad from ambcomp.c */
					/* use vector for azimuth + 90deg */
			const float3 vd = current.p - hit;
					/* brightness over cosine factor */
			const float gfact = current.v.y / dot( normal, vd );
					/* sine = proj_radius/vd_length */
			dgsum += vd * gfact;

			if (j)
				prevrow(j - 1) = prev;
			else
				prevrow(hp->ns - 1) = prev;
			prev = current;

#ifdef CORRAL
			/* ambcorral from ambcomp.c */
			if ( !i || !j || i == hp->ns - 1 || j == hp->ns - 1 ) {
				if ( ( current.d <= FTINY ) | ( current.d >= max_d ) )
					continue;	/* too far or too near */
				corral_u(corral_count) = *uv * vd;
				corral_d(corral_count++) = current.d * current.d;
			}
#endif /* CORRAL */
		}
	}
#else /* AMB_SAVE_MEM */
					/* sample divisions */
	for (i = hp->ns; i--; )
	    for (j = hp->ns; j--; )
			hp->sampOK += ambsample(hp, &ambsam(i, j), i, j, 0, normal, hit);
#endif /* AMB_SAVE_MEM */
	*rcol = hp->acol;

	if (!hp->sampOK) {		/* utter failure? */
		return( 0 );
	}
	if (hp->sampOK < hp->ns * hp->ns) {
		hp->sampOK *= -1;	/* soft failure */
		return( 1 );
	}

#ifdef AMB_SAVE_MEM
	/* doambient from ambcomp.c */
	if ((d = bright(*rcol)) > FTINY) {	/* normalize Y values */
		d = 0.99f * ( hp->ns * hp->ns ) / d;
		hessian = d * hessianY + 0.01f * hessian;
		gradient = d * gradientY + 0.01f * gradient;
	} else {
		pg = dg = NULL;
		crlp = NULL;
	}

	/* ambHessian from ambcomp.c */
	optix::Matrix<2,2> ab;
	if ( ra )			/* extract eigenvectors & radii */
		ab = eigenvectors( uv, ra, &hessian );

	/* ambHessian from ambcomp.c */
	if ( pg )
		*pg = *uv * gradient;

	/* ambdirgrad from ambcomp.c */
	if ( dg ) {
		optix::Matrix<2,2> rotate;
		rotate[0] = rotate[3] = 0.0f;
		rotate[1] = -1.0f;
		rotate[2] = 1.0f;
		*dg = rotate * *uv * dgsum / (hp->ns*hp->ns);
	}

	/* ambcorral from ambcomp.c */
	if ( ra ) {
		if ( pg ) {
			if ( ra->x * (d = fabsf( pg->x ) ) > 1.0f )
				ra->x = 1.0f / d;
			if ( ra->y * (d = fabsf( pg->y ) ) > 1.0f )
				ra->y = 1.0f / d;
			if ( ra->x > ra->y )
				ra->x = ra->y;
		}
		if ( ra->x < minarad ) {
			ra->x = minarad;
			if ( ra->y < minarad )
				ra->y = minarad;
		}
		*ra *= 1.0f / sqrtf( prd.result.weight );
		if ( ra->y > 2.0f * ra->x )
			ra->y = 2.0f * ra->x;
		if ( ra->y > maxarad ) {
			ra->y = maxarad;
			if ( ra->x > maxarad )
				ra->x = maxarad;
		}

#ifdef CORRAL
		if ( crlp ) {
			unsigned int flgs = 0u;
			const float2 r = *ra * ambacc;
			avg_d *= 4.0f / ( hp->ns * hp->ns );
			if ( ( hp->ns >= 12 ) && ( avg_d * r.x < 1.0f )	&& ( avg_d < max_d ) ) {
						/* else circle around perimeter */
				for ( i = 0; i < corral_count; i++ ) {
					float2 u = ab * corral_u(i);
					if ( ( r.x*r.x * u.x*u.x + r.y*r.y * u.y*u.y ) * corral_d(i) <= dot( u, u ) )
						continue;	/* occluder outside ellipse */
					float ang = atan2f( u.y, u.x );	/* else set direction flags */
					for ( float a1 = ang - ang_res; a1 <= ang + ang_res; a1 += ang_step )
						flgs |= 1L<<(int)( 16.0f * M_1_PIf * ( a1 + 2.0f * M_PIf * ( a1 < 0.0f ) ) );
				}
						/* add low-angle incident (< 20deg) */
				if ( fabsf( dot( ray.direction, normal ) ) <= 0.342f ) {
					const float2 u = *uv * ray.direction;
					if ( ( r.x*r.x * u.x*u.x + r.y*r.y * u.y*u.y ) > t_hit * t_hit ) {
						float ang = atan2f( -u.y, -u.x );
						ang += 2.0f * M_PIf * ( ang < 0.0f );
						ang *= 16.0f * M_1_PIf;
						if ( ( ang < 0.5f ) | ( ang >= 31.5f ) )
							flgs |= 0x80000001;
						else
							flgs |= 3L<<(int)( ang - 0.5f );
					}
				}
				*crlp = flgs;
			}
		}
#endif /* CORRAL */

		if ( pg ) {	/* cap gradient if necessary */
			d = pg->x*pg->x * ra->x*ra->x + pg->y*pg->y * ra->y*ra->y;
			if ( d > 1.0f )
				*pg *= 1.0f / sqrtf(d);
		}
	}
#else /* AMB_SAVE_MEM */
#ifdef AMB_SUPER_SAMPLE
	n = ambssamp * wt + 0.5f;
	if (n > 8) {			/* perform super-sampling? */
		ambsupersamp(hp, n, normal, hit);
		*rcol = hp->acol;
	}
#endif
#endif /* AMB_SAVE_MEM */

	return( 1 );			/* all is well */
}

RT_METHOD int ambsample(AMBHEMI *hp, AmbientSample *ap, const int& i, const int& j, const int& n, const float3& normal, const float3& hit)
{
#ifdef AMB_PARALLEL
	if (!n) {
		if (ap->d == -1.0f) // An exception occurred
			rtThrow((int)(ap->v.x));
		if (ap->d == 0.0f) // No exception, but bad data
			return(0);

		ap->v *= hp->acoef;	/* apply coefficient */
		hp->acol += ap->v;	/* add to our sum */
#ifdef DAYSIM_COMPATIBLE
		DaysimCoef sample_dc = make_uint3(0, i + hp->ns * j, prd.dc.z);
		daysimAddScaled(prd.dc, sample_dc, hp->acoef.x);
#endif
#ifdef RAY_COUNT
		prd.result.ray_count += ap->ray_count;
#endif
#ifdef HIT_COUNT
		prd.result.hit_count += ap->hit_count;
#endif
		return(1);
	}
#endif /* AMB_PARALLEL */
#if defined AMB_SUPER_SAMPLE || !defined AMB_PARALLEL
	PerRayData_radiance new_prd;
	float b2;
					/* generate hemispherical sample */
					/* ambient coefficient for weight */
	if (ambacc > FTINY)
		b2 = AVGREFL; // Reusing this variable
	else
		b2 = fmaxf(hp->acoef);
	new_prd.weight = prd.result.weight * b2;
	if (new_prd.weight < minweight) //if (rayorigin(&ar, AMBIENT, r, ar.rcoef) < 0)
		return(0);
	//if (ambacc > FTINY) {
	//	rcoef *= h->acoef;
	//	rcoef *= 1.0f / AVGREFL; // This all seems unnecessary
	//}
	//hlist[0] = hp->rp->rno;
	//hlist[1] = j;
	//hlist[2] = i;
	//multisamp(spt, 2, urand(ilhash(hlist,3)+n));
	float2 spt = make_float2( hiprand_uniform( prd.state ), hiprand_uniform( prd.state ) );
	if (!n) /* avoid coincident samples */
		spt = 0.1f + 0.8f * spt;
	SDsquare2disk( spt, (j+spt.y) / hp->ns, (i+spt.x) / hp->ns );
	float zd = sqrtf( 1.0f - dot( spt, spt ) );
	float3 rdir = normalize( spt.x*hp->ux + spt.y*hp->uy + zd*normal );
	//dimlist[ndims++] = AI(hp,i,j) + 90171;

	new_prd.depth = prd.result.lvl + 1;//prd.depth + 1;
	new_prd.ambient_depth = prd.result.lvl + 1;//prd.ambient_depth + 1;
	//new_prd.seed = prd.seed;//lcg( prd.seed );
	new_prd.state = prd.state;
#ifdef ANTIMATTER
	new_prd.mask = 0u; //TODO check if we are in an antimatter volume
	new_prd.inside = 0;
#endif
#ifdef DAYSIM_COMPATIBLE
	new_prd.dc = daysimNext(prd.dc);
#endif
	setupPayload(new_prd);
	Ray amb_ray = make_Ray( hit, rdir, radiance_ray_type, ray_start( hit, rdir, normal, RAY_START ), RAY_END );
	rtTrace(top_object, amb_ray, new_prd);
#ifdef RAY_COUNT
	prd.result.ray_count += new_prd.ray_count;
#endif
#ifdef HIT_COUNT
	prd.result.hit_count += new_prd.hit_count;
#endif

	//ndims--;
	if ( isnan( new_prd.result ) ) // TODO How does this happen?
		return(0);
	if ( new_prd.distance <= FTINY )
		return(0);		/* should never happen */
	new_prd.result *= hp->acoef;	/* apply coefficient */
	if (!n || new_prd.distance * ap->d < 1.0f )		/* new/closer distance? */
		ap->d = 1.0f / new_prd.distance;
	if (!n) {			/* record first vertex & value */
		if ( new_prd.distance > 10.0f * maxarad ) // 10 * thescene.cusize
			new_prd.distance = 10.0f * maxarad;
		ap->p = hit + rdir * new_prd.distance;
		ap->v = new_prd.result; // only one AmbientSample, otherwise would need +=
#ifdef DAYSIM_COMPATIBLE
		daysimAddScaled(prd.dc, new_prd.dc, hp->acoef.x);
#endif
#ifdef AMB_SUPER_SAMPLE
	} else {			/* else update recorded value */
		hp->acol -= ap->v;
		zd = 1.0f / (n+1);
		new_prd.result *= zd;
		zd *= n;
		ap->v *= zd;
		ap->v += new_prd.result;
		// TODO daysim compatibility
#endif
	}
	hp->acol += ap->v;	/* add to our sum */
#endif /* AMB_SUPER_SAMPLE || !AMB_PARALLEL */
	return(1);
}

#ifdef AMB_SUPER_SAMPLE
/* Estimate errors based on ambient division differences */
RT_METHOD void getambdiffs(AMBHEMI *hp)
{
	/* compute squared neighbor diffs */
	for (unsigned int i = 0u; i < hp->ns; i++)
		for (unsigned int j = 0u; j < hp->ns; j++) {
			earr(i, j) = 0.0f;
			float b = bright(ambsam(i, j).v);
			if (i) {		/* from above */
				float d2 = b - bright(ambsam(i - 1, j).v);
				d2 *= d2;
				earr(i, j) += d2;
				earr(i - 1, j) += d2;
			}
			if (!j) continue;
			/* from behind */
			float d2 = b - bright(ambsam(i, j - 1).v);
			d2 *= d2;
			earr(i, j) += d2;
			earr(i, j - 1) += d2;
			if (!i) continue;
			/* diagonal */
			d2 = b - bright(ambsam(i - 1, j - 1).v);
			d2 *= d2;
			earr(i, j) += d2;
			earr(i - 1, j - 1) += d2;
		}

	/* correct for number of neighbors */
	earr(0, 0) *= 8.0f / 3.0f;
	earr(0, hp->ns - 1) *= 8.0f / 3.0f;
	earr(hp->ns - 1, 0) *= 8.0f / 3.0f;
	earr(hp->ns - 1, hp->ns - 1) *= 8.0f / 3.0f;
	for (unsigned int i = 1u; i < hp->ns - 1; i++) {
		earr(i, 0) *= 8.0f / 5.0f;
		earr(i, hp->ns - 1) *= 8.0f / 5.0f;
		earr(0, i) *= 8.0f / 5.0f;
		earr(hp->ns - 1, i) *= 8.0f / 5.0f;
	}
}

/* Perform super-sampling on hemisphere (introduces bias) */
RT_METHOD void ambsupersamp(AMBHEMI *hp, int cnt, const float3& normal, const float3& hit)
{
	getambdiffs(hp);
	float e2rem = 0.0f;
	float *ep = &earr(0, 0);

	/* accumulate estimated variances */
	for (unsigned int i = hp->ns * hp->ns; i--; )
		e2rem += *ep++;

	/* perform super-sampling */
	ep = &earr(0, 0);
	for (unsigned int i = 0u; i < hp->ns; i++)
		for (unsigned int j = 0u; j < hp->ns; j++) {
			if (e2rem <= FTINY)
				return;	/* nothing left to do */
			int nss = *ep / e2rem * cnt + hiprand_uniform( prd.state );
			for (int n = 1; n <= nss && ambsample(hp, &ambsam(i, j), i, j, n, normal, hit); n++)
				--cnt;
			e2rem -= *ep++;		/* update remainder */
		}
}
#endif /* AMB_SUPER_SAMPLE */

/* Return brightness of farthest ambient sample */
RT_METHOD float back_ambval( const AmbientSample *n1, const AmbientSample *n2, const AmbientSample *n3 )
{
	if (n1->d <= n2->d) {
		if (n1->d <= n3->d)
			return(n1->v.y);
		return(n3->v.y);
	}
	if (n2->d <= n3->d)
		return(n2->v.y);
	return(n3->v.y);
}

/* Compute vectors and coefficients for Hessian/gradient calcs */
RT_METHOD void comp_fftri( FFTRI *ftp, const AmbientSample *n0, const AmbientSample *n1, const float3& hit )
{
	ftp->r_i = n0->p - hit;
	ftp->r_i1 = n1->p - hit;
	ftp->e_i = n1->p - n0->p;
	ftp->rcp = cross( ftp->r_i, ftp->r_i1 );
	const float rdot_cp = 1.0f / dot( ftp->rcp, ftp->rcp );
	const float dot_e = dot( ftp->e_i, ftp->e_i );
	const float dot_er = dot( ftp->e_i, ftp->r_i );
	const float rdot_r = 1.0f / dot( ftp->r_i, ftp->r_i );
	const float rdot_r1 = 1.0f / dot( ftp->r_i1, ftp->r_i1 );
	ftp->I1 = acosf( dot( ftp->r_i, ftp->r_i1 ) * sqrtf( rdot_r * rdot_r1 ) ) * sqrtf( rdot_cp );
	ftp->I2 = ( dot( ftp->e_i, ftp->r_i1 ) * rdot_r1 - dot_er * rdot_r + dot_e * ftp->I1 ) * 0.5f * rdot_cp;
	const float J2 =  ( 0.5f * ( rdot_r - rdot_r1 ) - dot_er * ftp->I2 ) / dot_e;
	ftp->rI2_eJ2 = ftp->I2 * ftp->r_i + J2 * ftp->e_i;
}

/* Compose 3x3 matrix from two vectors */
RT_METHOD optix::Matrix<3,3> compose_matrix( const float3& va, const float3& vb )
{
	optix::Matrix<3,3> mat;
	mat.setRow( 0, va * vb.x + vb * va.x );
	mat.setRow( 1, va * vb.y + vb * va.y );
	mat.setRow( 2, va * vb.z + vb * va.z );
	//mat += mat.transpose();
	return mat;
}

/* Compute partial 3x3 Hessian matrix for edge */
RT_METHOD optix::Matrix<3,3> comp_hessian( FFTRI *ftp, const float3& normal )
{
					/* compute intermediate coefficients */
	float d1 = 1.0f / dot( ftp->r_i, ftp->r_i );
	float d2 = 1.0f / dot( ftp->r_i1, ftp->r_i1 );
	float d3 = 1.0f / dot( ftp->e_i, ftp->e_i );
	float d4 = dot( ftp->e_i, ftp->r_i );
	const float I3 = ( dot( ftp->e_i, ftp->r_i1 ) * d2 * d2 - d4 * d1 * d1 + 3.0f / d3 * ftp->I2 ) / ( 4.0f * dot( ftp->rcp, ftp->rcp ) );
	const float J3 = 0.25f * d3 * ( d1 * d1 - d2 * d2 ) - d4 * d3 * I3;
	const float K3 = d3 * ( ftp->I2 - I3 / d1 - 2.0f * d4 * J3);
					/* intermediate matrices */
	const float3 ncp = cross( normal, ftp->e_i );
	const optix::Matrix<3,3> m1 = compose_matrix( ncp, ftp->rI2_eJ2 );
	const optix::Matrix<3,3> m2 = compose_matrix( ftp->r_i, ftp->r_i );
	const optix::Matrix<3,3> m3 = compose_matrix( ftp->e_i, ftp->e_i );
	const optix::Matrix<3,3> m4 = compose_matrix( ftp->r_i, ftp->e_i );
	d1 = dot( normal, ftp->rcp );
	d2 = -d1 * ftp->I2;
	d1 *= 2.0f;
					/* final matrix sum */
	optix::Matrix<3,3> hess = m1 + d1 * ( I3 * m2 + K3 * m3 + 2.0f * J3 * m4 );
	hess += d2 * Matrix<3,3>::identity();
	hess *= -M_1_PIf;
	return hess;
}

/* Compute partial displacement form factor gradient for edge */
RT_METHOD float3 comp_gradient( FFTRI *ftp, const float3& normal )
{
	const float f1 = 2.0f * dot( normal, ftp->rcp );
	const float3 ncp = cross( normal, ftp->e_i );
	return ( 0.5f * M_1_PIf ) * ( ftp->I1 * ncp + f1 * ftp->rI2_eJ2 );
}

/* Compute anisotropic radii and eigenvector directions */
RT_METHOD optix::Matrix<2,2> eigenvectors( optix::Matrix<2,3> *uv, float2 *ra, optix::Matrix<3,3> *hessian )
{
					/* project Hessian to sample plane */
	const optix::Matrix<2,2> hess2 = *uv * *hessian * uv->transpose();
					/* compute eigenvalue(s) */
	float2 evalue;
	const unsigned int i = quadratic( &evalue, 1.0f, -hess2[0] - hess2[3], hess2[0] * hess2[3] - hess2[1] * hess2[2] );
	//if (i == 1u)			/* double-root (circle) */
	//	evalue.y = evalue.x;
	if (!i || ((evalue.x = fabsf(evalue.x)) <= FTINY*FTINY) | ((evalue.y = fabsf(evalue.y)) <= FTINY*FTINY) ) {
		*ra = make_float2( maxarad );
		return optix::Matrix<2,2>::identity();
	}
	float slope1;
	if ( evalue.x > evalue.y ) {
		*ra = sqrtf( sqrtf ( 4.0f / evalue ) );
		slope1 = evalue.y;
	} else {
		*ra = make_float2( sqrtf( sqrtf ( 4.0f / evalue.y ) ), sqrtf( sqrtf ( 4.0f / evalue.x ) ) );
		slope1 = evalue.x;
	}
					/* compute unit eigenvectors */
	if ( fabsf( hess2[1] ) <= FTINY )
		return optix::Matrix<2,2>::identity();			/* uv OK as is */
	slope1 = ( slope1 - hess2[0] ) / hess2[1];
	const float xmag1 = sqrtf( 1.0f / ( 1.0f + slope1 * slope1 ) );
	optix::Matrix<2,2> ab;
	ab[0] = ab[3] = slope1 * xmag1;
	ab[1] = -xmag1;
	ab[2] = xmag1;
	*uv = ab * *uv;

	/* needed for ambcorral */
	return ab;
}

#ifndef AMB_SAVE_MEM
RT_METHOD void ambHessian( AMBHEMI *hp, optix::Matrix<2,3> *uv, float2 *ra, float2 *pg, const float3& normal, const float3& hit )
{
	optix::Matrix<3,3> hessian;
	float3 gradient = make_float3( 0.0f );
	hessian.setRow( 0, gradient ); // Set zero matrix
	hessian.setRow( 1, gradient );
	hessian.setRow( 2, gradient );
	FFTRI fftr;
	int i, j;
					/* be sure to assign unit vectors */
	uv->setRow( 0, hp->ux );
	uv->setRow( 1, hp->uy );
			/* clock-wise vertex traversal from sample POV */
	//if (ra != NULL) {		/* initialize Hessian row buffer */
	//	hessrow = (FVECT (*)[3])malloc(sizeof(FVECT)*3*(hp->ns-1)); //TODO set memory size
	//	if (hessrow == NULL)
	//		error(SYSTEM, memerrmsg);
	//	memset(hessian, 0, sizeof(hessian));
	//} else if (pg == NULL)		/* bogus call? */
	//	return;
	//if (pg != NULL) {		/* initialize form factor row buffer */
	//	gradrow = (FVECT *)malloc(sizeof(FVECT)*(hp->ns-1));
	//	if (gradrow == NULL)
	//		error(SYSTEM, memerrmsg);
	//	memset(gradient, 0, sizeof(gradient));
	//}
					/* compute first row of edges */
	for (j = 0; j < hp->ns-1; j++) {
		comp_fftri(&fftr, &ambsam(0, j), &ambsam(0, j + 1), hit);
		if (ra != NULL)
			hessrow(j) = comp_hessian( &fftr, normal );
		if (pg != NULL)
			gradrow(j) = comp_gradient( &fftr, normal );
	}
					/* sum each row of triangles */
	for (i = 0; i < hp->ns-1; i++) {
	    optix::Matrix<3,3> hesscol;	/* compute first vertical edge */
	    float3 gradcol;
		comp_fftri(&fftr, &ambsam(i, 0), &ambsam(i + 1, 0), hit);
		if (ra != NULL)
			hesscol = comp_hessian( &fftr, normal );
		if (pg != NULL)
			gradcol = comp_gradient( &fftr, normal );
	    for (j = 0; j < hp->ns-1; j++) {
			optix::Matrix<3,3> hessdia;	/* compute triangle contributions */
			float3 graddia;
			float backg = back_ambval(&ambsam(i, j), &ambsam(i, j + 1), &ambsam(i + 1, j));
						/* diagonal (inner) edge */
			comp_fftri(&fftr, &ambsam(i, j + 1), &ambsam(i + 1, j), hit);
			if (ra != NULL) {
				hessdia = comp_hessian( &fftr, normal );
				//hesscol = -hesscol;
				hessian += backg * ( hessrow(j) + hessdia - hesscol );
			}
			if (pg != NULL) {
				graddia = comp_gradient( &fftr, normal );
				//gradcol = -gradcol;
				gradient += backg * ( gradrow(j) + graddia - gradcol );
			}
						/* initialize edge in next row */
			comp_fftri(&fftr, &ambsam(i + 1, j + 1), &ambsam(i + 1, j), hit);
			if (ra != NULL)
				hessrow(j) = comp_hessian( &fftr, normal );
			if (pg != NULL)
				gradrow(j) = comp_gradient( &fftr, normal );
						/* new column edge & paired triangle */
			backg = back_ambval(&ambsam(i + 1, j + 1), &ambsam(i + 1, j), &ambsam(i, j + 1));
			comp_fftri(&fftr, &ambsam(i, j + 1), &ambsam(i + 1, j + 1), hit);
			if (ra != NULL) {
				hesscol = comp_hessian( &fftr, normal );
				//hessdia = -hessdia;
				hessian += backg * ( hessrow(j) - hessdia + hesscol );
				if ( i < hp->ns-2 )
					hessrow(j) *= -1.0f;
			}
			if (pg != NULL) {
				gradcol = comp_gradient( &fftr, normal );
				//graddia = -graddia;
				gradient += backg * ( gradrow(j) - graddia + gradcol );
				if ( i < hp->ns-2 )
					gradrow(j) = -gradrow(j);
			}
	    }
	}
					/* release row buffers */
	//if (hessrow != NULL) free(hessrow);
	//if (gradrow != NULL) free(gradrow);
	
	if (ra != NULL)			/* extract eigenvectors & radii */
		eigenvectors( uv, ra, &hessian );
	if (pg != NULL) {		/* tangential position gradient */
		*pg = *uv * gradient;
	}
}

/* Compute direction gradient from a hemispherical sampling */
RT_METHOD void ambdirgrad( AMBHEMI *hp, const float3& u, const float3& v, float2 *dg, const float3& normal, const float3& hit )
{
	float2 dgsum = make_float2( 0.0f );	/* sum values times -tan(theta) */
	for (int i = 0; i < hp->ns; i++)
		for (int j = 0; j < hp->ns; j++) {
			AmbientSample *ap = &ambsam(i, j);
					/* use vector for azimuth + 90deg */
			float3 vd = ap->p - hit;
					/* brightness over cosine factor */
			float gfact = ap->v.y / dot( normal, vd );
					/* sine = proj_radius/vd_length */
			dgsum.x -= dot( v, vd ) * gfact;
			dgsum.y += dot( u, vd ) * gfact;
		}
	*dg = dgsum / (hp->ns*hp->ns);
}

/* Compute potential light leak direction flags for cache value */
RT_METHOD unsigned int ambcorral( AMBHEMI *hp, optix::Matrix<2,3> *uv, const float2& r, const float3& normal, const float3& hit )
{
	const float max_d = 1.0f / ( minarad * ambacc + 0.001f );
	const float ang_res = M_PI_2f / hp->ns;
	const float ang_step = ang_res / ( (int)( 16.0f * M_1_PIf * ang_res ) + ( 1.01f ) );
	float avg_d = 0.0f;
	unsigned int flgs = 0u;
	int i, j;
					/* don't bother for a few samples */
	if ( hp->ns < 12 )
		return(0u);
					/* check distances overhead */
	for ( i = hp->ns * 3 / 4; i-- > hp->ns>>2; )
	    for ( j = hp->ns * 3 / 4; j-- > hp->ns>>2; )
			avg_d += ambsam(i, j).d;
	avg_d *= 4.0f / ( hp->ns * hp->ns );
	if ( avg_d * r.x >= 1.0f )		/* ceiling too low for corral? */
		return(0u);
	if ( avg_d >= max_d )		/* insurance */
		return(0u);
					/* else circle around perimeter */
	for ( i = 0; i < hp->ns; i++ )
	    for ( j = 0; j < hp->ns; j += !i|(i==hp->ns-1) ? 1 : hp->ns-1 ) {
			AmbientSample *ap = &ambsam(i, j);
			if ( ( ap->d <= FTINY ) | ( ap->d >= max_d ) )
				continue;	/* too far or too near */
			const float2 u = *uv * ( ap->p - hit );
			if ( ( r.x*r.x * u.x*u.x + r.y*r.y * u.y*u.y ) * ap->d*ap->d <= u.x*u.x + u.y*u.y )
				continue;	/* occluder outside ellipse */
			const float ang = atan2f( u.y, u.x );	/* else set direction flags */
			for ( float a1 = ang - ang_res; a1 <= ang + ang_res; a1 += ang_step )
				flgs |= 1L<<(int)( 16.0f * M_1_PIf * ( a1 + 2.0f * M_PIf * ( a1 < 0.0f ) ) );
	    }
					/* add low-angle incident (< 20deg) */
	if ( fabsf( dot( ray.direction, normal ) ) <= 0.342f ) {
		const float2 u = *uv * ray.direction;
		if ( ( r.x*r.x * u.x*u.x + r.y*r.y * u.y*u.y ) > t_hit * t_hit ) {
			float ang = atan2f( -u.y, -u.x );
			ang += 2.0f * M_PIf * ( ang < 0.0f );
			ang *= 16.0f * M_1_PIf;
			if ( ( ang < 0.5f ) | ( ang >= 31.5f ) )
				flgs |= 0x80000001;
			else
				flgs |= 3L<<(int)( ang - 0.5f );
		}
	}
	return(flgs);
}
#endif /* AMB_SAVE_MEM */
#else /* OLDAMB */

RT_METHOD float doambient( float3 *rcol, float3 *pg, float3 *dg, const float3& nrm, const float3& hit_point )
{
	float  b, d;
	AMBHEMI  hemi;
	AMBSAMP  *div;
	AMBSAMP  dnew;
	float3  acol;
	AMBSAMP  *dp;
	float  arad;
	int  divcnt;
	unsigned int  i, j;
					/* initialize hemisphere */
	inithemi(&hemi, *rcol, nrm);
	divcnt = hemi.nt * hemi.np;
					/* initialize */
	//if (pg != NULL)
	//*pg = make_float3( 0.0f );
	//if (dg != NULL)
	//*dg = make_float3( 0.0f );
	*rcol = make_float3( 0.0f );
	if (divcnt == 0)
		return(0.0f); //TODO does this change the value of rcol in the calling method?
					/* allocate super-samples */
	//if (hemi.ns > 0) {// || pg != NULL || dg != NULL) {
	//	div = (AMBSAMP *)malloc(divcnt*sizeof(AMBSAMP));
	//	//if (div == NULL) // This is 0
	//	//	error(SYSTEM, "out of memory in doambient");
	//} else
		div = NULL; // This is 0
					/* sample the divisions */
	arad = 0.0f;
	acol = make_float3( 0.0f );
	if ((dp = div) == NULL)
		dp = &dnew;
	divcnt = 0;

	/* Set-up from posgradient in ambcomp.c */
	float xdp = 0.0f;
	float ydp = 0.0f;

	/* Set-up from dirgradient in ambcomp.c */
	float xdd = 0.0f;
	float ydd = 0.0f;

	for (j = 0u; j < hemi.np; j++) {
		/* Set-up from posgradient in ambcomp.c */
		float mag0 = 0.0f;
		float mag1 = 0.0f;
		float lastsine = 0.0f;
		float rprev, bprev;

		/* Set-up from dirgradient in ambcomp.c */
		float mag = 0.0f;

		for (i = 0u; i < hemi.nt; i++) {
			dp->t = i; dp->p = j;
			dp->v = make_float3( 0.0f );
			dp->r = 0.0f;
			dp->n = 0;
			if (divsample(dp, &hemi, hit_point, nrm) < 0) {
				rprevrow(i) = rprev = dp->r; // Set values for posgradient to avoid NaN
				bprevrow(i) = bprev = bright(dp->v);
				if (div != NULL)
					dp++;
				continue;
			}
			arad += dp->r;
			divcnt++;
			if (div != NULL)
				dp++;
			else
				acol += dp->v;

			/* Processing from posgradient in ambcomp.c */
			b = bright(dp->v);
			if (i > 0u) {
				d = rprev;//dp[-hp->np].r;
				if ( dp->r > d ) d = dp->r;
				d *= lastsine * ( 1.0f - (float)i / hemi.nt ); /* sin(t)*cos(t)^2 */
				mag0 += d * ( b - bprev ); // bright(dp[-hp->np].v)
			}
			float nextsine = sqrtf( (float)(i+1) / hemi.nt );
			if (j > 0u) {
				d = rprevrow(i);//dp[-1].r;
				if ( dp->r > d ) d = dp->r;
				mag1 += d * ( nextsine - lastsine ) * ( b - bprevrow(i) ); // bright(dp[-1].v)
			//} else {
			//	d = dp[hp->np-1].r;
			//	if ( dp->r > d ) d = dp->r;
			//	mag1 += d * (nextsine - lastsine) * (b - bright(dp[hp->np-1].v));
			}
			lastsine = nextsine;
			rprevrow(i) = rprev = dp->r;
			bprevrow(i) = bprev = b;

			/* Processing from dirgradient in ambcomp.c */
			mag += b / sqrtf( hemi.nt / ( i + 0.5f ) - 1.0f );
		}

		/* Processing from posgradient in ambcomp.c */
		mag0 *= 2.0f * M_PIf / hemi.np;
		float phi = 2.0f * M_PIf * (float)j / hemi.np;
		float cosp = cosf(phi);
		float sinp = sinf(phi);
		xdp += mag0 * cosp - mag1 * sinp;
		ydp += mag0 * sinp + mag1 * cosp;

		/* Processing from dirgradient in ambcomp.c */
		phi = 2.0f * M_PIf * ( j + 0.5f ) / hemi.np + M_PIf * 0.5f;
		xdd += mag * cosf(phi);
		ydd += mag * sinf(phi);
	}
	if (!divcnt) {
		//if (div != NULL)
		//	free((void *)div);
		return(0.0f);		/* no samples taken */
	}
	//if (divcnt < hemi.nt*hemi.np) {
	//	//pg = dg = NULL;		/* incomplete sampling */
	//	hemi.ns = 0;
	//} else if (arad > FTINY && divcnt/arad < minarad) {
	//	hemi.ns = 0;		/* close enough */
	//} else if (hemi.ns > 0) {	/* else perform super-sampling? */
	//	comperrs(div, &hemi);			/* compute errors */
	//	//qsort(div, divcnt, sizeof(AMBSAMP), ambcmp);	/* sort divs */ TODO necessary?
	//					/* super-sample */
	//	for (i = hemi.ns; i > 0u; i--) {
	//		dnew = *div;
	//		if (divsample(&dnew, &hemi, hit_point, nrm) < 0) {
	//			dp++;
	//			continue;
	//		}
	//		dp = div;		/* reinsert */
	//		j = divcnt < i ? divcnt : i;
	//		while (--j > 0 && dnew.k < dp[1].k) {
	//			*dp = *(dp+1);
	//			dp++;
	//		}
	//		*dp = dnew;
	//	}
	//	//if (pg != NULL || dg != NULL)	/* restore order */
	//	//	qsort(div, divcnt, sizeof(AMBSAMP), ambnorm);
	//}
					/* compute returned values */
	//if (div != NULL) {
		//arad = 0.0f;		/* note: divcnt may be < nt*np */
		//for (i = hemi.nt*hemi.np, dp = div; i-- > 0u; dp++) {
		//	arad += dp->r;
		//	if (dp->n > 1) {
		//		b = 1.0f/dp->n;
		//		dp->v *= b;
		//		dp->r *= b;
		//		dp->n = 1;
		//	}
		//	acol += dp->v;
		//}
		b = bright(acol);
		if (b > FTINY) {
			b = 1.0f / b;	/* compute & normalize gradient(s) */
			//if (pg != NULL) {
				//posgradient(pg, div, &hemi);
				*pg = ( xdp * hemi.ux + ydp * hemi.uy ) * ( hemi.nt * hemi.np * M_1_PIf );
				*pg *= b;
			//}
			//if (dg != NULL) {
				//dirgradient(dg, div, &hemi);
				*dg = xdd * hemi.ux + ydd * hemi.uy;
				*dg *= b;
			//}
		}
		//free((void *)div);
	//}
	*rcol = acol;
	if (arad <= FTINY)
		arad = maxarad;
	else
		arad = (divcnt+hemi.ns)/arad;
	//if (pg != NULL) {		/* reduce radius if gradient large */
		d = dot( *pg, *pg );
		if ( d * arad * arad > 1.0f )
			arad = 1.0f / sqrtf(d);
	//}
	if (arad < minarad) {
		arad = minarad;
		if ( /*pg != NULL &&*/ d * arad * arad > 1.0f ) {	/* cap gradient */
			d = 1.0f / arad / sqrtf(d);
			*pg *= d;
		}
	}
	if ((arad /= sqrtf(prd.result.weight)) > maxarad) //TODO check that weight is correct
		arad = maxarad;
	return(arad);
}

/* initialize sampling hemisphere */
RT_METHOD void inithemi( AMBHEMI  *hp, const float3& ac, const float3& nrm )
{
	float	d;
	int  i;
	float wt = prd.result.weight;
					/* set number of divisions */
	if (ambacc <= FTINY && wt > (d = 0.8f * fmaxf(ac) * wt / (ambdiv*minweight)))
		wt = d;			/* avoid ray termination */
	hp->nt = sqrtf(ambdiv * wt * M_1_PIf) + 0.5f;
	i = ambacc > FTINY ? 3 : 1;	/* minimum number of samples */
	if (hp->nt < i)
		hp->nt = i;
	hp->np = M_PIf * hp->nt + 0.5f;
					/* set number of super-samples */
	hp->ns = ambssamp * wt + 0.5f;
					/* assign coefficient */
	hp->acoef = ac;
	d = 1.0f/(hp->nt*hp->np);
	hp->acoef *= d;
					/* make axes */
	hp->uz = nrm;
	hp->uy = cross_direction( hp->uz );
	hp->ux = normalize( cross(hp->uy, hp->uz) );
	hp->uy = normalize( cross(hp->uz, hp->ux) );
}

/* sample a division */
RT_METHOD int divsample( AMBSAMP  *dp, AMBHEMI  *h, const float3& hit_point, const float3& normal )
{
	PerRayData_radiance new_prd;
	//RAY  ar;
	//float3 rcoef; /* contribution coefficient w.r.t. parent */
	//int3  hlist;
	float2  spt;
	float  xd, yd, zd;
	float  b2;
	float  phi;
					/* ambient coefficient for weight */
	if (ambacc > FTINY)
		b2 = AVGREFL; // Reusing this variable
	else
		b2 = fmaxf(h->acoef);
	new_prd.weight = prd.result.weight * b2;
	if (new_prd.weight < minweight) //if (rayorigin(&ar, AMBIENT, r, ar.rcoef) < 0)
		return(-1);
	//if (ambacc > FTINY) {
	//	rcoef *= h->acoef;
	//	rcoef *= 1.0f / AVGREFL; // This all seems unnecessary
	//}
	//hlist = make_int3( prd.seed, dp->t, dp->p );
	//multisamp(spt, 2, urand(ilhash(hlist,3)+dp->n));//TODO implement
	//spt = multisamp2( frandom() );
	//int il = ilhash( hlist );
	//spt = make_float2( rnd( il ) );
	//spt = make_float2( rnd( prd.seed ) );
	spt = make_float2( hiprand_uniform( prd.state ), hiprand_uniform( prd.state ) );
	zd = sqrtf((dp->t + spt.x)/h->nt);
	phi = 2.0f*M_PIf * (dp->p + spt.y)/h->np;
	xd = cosf(phi) * zd;
	yd = sinf(phi) * zd;
	zd = sqrtf(1.0f - zd*zd);
	float3 rdir = normalize( xd*h->ux + yd*h->uy + zd*h->uz );
	//dimlist[ndims++] = dp->t*h->np + dp->p + 90171;

	new_prd.depth = prd.result.lvl + 1;//prd.depth + 1;
	new_prd.ambient_depth = prd.result.lvl + 1;//prd.ambient_depth + 1;
	//new_prd.seed = prd.seed;//lcg( prd.seed );
	new_prd.state = prd.state;
#ifdef ANTIMATTER
	new_prd.mask = prd.mask;
	new_prd.inside = prd.inside;
#endif
#ifdef DAYSIM_COMPATIBLE
	new_prd.dc = daysimNext(prd.dc);
#endif
	setupPayload(new_prd);
	Ray amb_ray = make_Ray( hit_point, rdir, radiance_ray_type, ray_start( hit_point, rdir, normal, RAY_START ), RAY_END );
	rtTrace(top_object, amb_ray, new_prd);
#ifdef RAY_COUNT
	prd.result.ray_count += new_prd.ray_count;
#endif
#ifdef HIT_COUNT
	prd.result.hit_count += new_prd.hit_count;
#endif

	//ndims--;
	if ( isnan( new_prd.result ) ) // TODO How does this happen?
		return(-1);
	new_prd.result *= h->acoef;	/* apply coefficient */
	dp->v += new_prd.result;
#ifdef DAYSIM_COMPATIBLE
	daysimAddScaled(prd.dc, new_prd.dc, h->acoef.x);
#endif
	/* use rt to improve gradient calc */
	if (new_prd.distance > FTINY && new_prd.distance < RAY_END)
		dp->r += 1.0f/new_prd.distance;

					/* (re)initialize error */
	if (dp->n++) {
		b2 = bright(dp->v)/dp->n - bright(new_prd.result);
		b2 = b2*b2 + dp->k*((dp->n-1)*(dp->n-1));
		dp->k = b2/(dp->n*dp->n);
	} else
		dp->k = 0.0f;
	return(0);
}

/* compute initial error estimates */
//RT_METHOD void comperrs( AMBSAMP *da, AMBHEMI *hp )
//{
//	float  b, b2;
//	int  i, j;
//	AMBSAMP  *dp;
//				/* sum differences from neighbors */
//	dp = da;
//	for (i = 0; i < hp->nt; i++)
//		for (j = 0; j < hp->np; j++) {
////#ifdef  DEBUG
////			if (dp->t != i || dp->p != j)
////				error(CONSISTENCY,
////					"division order in comperrs");
////#endif
//			b = bright(dp[0].v);
//			if (i > 0) {		/* from above */
//				b2 = bright(dp[-hp->np].v) - b;
//				b2 *= b2 * 0.25f;
//				dp[0].k += b2;
//				dp[-hp->np].k += b2;
//			}
//			if (j > 0) {		/* from behind */
//				b2 = bright(dp[-1].v) - b;
//				b2 *= b2 * 0.25f;
//				dp[0].k += b2;
//				dp[-1].k += b2;
//			} else {		/* around */
//				b2 = bright(dp[hp->np-1].v) - b;
//				b2 *= b2 * 0.25f;
//				dp[0].k += b2;
//				dp[hp->np-1].k += b2;
//			}
//			dp++;
//		}
//				/* divide by number of neighbors */
//	dp = da;
//	for (j = 0; j < hp->np; j++)		/* top row */
//		(dp++)->k *= 1.0f/3.0f;
//	if (hp->nt < 2)
//		return;
//	for (i = 1; i < hp->nt-1; i++)		/* central region */
//		for (j = 0; j < hp->np; j++)
//			(dp++)->k *= 0.25f;
//	for (j = 0; j < hp->np; j++)		/* bottom row */
//		(dp++)->k *= 1.0f/3.0f;
//}

/* decreasing order */
//RT_METHOD int ambcmp( const void *p1, const void *p2 )
//{
//	const AMBSAMP	*d1 = (const AMBSAMP *)p1;
//	const AMBSAMP	*d2 = (const AMBSAMP *)p2;
//
//	if (d1->k < d2->k)
//		return(1);
//	if (d1->k > d2->k)
//		return(-1);
//	return(0);
//}
#endif /* OLDAMB */

/* convert 1-dimensional sample to 2 dimensions, based on multisamp.c */
//RT_METHOD float2 multisamp2(float r)	/* 1-dimensional sample [0,1) */
//{
//	int	j;
//	register int	k;
//	int2	ti;
//	float	s;
//
//	ti = make_int2( 0 );
//	j = 8;
//	while (j--) {
//		k = s = r*(1<<2);
//		r = s - k;
//		ti += ti + make_int2( ((k>>2) & 1), ((k>>1) & 1) );
//	}
//	ti += make_int2( frandom() );
//	ti *= 1.0f/256.0f;
//}

/* hash a set of integer values */
//RT_METHOD int ilhash(int3 d)
//{
//	register int  hval;
//
//	hval = 0;
//	hval ^= d.x * 73771;
//	hval ^= d.y * 96289;
//	hval ^= d.z * 103699;
//	return(hval & 0x7fffffff);
//}
