#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2016 Nathaniel Jones
 * Massachusetts Institute of Technology
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>
#include "optix_shader_common.h"

using namespace optix;

rtBuffer<AmbientRecord> ambient_records;
#ifdef DAYSIM_COMPATIBLE
rtBuffer<DC, 2> ambient_dc;
#endif

rtDeclareVariable(float,        ambacc, , ); /* Ambient accuracy (aa). This value will approximately equal the error from indirect illuminance interpolation */
rtDeclareVariable(float,        minarad, , ); /* minimum ambient radius */

//rtDeclareVariable(float3, ambient_value, attribute ambient_value, );
//rtDeclareVariable(float, weight, attribute weight_attribute, );
//rtDeclareVariable(float, extrapolation, attribute extrapolation_attribute, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(PerRayData_ambient, prd, rtPayload, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );


#ifndef OLDAMB
RT_METHOD int plugaleak(const AmbientRecord* record, const float3& anorm, const float3& normal, float ang);
#endif


// Ignore the intersection so that the intersection program will continue to run for all overlapping recrods.
RT_PROGRAM void ambient_record_any_hit()
{
	//prd.wsum += weight;

	//if (extrapolation > 0.0f) {
	//	prd.result += ambient_value * (extrapolation * weight);
	//}
	rtIgnoreIntersection();
}

// based on makeambient from ambient.c
RT_PROGRAM void ambient_miss()
{
	//if ( prd.wsum == 0.0f )
	//	rtThrow( RT_EXCEPTION_USER );
}

// based on sumambient from ambient.c
RT_PROGRAM void ambient_record_intersect( int primIdx )
{
#ifdef HIT_COUNT
	prd.hit_count++;
#endif

	const AmbientRecord record = ambient_records[primIdx];

	/* Ambient level test. */
	if ( record.lvl > prd.ambient_depth )
		return;
	if (record.lvl == prd.ambient_depth && record.weight < 0.9f * prd.weight)
		return;

	const float3 normal = ray.direction;

#ifndef OLDAMB
	/* Direction test using unperturbed normal */
	float3 w = decodedir( record.ndir );
	float d = dot(w, normal); // Ray direction is unperturbed surface normal
	if ( d <= 0.0f )		/* >= 90 degrees */
		return;
	if (d > 1.0f)
		d = 1.0f;

	float delta_r2 = 2.0f - 2.0f * d;	/* approx. radians^2 */
	const float minangle = 10.0f * M_PIf / 180.0f;
	float maxangle = minangle + ambacc;
					/* adjust maximum angle */
	if (prd.weight < 0.6f)
		maxangle = (maxangle - M_PI_2f) * powf(prd.weight, 0.13f) + M_PI_2f;
	if ( delta_r2 >= maxangle * maxangle )
		return;

	/* Modified ray behind test */
	float3 ck0 = ray.origin - record.pos;
	d = dot( ck0, w );
	if ( d < -minarad * ambacc - 0.001f )
		return;
	d /= record.rad.x;
	float delta_t2 = d * d;
	if ( delta_t2 >= ambacc * ambacc )
		return;
	
	/* Elliptical radii test based on Hessian */
	float3 u = decodedir( record.udir );
	float3 v = cross( w, u );
	float uu, vv;
	d = (uu = dot( ck0, u )) / record.rad.x;
	delta_t2 += d * d;
	d = (vv = dot( ck0, v )) / record.rad.y;
	delta_t2 += d * d;
	if ( delta_t2 >= ambacc * ambacc )
		return;
	
	/* Test for potential light leak */
	if (record.corral && plugaleak(&record, w, normal, atan2f(vv, uu)))
		return;

	/* Extrapolate value and compute final weight (hat function) */
	/* This is extambient from ambient.c */
	/* gradient due to translation */
	d = 1.0f + dot( ck0, record.gpos.x * u + record.gpos.y * v );

	/* gradient due to rotation */
	ck0 = cross( w, prd.surface_normal );
	d += dot( ck0, record.gdir.x * u + record.gdir.y * v );

	//if (d < min_d)			/* should not use if we can avoid it */
	//	d = min_d;
	if ( d <= 0.05f )
		return;

	if (rtPotentialIntersection(-dot(ck0, normal))) {
		float wt = ( 1.0f - sqrtf(delta_r2) / maxangle ) * ( 1.0f - sqrtf(delta_t2) / ambacc );
		prd.wsum += wt;

		// This assignment to the prd would take place in the any-hit program if there were one
		prd.result += record.val * ( d * wt );
#ifdef DAYSIM_COMPATIBLE
		if (ambient_dc.size().x)
			daysimAddScaled(prd.dc, &ambient_dc[make_uint2(0, primIdx)], d * wt);
#endif

		rtReportIntersection( 0 ); // There is only one material for ambient geometry group
	}
#else /* OLDAMB */
	/* Ambient radius test. */
	float3 ck0 = record.pos - ray.origin;
	float e1 = dot( ck0, ck0 ) / ( record.rad * record.rad );
	float acc = ambacc * ambacc * 1.21f;
	if ( e1 > acc )
		return;

	/* Direction test using closest normal. */
	float d = dot( record.dir, normal ); // Ray direction is unperturbed surface normal
	//if (rn != r->ron) {
	//	rn_dot = DOT(av->dir, rn);
	//	if (rn_dot > 1.0-FTINY)
	//		rn_dot = 1.0-FTINY;
	//	if (rn_dot >= d-FTINY) {
	//		d = rn_dot;
	//		rn_dot = -2.0;
	//	}
	//}
	float e2 = (1.0f - d) * prd.weight;
	if (e2 < 0.0f)
		e2 = 0.0f;
	else if (e1 + e2 > acc)
		return;

	/* Ray behind test. */
	d = dot( ck0, record.dir + normal );
	if (d * 0.5f > minarad * ambacc + 0.001f )
		return;

	/* Jittering final test reduces image artifacts. */
	e1 = sqrtf(e1);
	e2 = sqrtf(e2);
	float wt = e1 + e2;
	if (wt > ambacc * ( 0.9f + 0.2f * hiprand_uniform( prd.state ) ) )
		return;

	if (rtPotentialIntersection(dot(ck0, normal))) {
		/* Recompute directional error using perturbed normal */
		//if (rn_dot > 0.0) {
		//	e2 = sqrtf( ( 1.0f - rn_dot ) * prd.weight);
		//	wt = e1 + e2;
		//}
		if (wt <= 1e-3f)
			wt = 1e3f;
		else
			wt = 1.0f / wt;
		prd.wsum += wt; // This assignment to the prd would take place in the any-hit program if there were one

		/* This is extambient from ambient.c */
		//float d = 1.0f;			/* zeroeth order */

		/* gradient due to translation */
		d = 1.0f - dot( record.gpos, ck0 );

		/* gradient due to rotation */
		ck0 = cross( record.dir, prd.surface_normal );
		d += dot( record.gdir, ck0 );

		if (d > 0.0f) {
			// This assignment to the prd would take place in the any-hit program if there were one
			prd.result += record.val * (d * wt);
		}

		rtReportIntersection( 0 ); // There is only one material for ambient geometry group
	}
#endif /* OLDAMB */
}

RT_PROGRAM void ambient_record_bounds (int primIdx, float result[6])
{
	optix::Aabb* aabb = (optix::Aabb*)result;
	const AmbientRecord record = ambient_records[primIdx];

#ifndef OLDAMB
	const float2 rad = record.rad * ambacc; // Acceleration structure becomes dirty when ambacc is increased.

	if( rad.x > FTINY && isfinite(rad.y) ) {
		const float3 w = decodedir( record.ndir );
		const float3 u = decodedir( record.udir );
		const float3 v = cross( w, u );
		const float3 dims = sqrtf( u*u*(rad.x*rad.x) + v*v*(rad.y*rad.y) );// + FTINY; // Expanding by FTINY seems to help prevent misses
		//const float3 dims = sqrtf( u*u*(rad.x*rad.x) + v*v*(rad.y*rad.y) + w*w*(rad.x*rad.x) );// + FTINY;
		aabb->m_min = record.pos - dims;
		aabb->m_max = record.pos + dims;
	} else {
		aabb->invalidate();
	}
#else /* OLDAMB */
	const float rad = record.rad * 1.1f * ambacc; // Acceleration structure becomes dirty when ambacc is increased.

	if( rad > FTINY && isfinite(rad) ) {
		const float3 dims = rad * sqrtf( 1.0f - record.dir * record.dir );// + FTINY; // Expanding by FTINY seems to help prevent misses
		aabb->m_min = record.pos - dims;
		aabb->m_max = record.pos + dims;
	} else {
		aabb->invalidate();
	}
#endif /* OLDAMB */
}

#ifndef OLDAMB
/* Plug a potential leak where ambient cache value is occluded */
RT_METHOD int plugaleak(const AmbientRecord* record, const float3& anorm, const float3& normal, float ang)
{
	const float cost70sq = 0.1169778f;	/* cos(70deg)^2 */
	float2 t;

	ang += 2.0f * M_PIf * (ang < 0);			/* check direction flags */
	if ( !(record->corral>>(int)(ang * 16.0f * M_1_PIf) & 1) )
		return(0);
	/*
	 * Generate test ray, targeting 20 degrees above sample point plane
	 * along surface normal from cache position.  This should be high
	 * enough to miss local geometry we don't really care about.
	 */
	float3 vdif = record->pos - ray.origin;
	float normdot = dot(anorm, normal);
	float ndotd = dot(vdif, normal);
	float nadotd = dot( vdif, anorm );
	float a = normdot * normdot - cost70sq;
	float b = 2.0f * ( normdot * ndotd - nadotd * cost70sq );
	float c = ndotd * ndotd - dot( vdif, vdif ) * cost70sq;
	if ( quadratic( &t, a, b, c ) != 2 )
		return(1);			/* should rarely happen */
	//if ( t.y <= FTINY )
		return(0);			/* should fail behind test */

	/* Can't shoot rays from an intersection program. */
	//float3 rdir = vdif + anorm * t.y;	/* further dist. > plane */
	//Ray shadow_ray = make_Ray( ray.origin, normalize( rdir ), shadow_ray_type, RAY_START, length( rdir ) );
	//PerRayData_shadow shadow_prd;
	//shadow_prd.result = make_float3( 1.0f );
#ifdef ANTIMATTER
	//shadow_prd.mask = prd.mask;
	//shadow_prd.inside = prd.inside;
#endif
	//rtTrace( top_object, shadow_ray, shadow_prd );
	//return( dot( shadow_prd.result, shadow_prd.result ) < 1.0f );	/* check for occluder */
}
#endif /* OLDAMB */