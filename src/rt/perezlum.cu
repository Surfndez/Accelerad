#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2014 Nathaniel Jones
 * Massachusetts Institute of Technology
 */

#include <optix_world.h>
//#include "optix_shader_common.h"

/* Program variables */
rtDeclareVariable(float,  diffuse, , ); /* diffuse normalization */
rtDeclareVariable(float,  ground, , ); /* ground plane brightness */
rtDeclareVariable(float3, coef0, , ); /* coefficients for the Perez model */
rtDeclareVariable(float2, coef1, , ); /* coefficients for the Perez model */
rtDeclareVariable(float3, sun, , ); /* sun direction */

// Calculate the All-weather Angular Sky Luminance Distribution value for the current ray direction.
// This function replicates the algorithm in perezlum.cal.
RT_CALLABLE_PROGRAM float perez_lum( const float3 direction )
{
	rtPrintf("PerezLum Recieved (%f, %f, %f)\n", direction.x, direction.y, direction.z);
	float cosgamma = optix::dot( direction, sun ); // cosgamma = Dx*A8 + Dy*A9 + Dz*A10;
	float gamma = acosf(cosgamma); // gamma = Acos(cosgamma);		{ angle from sun to this point in sky }
	//float zt = acos(sun.z); // zt = Acos(A10);			{ angle from zenith to sun }
	//float eta = acos(direction.z); // eta = Acos(Dz);			{ angle from zenith to this point in sky }

	float dz = direction.z;
	if (dz < 0.01f) {
		dz = 0.01f;
	}

	float intersky = diffuse * (1.0f + coef0.x * expf( coef0.y / dz ) ) * ( 1.0f + coef0.z * expf(coef1.x * gamma) + coef1.y * cosgamma * cosgamma );
	//intersky = if( (Dz-0.01),  
	//		A1 * (1 + A3*Exp(A4/Dz) ) * ( 1 + A5*Exp(A6*gamma) + A7*cos(gamma)*cos(gamma) ),
	//		A1 * (1 + A3*Exp(A4/0.01) ) * ( 1 + A5*Exp(A6*gamma) + A7*cos(gamma)*cos(gamma) ) );

	float a = powf(direction.z + 1.01f, 10.0f);
	float b = powf(direction.z + 1.01f, -10.0f);

	float skybright = (a * intersky + b * ground) / (a + b); // wmean(a, x, b, y) = (a*x+b*y)/(a+b);
	return skybright;
}