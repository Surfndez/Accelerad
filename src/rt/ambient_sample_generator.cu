#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2015 Nathaniel Jones
 * Massachusetts Institute of Technology
 */

#include <optix_world.h>
#include "optix_shader_common.h"
#include "optix_ambient_common.h"

#ifdef AMB_PARALLEL

using namespace optix;

/* Contex variables */
rtBuffer<PointDirection, 1>     cluster_buffer; /* input */
rtBuffer<AmbientSample, 3>      amb_samp_buffer; /* ambient sample output */
rtDeclareVariable(rtObject,     top_object, , );
rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(unsigned int, level, , ) = 0u;
rtDeclareVariable(unsigned int, segment_offset, , ) = 0u;

rtDeclareVariable(float,        ambacc, , ); /* Ambient accuracy (aa). This value will approximately equal the error from indirect illuminance interpolation */
rtDeclareVariable(float,        maxarad, , ); /* maximum ambient radius */
rtDeclareVariable(float,        minweight, , ); /* minimum ray weight (lw) */

/* OptiX variables */
rtDeclareVariable(uint3, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint3, launch_dim,   rtLaunchDim, );

// Initialize the random state
RT_METHOD void init_state( PerRayData_radiance* prd )
{
	rand_state state;
	prd->state = &state;
	hiprand_init(launch_index.x + launch_dim.x * (launch_index.y + launch_dim.y * (launch_index.z + launch_dim.z * level)), 0, 0, prd->state);
}

RT_PROGRAM void ambient_sample_camera()
{
	uint3 index = launch_index;
	index.z += segment_offset;
	PointDirection cluster = cluster_buffer[index.z];

	PerRayData_radiance prd;
	init_state(&prd);
	float b2;
					/* generate hemispherical sample */
					/* ambient coefficient for weight */
//	if (ambacc > FTINY)
		b2 = AVGREFL; // Reusing this variable
//	else
//		b2 = fmaxf(hp->acoef); //TODO

	prd.weight = b2;
	for ( int i = level; i--; )
		prd.weight *= AVGREFL; // Compute weight as in makeambient() from ambient.c

	if (prd.weight < minweight) { //if (rayorigin(&ar, AMBIENT, r, ar.rcoef) < 0)
		amb_samp_buffer[index].d = 0.0f;
		return;
	}
	//if (ambacc > FTINY) {
	//	rcoef *= h->acoef;
	//	rcoef *= 1.0f / AVGREFL; // This all seems unnecessary
	//}
	//hlist[0] = hp->rp->rno;
	//hlist[1] = j;
	//hlist[2] = i;
	//multisamp(spt, 2, urand(ilhash(hlist,3)+n));
	float3 ux = getperpendicular(cluster.dir); // Can't be random, must be same for all threads for this point
	float3 uy = cross(cluster.dir, ux);
					/* avoid coincident samples */
	float2 spt = 0.1f + 0.8f * make_float2(hiprand_uniform(prd.state), hiprand_uniform(prd.state));
	SDsquare2disk(spt, (launch_index.y + spt.y) / launch_dim.y, (launch_index.x + spt.x) / launch_dim.x);
	float zd = sqrtf(1.0f - dot(spt, spt));
	float3 rdir = normalize(spt.x * ux + spt.y * uy + zd * cluster.dir);
	//dimlist[ndims++] = AI(hp,i,j) + 90171;

	prd.depth = level + 1;//prd.depth + 1;
	prd.ambient_depth = level + 1;//prd.ambient_depth + 1;
#ifdef DAYSIM_COMPATIBLE
	prd.dc = make_uint3(0, launch_index.x + launch_dim.x * launch_index.y, launch_index.z);
	prd.dc = daysimNext(prd.dc); // Skip ahead one
	daysimSet(prd.dc, 0.0f);
#endif
	setupPayload(prd, 1);
	Ray ray = make_Ray(cluster.pos, rdir, radiance_ray_type, ray_start(cluster.pos, rdir, cluster.dir, RAY_START), RAY_END);
	rtTrace(top_object, ray, prd);
#ifdef RAY_COUNT
	amb_samp_buffer[index].ray_count = prd.ray_count;
#endif
#ifdef HIT_COUNT
	amb_samp_buffer[index].hit_count = prd.hit_count;
#endif

	//ndims--;
	checkFinite(prd.result);
	if (prd.distance <= FTINY) {
		amb_samp_buffer[index].d = 0.0f;
		return;
	}

	//if ( new_prd.distance * ap->d < 1.0f )		/* new/closer distance? */ //TODO where did this value come from?
		amb_samp_buffer[index].d = 1.0f / prd.distance;
	//if (!n) {			/* record first vertex & value */
		if ( prd.distance > 10.0f * maxarad ) // 10 * thescene.cusize
			prd.distance = 10.0f * maxarad;
		amb_samp_buffer[index].p = cluster.pos + rdir * prd.distance;
		amb_samp_buffer[index].v = prd.result; // only one AmbientSample, otherwise would need +=
	//} else {			/* else update recorded value */
	//	hp->acol -= ap->v;
	//	zd = 1.0f / (float)(n+1);
	//	prd.result *= zd;
	//	zd *= (float)n;
	//	ambient_sample_buffer[index].v *= zd;
	//	ambient_sample_buffer[index].v += new_prd.result;
	//}
}

RT_PROGRAM void exception()
{
	const unsigned int code = rtGetExceptionCode();
	rtPrintf("Caught exception 0x%X at launch index (%d,%d,%d)\n", code, launch_index.x, launch_index.y, launch_index.z);
	uint3 index = launch_index;
	index.z += segment_offset;
	amb_samp_buffer[index].d = -1.0f;
	amb_samp_buffer[index].v = exceptionToFloat3( code );
	amb_samp_buffer[index].p = exceptionToFloat3( code );
#ifdef RAY_COUNT
	amb_samp_buffer[index].ray_count = 0;
#endif
#ifdef HIT_COUNT
	amb_samp_buffer[index].hit_count = 0;
#endif
}

#endif /* AMB_PARALLEL */
