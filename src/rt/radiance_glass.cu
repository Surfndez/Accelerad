#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2015 Nathaniel Jones
 * Massachusetts Institute of Technology
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "optix_shader_common.h"

using namespace optix;

/* Context variables */
rtDeclareVariable(rtObject,     top_object, , );

rtDeclareVariable(float,        minweight, , ); /* minimum ray weight */
rtDeclareVariable(int,          maxdepth, , ); /* maximum recursion depth */

/* Material variables */
#ifdef HIT_TYPE
rtDeclareVariable(unsigned int, type, , ); /* The material type representing "glass" or "dielectric" */
#endif
rtDeclareVariable(float,        r_index, , ) = 1.52f; /* Refractive index, usually 1.52 */
rtDeclareVariable(float3,       color, , ); /* The material color given by the rad file "glass" object */

/* OptiX variables */
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(PerRayData_radiance, prd, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow, rtPayload, );

/* Attributes */
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );


RT_PROGRAM void closest_hit_shadow()
{
	float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
	float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );

	float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );
	float3 snormal = faceforward( world_geometric_normal, -ray.direction, world_geometric_normal );

	PerRayData_shadow new_prd;             
	float3 result = make_float3( 0.0f );
	float3 hit_point = ray.origin + t_hit * ray.direction;
	float3 mcolor = color;

	/* check transmission */
	bool hastrans = fmaxf( mcolor ) > 1e-15f;
	if (!hastrans) {
		return;
	}
	mcolor = fmaxf( mcolor, make_float3( 1e-15f ) ); // no color channel should be smaller than 1e-15

	/* get modifiers */
	// we'll skip this for now

	/* perturb normal */
	// if there's a bump map, we use that, else
	float pdot = -dot( ray.direction, ffnormal );
	if (pdot < 0.0f) {		/* fix orientation from raynormal in raytrace.c */
		ffnormal += 2.0f * pdot * ray.direction;
		pdot = -pdot;
	}

	/* angular transmission */
	float cos2 = sqrtf(1.0f + (pdot * pdot - 1.0f) / (r_index * r_index));
	//if (hastrans) {
	mcolor = make_float3( powf( mcolor.x, 1.0f / cos2 ), powf( mcolor.y, 1.0f / cos2 ), powf( mcolor.z, 1.0f / cos2 ) );
	//}

	/* compute reflection */
	float r1e = (pdot - r_index * cos2) / (pdot + r_index * cos2);
	r1e *= r1e;
	float r1m = (1.0f / pdot - r_index / cos2) / (1.0f / pdot + r_index / cos2);
	r1m *= r1m;

	/* compute transmission */
	//if (hastrans) {
		float3 trans = 0.5f * (1.0f-r1e) * (1.0f-r1e) * mcolor / (1.0f - r1e * r1e * mcolor * mcolor);
		trans       += 0.5f * (1.0f-r1m) * (1.0f-r1m) * mcolor / (1.0f - r1m * r1m * mcolor * mcolor);

		/* modify by pattern */
		//trans *= pcol;

		/* transmitted ray */
		//new_prd.depth = prd.depth + 1;
		new_prd.target = prd_shadow.target;
		new_prd.result = make_float3( 0.0f );
#ifdef DAYSIM_COMPATIBLE
		new_prd.dc = daysimNext(prd_shadow.dc);
		daysimSet(new_prd.dc, 0.0f);
#endif
		Ray trans_ray = make_Ray(hit_point, ray.direction, ray.ray_type, ray_start(hit_point, ray.direction, snormal, RAY_START), RAY_END);
		rtTrace(top_object, trans_ray, new_prd);
		result += new_prd.result * trans;
#ifdef DAYSIM_COMPATIBLE
		daysimAddScaled(prd_shadow.dc, new_prd.dc, trans.x);
#endif
	//}

	// pass the color back up the tree
	prd_shadow.result = result;
}


RT_PROGRAM void closest_hit_radiance()
{
	float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
	float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );

	float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );
	float3 snormal = faceforward( world_geometric_normal, -ray.direction, world_geometric_normal );

	PerRayData_radiance new_prd;             
	float3 result = make_float3( 0.0f );
	float3 hit_point = ray.origin + t_hit * ray.direction;
	float3 mcolor = color;

	/* check transmission */
	bool hastrans = fmaxf( mcolor ) > 1e-15f;
	if (hastrans) {
		mcolor = fmaxf( mcolor, make_float3( 1e-15f ) ); // no color channel should be smaller than 1e-15
	} // else we return if it's a shadow ray, which it isn't

	/* get modifiers */
	// we'll skip this for now

	float transtest = 0.0f, transdist = t_hit;
	float mirtest = 0.0f, mirdist = t_hit;

	/* perturb normal */
	float3 pert = snormal - ffnormal;
	int hastexture = dot(pert, pert) > FTINY * FTINY;
	float pdot = -dot(ray.direction, ffnormal);
	if (pdot < 0.0f) {		/* fix orientation from raynormal in raytrace.c */
		ffnormal += 2.0f * pdot * ray.direction;
		pdot = -pdot;
	}

	/* angular transmission */
	float cos2 = sqrtf(1.0f + (pdot * pdot - 1.0f) / (r_index * r_index));
	if (hastrans) {
		mcolor = make_float3( powf( mcolor.x, 1.0f / cos2 ), powf( mcolor.y, 1.0f / cos2 ), powf( mcolor.z, 1.0f / cos2 ) );
	}

	/* compute reflection */
	float r1e = (pdot - r_index * cos2) / (pdot + r_index * cos2);
	r1e *= r1e;
	float r1m = (1.0f / pdot - r_index / cos2) / (1.0f / pdot + r_index / cos2);
	r1m *= r1m;

	/* compute transmission */
	if (hastrans) {
		float3 trans = 0.5f * (1.0f-r1e) * (1.0f-r1e) * mcolor / (1.0f - r1e * r1e * mcolor * mcolor);
		trans       += 0.5f * (1.0f-r1m) * (1.0f-r1m) * mcolor / (1.0f - r1m * r1m * mcolor * mcolor);

		/* modify by pattern */
		//trans *= pcol;

		/* transmitted ray */
		new_prd.weight = prd.weight * fmaxf(trans);
		if (new_prd.weight >= minweight) {
			new_prd.result = make_float3(0.0f);
			new_prd.depth = prd.depth;
			new_prd.ambient_depth = prd.ambient_depth;
			//new_prd.seed = prd.seed;//lcg( prd.seed );
			new_prd.state = prd.state;
#ifdef DAYSIM_COMPATIBLE
			new_prd.dc = daysimNext(prd.dc);
#endif
			float3 R = ray.direction;

			if (!new_prd.ambient_depth && hastexture) {
				R = normalize(ray.direction + pert * (2.0f * (1.0f - r_index)));
				if (isnan(R))
					R = ray.direction;
			} else {
				transtest = 2;
			}

			setupPayload(new_prd, 0);
			Ray trans_ray = make_Ray(hit_point, R, ray.ray_type, ray_start(hit_point, R, snormal, RAY_START), RAY_END);
			rtTrace(top_object, trans_ray, new_prd);
			float3 rcol = new_prd.result * trans;
			result += rcol;
			transtest *= bright( rcol );
			transdist = t_hit + new_prd.distance;
#ifdef DAYSIM_COMPATIBLE
			daysimAddScaled(prd.dc, new_prd.dc, trans.x);
#endif
			resolvePayload(prd, new_prd);
		}
	}
	// stop if it's a shadow ray, which it isn't

	/* compute reflectance */
	float3 refl = 0.5f * r1e * ( 1.0f + (1.0f-2.0f*r1e) * mcolor * mcolor ) / (1.0f - r1e * r1e * mcolor * mcolor );
	refl       += 0.5f * r1m * ( 1.0f + (1.0f-2.0f*r1m) * mcolor * mcolor ) / (1.0f - r1m * r1m * mcolor * mcolor );

	/* reflected ray */
	new_prd.weight = prd.weight * fmaxf(refl);
	new_prd.depth = prd.depth + 1;
	if (new_prd.weight >= minweight && new_prd.depth <= abs(maxdepth)) {
		new_prd.result = make_float3(0.0f);
		new_prd.ambient_depth = prd.ambient_depth;
		//new_prd.seed = prd.seed;//lcg( prd.seed );
		new_prd.state = prd.state;
#ifdef DAYSIM_COMPATIBLE
		new_prd.dc = daysimNext(prd.dc);
#endif
		setupPayload(new_prd, 0);
		float3 R = reflect( ray.direction, ffnormal );
		Ray refl_ray = make_Ray(hit_point, R, ray.ray_type, ray_start(hit_point, R, snormal, RAY_START), RAY_END);
		rtTrace(top_object, refl_ray, new_prd);
		float3 rcol = new_prd.result * refl;
		result += rcol;
		if (prd.ambient_depth || !hastexture) {
			mirtest = 2.0f * bright(rcol);
			mirdist = t_hit + new_prd.distance;
		}
#ifdef DAYSIM_COMPATIBLE
		daysimAddScaled(prd.dc, new_prd.dc, refl.x);
#endif
		resolvePayload(prd, new_prd);
	}
  
	/* check distance */
	float d = bright( result );
	if (transtest > d)
		prd.distance = transdist;
	else if (mirtest > d)
		prd.distance = mirdist;
	else
		prd.distance = t_hit;

	// pass the color back up the tree
	prd.result = result;
	
#ifdef HIT_TYPE
	prd.hit_type = type;
#endif
}
