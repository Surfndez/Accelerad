#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2014 Nathaniel Jones
 * Massachusetts Institute of Technology
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "optix_shader_common.h"

using namespace optix;

/* Context variables */
rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(unsigned int, shadow_ray_type, , );
rtDeclareVariable(rtObject,     top_object, , );
//rtDeclareVariable(rtObject,     top_shadower, , );

rtDeclareVariable(float3,       CIE_rgbf, , ); /* This is the value [ CIE_rf, CIE_gf, CIE_bf ] from color.h */

rtDeclareVariable(float,        minweight, , ); /* minimum ray weight */
rtDeclareVariable(int,          maxdepth, , ); /* maximum recursion depth */

/* Material variables */
#ifdef HIT_TYPE
rtDeclareVariable(unsigned int, type, , ); /* The material type representing "glass" or "dielectric" */
#endif
rtDeclareVariable(float,        rindex, , ) = 1.52f; /* Refractive index, usually 1.52 */
rtDeclareVariable(float3,       color, , ); /* The material color given by the rad file "glass" object */

/* OptiX variables */
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(PerRayData_radiance, prd, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow, rtPayload, );

/* Attributes */
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );


RT_METHOD float bright( const float3 &rgb );


RT_PROGRAM void closest_hit_shadow()
{
	float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
	float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );

	float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

	PerRayData_shadow new_prd;             
	float3 result = make_float3( 0.0f );
	float3 hit_point = ray.origin + t_hit * ray.direction;
	float3 mcolor = color;

	/* check transmission */
	bool hastrans = optix::fmaxf( mcolor ) > 1e-15f;
	if (!hastrans) {
		return;
	}
	mcolor = optix::fmaxf( mcolor, make_float3( 1e-15f ) ); // no color channel should be smaller than 1e-15

	/* get modifiers */
	// we'll skip this for now

	/* perturb normal */
	// if there's a bump map, we use that, else
	float pdot = -optix::dot( ray.direction, ffnormal );

	/* angular transmission */
	float cos2 = sqrtf( 1.0f + ( pdot * pdot - 1.0f ) / ( rindex * rindex ) );
	//if (hastrans) {
	mcolor = make_float3( powf( mcolor.x, 1.0f / cos2 ), powf( mcolor.y, 1.0f / cos2 ), powf( mcolor.z, 1.0f / cos2 ) );
	//}

	/* compute reflection */
	float r1e = (pdot - rindex*cos2) / (pdot + rindex*cos2);
	r1e *= r1e;
	float r1m = (1.0f/pdot - rindex/cos2) / (1.0f/pdot + rindex/cos2);
	r1m *= r1m;

	/* compute transmission */
	//if (hastrans) {
		float3 trans = 0.5f * (1.0f-r1e) * (1.0f-r1e) * mcolor / (1.0f - r1e * r1e * mcolor * mcolor);
		trans       += 0.5f * (1.0f-r1m) * (1.0f-r1m) * mcolor / (1.0f - r1m * r1m * mcolor * mcolor);

		/* modify by pattern */
		//trans *= pcol;

		/* transmitted ray */
		//new_prd.depth = prd.depth + 1;
		new_prd.target = prd_shadow.target;
		new_prd.result = make_float3( 0.0f );
		optix::Ray trans_ray = optix::make_Ray( hit_point, ray.direction, shadow_ray_type, ray_start( hit_point, ray.direction, ffnormal, RAY_START ), RAY_END );
		rtTrace(top_object, trans_ray, new_prd);
		result += new_prd.result * trans;
	//}

	// pass the color back up the tree
	prd_shadow.result = result;
}


RT_PROGRAM void closest_hit_radiance()
{
	float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
	float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );

	float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

	PerRayData_radiance new_prd;             
	float3 result = make_float3( 0.0f );
	float3 hit_point = ray.origin + t_hit * ray.direction;
	float3 mcolor = color;

	/* check transmission */
	bool hastrans = optix::fmaxf( mcolor ) > 1e-15f;
	if (hastrans) {
		mcolor = optix::fmaxf( mcolor, make_float3( 1e-15f ) ); // no color channel should be smaller than 1e-15
	} // else we return if it's a shadow ray, which it isn't

	/* get modifiers */
	// we'll skip this for now

	float transtest = 0.0f, transdist = 0.0f;
	float mirtest = 0.0f, mirdist = 0.0f;

	/* perturb normal */
	// if there's a bump map, we use that, else
	float pdot = -optix::dot( ray.direction, ffnormal );

	/* angular transmission */
	float cos2 = sqrtf( 1.0f + ( pdot * pdot - 1.0f ) / ( rindex * rindex ) );
	if (hastrans) {
		mcolor = make_float3( powf( mcolor.x, 1.0f / cos2 ), powf( mcolor.y, 1.0f / cos2 ), powf( mcolor.z, 1.0f / cos2 ) );
	}

	/* compute reflection */
	float r1e = (pdot - rindex*cos2) / (pdot + rindex*cos2);
	r1e *= r1e;
	float r1m = (1.0f/pdot - rindex/cos2) / (1.0f/pdot + rindex/cos2);
	r1m *= r1m;

	/* compute transmission */
	if (hastrans) {
		float3 trans = 0.5f * (1.0f-r1e) * (1.0f-r1e) * mcolor / (1.0f - r1e * r1e * mcolor * mcolor);
		trans       += 0.5f * (1.0f-r1m) * (1.0f-r1m) * mcolor / (1.0f - r1m * r1m * mcolor * mcolor);

		/* modify by pattern */
		//trans *= pcol;

		/* transmitted ray */
		new_prd.weight = prd.weight * fmaxf(trans);
		if (new_prd.weight >= minweight) {
			new_prd.depth = prd.depth;
			new_prd.ambient_depth = prd.ambient_depth;
			//new_prd.seed = prd.seed;//lcg( prd.seed );
			new_prd.state = prd.state;
#ifdef FILL_GAPS
			new_prd.primary = 0;
#endif
#ifdef RAY_COUNT
			new_prd.ray_count = 1;
#endif
#ifdef HIT_COUNT
			new_prd.hit_count = 0;
#endif
			Ray trans_ray = make_Ray( hit_point, ray.direction, radiance_ray_type, ray_start( hit_point, ray.direction, ffnormal, RAY_START ), RAY_END );
			rtTrace(top_object, trans_ray, new_prd);
			float3 rcol = new_prd.result * trans;
			result += rcol;
			transtest = 2.0f * bright( rcol );
			transdist = t_hit + new_prd.distance;
#ifdef RAY_COUNT
			prd.ray_count += new_prd.ray_count;
#endif
#ifdef HIT_COUNT
			prd.hit_count += new_prd.hit_count;
#endif
		}
	}
	// stop if it's a shadow ray, which it isn't

	/* compute reflectance */
	float3 refl = 0.5f * r1e * ( 1.0f + (1.0f-2.0f*r1e) * mcolor * mcolor ) / (1.0f - r1e * r1e * mcolor * mcolor );
	refl       += 0.5f * r1m * ( 1.0f + (1.0f-2.0f*r1m) * mcolor * mcolor ) / (1.0f - r1m * r1m * mcolor * mcolor );

	/* reflected ray */
	new_prd.weight = prd.weight * fmaxf(refl);
	new_prd.depth = prd.depth + 1;
	if (new_prd.weight >= minweight && new_prd.depth <= abs(maxdepth)) {
		new_prd.ambient_depth = prd.ambient_depth;
		//new_prd.seed = prd.seed;//lcg( prd.seed );
		new_prd.state = prd.state;
#ifdef FILL_GAPS
		new_prd.primary = 0;
#endif
#ifdef RAY_COUNT
		new_prd.ray_count = 1;
#endif
#ifdef HIT_COUNT
		new_prd.hit_count = 0;
#endif
		float3 R = reflect( ray.direction, ffnormal );
		Ray refl_ray = make_Ray( hit_point, R, radiance_ray_type, ray_start( hit_point, R, ffnormal, RAY_START ), RAY_END );
		rtTrace(top_object, refl_ray, new_prd);
		float3 rcol = new_prd.result * refl;
		result += rcol;
		mirtest = 2.0f * bright( rcol );
		mirdist = t_hit + new_prd.distance;
#ifdef RAY_COUNT
		prd.ray_count += new_prd.ray_count;
#endif
#ifdef HIT_COUNT
		prd.hit_count += new_prd.hit_count;
#endif
	}
  
	/* check distance */
	float d = bright( result );
	if (transtest > d)
		prd.distance = transdist;
	else if (mirtest > d)
		prd.distance = mirdist;
	else
		prd.distance = t_hit;

	// pass the color back up the tree
	prd.result = result;
	
#ifdef HIT_TYPE
	prd.hit_type = type;
#endif
}

RT_METHOD float bright( const float3 &rgb )
{
	return dot( rgb, CIE_rgbf );
}
