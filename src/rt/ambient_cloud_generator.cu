#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2016 Nathaniel Jones
 * Massachusetts Institute of Technology
 */

#include <optix_world.h>
#include "optix_shader_common.h"
#include "optix_point_common.h"

using namespace optix;

#ifdef DAYSIM_COMPATIBLE
#define threadIndex()	((launch_index.x + launch_dim.x * launch_index.y) / stride + segment_offset)
#else
#define threadIndex()	((launch_index.x + launch_dim.x * launch_index.y) / stride)
#endif

/* Program variables */
rtDeclareVariable(unsigned int,  stride, , ) = 1u; /* Spacing between used threads in warp. */

/* Contex variables */
rtBuffer<PointDirection, 1>      cluster_buffer; /* input */
rtBuffer<AmbientRecord, 1>       ambient_record_buffer; /* ambient record output */
#ifdef DAYSIM_COMPATIBLE
rtBuffer<DC, 2>                  ambient_dc_buffer; /* daylight coefficient output */
#endif
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(rtObject,      top_irrad, , );
rtDeclareVariable(unsigned int,  ambient_record_ray_type, , );
rtDeclareVariable(unsigned int,  level, , ) = 0u;
#ifdef DAYSIM_COMPATIBLE
rtDeclareVariable(unsigned int,  segment_offset, , ) = 0u; /* Offset into data if computed with multiple segments */
#endif /* DAYSIM_COMPATIBLE */
rtDeclareVariable(unsigned int,  imm_irrad, , ) = 0u; /* Immediate irradiance (-I) */

/* OptiX variables */
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );
//rtDeclareVariable(unsigned int, launch_index, rtLaunchIndex, );
//rtDeclareVariable(unsigned int, launch_dim,   rtLaunchDim, );


RT_PROGRAM void ambient_cloud_camera()
{
	// Check stride
	if ((launch_index.x + launch_dim.x * launch_index.y) % stride)
		return;
	const unsigned int index = threadIndex();
	if (index >= cluster_buffer.size())
		return;

	PerRayData_ambient_record prd;
	init_rand(&prd.state, launch_index.x + launch_dim.x * (launch_index.y + launch_dim.y * level));
	prd.parent = NULL;
	prd.result.pos = prd.result.val = make_float3( 0.0f );
	prd.result.lvl = level;
	prd.result.weight = 1.0f;
	for ( int i = level; i--; )
		prd.result.weight *= AVGREFL; // Compute weight as in makeambient() from ambient.c
#ifndef OLDAMB
	prd.result.rad = make_float2( 0.0f );
	prd.result.udir = 0; // Initialize in case something goes wrong
#else
	prd.result.rad = 0.0f;
	prd.result.dir = make_float3( 0.0f ); // Initialize in case something goes wrong
#endif
#ifdef DAYSIM_COMPATIBLE
	prd.dc = make_uint3(0, 0, index - segment_offset);
	daysimSet(prd.dc, 0.0f);
#endif
#ifdef RAY_COUNT
	prd.result.ray_count = 1;
#endif
#ifdef HIT_COUNT
	prd.result.hit_count = 0;
#endif

	// Get the position and normal of the ambient record to be created
	PointDirection cluster = cluster_buffer[index];

	if ( dot( cluster.dir, cluster.dir ) > FTINY ) { // Check that this is a valid ray
		float3 ray_direction = -normalize( cluster.dir ); // Ray will face opposite the normal direction
		const float tmin = ray_start( cluster.pos, RAY_START );
		Ray ray = make_Ray(cluster.pos, ray_direction, ambient_record_ray_type, -tmin, tmin);
		if ( imm_irrad && !level )
			rtTrace(top_irrad, ray, prd);
		else
			rtTrace(top_object, ray, prd);
	}

	checkFinite(prd.result.val);
#ifndef OLDAMB
	checkFinite(prd.result.gdir);
#endif

	ambient_record_buffer[index] = prd.result;
#ifdef DAYSIM_COMPATIBLE
	if (ambient_dc_buffer.size().x)
		daysimCopy(&ambient_dc_buffer[make_uint2(0, index)], prd.dc);
#endif
}

RT_PROGRAM void exception()
{
	// Check stride
	if ((launch_index.x + launch_dim.x * launch_index.y) % stride)
		return;
	const unsigned int index = threadIndex();
	if (index >= ambient_record_buffer.size())
		return;

	const unsigned int code = rtGetExceptionCode();
	rtPrintf("Caught exception 0x%X at launch index (%d,%d)\n", code, launch_index.x, launch_index.y);
	ambient_record_buffer[index].lvl = level;
	ambient_record_buffer[index].val = exceptionToFloat3( code );
	ambient_record_buffer[index].weight = -1.0f;
}
