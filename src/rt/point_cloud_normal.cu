#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2015 Nathaniel Jones
 * Massachusetts Institute of Technology
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "optix_shader_common.h"

using namespace optix;

/* OptiX variables */
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(PerRayData_point_cloud, prd, rtPayload, );

/* Attributes */
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
#ifdef ANTIMATTER
rtDeclareVariable(int, mat_id, attribute mat_id, );

/* Context variables */
rtDeclareVariable(rtObject, top_object, , );
#endif

#ifdef AMBIENT_CELL
/* Context variables */
rtDeclareVariable(float3, cuorg, , ); /* bounding box minimum */
rtDeclareVariable(float, cell_size, , ); /* cell side dimension */
rtDeclareVariable(unsigned int, level, , ) = 0u;

rtDeclareVariable(unsigned int, ambient_ray_type, , );
rtDeclareVariable(rtObject, top_ambient, , );


RT_METHOD uint2 cell_hash(const float3& pos, const float3& dir)
{
	uint2 cell;
	float3 absdir = make_float3(fabsf(dir.x), fabsf(dir.y), fabsf(dir.z));
	if (absdir.x > absdir.y) {
		if (absdir.x > absdir.z)
			cell.x = dir.x > 0 ? 0 : 0x10000;
		else
			cell.x = dir.z > 0 ? 0x40000 : 0x50000;
	}
	else {
		if (absdir.y > absdir.z)
			cell.x = dir.y > 0 ? 0x20000 : 0x30000;
		else
			cell.x = dir.z > 0 ? 0x40000 : 0x50000;
	}
	float3 cell_index = (pos - cuorg) / cell_size;
	cell.x += ((unsigned int)cell_index.x) & 0xffff;
	cell.y = (((unsigned int)cell_index.y) << 16) + (((unsigned int)cell_index.z) & 0xffff);
	return cell;
}

RT_METHOD int occupied(const float3& pos, const float3& dir, const float3& world)
{
	float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));

	PerRayData_ambient ambient_prd;
	ambient_prd.result = make_float3(0.0f);
	ambient_prd.surface_normal = faceforward(world_shading_normal, -ray.direction, world);
	ambient_prd.ambient_depth = level;
	ambient_prd.wsum = 0.0f;
	ambient_prd.weight = 1.0f;
	for ( int i = level; i--; )
		ambient_prd.weight *= AVGREFL; // Compute weight as in makeambient() from ambient.c

#ifdef OLDAMB
	ambient_prd.state = prd.state; // TODO make available here
#endif
#ifdef DAYSIM_COMPATIBLE
	ambient_prd.dc = daysimNext(prd.dc);
	daysimSet(ambient_prd.dc, 0.0f);
#endif
#ifdef HIT_COUNT
	ambient_prd.hit_count = 0;
#endif
	const float tmin = ray_start(pos, AMBIENT_RAY_LENGTH);
	Ray ambient_ray = make_Ray(pos, dir, ambient_ray_type, -tmin, tmin);
	rtTrace(top_ambient, ambient_ray, ambient_prd);
#ifdef HIT_COUNT
	prd.hit_count += ambient_prd.hit_count;
#endif
	return ambient_prd.wsum > FTINY;
}
#endif /* AMBIENT_CELL */

RT_PROGRAM void any_hit_point_cloud_glass()
{
#ifdef ANTIMATTER
	if (prd.mask & (1 << mat_id)) {
		rtIgnoreIntersection();
		return;
	}
#endif /* ANTIMATTER */

	float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );

	prd.backup.pos = ray.origin + t_hit * ray.direction;
	prd.backup.dir = faceforward(world_geometric_normal, -ray.direction, world_geometric_normal);
#ifdef AMBIENT_CELL
	prd.backup.cell = cell_hash(prd.backup.pos, prd.backup.dir);
#endif

	//TODO should probably use first intersection only and send transmitted ray
	rtIgnoreIntersection();
}

RT_PROGRAM void closest_hit_point_cloud_normal()
{
	float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );

#ifdef ANTIMATTER
	if (prd.mask & (1 << mat_id)) {
		prd.inside += dot(world_geometric_normal, ray.direction) < 0.0f ? 1 : -1;

		/* Continue the ray */
		float3 snormal = faceforward(world_geometric_normal, -ray.direction, world_geometric_normal);
		Ray new_ray = make_Ray(ray.origin, ray.direction, ray.ray_type, ray_start(ray.origin + t_hit * ray.direction, ray.direction, snormal, RAY_START) + t_hit, RAY_END);
		rtTrace(top_object, new_ray, prd);
		return;
	}
#endif /* ANTIMATTER */

	prd.result.pos = ray.origin + t_hit * ray.direction;
	prd.result.dir = faceforward(world_geometric_normal, -ray.direction, world_geometric_normal);

	/* Don't reflect off occluded surfaces */
	if (dot(ray.direction, prd.backup.pos - prd.result.pos) > 0)
		clear(prd.backup);

#ifdef AMBIENT_CELL
	if (occupied(prd.result.pos, prd.result.dir, world_geometric_normal))
		clear(prd.result);
	else
		prd.result.cell = cell_hash(prd.result.pos, prd.result.dir);
#endif
}

RT_PROGRAM void closest_hit_point_cloud_light()
{
#ifdef ANTIMATTER
	if (prd.mask & (1 << mat_id)) {
		float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
		prd.inside += dot(world_geometric_normal, ray.direction) < 0.0f ? 1 : -1;

		/* Continue the ray */
		float3 snormal = faceforward(world_geometric_normal, -ray.direction, world_geometric_normal);
		Ray new_ray = make_Ray(ray.origin, ray.direction, ray.ray_type, ray_start(ray.origin + t_hit * ray.direction, ray.direction, snormal, RAY_START) + t_hit, RAY_END);
		rtTrace(top_object, new_ray, prd);
		return;
	}
#endif /* ANTIMATTER */

	/* Don't reflect off occluded surfaces */
	if (dot(ray.direction, prd.backup.pos - prd.result.pos) > 0)
		clear(prd.backup);

#ifdef AMBIENT_CELL
	clear(prd.result);
#else
	prd.result = prd.backup;
#endif
}

RT_PROGRAM void point_cloud_miss()
{
#ifdef AMBIENT_CELL
	clear(prd.result);
#else
	prd.result = prd.backup;
#endif
}
