#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2016 Nathaniel Jones
 * Massachusetts Institute of Technology
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "optix_shader_common.h"
#include "optix_point_common.h"

using namespace optix;

/* Material variables */
rtDeclareVariable(float, spec, , ) = 0.0f;	/* The material specularity given by the rad file "plastic", "metal", or "trans" object */
rtDeclareVariable(float, rough, , ) = 0.0f;	/* The material roughness given by the rad file "plastic", "metal", or "trans" object */
rtDeclareVariable(float, transm, , ) = 0.0f;	/* The material transmissivity given by the rad file "trans" object */
rtDeclareVariable(float, tspecu, , ) = 0.0f;	/* The material transmitted specular component given by the rad file "trans" object */
rtDeclareVariable(unsigned int, ambincl, , ) = 1u;	/* Flag to skip ambient calculation and use default (ae, aE, ai, aI) */

/* OptiX variables */
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(PerRayData_point_cloud, prd, rtPayload, );

/* Attributes */
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
#ifdef ANTIMATTER
rtDeclareVariable(int, mat_id, attribute mat_id, );

/* Context variables */
rtDeclareVariable(rtObject, top_object, , );
#endif

/* Context variables */
rtBuffer<PointDirection, 3>      seed_buffer; /* output */

rtDeclareVariable(float, specthresh, , );	/* This is the minimum fraction of reflection or transmission, under which no specular sampling is performed */
rtDeclareVariable(float, specjitter, , );	/* specular sampling (ss) */

#ifdef AMBIENT_CELL
rtDeclareVariable(float3, cuorg, , ); /* bounding box minimum */
rtDeclareVariable(float, cell_size, , ); /* cell side dimension */
rtDeclareVariable(unsigned int, level, , ) = 0u;

rtDeclareVariable(unsigned int, ambient_ray_type, , );
rtDeclareVariable(rtObject, top_ambient, , );


RT_METHOD uint2 cell_hash(const float3& pos, const float3& dir)
{
	uint2 cell;
	float3 absdir = make_float3(fabsf(dir.x), fabsf(dir.y), fabsf(dir.z));
	if (absdir.x > absdir.y) {
		if (absdir.x > absdir.z)
			cell.x = dir.x > 0 ? 0 : 0x10000;
		else
			cell.x = dir.z > 0 ? 0x40000 : 0x50000;
	}
	else {
		if (absdir.y > absdir.z)
			cell.x = dir.y > 0 ? 0x20000 : 0x30000;
		else
			cell.x = dir.z > 0 ? 0x40000 : 0x50000;
	}
	float3 cell_index = (pos - cuorg) / cell_size;
	cell.x += ((unsigned int)cell_index.x) & 0xffff;
	cell.y = (((unsigned int)cell_index.y) << 16) + (((unsigned int)cell_index.z) & 0xffff);
	return cell;
}

RT_METHOD int occupied(const float3& pos, const float3& dir, const float3& world)
{
	float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));

	PerRayData_ambient ambient_prd;
	ambient_prd.result = make_float3(0.0f);
	ambient_prd.surface_normal = faceforward(world_shading_normal, -ray.direction, world);
	ambient_prd.ambient_depth = level;
	ambient_prd.wsum = 0.0f;
	ambient_prd.weight = 1.0f;
	for ( int i = level; i--; )
		ambient_prd.weight *= AVGREFL; // Compute weight as in makeambient() from ambient.c

#ifdef OLDAMB
	ambient_prd.state = prd.state;
#endif
#ifdef DAYSIM_COMPATIBLE
	ambient_prd.dc = make_uint3(0u); // Mark as null (TODO check this)
#endif
#ifdef HIT_COUNT
	ambient_prd.hit_count = 0;
#endif
	const float tmin = ray_start(pos, AMBIENT_RAY_LENGTH);
	Ray ambient_ray = make_Ray(pos, dir, ambient_ray_type, -tmin, tmin);
	rtTrace(top_ambient, ambient_ray, ambient_prd);
#ifdef HIT_COUNT
	prd.hit_count += ambient_prd.hit_count;
#endif
	return ambient_prd.wsum > FTINY;
}
#else /* AMBIENT_CELL */
RT_METHOD float3 sample_hemisphere(const float3& uz)
{
	float3 ux = getperpendicular(uz);
	float3 uy = normalize(cross(uz, ux));
	float zd = sqrtf(hiprand_uniform(prd.state));
	float phi = 2.0f*M_PIf * hiprand_uniform(prd.state);
	float xd = cosf(phi) * zd;
	float yd = sinf(phi) * zd;
	zd = sqrtf(1.0f - zd*zd);
	return normalize(xd*ux + yd*uy + zd*uz);
}
#endif /* AMBIENT_CELL */

RT_PROGRAM void closest_hit_point_cloud_glass()
{
	float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	float3 snormal = faceforward(world_geometric_normal, -ray.direction, world_geometric_normal);

#ifdef ANTIMATTER
	if (prd.mask & (1 << mat_id)) {
		prd.inside += dot(world_geometric_normal, ray.direction) < 0.0f ? 1 : -1;

		/* Continue the ray */
		Ray new_ray = make_Ray(ray.origin, ray.direction, ray.ray_type, ray_start(ray.origin + t_hit * ray.direction, ray.direction, snormal, RAY_START) + t_hit, RAY_END);
		rtTrace(top_object, new_ray, prd);
		return;
	}
#endif /* ANTIMATTER */

	prd.point = ray.origin + t_hit * ray.direction;

	/* Transmission */
#ifdef AMBIENT_CELL
	prd.forward = ray.direction;
#else
	prd.forward = sample_hemisphere(-snormal);
#endif

	/* Reflection */
#ifdef AMBIENT_CELL
	prd.reverse = reflect(ray.direction, snormal);
#else
	prd.reverse = sample_hemisphere(snormal);
#endif
}

RT_PROGRAM void closest_hit_point_cloud_normal()
{
	float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
	float3 snormal = faceforward(world_geometric_normal, -ray.direction, world_geometric_normal);

#ifdef ANTIMATTER
	if (prd.mask & (1 << mat_id)) {
		prd.inside += dot(world_geometric_normal, ray.direction) < 0.0f ? 1 : -1;

		/* Continue the ray */
		Ray new_ray = make_Ray(ray.origin, ray.direction, ray.ray_type, ray_start(ray.origin + t_hit * ray.direction, ray.direction, snormal, RAY_START) + t_hit, RAY_END);
		rtTrace(top_object, new_ray, prd);
		return;
	}
#endif /* ANTIMATTER */

	float trans = transm * (1.0f - spec);
	float tspec = trans * tspecu;
	float alpha2 = rough * rough;

	/* Record new origin */
	prd.point = ray.origin + t_hit * ray.direction;

	/* Transmitted ambient */
	if (ambincl && trans - tspec > FTINY && prd.index.z < prd.seeds) {
#ifdef AMBIENT_CELL
		if (!occupied(prd.point, -snormal, world_geometric_normal)) {
			seed_buffer[prd.index].cell = cell_hash(prd.point, -snormal);
#endif

			/* Store seed point */
			seed_buffer[prd.index].pos = prd.point;
			seed_buffer[prd.index].dir = -snormal;
			prd.index.z++;

#ifdef AMBIENT_CELL
		}
#endif
	}

	/* Reflected ambient */
	if (ambincl && 1.0f - trans - spec > FTINY && prd.index.z < prd.seeds) {
#ifdef AMBIENT_CELL
		if (!occupied(prd.point, snormal, world_geometric_normal)) {
			seed_buffer[prd.index].cell = cell_hash(prd.point, snormal);
#endif

			/* Store seed point */
			seed_buffer[prd.index].pos = prd.point;
			seed_buffer[prd.index].dir = snormal;
			prd.index.z++;

#ifdef AMBIENT_CELL
		}
#endif
	}

	if (prd.index.z >= prd.seeds) return;

	/* Transmitted ray */
	if (tspec > FTINY && (alpha2 <= FTINY || specthresh < tspec - FTINY)) {
#ifdef AMBIENT_CELL
		prd.forward = ray.direction;

		if (alpha2 > FTINY) {
			float3 u = getperpendicular(-snormal); //TODO should be pnormal
			float3 v = cross(-snormal, u);
			float2 rv = make_float2(hiprand_uniform(prd.state), hiprand_uniform(prd.state)); // should be evenly distributed in both dimensions
			float d = 2.0f * M_PIf * rv.x;
			float cosp = cosf(d);
			float sinp = sinf(d);
			if ((0.0f <= specjitter) && (specjitter < 1.0f))
				rv.y = 1.0f - specjitter * rv.y;
			if (rv.y <= FTINY)
				d = 1.0f;
			else
				d = sqrtf(alpha2 * -logf(rv.y));
			float3 h = d * (cosp * u + sinp * v) - snormal; //TODO should be pnormal
			d = -2.0f * dot(h, prd.forward) / (1.0f + d*d);
			h = prd.forward + h * d;

			/* sample rejection test */
			if (dot(h, snormal) < -FTINY)
				prd.forward = h;
		}
#else
		prd.forward = sample_hemisphere(-snormal);
#endif
	}

	/* Reflected ray */
	if (spec > FTINY && (alpha2 <= FTINY || specthresh < spec - FTINY)) {
#ifdef AMBIENT_CELL
		prd.reverse = reflect(ray.direction, snormal);

		if (alpha2 > FTINY) {
			float3 u = getperpendicular(snormal); //TODO should be pnormal
			float3 v = cross(snormal, u);
			float2 rv = make_float2(hiprand_uniform(prd.state), hiprand_uniform(prd.state)); // should be evenly distributed in both dimensions
			float d = 2.0f * M_PIf * rv.x;
			float cosp = cosf(d);
			float sinp = sinf(d);
			if ((0.0f <= specjitter) && (specjitter < 1.0f))
				rv.y = 1.0f - specjitter * rv.y;
			if (rv.y <= FTINY)
				d = 1.0f;
			else
				d = sqrtf(alpha2 * -logf(rv.y));
			float3 h = d * (cosp * u + sinp * v) + snormal; //TODO should be pnormal
			d = -2.0f * dot(h, prd.reverse) / (1.0f + d*d);
			h = prd.reverse + h * d;

			/* sample rejection test */
			if (dot(h, snormal) > FTINY)
				prd.reverse = h;
		}
#else
		prd.reverse = sample_hemisphere(snormal);
#endif
	}
}

RT_PROGRAM void closest_hit_point_cloud_light()
{
#ifdef ANTIMATTER
	if (prd.mask & (1 << mat_id)) {
		float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
		prd.inside += dot(world_geometric_normal, ray.direction) < 0.0f ? 1 : -1;

		/* Continue the ray */
		float3 snormal = faceforward(world_geometric_normal, -ray.direction, world_geometric_normal);
		Ray new_ray = make_Ray(ray.origin, ray.direction, ray.ray_type, ray_start(ray.origin + t_hit * ray.direction, ray.direction, snormal, RAY_START) + t_hit, RAY_END);
		rtTrace(top_object, new_ray, prd);
		return;
	}
#endif /* ANTIMATTER */
}
