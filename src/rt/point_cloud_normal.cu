#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013-2015 Nathaniel Jones
 * Massachusetts Institute of Technology
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "optix_shader_common.h"

using namespace optix;

/* OptiX variables */
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(PerRayData_point_cloud, prd, rtPayload, );

/* Attributes */
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );


RT_PROGRAM void any_hit_point_cloud_glass()
{
	float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
	float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );

	prd.backup.pos = ray.origin + t_hit * ray.direction;
	prd.backup.dir = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

	//TODO should probably use first intersection only and send transmitted ray
	rtIgnoreIntersection();
}

RT_PROGRAM void closest_hit_point_cloud()
{
	float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
	float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );

	prd.result.pos = ray.origin + t_hit * ray.direction;
	prd.result.dir = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );
}

RT_PROGRAM void point_cloud_miss()
{
	prd.result = prd.backup;
}
