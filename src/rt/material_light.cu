#include "hip/hip_runtime.h"
/*
 *  material_light.cu - hit programs for light materials on GPUs.
 */

#include "accelerad_copyright.h"

#include "otypes.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "optix_shader_ray.h"


using namespace optix;

/* Context variables */
rtDeclareVariable(int,          directvis, , );		/* Boolean switch for light source visibility (dv) */


RT_METHOD int spotout(const IntersectData &data);


RT_CALLABLE_PROGRAM PerRayData_shadow closest_hit_light_shadow(IntersectData const&data, PerRayData_shadow prd_shadow)
{
	if (data.t > data.mat.params.l.maxrad || spotout(data) || dot(data.world_shading_normal, data.ray_direction) > 0.0f || data.surface_id != -prd_shadow.target - 1)
		prd_shadow.result = make_float3( 0.0f );
	else if (data.mat.params.l.function > RT_PROGRAM_ID_NULL)
		prd_shadow.result = data.mat.color * ((rtCallableProgramId<float3(const float3, const float3)>)data.mat.params.l.function)(data.ray_direction, data.world_shading_normal);
	else
		prd_shadow.result = data.mat.color;
	return prd_shadow;
}

RT_CALLABLE_PROGRAM PerRayData_radiance closest_hit_light_radiance(IntersectData const&data, PerRayData_radiance prd)
{
	// no contribution to ambient calculation
	if (!directvis || data.t > data.mat.params.l.maxrad && prd.depth > 0 || prd.ambient_depth > 0 || spotout(data) || dot(data.world_shading_normal, data.ray_direction) > 0.0f) //TODO need a better ambient test
		prd.result = make_float3( 0.0f );
	else if (data.mat.params.l.function > RT_PROGRAM_ID_NULL)
		prd.result = data.mat.color * ((rtCallableProgramId<float3(const float3, const float3)>)data.mat.params.l.function)(data.ray_direction, data.world_shading_normal);
	else
		prd.result = data.mat.color;
	prd.mirror = make_float3(0.0f);
	prd.distance = prd.mirror_distance = data.t;
	return prd;
}

RT_METHOD int spotout(const IntersectData &data)
{
	if (data.mat.type != MAT_SPOT)
		return(0); /* Not a spotlight */
	if (data.mat.params.l.flen < -FTINY) {		/* distant source */
		const float3 ray_origin = data.hit - data.t * data.ray_direction;
		const float3 vd = data.mat.params.l.aim - ray_origin;
		float d = dot(data.ray_direction, vd);
		/*			wrong side?
		if (d <= FTINY)
			return(1);	*/
		d = dot( vd, vd ) - d * d;
		return (M_PIf * d > data.mat.params.l.siz); /* If true then out */
	}
					/* local source */
	return (data.mat.params.l.siz < 2.0f * M_PIf * (1.0f + dot(data.mat.params.l.aim, data.ray_direction)));	/* If true then out */
}
